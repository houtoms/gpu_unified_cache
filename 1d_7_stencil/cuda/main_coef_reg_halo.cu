#include "hip/hip_runtime.h"
#include <iostream>
#include "metrics.h"
using namespace std;
// #define DATA_TYPE float
// #define DATA_TYPE double
#define warpSize 32 

#define  IN_1D(_x)  in[_x]
#define OUT_1D(_x) out[_x]

// #define __DEBUG

#ifdef __DEBUG
#define ITER 1
#else
#define ITER 100
#endif

void Init_Input_1D(DATA_TYPE *in, int n, int halo, unsigned int seed)
{
    srand(seed);
    for(int i = 0; i < n+2*halo; i++)
    {
        if(i < halo || i >= n+halo)
            IN_1D(i) = 0.0;
        else
#ifdef __DEBUG
            IN_1D(i) = 1.0; 
                // IN_2D(i,j) = (DATA_TYPE)rand()*100.0 / ((long)RAND_MAX);
#else
            IN_1D(i) = (DATA_TYPE)rand()*10.0 / ((long)RAND_MAX);
#endif
    }
}

void Clear_Output_1D(DATA_TYPE *in, int n, int halo)
{
    for(int i = 0; i < n+2*halo; i++)
    {
        IN_1D(i) = 0.0;
    }
}

void Show_Me(DATA_TYPE *in, int n, int halo, string prompt)
{
    cout << prompt << endl;
    for(int i = 0; i < n+2*halo; i++)
    {
        std::cout << IN_1D(i) << ",";
    }
    std::cout << std::endl;
}

inline double tol_finder(int error_tol)
{
    double val = 1.0;
    for(; error_tol > 0; error_tol--)
        val *= 10;
    return 1.0/(double)val;
}

bool Verify(DATA_TYPE *test, DATA_TYPE *ref, int n)
{
    bool flag = true;
    double precision = tol_finder(2);

    for(int i = 0; i < n; i++)
    {
        if(fabs(test[i]-ref[i]) > precision)
        {
            std::cout << "wrong at " << i << " test:" << test[i] << " (ref: " << ref[i] << ")";
            std::cout << std::endl;
            flag = false;
            break;
        }
    }
    return flag;
}


void Stencil_Seq(DATA_TYPE *in, DATA_TYPE *out, 
        DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2,
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int n, int halo)
{
#pragma omp parallel for
    for(int i = halo; i < n+halo; i++)
    {
        OUT_1D(i) = a0*IN_1D(i-3) + 
                    a1*IN_1D(i-2) + 
                    a2*IN_1D(i-1) + 
                    a3*IN_1D(i  ) + 
                    a4*IN_1D(i+1) + 
                    a5*IN_1D(i+2) + 
                    a6*IN_1D(i+3) ;
    }
}

__global__ void Stencil_Cuda(DATA_TYPE *in, DATA_TYPE *out, 
        DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2,
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int n, int halo) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;  
    OUT_1D(i) = a0*IN_1D(i-3) + 
                a1*IN_1D(i-2) + 
                a2*IN_1D(i-1) + 
                a3*IN_1D(i  ) + 
                a4*IN_1D(i+1) + 
                a5*IN_1D(i+2) + 
                a6*IN_1D(i+3) ;
}

__global__ void Stencil_Cuda_Sm_Branch(DATA_TYPE *in, DATA_TYPE *out, 
        DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2,
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int n, int halo) 
{
    __shared__ DATA_TYPE local[256+2*3];
    unsigned int tid = threadIdx.x;
    unsigned int gid = threadIdx.x + blockIdx.x * blockDim.x + halo;  
    int local_id = tid + halo;
    local[local_id] = IN_1D(gid);
    if(tid == 0)
    {
        local[local_id-1] = IN_1D(gid-1);
        local[local_id-2] = IN_1D(gid-2);
        local[local_id-3] = IN_1D(gid-3);
    }
    if(tid == blockDim.x - 1)
    {
        local[local_id+1] = IN_1D(gid+1);
        local[local_id+2] = IN_1D(gid+2);
        local[local_id+3] = IN_1D(gid+3);
    }
    __syncthreads();

    OUT_1D(gid) = a0*local[local_id-3] + 
                  a1*local[local_id-2] + 
                  a2*local[local_id-1] + 
                  a3*local[local_id  ] + 
                  a4*local[local_id+1] + 
                  a5*local[local_id+2] + 
                  a6*local[local_id+3] ;
}

__global__ void Stencil_Cuda_Sm_Cyclic(DATA_TYPE *in, DATA_TYPE *out, 
        DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2,
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int n, int halo) 
{
    __shared__ DATA_TYPE local[256+2*3];
    unsigned int tid = threadIdx.x;
    unsigned int gid = threadIdx.x + blockIdx.x * blockDim.x + halo;  
    int local_id = tid + halo;

    unsigned int lane_id = threadIdx.x;
    int lane_id_it = lane_id;
    int blk_id_x = blockIdx.x;
    int new_i  = (blk_id_x<<8) + lane_id_it%262;
    int new_li = lane_id_it%262;
    local[new_li] = IN_1D(new_i);
    lane_id_it += 256;
    new_i  = (blk_id_x<<8) + (lane_id_it/262)*262 + lane_id_it%262;
    new_li = (lane_id_it/262)*262 + lane_id_it%262;
    if(new_li < 262)
        local[new_li] = IN_1D(new_i);
    
    __syncthreads();

    OUT_1D(gid) = a0*local[local_id-3] + 
                  a1*local[local_id-2] + 
                  a2*local[local_id-1] + 
                  a3*local[local_id  ] + 
                  a4*local[local_id+1] + 
                  a5*local[local_id+2] + 
                  a6*local[local_id+3] ;
}

__global__ void Stencil_Cuda_Shfl(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2,
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6 , int n, int halo) 
{
    unsigned int gid = threadIdx.x + blockIdx.x * blockDim.x + halo;  
    unsigned int tid = threadIdx.x;
    unsigned int lane_id = tid % warpSize;

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>5;

    DATA_TYPE reg0, reg1;
    int lane_id_it = lane_id;
    int new_i = (warp_id_x<<5) + lane_id_it%38;
    reg0 = IN_1D(new_i);
    lane_id_it += warpSize;
    new_i = (warp_id_x<<5) + (lane_id_it/38)*38 + lane_id_it%38;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    reg1 = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    int friend_id0;
    DATA_TYPE tx0, ty0;

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    sum0 += a0*tx0;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    sum0 += a1*((lane_id < 31)? tx0: ty0);

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    sum0 += a2*((lane_id < 30)? tx0: ty0);

    friend_id0 = (lane_id+3 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    sum0 += a3*((lane_id < 29)? tx0: ty0);

    friend_id0 = (lane_id+4 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    sum0 += a4*((lane_id < 28)? tx0: ty0);

    friend_id0 = (lane_id+5 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    sum0 += a5*((lane_id < 27)? tx0: ty0);

    friend_id0 = (lane_id+6 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    sum0 += a6*((lane_id < 26)? tx0: ty0);

    OUT_1D(gid) = sum0; 
}

__global__ void Stencil_Cuda_Shfl2(DATA_TYPE *in, DATA_TYPE *out, 
        DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2,
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int n, int halo) 
{
    unsigned int tid = threadIdx.x;
    unsigned int lane_id = tid % warpSize;
    unsigned int gid = (((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<6) + lane_id + halo;  
    int warp_id_x = ((((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<6) + lane_id)>>5;

    DATA_TYPE reg0, reg1, reg2;
    int lane_id_it = lane_id;
    int new_i = (warp_id_x<<5) + lane_id_it%38;
    reg0 = IN_1D(new_i);
    lane_id_it += warpSize;
    new_i = (warp_id_x<<5) + (lane_id_it/38)*38 + lane_id_it%38;
    reg1 = IN_1D(new_i);
    lane_id_it += warpSize;
    new_i = (warp_id_x<<5) + (lane_id_it/38)*38 + lane_id_it%38;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    reg2 = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    int friend_id0;
    // int friend_id1;
    DATA_TYPE tx0, ty0, tx1, ty1;

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    sum0 += a0*tx0;
    sum1 += a0*tx1;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    sum0 += a1*((lane_id < 31)? tx0: ty0);
    sum1 += a1*((lane_id < 31)? tx1: ty1);

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    sum0 += a2*((lane_id < 30)? tx0: ty0);
    sum1 += a2*((lane_id < 30)? tx1: ty1);

    friend_id0 = (lane_id+3 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    sum0 += a3*((lane_id < 29)? tx0: ty0);
    sum1 += a3*((lane_id < 29)? tx1: ty1);

    friend_id0 = (lane_id+4 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    sum0 += a4*((lane_id < 28)? tx0: ty0);
    sum1 += a4*((lane_id < 28)? tx1: ty1);

    friend_id0 = (lane_id+5 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    sum0 += a5*((lane_id < 27)? tx0: ty0);
    sum1 += a5*((lane_id < 27)? tx1: ty1);

    friend_id0 = (lane_id+6 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    sum0 += a6*((lane_id < 26)? tx0: ty0);
    sum1 += a6*((lane_id < 26)? tx1: ty1);

    OUT_1D(gid   ) = sum0; 
    OUT_1D(gid+32) = sum1; 
}

__global__ void Stencil_Cuda_Shfl4(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2,
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6 , int n, int halo) 
{
    unsigned int tid = threadIdx.x;
    unsigned int lane_id = tid % warpSize;
    unsigned int gid = (((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<7) + lane_id + halo;  
    int warp_id_x = ((((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<7) + lane_id)>>5;

    DATA_TYPE reg0, reg1, reg2, reg3, reg4;
    int lane_id_it = lane_id;
    int new_i = (warp_id_x<<5) + lane_id_it%38;
    reg0 = IN_1D(new_i);
    lane_id_it += warpSize;
    new_i = (warp_id_x<<5) + (lane_id_it/38)*38 + lane_id_it%38;
    reg1 = IN_1D(new_i);
    lane_id_it += warpSize;
    new_i = (warp_id_x<<5) + (lane_id_it/38)*38 + lane_id_it%38;
    reg2 = IN_1D(new_i);
    lane_id_it += warpSize;
    new_i = (warp_id_x<<5) + (lane_id_it/38)*38 + lane_id_it%38;
    reg3 = IN_1D(new_i);
    lane_id_it += warpSize;
    new_i = (warp_id_x<<5) + (lane_id_it/38)*38 + lane_id_it%38;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    reg4 = IN_1D(new_i);


    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    int friend_id0;
    // int friend_id1;
    DATA_TYPE tx0, ty0, tx1, ty1;
    DATA_TYPE tx2, ty2, tx3, ty3;

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    sum0 += a0*tx0;
    sum1 += a0*tx1;
    sum2 += a0*tx2;
    sum3 += a0*tx3;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    sum0 += a1*((lane_id < 31)? tx0: ty0);
    sum1 += a1*((lane_id < 31)? tx1: ty1);
    sum2 += a1*((lane_id < 31)? tx2: ty2);
    sum3 += a1*((lane_id < 31)? tx3: ty3);

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    sum0 += a2*((lane_id < 30)? tx0: ty0);
    sum1 += a2*((lane_id < 30)? tx1: ty1);
    sum2 += a2*((lane_id < 30)? tx2: ty2);
    sum3 += a2*((lane_id < 30)? tx3: ty3);

    friend_id0 = (lane_id+3 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    sum0 += a3*((lane_id < 29)? tx0: ty0);
    sum1 += a3*((lane_id < 29)? tx1: ty1);
    sum2 += a3*((lane_id < 29)? tx2: ty2);
    sum3 += a3*((lane_id < 29)? tx3: ty3);

    friend_id0 = (lane_id+4 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    sum0 += a4*((lane_id < 28)? tx0: ty0);
    sum1 += a4*((lane_id < 28)? tx1: ty1);
    sum2 += a4*((lane_id < 28)? tx2: ty2);
    sum3 += a4*((lane_id < 28)? tx3: ty3);

    friend_id0 = (lane_id+5 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    sum0 += a5*((lane_id < 27)? tx0: ty0);
    sum1 += a5*((lane_id < 27)? tx1: ty1);
    sum2 += a5*((lane_id < 27)? tx2: ty2);
    sum3 += a5*((lane_id < 27)? tx3: ty3);

    friend_id0 = (lane_id+6 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    sum0 += a6*((lane_id < 26)? tx0: ty0);
    sum1 += a6*((lane_id < 26)? tx1: ty1);
    sum2 += a6*((lane_id < 26)? tx2: ty2);
    sum3 += a6*((lane_id < 26)? tx3: ty3);

    OUT_1D(gid   ) = sum0; 
    OUT_1D(gid+32) = sum1; 
    OUT_1D(gid+64) = sum2; 
    OUT_1D(gid+96) = sum3; 
}

__global__ void Stencil_Cuda_Shfl8(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2,
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6 , int n, int halo) 
{
    unsigned int tid = threadIdx.x;
    unsigned int lane_id = tid % warpSize;
    unsigned int gid = (((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<8) + lane_id + halo;  
    int warp_id_x = ((((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<8) + lane_id)>>5;

    DATA_TYPE reg0, reg1, reg2, reg3, reg4, reg5,
              reg6, reg7, reg8;
    int new_i = (warp_id_x<<5) + lane_id%34;
    reg0 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+32)/34)*34 + (lane_id+32)%34;
    reg1 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+64)/34)*34 + (lane_id+64)%34;
    reg2 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+96)/34)*34 + (lane_id+96)%34;
    reg3 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+128)/34)*34 + (lane_id+128)%34;
    reg4 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+160)/34)*34 + (lane_id+160)%34;
    reg5 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+192)/34)*34 + (lane_id+192)%34;
    reg6 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+224)/34)*34 + (lane_id+224)%34;
    reg7 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+256)/34)*34 + (lane_id+256)%34;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    reg8 = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    DATA_TYPE sum4 = 0.0;
    DATA_TYPE sum5 = 0.0;
    DATA_TYPE sum6 = 0.0;
    DATA_TYPE sum7 = 0.0;
    int friend_id0;
    // int friend_id1;
    DATA_TYPE tx0, ty0, tx1, ty1;
    DATA_TYPE tx2, ty2, tx3, ty3;
    DATA_TYPE tx4, ty4, tx5, ty5;
    DATA_TYPE tx6, ty6, tx7, ty7;

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    tx4 = __shfl(reg4, friend_id0);
    tx5 = __shfl(reg5, friend_id0);
    tx6 = __shfl(reg6, friend_id0);
    tx7 = __shfl(reg7, friend_id0);
    sum0 += a0*tx0;
    sum1 += a0*tx1;
    sum2 += a0*tx2;
    sum3 += a0*tx3;
    sum4 += a0*tx4;
    sum5 += a0*tx5;
    sum6 += a0*tx6;
    sum7 += a0*tx7;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    tx4 = __shfl(reg4, friend_id0);
    ty4 = __shfl(reg5, friend_id0);
    tx5 = __shfl(reg5, friend_id0);
    ty5 = __shfl(reg6, friend_id0);
    tx6 = __shfl(reg6, friend_id0);
    ty6 = __shfl(reg7, friend_id0);
    tx7 = __shfl(reg7, friend_id0);
    ty7 = __shfl(reg8, friend_id0);
    sum0 += a1*((lane_id < 31)? tx0: ty0);
    sum1 += a1*((lane_id < 31)? tx1: ty1);
    sum2 += a1*((lane_id < 31)? tx2: ty2);
    sum3 += a1*((lane_id < 31)? tx3: ty3);
    sum4 += a1*((lane_id < 31)? tx4: ty4);
    sum5 += a1*((lane_id < 31)? tx5: ty5);
    sum6 += a1*((lane_id < 31)? tx6: ty6);
    sum7 += a1*((lane_id < 31)? tx7: ty7);

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    tx4 = __shfl(reg4, friend_id0);
    ty4 = __shfl(reg5, friend_id0);
    tx5 = __shfl(reg5, friend_id0);
    ty5 = __shfl(reg6, friend_id0);
    tx6 = __shfl(reg6, friend_id0);
    ty6 = __shfl(reg7, friend_id0);
    tx7 = __shfl(reg7, friend_id0);
    ty7 = __shfl(reg8, friend_id0);
    sum0 += a2*((lane_id < 30)? tx0: ty0);
    sum1 += a2*((lane_id < 30)? tx1: ty1);
    sum2 += a2*((lane_id < 30)? tx2: ty2);
    sum3 += a2*((lane_id < 30)? tx3: ty3);
    sum4 += a2*((lane_id < 30)? tx4: ty4);
    sum5 += a2*((lane_id < 30)? tx5: ty5);
    sum6 += a2*((lane_id < 30)? tx6: ty6);
    sum7 += a2*((lane_id < 30)? tx7: ty7);

    friend_id0 = (lane_id+3 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    tx4 = __shfl(reg4, friend_id0);
    ty4 = __shfl(reg5, friend_id0);
    tx5 = __shfl(reg5, friend_id0);
    ty5 = __shfl(reg6, friend_id0);
    tx6 = __shfl(reg6, friend_id0);
    ty6 = __shfl(reg7, friend_id0);
    tx7 = __shfl(reg7, friend_id0);
    ty7 = __shfl(reg8, friend_id0);
    sum0 += a3*((lane_id < 29)? tx0: ty0);
    sum1 += a3*((lane_id < 29)? tx1: ty1);
    sum2 += a3*((lane_id < 29)? tx2: ty2);
    sum3 += a3*((lane_id < 29)? tx3: ty3);
    sum4 += a3*((lane_id < 29)? tx4: ty4);
    sum5 += a3*((lane_id < 29)? tx5: ty5);
    sum6 += a3*((lane_id < 29)? tx6: ty6);
    sum7 += a3*((lane_id < 29)? tx7: ty7);

    friend_id0 = (lane_id+4 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    tx4 = __shfl(reg4, friend_id0);
    ty4 = __shfl(reg5, friend_id0);
    tx5 = __shfl(reg5, friend_id0);
    ty5 = __shfl(reg6, friend_id0);
    tx6 = __shfl(reg6, friend_id0);
    ty6 = __shfl(reg7, friend_id0);
    tx7 = __shfl(reg7, friend_id0);
    ty7 = __shfl(reg8, friend_id0);
    sum0 += a4*((lane_id < 28)? tx0: ty0);
    sum1 += a4*((lane_id < 28)? tx1: ty1);
    sum2 += a4*((lane_id < 28)? tx2: ty2);
    sum3 += a4*((lane_id < 28)? tx3: ty3);
    sum4 += a4*((lane_id < 28)? tx4: ty4);
    sum5 += a4*((lane_id < 28)? tx5: ty5);
    sum6 += a4*((lane_id < 28)? tx6: ty6);
    sum7 += a4*((lane_id < 28)? tx7: ty7);

    friend_id0 = (lane_id+5 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    tx4 = __shfl(reg4, friend_id0);
    ty4 = __shfl(reg5, friend_id0);
    tx5 = __shfl(reg5, friend_id0);
    ty5 = __shfl(reg6, friend_id0);
    tx6 = __shfl(reg6, friend_id0);
    ty6 = __shfl(reg7, friend_id0);
    tx7 = __shfl(reg7, friend_id0);
    ty7 = __shfl(reg8, friend_id0);
    sum0 += a5*((lane_id < 27)? tx0: ty0);
    sum1 += a5*((lane_id < 27)? tx1: ty1);
    sum2 += a5*((lane_id < 27)? tx2: ty2);
    sum3 += a5*((lane_id < 27)? tx3: ty3);
    sum4 += a5*((lane_id < 27)? tx4: ty4);
    sum5 += a5*((lane_id < 27)? tx5: ty5);
    sum6 += a5*((lane_id < 27)? tx6: ty6);
    sum7 += a5*((lane_id < 27)? tx7: ty7);

    friend_id0 = (lane_id+6 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    tx4 = __shfl(reg4, friend_id0);
    ty4 = __shfl(reg5, friend_id0);
    tx5 = __shfl(reg5, friend_id0);
    ty5 = __shfl(reg6, friend_id0);
    tx6 = __shfl(reg6, friend_id0);
    ty6 = __shfl(reg7, friend_id0);
    tx7 = __shfl(reg7, friend_id0);
    ty7 = __shfl(reg8, friend_id0);
    sum0 += a6*((lane_id < 26)? tx0: ty0);
    sum1 += a6*((lane_id < 26)? tx1: ty1);
    sum2 += a6*((lane_id < 26)? tx2: ty2);
    sum3 += a6*((lane_id < 26)? tx3: ty3);
    sum4 += a6*((lane_id < 26)? tx4: ty4);
    sum5 += a6*((lane_id < 26)? tx5: ty5);
    sum6 += a6*((lane_id < 26)? tx6: ty6);
    sum7 += a6*((lane_id < 26)? tx7: ty7);



    OUT_1D(gid    ) = sum0; 
    OUT_1D(gid+32 ) = sum1; 
    OUT_1D(gid+64 ) = sum2; 
    OUT_1D(gid+96 ) = sum3; 
    OUT_1D(gid+128) = sum4; 
    OUT_1D(gid+160) = sum5; 
    OUT_1D(gid+192) = sum6; 
    OUT_1D(gid+224) = sum7; 
}

int main(int argc, char **argv)
{
#ifdef __DEBUG
    int n = 512;
#else
    int n = 33554432; // 2^25
#endif
    int halo = 3; 
    int total = (n+2*halo);
    const int K = 7;
#ifdef __DEBUG
    DATA_TYPE args[K] = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
#else
    DATA_TYPE args[K] = {0.14, 0.14, 0.14, 0.14, 0.14, 0.14, 0.14};
#endif
    DATA_TYPE *in = new DATA_TYPE[total];
    DATA_TYPE *out_ref = new DATA_TYPE[total];
    unsigned int seed = time(NULL);
    Clear_Output_1D(in, n, halo);
    Clear_Output_1D(out_ref, n, halo);
    Init_Input_1D(in, n, halo, seed);

    // Show_Me(in, n, halo, "Input:");
    for(int i=0; i< ITER; i++)
    {
        Stencil_Seq(in, out_ref, 
                args[0], args[1], args[2], args[3], args[4], args[5], args[6], 
                n, halo);
        swap(in, out_ref);
    }
    swap(in, out_ref);
    // Show_Me(out_ref, n, halo, "Output:");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    float time_wo_pci;

    DATA_TYPE *in_d;
    DATA_TYPE *out_d;
    DATA_TYPE *out = new DATA_TYPE[total];
    hipMalloc((void**)&in_d, total*sizeof(DATA_TYPE));
    hipMalloc((void**)&out_d, total*sizeof(DATA_TYPE));
    dim3 dimGrid;
    dim3 dimBlock;
    
    // Cuda version
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dimGrid.x = (n)/256;
    dimGrid.y = 1;
    dimGrid.z = 1;
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda<<<dimGrid, dimBlock>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], args[5], args[6], 
                n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D7, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shared Memory with Branch
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dimGrid.x = (n)/256;
    dimGrid.y = 1;
    dimGrid.z = 1;
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Sm_Branch<<<dimGrid, dimBlock>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], args[5], args[6], 
                n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sm_Branch: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sm_Branch Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D7, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shared Memory with Cyclic
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dimGrid.x = (n)/256;
    dimGrid.y = 1;
    dimGrid.z = 1;
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Sm_Cyclic<<<dimGrid, dimBlock>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], args[5], args[6], 
                n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sm_Cyclic: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sm_Cyclic Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D7, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda Shfl 1D-Warp 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dimGrid.x = (n)/256;
    dimGrid.y = 1;
    dimGrid.z = 1;
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl<<<dimGrid, dimBlock>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], args[5], args[6], 
                n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D7, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shfl2 version 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dimGrid.x = (n)/(256*2);
    dimGrid.y = 1;
    dimGrid.z = 1;
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl2<<<dimGrid, dimBlock>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], args[5], args[6], 
                n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl2: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl2 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D7, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shfl4 version 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dimGrid.x = (n)/(256*4);
    dimGrid.y = 1;
    dimGrid.z = 1;
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl4<<<dimGrid, dimBlock>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], args[5], args[6], 
                n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl4: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl4 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D7, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    /*

    // Cuda Shfl8 version 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid5((n)/(256*8), 1, 1);
    dim3 dimBlock5(256, 1, 1);

    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl8<<<dimGrid5, dimBlock5>>>(in_d, out_d, args[0], args[1], args[2], args[3], args[4], args[5], 
                args[6] , n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, n, halo,  "Output(Device):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl8: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D7, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));
    */


    hipFree(in_d);
    hipFree(out_d);

    delete[] in;
    delete[] out;
    delete[] out_ref;
}

