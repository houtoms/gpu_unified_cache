#include "hip/hip_runtime.h"
#include <iostream>
#include <metrics.h>
using namespace std;
#define IN_3D(_z,_y,_x)   in[(_z)*(M)*(N)+(_y)*(N)+(_x)]
#define OUT_3D(_z,_y,_x) out[(_z)*(M)*(N)+(_y)*(N)+(_x)]
// #define ARG_3D(_l,_w,_x,_y)   args[(_l)*(Z)*(N)*(M)+(_w)*(N)*(M)+(_x)*(N)+(_y)]

#define SM_Z (8+2)
#define SM_M (4+2)
#define SM_N (8+2)
#define LOC_3D(_z,_y,_x) local[(_z)*(SM_M)*(SM_N)+(_y)*(SM_N)+(_x)]

#define SM_2D_M (4+2)
#define SM_2D_N (64+2)
#define LOC_2D(_y,_x) local[(_y)*(SM_2D_N)+(_x)]
// #define LOC_2D2(_y,_x) local[(_y)*(SM_2D_N2+2*halo)+(_x)]
#define LOC_L_2D(_z,_y,_x) local[(_z)*(SM_2D_M*SM_2D_N)+(_y)*(SM_2D_N)+(_x)]

// #define DATA_TYPE float
// #define DATA_TYPE double
#define warpSize 32 

// #define __DEBUG

#ifdef __DEBUG
#define ITER 1
#else
#define ITER 100
#endif


// #define TEMP
#define SPAC1

__device__ __inline__ double shfl(double x, int lane)
{
    // Split the double number into 2 32b registers.
    int lo, hi;
    asm volatile( "mov.b32 {%0,%1}, %2;" : "=r"(lo), "=r"(hi) : "d"(x));
    // Shuffle the two 32b registers.
    lo = __shfl(lo, lane);
    hi = __shfl(hi, lane);
    // Recreate the 64b number.
    asm volatile( "mov.b64 %0, {%1,%2};" : "=d"(x) : "r"(lo), "r"(hi));
    return x;
}

void Init_Input_3D(DATA_TYPE *in, int Z, int M, int N, unsigned int seed)
{
    srand(seed);

    for(int k = 0; k < Z; k++)
        for(int j = 0; j < M; j++)
            for(int i = 0; i < N; i++)
#ifdef __DEBUG
                IN_3D(k,j,i) = 1; 
                // IN_3D(k,j,i) = (DATA_TYPE)rand()*100.0 / ((long)RAND_MAX);
#else
                IN_3D(k,j,i) = (DATA_TYPE)rand()*10.0 / ((long)RAND_MAX);
#endif
}

/*
void Init_Args_3D(DATA_TYPE *args, int l, int Z, int M, int N, DATA_TYPE val)
{
    for(int k = 0; k < l; k++)
    {
        for(int w = 0; w < Z; w++)
        {
            for(int i = 0; i < M; i++)
            {
                for(int j = 0; j < N; j++)
                {
                    ARG_3D(k,w,i,j) = val; 
                }
            }
        }
    }
}
*/

void Clear_Output_3D(DATA_TYPE *in, int Z, int M, int N)
{
    for(int k = 0; k < Z; k++)
        for(int j = 0; j < M; j++)
            for(int i = 0; i < N; i++)
                IN_3D(k,j,i) = 0;
}

void Show_Me(DATA_TYPE *in, int Z, int M, int N, std::string prompt)
{
    std::cout << prompt << std::endl;
    for(int k = 0; k < Z; k++)
    {
        for(int j = 0; j < M; j++)
        {
            for(int i = 0; i < N; i++)
                std::cout << IN_3D(k,j,i) << ",";
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
}

void Stencil_Seq(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, 
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, DATA_TYPE a7, 
        DATA_TYPE a8, DATA_TYPE a9, DATA_TYPE a10, DATA_TYPE a11, DATA_TYPE a12, 
        DATA_TYPE a13, DATA_TYPE a14, DATA_TYPE a15, DATA_TYPE a16, DATA_TYPE a17, 
        DATA_TYPE a18, DATA_TYPE a19, DATA_TYPE a20, DATA_TYPE a21, DATA_TYPE a22, 
        DATA_TYPE a23, DATA_TYPE a24, DATA_TYPE a25, DATA_TYPE a26 , 
        int Z, int M, int N)
{
#pragma omp parallel for 
    for(int k = 0; k < Z; k++)
    {
        int b = (k == 0)      ? k : k - 1;
        int t = (k == Z-1)    ? k : k + 1;
        for(int j = 0; j < M; j++)
        {
            int n = (j == 0)      ? j : j - 1;
            int s = (j == M-1)    ? j : j + 1;
            for(int i = 0; i < N; i++)
            {
                int w = (i == 0)      ? i : i - 1;
                int e = (i == N-1)    ? i : i + 1;
                OUT_3D(k,j,i) = a0  * IN_3D(b,n,w) +
                                a1  * IN_3D(b,n,i) +
                                a2  * IN_3D(b,n,e) +
                                a3  * IN_3D(b,j,w) +
                                a4  * IN_3D(b,j,i) +
                                a5  * IN_3D(b,j,e) +
                                a6  * IN_3D(b,s,w) + 
                                a7  * IN_3D(b,s,i) + 
                                a8  * IN_3D(b,s,e) + 
                                a9  * IN_3D(k,n,w) + 
                                a10 * IN_3D(k,n,i) + 
                                a11 * IN_3D(k,n,e) + 
                                a12 * IN_3D(k,j,w) + 
                                a13 * IN_3D(k,j,i) + 
                                a14 * IN_3D(k,j,e) + 
                                a15 * IN_3D(k,s,w) + 
                                a16 * IN_3D(k,s,i) + 
                                a17 * IN_3D(k,s,e) + 
                                a18 * IN_3D(t,n,w) + 
                                a19 * IN_3D(t,n,i) + 
                                a20 * IN_3D(t,n,e) + 
                                a21 * IN_3D(t,j,w) + 
                                a22 * IN_3D(t,j,i) + 
                                a23 * IN_3D(t,j,e) + 
                                a24 * IN_3D(t,s,w) + 
                                a25 * IN_3D(t,s,i) + 
                                a26 * IN_3D(t,s,e) ;
            }
        }
    }
}

inline double tol_finder(int error_tol)
{
    double val = 1.0;
    for(; error_tol > 0; error_tol--)
        val *= 10;
    return 1.0/(double)val;
}

bool Verify(DATA_TYPE *test, DATA_TYPE *ref, int n)
{
    bool flag = true;
    double precision = tol_finder(2);

    for(int i = 0; i < n; i++)
    {
        if(fabs(test[i]-ref[i]) > precision)
        {
            std::cout << "difference: " << fabs(test[i]-ref[i])-precision << std::endl;
            std::cout << "wrong at " << i << " test:" << test[i] << " (ref: " << ref[i] << ")";
            std::cout << std::endl;
            flag = false;
            break;
        }
    }
    return flag;
}

__global__ void Stencil_Cuda(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, 
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, DATA_TYPE a7, 
        DATA_TYPE a8, DATA_TYPE a9, DATA_TYPE a10, DATA_TYPE a11, DATA_TYPE a12, 
        DATA_TYPE a13, DATA_TYPE a14, DATA_TYPE a15, DATA_TYPE a16, DATA_TYPE a17, 
        DATA_TYPE a18, DATA_TYPE a19, DATA_TYPE a20, DATA_TYPE a21, DATA_TYPE a22, 
        DATA_TYPE a23, DATA_TYPE a24, DATA_TYPE a25, DATA_TYPE a26 , 
        int Z, int M, int N) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.z + blockIdx.z * blockDim.z ;

    int b = (k == 0)      ? k : k - 1;
    int t = (k == Z-1)    ? k : k + 1;
    int n = (j == 0)      ? j : j - 1;
    int s = (j == M-1)    ? j : j + 1;
    int w = (i == 0)      ? i : i - 1;
    int e = (i == N-1)    ? i : i + 1;

    OUT_3D(k,j,i) = a0  * IN_3D(b,n,w) +
                    a1  * IN_3D(b,n,i) +
                    a2  * IN_3D(b,n,e) +
                    a3  * IN_3D(b,j,w) +
                    a4  * IN_3D(b,j,i) +
                    a5  * IN_3D(b,j,e) +
                    a6  * IN_3D(b,s,w) + 
                    a7  * IN_3D(b,s,i) + 
                    a8  * IN_3D(b,s,e) + 
                    a9  * IN_3D(k,n,w) + 
                    a10 * IN_3D(k,n,i) + 
                    a11 * IN_3D(k,n,e) + 
                    a12 * IN_3D(k,j,w) + 
                    a13 * IN_3D(k,j,i) + 
                    a14 * IN_3D(k,j,e) + 
                    a15 * IN_3D(k,s,w) + 
                    a16 * IN_3D(k,s,i) + 
                    a17 * IN_3D(k,s,e) + 
                    a18 * IN_3D(t,n,w) + 
                    a19 * IN_3D(t,n,i) + 
                    a20 * IN_3D(t,n,e) + 
                    a21 * IN_3D(t,j,w) + 
                    a22 * IN_3D(t,j,i) + 
                    a23 * IN_3D(t,j,e) + 
                    a24 * IN_3D(t,s,w) + 
                    a25 * IN_3D(t,s,i) + 
                    a26 * IN_3D(t,s,e) ;
}

__global__ void Stencil_Cuda_Sweep(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, 
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, DATA_TYPE a7, 
        DATA_TYPE a8, DATA_TYPE a9, DATA_TYPE a10, DATA_TYPE a11, DATA_TYPE a12, 
        DATA_TYPE a13, DATA_TYPE a14, DATA_TYPE a15, DATA_TYPE a16, DATA_TYPE a17, 
        DATA_TYPE a18, DATA_TYPE a19, DATA_TYPE a20, DATA_TYPE a21, DATA_TYPE a22, 
        DATA_TYPE a23, DATA_TYPE a24, DATA_TYPE a25, DATA_TYPE a26 , 
        int Z, int M, int N) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;

    const int block_z = Z / gridDim.z;
    int k = block_z * blockIdx.z ;
    const int k_end = k + block_z;

    int n = (j == 0)      ? j : j - 1;
    int s = (j == M-1)    ? j : j + 1;
    int w = (i == 0)      ? i : i - 1;
    int e = (i == N-1)    ? i : i + 1;
#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        int b = (k == 0)      ? k : k - 1;
        int t = (k == Z-1)    ? k : k + 1;

        OUT_3D(k,j,i) = a0  * IN_3D(b,n,w) +
                        a1  * IN_3D(b,n,i) +
                        a2  * IN_3D(b,n,e) +
                        a3  * IN_3D(b,j,w) +
                        a4  * IN_3D(b,j,i) +
                        a5  * IN_3D(b,j,e) +
                        a6  * IN_3D(b,s,w) + 
                        a7  * IN_3D(b,s,i) + 
                        a8  * IN_3D(b,s,e) + 
                        a9  * IN_3D(k,n,w) + 
                        a10 * IN_3D(k,n,i) + 
                        a11 * IN_3D(k,n,e) + 
                        a12 * IN_3D(k,j,w) + 
                        a13 * IN_3D(k,j,i) + 
                        a14 * IN_3D(k,j,e) + 
                        a15 * IN_3D(k,s,w) + 
                        a16 * IN_3D(k,s,i) + 
                        a17 * IN_3D(k,s,e) + 
                        a18 * IN_3D(t,n,w) + 
                        a19 * IN_3D(t,n,i) + 
                        a20 * IN_3D(t,n,e) + 
                        a21 * IN_3D(t,j,w) + 
                        a22 * IN_3D(t,j,i) + 
                        a23 * IN_3D(t,j,e) + 
                        a24 * IN_3D(t,s,w) + 
                        a25 * IN_3D(t,s,i) + 
                        a26 * IN_3D(t,s,e) ;
    }
}

__global__ void Stencil_Cuda_Sweep_Sm(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, 
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, DATA_TYPE a7, 
        DATA_TYPE a8, DATA_TYPE a9, DATA_TYPE a10, DATA_TYPE a11, DATA_TYPE a12, 
        DATA_TYPE a13, DATA_TYPE a14, DATA_TYPE a15, DATA_TYPE a16, DATA_TYPE a17, 
        DATA_TYPE a18, DATA_TYPE a19, DATA_TYPE a20, DATA_TYPE a21, DATA_TYPE a22, 
        DATA_TYPE a23, DATA_TYPE a24, DATA_TYPE a25, DATA_TYPE a26 , 
        int Z, int M, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    const int block_z = Z / gridDim.z;
    int k = block_z * blockIdx.z ;
    const int k_end = k + block_z;
    int li = threadIdx.x + 1;
    int lj = threadIdx.y + 1;

    extern __shared__ DATA_TYPE local[];

    // DATA_TYPE t1, t2, t3;
    // DATA_TYPE r1, r2, r3, r4;
    // DATA_TYPE sum = 0.0;
    int n = (j == 0)      ? j : j - 1;
    int s = (j == M-1)    ? j : j + 1;
    int w = (i == 0)      ? i : i - 1;
    int e = (i == N-1)    ? i : i + 1;
    // load current layer
    // t3 = IN_3D(k, j, i); 
    // if(threadIdx.x == 0)            r1 = IN_3D(k,j,w);
    // if(threadIdx.x == blockDim.x-1) r2 = IN_3D(k,j,e);
    // if(threadIdx.y == 0)            r3 = IN_3D(k,n,i);
    // if(threadIdx.y == blockDim.y-1) r4 = IN_3D(k,s,i);

    // load previous layer (same with k)
    int b = (k == 0)      ? k : k - 1;
    // t2 = IN_3D(b, j, i);

    int t1, t2, t3;
    t3 = 2; t2 = 1;
    LOC_L_2D(t3,lj,li) = IN_3D(k,j,i);
    LOC_L_2D(t2,lj,li) = IN_3D(b,j,i);
    if(li == 1)                                   
    {
        LOC_L_2D(t3,lj,li-1) = IN_3D(k,j,w); 
        LOC_L_2D(t2,lj,li-1) = IN_3D(b,j,w);
    }
    if(li == SM_2D_N-2)                         
    {
        LOC_L_2D(t3,lj,li+1) = IN_3D(k,j,e); 
        LOC_L_2D(t2,lj,li+1) = IN_3D(b,j,e);
    }
    if(lj == 1)                                   
    {
        LOC_L_2D(t3,lj-1,li) = IN_3D(k,n,i); 
        LOC_L_2D(t2,lj-1,li) = IN_3D(b,n,i);
    }
    if(lj == SM_2D_M-2)                         
    {
        LOC_L_2D(t3,lj+1,li) = IN_3D(k,s,i); 
        LOC_L_2D(t2,lj+1,li) = IN_3D(b,s,i);
    }
    if(li == 1 && lj == 1)                     
    {
        LOC_L_2D(t3,lj-1,li-1) = IN_3D(k,n,w); 
        LOC_L_2D(t2,lj-1,li-1) = IN_3D(b,n,w);
    }
    if(li == SM_2D_N-2 && lj == 1)           
    {
        LOC_L_2D(t3,lj-1,li+1) = IN_3D(k,n,e); 
        LOC_L_2D(t2,lj-1,li+1) = IN_3D(b,n,e);
    }
    if(li == 1 && lj == SM_2D_M-2)           
    { 
        LOC_L_2D(t3,lj+1,li-1) = IN_3D(k,s,w); 
        LOC_L_2D(t2,lj+1,li-1) = IN_3D(b,s,w);
    }
    if(li == SM_2D_N-2 && lj == SM_2D_M-2) 
    {
        LOC_L_2D(t3,lj+1,li+1) = IN_3D(k,s,e); 
        LOC_L_2D(t2,lj+1,li+1) = IN_3D(b,s,e);
    }

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        // sum = 0.0;
        t1 = t2;
        t2 = t3;
        t3 = (t3+1)%3;
        // load next layer
        int t = (k == Z-1)    ? k : k+1;
        LOC_L_2D(t3,lj,li) = IN_3D(t,j,i);
        if(li == 1)                            {LOC_L_2D(t3,lj  ,li-1) = IN_3D(t,j,w);}
        if(li == SM_2D_N-2)                    {LOC_L_2D(t3,lj  ,li+1) = IN_3D(t,j,e);}
        if(lj == 1)                            {LOC_L_2D(t3,lj-1,li  ) = IN_3D(t,n,i);}
        if(lj == SM_2D_M-2)                    {LOC_L_2D(t3,lj+1,li  ) = IN_3D(t,s,i);}
        if(li == 1 && lj == 1)                 {LOC_L_2D(t3,lj-1,li-1) = IN_3D(t,n,w);}
        if(li == SM_2D_N-2 && lj == 1)         {LOC_L_2D(t3,lj-1,li+1) = IN_3D(t,n,e);}
        if(li == 1 && lj == SM_2D_M-2)         {LOC_L_2D(t3,lj+1,li-1) = IN_3D(t,s,w);}
        if(li == SM_2D_N-2 && lj == SM_2D_M-2) {LOC_L_2D(t3,lj+1,li+1) = IN_3D(t,s,e);}
        __syncthreads();

        
        OUT_3D(k,j,i) = a0  * LOC_L_2D(t1,lj-1,li-1) +
                        a1  * LOC_L_2D(t1,lj-1,li  ) +
                        a2  * LOC_L_2D(t1,lj-1,li+1) +
                        a3  * LOC_L_2D(t1,lj  ,li-1) +
                        a4  * LOC_L_2D(t1,lj  ,li  ) +
                        a5  * LOC_L_2D(t1,lj  ,li+1) +
                        a6  * LOC_L_2D(t1,lj+1,li-1) + 
                        a7  * LOC_L_2D(t1,lj+1,li  ) + 
                        a8  * LOC_L_2D(t1,lj+1,li+1) + 
                        a9  * LOC_L_2D(t2,lj-1,li-1) + 
                        a10 * LOC_L_2D(t2,lj-1,li  ) + 
                        a11 * LOC_L_2D(t2,lj-1,li+1) + 
                        a12 * LOC_L_2D(t2,lj  ,li-1) + 
                        a13 * LOC_L_2D(t2,lj  ,li  ) + 
                        a14 * LOC_L_2D(t2,lj  ,li+1) + 
                        a15 * LOC_L_2D(t2,lj+1,li-1) + 
                        a16 * LOC_L_2D(t2,lj+1,li  ) + 
                        a17 * LOC_L_2D(t2,lj+1,li+1) + 
                        a18 * LOC_L_2D(t3,lj-1,li-1) + 
                        a19 * LOC_L_2D(t3,lj-1,li  ) + 
                        a20 * LOC_L_2D(t3,lj-1,li+1) + 
                        a21 * LOC_L_2D(t3,lj  ,li-1) + 
                        a22 * LOC_L_2D(t3,lj  ,li  ) + 
                        a23 * LOC_L_2D(t3,lj  ,li+1) + 
                        a24 * LOC_L_2D(t3,lj+1,li-1) + 
                        a25 * LOC_L_2D(t3,lj+1,li  ) + 
                        a26 * LOC_L_2D(t3,lj+1,li+1) ;
    }
}

__global__ void Stencil_Cuda_Sm(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, 
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, DATA_TYPE a7, 
        DATA_TYPE a8, DATA_TYPE a9, DATA_TYPE a10, DATA_TYPE a11, DATA_TYPE a12, 
        DATA_TYPE a13, DATA_TYPE a14, DATA_TYPE a15, DATA_TYPE a16, DATA_TYPE a17, 
        DATA_TYPE a18, DATA_TYPE a19, DATA_TYPE a20, DATA_TYPE a21, DATA_TYPE a22, 
        DATA_TYPE a23, DATA_TYPE a24, DATA_TYPE a25, DATA_TYPE a26 , 
        int Z, int M, int N) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.z + blockIdx.z * blockDim.z ;

    int li = threadIdx.x + 1;
    int lj = threadIdx.y + 1;
    int lk = threadIdx.z + 1;

    __shared__ DATA_TYPE local[SM_Z*SM_M*SM_N];

    LOC_3D(lk,lj,li) = IN_3D(k,j,i);

    int b = (k == 0)      ? k : k - 1;
    int t = (k == Z-1)    ? k : k + 1;
    int n = (j == 0)      ? j : j - 1;
    int s = (j == M-1)    ? j : j + 1;
    int w = (i == 0)      ? i : i - 1;
    int e = (i == N-1)    ? i : i + 1;
    if(threadIdx.x == 0)            LOC_3D(lk  ,lj  ,li-1) = IN_3D(k,j,w);
    if(threadIdx.x == blockDim.x-1) LOC_3D(lk  ,lj  ,li+1) = IN_3D(k,j,e);
    if(threadIdx.y == 0)            LOC_3D(lk  ,lj-1,li  ) = IN_3D(k,n,i);
    if(threadIdx.y == blockDim.y-1) LOC_3D(lk  ,lj+1,li  ) = IN_3D(k,s,i);
    if(threadIdx.z == 0)            LOC_3D(lk-1,lj  ,li  ) = IN_3D(b,j,i);
    if(threadIdx.z == blockDim.z-1) LOC_3D(lk+1,lj  ,li  ) = IN_3D(t,j,i);


    if(li == 1)      LOC_3D(lk,lj,li-1) = IN_3D(k,j,w);
    if(li == SM_N-2) LOC_3D(lk,lj,li+1) = IN_3D(k,j,e);
    if(lj == 1)      LOC_3D(lk,lj-1,li) = IN_3D(k,n,i);
    if(lj == SM_M-2) LOC_3D(lk,lj+1,li) = IN_3D(k,s,i);
    if(lk == 1)      LOC_3D(lk-1,lj,li) = IN_3D(b,j,i);
    if(lk == SM_Z-2) LOC_3D(lk+1,lj,li) = IN_3D(t,j,i);
    if(li == 1 && lj == 1) LOC_3D(lk  ,lj-1,li-1) = IN_3D(k,n,w);
    if(li == 1 && lk == 1) LOC_3D(lk-1,lj  ,li-1) = IN_3D(b,j,w);
    if(lj == 1 && lk == 1) LOC_3D(lk-1,lj-1,li  ) = IN_3D(b,n,i);
    if(li == SM_N-2 && lj == 1) LOC_3D(lk  ,lj-1,li+1) = IN_3D(k,n,e);
    if(li == SM_N-2 && lk == 1) LOC_3D(lk-1,lj  ,li+1) = IN_3D(b,j,e);
    if(lj == SM_M-2 && lk == 1) LOC_3D(lk-1,lj+1,li  ) = IN_3D(b,s,i);
    if(li == 1 && lj == SM_M-2) LOC_3D(lk  ,lj+1,li-1) = IN_3D(k,s,w);
    if(li == 1 && lk == SM_Z-2) LOC_3D(lk+1,lj  ,li-1) = IN_3D(t,j,w);
    if(lj == 1 && lk == SM_Z-2) LOC_3D(lk+1,lj-1,li  ) = IN_3D(t,n,i);
    if(li == SM_N-2 && lj == SM_M-2) LOC_3D(lk  ,lj+1,li+1) = IN_3D(k,s,e);
    if(li == SM_N-2 && lk == SM_Z-2) LOC_3D(lk+1,lj  ,li+1) = IN_3D(t,j,e);
    if(lj == SM_M-2 && lk == SM_Z-2) LOC_3D(lk+1,lj+1,li  ) = IN_3D(t,s,i);
    if(li == 1 && lj == 1 && lk == 1)      LOC_3D(lk-1,lj-1,li-1) = IN_3D(b,n,w);
    if(li == 1 && lj == 1 && lk == SM_Z-2) LOC_3D(lk+1,lj-1,li-1) = IN_3D(t,n,w);
    if(li == 1 && lj == SM_M-2 && lk == 1) LOC_3D(lk-1,lj+1,li-1) = IN_3D(b,s,w);
    if(li == 1 && lj == SM_M-2 && lk == SM_Z-2) LOC_3D(lk+1,lj+1,li-1) = IN_3D(t,s,w);
    if(li == SM_N-2 && lj == 1 && lk == 1)      LOC_3D(lk-1,lj-1,li+1) = IN_3D(b,n,e);
    if(li == SM_N-2 && lj == 1 && lk == SM_Z-2) LOC_3D(lk+1,lj-1,li+1) = IN_3D(t,n,e);
    if(li == SM_N-2 && lj == SM_M-2 && lk == 1) LOC_3D(lk-1,lj+1,li+1) = IN_3D(b,s,e);
    if(li == SM_N-2 && lj == SM_M-2 && lk == SM_Z-2) LOC_3D(lk+1,lj+1,li+1) = IN_3D(t,s,e);

    __syncthreads();

    OUT_3D(k,j,i) = a0  * LOC_3D(lk-1,lj-1,li-1) +
                    a1  * LOC_3D(lk-1,lj-1,li  ) +
                    a2  * LOC_3D(lk-1,lj-1,li+1) +
                    a3  * LOC_3D(lk-1,lj  ,li-1) +
                    a4  * LOC_3D(lk-1,lj  ,li  ) +
                    a5  * LOC_3D(lk-1,lj  ,li+1) +
                    a6  * LOC_3D(lk-1,lj+1,li-1) + 
                    a7  * LOC_3D(lk-1,lj+1,li  ) + 
                    a8  * LOC_3D(lk-1,lj+1,li+1) + 
                    a9  * LOC_3D(lk  ,lj-1,li-1) + 
                    a10 * LOC_3D(lk  ,lj-1,li  ) + 
                    a11 * LOC_3D(lk  ,lj-1,li+1) + 
                    a12 * LOC_3D(lk  ,lj  ,li-1) + 
                    a13 * LOC_3D(lk  ,lj  ,li  ) + 
                    a14 * LOC_3D(lk  ,lj  ,li+1) + 
                    a15 * LOC_3D(lk  ,lj+1,li-1) + 
                    a16 * LOC_3D(lk  ,lj+1,li  ) + 
                    a17 * LOC_3D(lk  ,lj+1,li+1) + 
                    a18 * LOC_3D(lk+1,lj-1,li-1) + 
                    a19 * LOC_3D(lk+1,lj-1,li  ) + 
                    a20 * LOC_3D(lk+1,lj-1,li+1) + 
                    a21 * LOC_3D(lk+1,lj  ,li-1) + 
                    a22 * LOC_3D(lk+1,lj  ,li  ) + 
                    a23 * LOC_3D(lk+1,lj  ,li+1) + 
                    a24 * LOC_3D(lk+1,lj+1,li-1) + 
                    a25 * LOC_3D(lk+1,lj+1,li  ) + 
                    a26 * LOC_3D(lk+1,lj+1,li+1) ;
}

__global__ void Stencil_Cuda_SmX(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, 
        int Z, int M, int N) 
{
    __shared__ DATA_TYPE local[SM_Z*SM_M*SM_N];
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.z + blockIdx.z * blockDim.z ;

    int li = threadIdx.x + 1;
    int lj = threadIdx.y + 1;
    int lk = threadIdx.z + 1;

    int lane_id = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;

    int blk_id_x = blockIdx.x; //(threadIdx.x + blockIdx.x * blockDim.x)>>3; 
    int blk_id_y = blockIdx.y; //(threadIdx.y + blockIdx.y * blockDim.y)>>2;
    int blk_id_z = blockIdx.z; //(threadIdx.z + blockIdx.z * blockDim.z)>>3; 

    int new_i = (blk_id_x<<3) + lane_id%10 - 1;     
    int new_j = (blk_id_y<<2) + (lane_id/10)%6 - 1; 
    int new_k = (blk_id_z<<3) + lane_id/60 - 1;     
    int new_li = lane_id%10;
    int new_lj = (lane_id/10)%6;
    int new_lk = lane_id/60;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    LOC_3D(new_lk,new_lj,new_li) = IN_3D(new_k,new_j,new_i);
    new_i = (blk_id_x<<3) + (lane_id+256)%10 -1;
    new_j = (blk_id_y<<2) + ((lane_id+256)/10)%6-1;
    new_k = (blk_id_z<<3) + (lane_id+256)/60-1;
    new_li = (lane_id+256)%10;
    new_lj = ((lane_id+256)/10)%6;
    new_lk = (lane_id+256)/60; 
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    LOC_3D(new_lk,new_lj,new_li) = IN_3D(new_k,new_j,new_i);
    new_i = (blk_id_x<<3) + (lane_id+512)%10 -1;
    new_j = (blk_id_y<<2) + ((lane_id+512)/10)%6-1;
    new_k = (blk_id_z<<3) + (lane_id+512)/60-1;
    new_li = (lane_id+512)%10;
    new_lj = ((lane_id+512)/10)%6;
    new_lk = (lane_id+512)/60; 
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    if(new_li < 10 &&  new_lj < 6 && new_lk < 10 )
        LOC_3D(new_lk,new_lj,new_li) = IN_3D(new_k,new_j,new_i);

    __syncthreads();

    OUT_3D(k,j,i) = a0 * LOC_3D(lk-1,lj  ,li  ) +
                    a1 * LOC_3D(lk  ,lj-1,li  ) +
                    a2 * LOC_3D(lk  ,lj  ,li-1) +
                    a3 * LOC_3D(lk  ,lj  ,li  ) +
                    a4 * LOC_3D(lk  ,lj  ,li+1) +
                    a5 * LOC_3D(lk  ,lj+1,li  ) +
                    a6 * LOC_3D(lk+1,lj  ,li  ) ;
}

__global__ void Stencil_Cuda_Shfl(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, 
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, DATA_TYPE a7, 
        DATA_TYPE a8, DATA_TYPE a9, DATA_TYPE a10, DATA_TYPE a11, DATA_TYPE a12, 
        DATA_TYPE a13, DATA_TYPE a14, DATA_TYPE a15, DATA_TYPE a16, DATA_TYPE a17, 
        DATA_TYPE a18, DATA_TYPE a19, DATA_TYPE a20, DATA_TYPE a21, DATA_TYPE a22, 
        DATA_TYPE a23, DATA_TYPE a24, DATA_TYPE a25, DATA_TYPE a26 , 
        int Z, int M, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = threadIdx.z + blockIdx.z * blockDim.z ;
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int lane_id = tid % warpSize;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    int warp_id_z = (threadIdx.z + blockIdx.z * blockDim.z)>>0; // there numbers
    int new_i = (warp_id_x<<3) + lane_id%10 - 1;     // 10 is extended dimension of i
    int new_j = (warp_id_y<<2) + (lane_id/10)%6 - 1; // 6  is extended dimension of j 
    int new_k = (warp_id_z<<0) + lane_id/60 - 1;     // 60 is extended area of ixj = 10x6
    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5;
    
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10 -1;
    new_j = (warp_id_y<<2) + ((lane_id+32)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+32)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+64)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+64)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+96)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+96)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+128)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+128)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput4 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+160)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+160)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput5 = IN_3D(new_k, new_j, new_i);

    DATA_TYPE sum = 0.0;
    int friend_id;
    DATA_TYPE tx, ty, tz;

    friend_id = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a0*((lane_id < 26)? tx: ty);

    friend_id = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a1*((lane_id < 25)? tx: ty);

    friend_id = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a2*((lane_id < 24)? tx: ty);

    friend_id = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a3*((lane_id < 18)? tx: ty);

    friend_id = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a4*((lane_id < 17)? tx: ty);

    friend_id = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a5*((lane_id < 16)? tx: ty);

    friend_id = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a6*((lane_id < 10)? tx: ty);

    friend_id = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a7*((lane_id < 9 )? tx: ty);

    friend_id = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a8*((lane_id < 8 )? tx: ty);

    friend_id = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput1, friend_id);
    ty = __shfl(threadInput2, friend_id);
    tz = __shfl(threadInput3, friend_id);
    sum += a9*((lane_id < 4 )? tx: ((lane_id < 30)? ty: tz));

    friend_id = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput1, friend_id);
    ty = __shfl(threadInput2, friend_id);
    tz = __shfl(threadInput3, friend_id);
    sum += a10*((lane_id < 3 )? tx: ((lane_id < 29)? ty: tz));

    friend_id = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput1, friend_id);
    ty = __shfl(threadInput2, friend_id);
    tz = __shfl(threadInput3, friend_id);
    sum += a11*((lane_id < 2 )? tx: ((lane_id < 28)? ty: tz));

    friend_id = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput2, friend_id);
    ty = __shfl(threadInput3, friend_id);
    sum += a12*((lane_id < 22)? tx: ty);

    friend_id = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput2, friend_id);
    ty = __shfl(threadInput3, friend_id);
    sum += a13*((lane_id < 21)? tx: ty);

    friend_id = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput2, friend_id);
    ty = __shfl(threadInput3, friend_id);
    sum += a14*((lane_id < 20)? tx: ty);

    friend_id = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput2, friend_id);
    ty = __shfl(threadInput3, friend_id);
    sum += a15*((lane_id < 14)? tx: ty);

    friend_id = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput2, friend_id);
    ty = __shfl(threadInput3, friend_id);
    sum += a16*((lane_id < 13)? tx: ty);

    friend_id = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput2, friend_id);
    ty = __shfl(threadInput3, friend_id);
    sum += a17*((lane_id < 12)? tx: ty);

    friend_id = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput3, friend_id);
    ty = __shfl(threadInput4, friend_id);
    sum += a18*((lane_id < 8 )? tx: ty);

    friend_id = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput3, friend_id);
    ty = __shfl(threadInput4, friend_id);
    sum += a19*((lane_id < 7 )? tx: ty);

    friend_id = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput3, friend_id);
    ty = __shfl(threadInput4, friend_id);
    sum += a20*((lane_id < 6 )? tx: ty);

    friend_id = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput4, friend_id);
    ty = __shfl(threadInput5, friend_id);
    sum += a21*((lane_id < 24)? tx: ty);

    friend_id = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput4, friend_id);
    ty = __shfl(threadInput5, friend_id);
    sum += a22*((lane_id < 24)? tx: ty);

    friend_id = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput4, friend_id);
    ty = __shfl(threadInput5, friend_id);
    sum += a23*((lane_id < 24)? tx: ty);

    friend_id = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput4, friend_id);
    ty = __shfl(threadInput5, friend_id);
    sum += a24*((lane_id < 16)? tx: ty);

    friend_id = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput4, friend_id);
    ty = __shfl(threadInput5, friend_id);
    sum += a25*((lane_id < 16)? tx: ty);

    friend_id = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput4, friend_id);
    ty = __shfl(threadInput5, friend_id);
    sum += a26*((lane_id < 16)? tx: ty);
    
    OUT_3D(k,j,i) = sum;

}

__global__ void Stencil_Cuda_Shfl2(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, 
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, DATA_TYPE a7, 
        DATA_TYPE a8, DATA_TYPE a9, DATA_TYPE a10, DATA_TYPE a11, DATA_TYPE a12, 
        DATA_TYPE a13, DATA_TYPE a14, DATA_TYPE a15, DATA_TYPE a16, DATA_TYPE a17, 
        DATA_TYPE a18, DATA_TYPE a19, DATA_TYPE a20, DATA_TYPE a21, DATA_TYPE a22, 
        DATA_TYPE a23, DATA_TYPE a24, DATA_TYPE a25, DATA_TYPE a26 , 
        int Z, int M, int N)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<1) + (lane_id>>5) ; 
    // thread coarsening: related to warp dimensions 1x4x8. 
    // We coarsen from dimension z from 2^0 to 2^1, also need to know there are how many values in dimension z

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    int warp_id_z = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<1) + (lane_id>>5); // these numbers
    int new_i = (warp_id_x<<3) + lane_id%10-1;     // 10 is extended dimension of i
    int new_j = (warp_id_y<<2) + (lane_id/10)%6-1; // 6  is extended dimension of j 
    int new_k = (warp_id_z<<0) + lane_id/60-1;     // 60 is extended area of ixj = 10x6
    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5,
              threadInput6, threadInput7;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+32)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+32)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+64)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+64)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+96)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+96)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+128)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+128)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput4 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+160)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+160)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput5 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+192)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+192)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+192)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput6 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+224)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+224)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+224)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput7 = IN_3D(new_k, new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    int friend_id0, friend_id1;
    DATA_TYPE tx0, ty0, tz0, tx1, ty1, tz1;
    friend_id0 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    sum0 += a0*((lane_id < 26)? tx0: ty0);
    sum1 += a0*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));

    friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    sum0 += a1*((lane_id < 25)? tx0: ty0);
    sum1 += a1*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));

    friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    sum0 += a2*((lane_id < 24)? tx0: ty0);
    sum1 += a2*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));

    friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    sum0 += a3*((lane_id < 18)? tx0: ty0);
    sum1 += a3*((lane_id < 22)? tx1: ty1);

    friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    sum0 += a4*((lane_id < 17)? tx0: ty0);
    sum1 += a4*((lane_id < 21)? tx1: ty1);

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    sum0 += a5*((lane_id < 16)? tx0: ty0);
    sum1 += a5*((lane_id < 20)? tx1: ty1);

    friend_id0 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    sum0 += a6*((lane_id < 10)? tx0: ty0);
    sum1 += a6*((lane_id < 14)? tx1: ty1);

    friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    sum0 += a7*((lane_id < 9 )? tx0: ty0);
    sum1 += a7*((lane_id < 13)? tx1: ty1);

    friend_id0 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    sum0 += a8*((lane_id < 8 )? tx0: ty0);
    sum1 += a8*((lane_id < 12)? tx1: ty1);

    friend_id0 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    sum0 += a9*((lane_id < 4 )? tx0: ((lane_id < 30)? ty0: tz0));
    sum1 += a9*((lane_id < 8)? tx1: ty1);

    friend_id0 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    sum0 += a10*((lane_id < 3 )? tx0: ((lane_id < 29)? ty0: tz0));
    sum1 += a10*((lane_id < 7)? tx1: ty1);

    friend_id0 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    sum0 += a11*((lane_id < 2 )? tx0: ((lane_id < 28)? ty0: tz0));
    sum1 += a11*((lane_id < 6)? tx1: ty1);

    friend_id0 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    sum0 += a12*((lane_id < 22)? tx0: ty0);
    sum1 += a12*((lane_id < 24)? tx1: ty1);

    friend_id0 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    sum0 += a13*((lane_id < 21)? tx0: ty0);
    sum1 += a13*((lane_id < 24)? tx1: ty1);

    friend_id0 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    sum0 += a14*((lane_id < 20)? tx0: ty0);
    sum1 += a14*((lane_id < 24)? tx1: ty1);

    friend_id0 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    sum0 += a15*((lane_id < 14)? tx0: ty0);
    sum1 += a15*((lane_id < 16)? tx1: ty1);

    friend_id0 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    sum0 += a16*((lane_id < 13)? tx0: ty0);
    sum1 += a16*((lane_id < 16)? tx1: ty1);

    friend_id0 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    sum0 += a17*((lane_id < 12)? tx0: ty0);
    sum1 += a17*((lane_id < 16)? tx1: ty1);

    friend_id0 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    sum0 += a18*((lane_id < 8 )? tx0: ty0);
    sum1 += a18*((lane_id < 10)? tx1: ty1);

    friend_id0 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    sum0 += a19*((lane_id < 7 )? tx0: ty0);
    sum1 += a19*((lane_id < 9 )? tx1: ty1);

    friend_id0 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    sum0 += a20*((lane_id < 6 )? tx0: ty0);
    sum1 += a20*((lane_id < 8 )? tx1: ty1);

    friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tz1 = __shfl(threadInput7, friend_id1);
    sum0 += a21*((lane_id < 24)? tx0: ty0);
    sum1 += a21*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));

    friend_id0 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tz1 = __shfl(threadInput7, friend_id1);
    sum0 += a22*((lane_id < 24)? tx0: ty0);
    sum1 += a22*((lane_id < 1 )? tx1: ((lane_id < 27)? ty1: tz1));

    friend_id0 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    sum0 += a23*((lane_id < 24)? tx0: ty0);
    sum1 += a23*((lane_id < 26)? tx1: ty1);

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    sum0 += a24*((lane_id < 16)? tx0: ty0);
    sum1 += a24*((lane_id < 20)? tx1: ty1);

    friend_id0 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    sum0 += a25*((lane_id < 16)? tx0: ty0);
    sum1 += a25*((lane_id < 19)? tx1: ty1);

    friend_id0 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    sum0 += a26*((lane_id < 16)? tx0: ty0);
    sum1 += a26*((lane_id < 18)? tx1: ty1);

    OUT_3D(k,j,i) = sum0;
    OUT_3D(k+1,j,i) = sum1;

}

__global__ void Stencil_Cuda_Shfl4(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, 
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, DATA_TYPE a7, 
        DATA_TYPE a8, DATA_TYPE a9, DATA_TYPE a10, DATA_TYPE a11, DATA_TYPE a12, 
        DATA_TYPE a13, DATA_TYPE a14, DATA_TYPE a15, DATA_TYPE a16, DATA_TYPE a17, 
        DATA_TYPE a18, DATA_TYPE a19, DATA_TYPE a20, DATA_TYPE a21, DATA_TYPE a22, 
        DATA_TYPE a23, DATA_TYPE a24, DATA_TYPE a25, DATA_TYPE a26 ,
        int Z, int M, int N)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<2) + (lane_id>>5) ; 
    // Thread coarsening: related to warp dimensions 1x4x8. 
    // We coarsen from dimension z from 2^0 to 2^2, also need to know there are how many values in dimension z,
    // which is (lane_id>>5) 

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    int warp_id_z = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<2) + (lane_id>>5); // these numbers
    int new_i = (warp_id_x<<3) + lane_id%10-1;     // 10 is extended dimension of i
    int new_j = (warp_id_y<<2) + (lane_id/10)%6-1; // 6  is extended dimension of j 
    int new_k = (warp_id_z<<0) + lane_id/60-1;     // 60 is extended area of ixj = 10x6
    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5,
              threadInput6, threadInput7, threadInput8, threadInput9, threadInput10, threadInput11;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+32)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+32)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+64)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+64)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+96)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+96)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+128)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+128)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput4 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+160)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+160)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput5 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+192)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+192)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+192)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput6 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+224)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+224)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+224)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput7 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+256)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+256)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+256)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput8 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+288)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+288)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+288)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput9 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+320)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+320)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+320)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput10 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+352)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+352)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+352)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput11 = IN_3D(new_k, new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    int friend_id0, friend_id1, friend_id2, friend_id3;
    DATA_TYPE tx0, ty0, tz0, tx1, ty1, tz1, tx2, ty2, tz2, tx3, ty3, tz3;

    friend_id0 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    sum0 += a0*((lane_id < 26)? tx0: ty0);
    sum1 += a0*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
    sum2 += a0*((lane_id < 8 )? tx2: ty2);
    sum3 += a0*((lane_id < 10)? tx3: ty3);

    friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    sum0 += a1*((lane_id < 25)? tx0: ty0);
    sum1 += a1*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
    sum2 += a1*((lane_id < 7 )? tx2: ty2);
    sum3 += a1*((lane_id < 9 )? tx3: ty3);

    friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    sum0 += a2*((lane_id < 24)? tx0: ty0);
    sum1 += a2*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
    sum2 += a2*((lane_id < 6 )? tx2: ty2);
    sum3 += a2*((lane_id < 8 )? tx3: ty3);

    friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    tz3 = __shfl(threadInput7, friend_id3);
    sum0 += a3*((lane_id < 18)? tx0: ty0);
    sum1 += a3*((lane_id < 22)? tx1: ty1);
    sum2 += a3*((lane_id < 24)? tx2: ty2);
    sum3 += a3*((lane_id < 2 )? tx3: ((lane_id < 28)? ty3: tz3));

    friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    tz3 = __shfl(threadInput7, friend_id3);
    sum0 += a4*((lane_id < 17)? tx0: ty0);
    sum1 += a4*((lane_id < 21)? tx1: ty1);
    sum2 += a4*((lane_id < 24)? tx2: ty2);
    sum3 += a4*((lane_id < 1 )? tx3: ((lane_id < 27)? ty3: tz3));

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    sum0 += a5*((lane_id < 16)? tx0: ty0);
    sum1 += a5*((lane_id < 20)? tx1: ty1);
    sum2 += a5*((lane_id < 24)? tx2: ty2);
    sum3 += a5*((lane_id < 26)? tx3: ty3);

    friend_id0 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    sum0 += a6*((lane_id < 10)? tx0: ty0);
    sum1 += a6*((lane_id < 14)? tx1: ty1);
    sum2 += a6*((lane_id < 16)? tx2: ty2);
    sum3 += a6*((lane_id < 20)? tx3: ty3);

    friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    sum0 += a7*((lane_id < 9 )? tx0: ty0);
    sum1 += a7*((lane_id < 13)? tx1: ty1);
    sum2 += a7*((lane_id < 16)? tx2: ty2);
    sum3 += a7*((lane_id < 19)? tx3: ty3);

    friend_id0 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    sum0 += a8*((lane_id < 8 )? tx0: ty0);
    sum1 += a8*((lane_id < 12)? tx1: ty1);
    sum2 += a8*((lane_id < 16)? tx2: ty2);
    sum3 += a8*((lane_id < 18)? tx3: ty3);

    friend_id0 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    sum0 += a9*((lane_id < 4 )? tx0: ((lane_id < 30)? ty0: tz0));
    sum1 += a9*((lane_id < 8)? tx1: ty1);
    sum2 += a9*((lane_id < 10)? tx2: ty2);
    sum3 += a9*((lane_id < 14)? tx3: ty3);

    friend_id0 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    sum0 += a10*((lane_id < 3 )? tx0: ((lane_id < 29)? ty0: tz0));
    sum1 += a10*((lane_id < 7 )? tx1: ty1);
    sum2 += a10*((lane_id < 9 )? tx2: ty2);
    sum3 += a10*((lane_id < 13)? tx3: ty3);

    friend_id0 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    sum0 += a11*((lane_id < 2 )? tx0: ((lane_id < 28)? ty0: tz0));
    sum1 += a11*((lane_id < 6 )? tx1: ty1);
    sum2 += a11*((lane_id < 8 )? tx2: ty2);
    sum3 += a11*((lane_id < 12)? tx3: ty3);

    friend_id0 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tz2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    sum0 += a12*((lane_id < 22)? tx0: ty0);
    sum1 += a12*((lane_id < 24)? tx1: ty1);
    sum2 += a12*((lane_id < 2 )? tx2: ((lane_id < 28)? ty2: tz2));
    sum3 += a12*((lane_id < 6 )? tx3: ty3);

    friend_id0 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tz2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    tz3 = __shfl(threadInput9, friend_id3);
    sum0 += a13*((lane_id < 21)? tx0: ty0);
    sum1 += a13*((lane_id < 24)? tx1: ty1);
    sum2 += a13*((lane_id < 1 )? tx2: ((lane_id < 27)? ty2: tz2));
    sum3 += a13*((lane_id < 5 )? tx3: ((lane_id < 31)? ty3: tz3));

    friend_id0 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    tz3 = __shfl(threadInput9, friend_id3);
    sum0 += a14*((lane_id < 20)? tx0: ty0);
    sum1 += a14*((lane_id < 24)? tx1: ty1);
    sum2 += a14*((lane_id < 26)? tx2: ty2);
    sum3 += a14*((lane_id < 4 )? tx3: ((lane_id < 30)? ty3: tz3));

    friend_id0 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput8, friend_id3);
    ty3 = __shfl(threadInput9, friend_id3);
    sum0 += a15*((lane_id < 14)? tx0: ty0);
    sum1 += a15*((lane_id < 16)? tx1: ty1);
    sum2 += a15*((lane_id < 20)? tx2: ty2);
    sum3 += a15*((lane_id < 24)? tx3: ty3);

    friend_id0 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput8, friend_id3);
    ty3 = __shfl(threadInput9, friend_id3);
    sum0 += a16*((lane_id < 13)? tx0: ty0);
    sum1 += a16*((lane_id < 16)? tx1: ty1);
    sum2 += a16*((lane_id < 19)? tx2: ty2);
    sum3 += a16*((lane_id < 23)? tx3: ty3);

    friend_id0 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput8, friend_id3);
    ty3 = __shfl(threadInput9, friend_id3);
    sum0 += a17*((lane_id < 12)? tx0: ty0);
    sum1 += a17*((lane_id < 16)? tx1: ty1);
    sum2 += a17*((lane_id < 18)? tx2: ty2);
    sum3 += a17*((lane_id < 22)? tx3: ty3);

    friend_id0 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    sum0 += a18*((lane_id < 8 )? tx0: ty0);
    sum1 += a18*((lane_id < 10)? tx1: ty1);
    sum2 += a18*((lane_id < 14)? tx2: ty2);
    sum3 += a18*((lane_id < 16)? tx3: ty3);

    friend_id0 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    sum0 += a19*((lane_id < 7 )? tx0: ty0);
    sum1 += a19*((lane_id < 9 )? tx1: ty1);
    sum2 += a19*((lane_id < 13)? tx2: ty2);
    sum3 += a19*((lane_id < 16)? tx3: ty3);

    friend_id0 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    sum0 += a20*((lane_id < 6 )? tx0: ty0);
    sum1 += a20*((lane_id < 8 )? tx1: ty1);
    sum2 += a20*((lane_id < 12)? tx2: ty2);
    sum3 += a20*((lane_id < 16)? tx3: ty3);

    friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tz1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    sum0 += a21*((lane_id < 24)? tx0: ty0);
    sum1 += a21*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
    sum2 += a21*((lane_id < 6 )? tx2: ty2);
    sum3 += a21*((lane_id < 8 )? tx3: ty3);

    friend_id0 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+23+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tz1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tz2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    sum0 += a22*((lane_id < 24)? tx0: ty0);
    sum1 += a22*((lane_id < 1 )? tx1: ((lane_id < 27)? ty1: tz1));
    sum2 += a22*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
    sum3 += a22*((lane_id < 8 )? tx3: ty3);

    friend_id0 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tz2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    sum0 += a23*((lane_id < 24)? tx0: ty0);
    sum1 += a23*((lane_id < 26)? tx1: ty1);
    sum2 += a23*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
    sum3 += a23*((lane_id < 8 )? tx3: ty3);

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput8, friend_id2);
    ty2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput10, friend_id3);
    ty3 = __shfl(threadInput11, friend_id3);
    sum0 += a24*((lane_id < 16)? tx0: ty0);
    sum1 += a24*((lane_id < 20)? tx1: ty1);
    sum2 += a24*((lane_id < 24)? tx2: ty2);
    sum3 += a24*((lane_id < 26)? tx3: ty3);

    friend_id0 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput8, friend_id2);
    ty2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput10, friend_id3);
    ty3 = __shfl(threadInput11, friend_id3);
    sum0 += a25*((lane_id < 16)? tx0: ty0);
    sum1 += a25*((lane_id < 19)? tx1: ty1);
    sum2 += a25*((lane_id < 23)? tx2: ty2);
    sum3 += a25*((lane_id < 25)? tx3: ty3);

    friend_id0 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput8, friend_id2);
    ty2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput10, friend_id3);
    ty3 = __shfl(threadInput11, friend_id3);
    sum0 += a26*((lane_id < 16)? tx0: ty0);
    sum1 += a26*((lane_id < 18)? tx1: ty1);
    sum2 += a26*((lane_id < 22)? tx2: ty2);
    sum3 += a26*((lane_id < 24)? tx3: ty3);


    OUT_3D(k,j,i) = sum0;
    OUT_3D(k+1,j,i) = sum1;
    OUT_3D(k+2,j,i) = sum2;
    OUT_3D(k+3,j,i) = sum3;
}

__global__ void Stencil_Cuda_Shfl8(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, 
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, DATA_TYPE a7, 
        DATA_TYPE a8, DATA_TYPE a9, DATA_TYPE a10, DATA_TYPE a11, DATA_TYPE a12, 
        DATA_TYPE a13, DATA_TYPE a14, DATA_TYPE a15, DATA_TYPE a16, DATA_TYPE a17, 
        DATA_TYPE a18, DATA_TYPE a19, DATA_TYPE a20, DATA_TYPE a21, DATA_TYPE a22, 
        DATA_TYPE a23, DATA_TYPE a24, DATA_TYPE a25, DATA_TYPE a26 , 
        int Z, int M, int N)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int k = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<3) + (lane_id>>5) ; 
    // Thread coarsening: related to warp dimensions 1x4x8. 
    // We coarsen from dimension z from 2^0 to 2^3, also need to know there are how many values in dimension z,
    // which is (lane_id>>5) 

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    int warp_id_z = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<3) + (lane_id>>5); // these numbers
    int new_i = (warp_id_x<<3) + lane_id%10-1;     // 10 is extended dimension of i
    int new_j = (warp_id_y<<2) + (lane_id/10)%6-1; // 6  is extended dimension of j 
    int new_k = (warp_id_z<<0) + lane_id/60-1;     // 60 is extended area of ixj = 10x6
    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5,
              threadInput6, threadInput7, threadInput8, threadInput9, threadInput10, threadInput11,
              threadInput12, threadInput13, threadInput14, threadInput15, threadInput16, threadInput17,
              threadInput18;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+32)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+32)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+64)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+64)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+96)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+96)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+128)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+128)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput4 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+160)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+160)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput5 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+192)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+192)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+192)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput6 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+224)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+224)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+224)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput7 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+256)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+256)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+256)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput8 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+288)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+288)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+288)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput9 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+320)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+320)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+320)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput10 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+352)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+352)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+352)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput11 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+384)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+384)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+384)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput12 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+416)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+416)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+416)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput13 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+448)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+448)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+448)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput14 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+480)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+480)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+480)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput15 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+512)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+512)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+512)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput16 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+544)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+544)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+544)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput17 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+576)%10-1;
    new_j = (warp_id_y<<2) + ((lane_id+576)/10)%6-1;
    new_k = (warp_id_z<<0) + (lane_id+576)/60-1;
    new_k = (new_k == -1)   ? 0   : new_k;
    new_k = (new_k >= Z)    ? Z-1 : new_k;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput18 = IN_3D(new_k, new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    DATA_TYPE sum4 = 0.0;
    DATA_TYPE sum5 = 0.0;
    DATA_TYPE sum6 = 0.0;
    DATA_TYPE sum7 = 0.0;
    int friend_id0, friend_id1, friend_id2, friend_id3;
    int friend_id4, friend_id5, friend_id6, friend_id7;
    DATA_TYPE tx0, ty0, tz0, tx1, ty1, tz1, tx2, ty2, tz2, tx3, ty3, tz3;
    DATA_TYPE rx0, ry0, rz0, rx1, ry1, rz1, rx2, ry2, rz2, rx3, ry3, rz3;
    friend_id0 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    rx0 = __shfl(threadInput7, friend_id4);
    ry0 = __shfl(threadInput8, friend_id4);
    rx1 = __shfl(threadInput9 , friend_id5);
    ry1 = __shfl(threadInput10, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += a0*((lane_id < 26)? tx0: ty0);
    sum1 += a0*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
    sum2 += a0*((lane_id < 8 )? tx2: ty2);
    sum3 += a0*((lane_id < 10)? tx3: ty3);
    sum4 += a0*((lane_id < 14)? rx0: ry0);
    sum5 += a0*((lane_id < 16)? rx1: ry1);
    sum6 += a0*((lane_id < 20)? rx2: ry2);
    sum7 += a0*((lane_id < 24)? rx3: ry3);

    friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    rx0 = __shfl(threadInput7, friend_id4);
    ry0 = __shfl(threadInput8, friend_id4);
    rx1 = __shfl(threadInput9 , friend_id5);
    ry1 = __shfl(threadInput10, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += a1*((lane_id < 25)? tx0: ty0);
    sum1 += a1*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
    sum2 += a1*((lane_id < 7 )? tx2: ty2);
    sum3 += a1*((lane_id < 9 )? tx3: ty3);
    sum4 += a1*((lane_id < 13)? rx0: ry0);
    sum5 += a1*((lane_id < 16)? rx1: ry1);
    sum6 += a1*((lane_id < 19)? rx2: ry2);
    sum7 += a1*((lane_id < 23)? rx3: ry3);

    friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    rx0 = __shfl(threadInput7, friend_id4);
    ry0 = __shfl(threadInput8, friend_id4);
    rx1 = __shfl(threadInput9 , friend_id5);
    ry1 = __shfl(threadInput10, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += a2*((lane_id < 24)? tx0: ty0);
    sum1 += a2*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
    sum2 += a2*((lane_id < 6 )? tx2: ty2);
    sum3 += a2*((lane_id < 8 )? tx3: ty3);
    sum4 += a2*((lane_id < 12)? rx0: ry0);
    sum5 += a2*((lane_id < 16)? rx1: ry1);
    sum6 += a2*((lane_id < 18)? rx2: ry2);
    sum7 += a2*((lane_id < 22)? rx3: ry3);

    friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    tz3 = __shfl(threadInput7, friend_id3);
    rx0 = __shfl(threadInput7, friend_id4);
    ry0 = __shfl(threadInput8, friend_id4);
    rx1 = __shfl(threadInput9 , friend_id5);
    ry1 = __shfl(threadInput10, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += a3*((lane_id < 18)? tx0: ty0);
    sum1 += a3*((lane_id < 22)? tx1: ty1);
    sum2 += a3*((lane_id < 24)? tx2: ty2);
    sum3 += a3*((lane_id < 2 )? tx3: ((lane_id < 28)? ty3: tz3));
    sum4 += a3*((lane_id < 6 )? rx0: ry0);
    sum5 += a3*((lane_id < 8 )? rx1: ry1);
    sum6 += a3*((lane_id < 12)? rx2: ry2);
    sum7 += a3*((lane_id < 16)? rx3: ry3);

    friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+23+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+15+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    tz3 = __shfl(threadInput7, friend_id3);
    rx0 = __shfl(threadInput7, friend_id4);
    ry0 = __shfl(threadInput8, friend_id4);
    rz0 = __shfl(threadInput9, friend_id4);
    rx1 = __shfl(threadInput9 , friend_id5);
    ry1 = __shfl(threadInput10, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += a4*((lane_id < 17)? tx0: ty0);
    sum1 += a4*((lane_id < 21)? tx1: ty1);
    sum2 += a4*((lane_id < 24)? tx2: ty2);
    sum3 += a4*((lane_id < 1 )? tx3: ((lane_id < 27)? ty3: tz3));
    sum4 += a4*((lane_id < 5 )? rx0: ((lane_id < 31)? ry0: rz0));
    sum5 += a4*((lane_id < 8 )? rx1: ry1);
    sum6 += a4*((lane_id < 11)? rx2: ry2);
    sum7 += a4*((lane_id < 15)? rx3: ry3);

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    rx0 = __shfl(threadInput7, friend_id4);
    ry0 = __shfl(threadInput8, friend_id4);
    rz0 = __shfl(threadInput9, friend_id4);
    rx1 = __shfl(threadInput9 , friend_id5);
    ry1 = __shfl(threadInput10, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += a5*((lane_id < 16)? tx0: ty0);
    sum1 += a5*((lane_id < 20)? tx1: ty1);
    sum2 += a5*((lane_id < 24)? tx2: ty2);
    sum3 += a5*((lane_id < 26)? tx3: ty3);
    sum4 += a5*((lane_id < 4 )? rx0: ((lane_id < 30)? ry0: rz0));
    sum5 += a5*((lane_id < 8 )? rx1: ry1);
    sum6 += a5*((lane_id < 10)? rx2: ry2);
    sum7 += a5*((lane_id < 14)? rx3: ry3);

    friend_id0 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    rx0 = __shfl(threadInput8, friend_id4);
    ry0 = __shfl(threadInput9, friend_id4);
    rx1 = __shfl(threadInput10, friend_id5);
    ry1 = __shfl(threadInput11, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rz2 = __shfl(threadInput13, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += a6*((lane_id < 10)? tx0: ty0);
    sum1 += a6*((lane_id < 14)? tx1: ty1);
    sum2 += a6*((lane_id < 16)? tx2: ty2);
    sum3 += a6*((lane_id < 20)? tx3: ty3);
    sum4 += a6*((lane_id < 24)? rx0: ry0);
    sum5 += a6*((lane_id < 26)? rx1: ry1);
    sum6 += a6*((lane_id < 4 )? rx2: ((lane_id < 30)? ry2: rz2));
    sum7 += a6*((lane_id < 8 )? rx3: ry3);

    friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    rx0 = __shfl(threadInput8, friend_id4);
    ry0 = __shfl(threadInput9, friend_id4);
    rx1 = __shfl(threadInput10, friend_id5);
    ry1 = __shfl(threadInput11, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rz2 = __shfl(threadInput13, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += a7*((lane_id < 9 )? tx0: ty0);
    sum1 += a7*((lane_id < 13)? tx1: ty1);
    sum2 += a7*((lane_id < 16)? tx2: ty2);
    sum3 += a7*((lane_id < 19)? tx3: ty3);
    sum4 += a7*((lane_id < 23)? rx0: ry0);
    sum5 += a7*((lane_id < 25)? rx1: ry1);
    sum6 += a7*((lane_id < 3 )? rx2: ((lane_id < 29)? ry2: rz2));
    sum7 += a7*((lane_id < 7 )? rx3: ry3);

    friend_id0 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    rx0 = __shfl(threadInput8, friend_id4);
    ry0 = __shfl(threadInput9, friend_id4);
    rx1 = __shfl(threadInput10, friend_id5);
    ry1 = __shfl(threadInput11, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rz2 = __shfl(threadInput13, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += a8*((lane_id < 8 )? tx0: ty0);
    sum1 += a8*((lane_id < 12)? tx1: ty1);
    sum2 += a8*((lane_id < 16)? tx2: ty2);
    sum3 += a8*((lane_id < 18)? tx3: ty3);
    sum4 += a8*((lane_id < 22)? rx0: ry0);
    sum5 += a8*((lane_id < 24)? rx1: ry1);
    sum6 += a8*((lane_id < 2 )? rx2: ((lane_id < 28)? ry2: rz2));
    sum7 += a8*((lane_id < 6 )? rx3: ry3);

    friend_id0 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    rx0 = __shfl(threadInput9 , friend_id4);
    ry0 = __shfl(threadInput10, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += a9*((lane_id < 4 )? tx0: ((lane_id < 30)? ty0: tz0));
    sum1 += a9*((lane_id < 8)? tx1: ty1);
    sum2 += a9*((lane_id < 10)? tx2: ty2);
    sum3 += a9*((lane_id < 14)? tx3: ty3);
    sum4 += a9*((lane_id < 16)? rx0: ry0);
    sum5 += a9*((lane_id < 20)? rx1: ry1);
    sum6 += a9*((lane_id < 24)? rx2: ry2);
    sum7 += a9*((lane_id < 26)? rx3: ry3);

    friend_id0 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    rx0 = __shfl(threadInput9 , friend_id4);
    ry0 = __shfl(threadInput10, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += a10*((lane_id < 3 )? tx0: ((lane_id < 29)? ty0: tz0));
    sum1 += a10*((lane_id < 7 )? tx1: ty1);
    sum2 += a10*((lane_id < 9 )? tx2: ty2);
    sum3 += a10*((lane_id < 13)? tx3: ty3);
    sum4 += a10*((lane_id < 16)? rx0: ry0);
    sum5 += a10*((lane_id < 19)? rx1: ry1);
    sum6 += a10*((lane_id < 23)? rx2: ry2);
    sum7 += a10*((lane_id < 25)? rx3: ry3);

    friend_id0 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    rx0 = __shfl(threadInput9 , friend_id4);
    ry0 = __shfl(threadInput10, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += a11*((lane_id < 2 )? tx0: ((lane_id < 28)? ty0: tz0));
    sum1 += a11*((lane_id < 6 )? tx1: ty1);
    sum2 += a11*((lane_id < 8 )? tx2: ty2);
    sum3 += a11*((lane_id < 12)? tx3: ty3);
    sum4 += a11*((lane_id < 16)? rx0: ry0);
    sum5 += a11*((lane_id < 18)? rx1: ry1);
    sum6 += a11*((lane_id < 22)? rx2: ry2);
    sum7 += a11*((lane_id < 24)? rx3: ry3);

    friend_id0 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tz2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    rx0 = __shfl(threadInput9 , friend_id4);
    ry0 = __shfl(threadInput10, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += a12*((lane_id < 22)? tx0: ty0);
    sum1 += a12*((lane_id < 24)? tx1: ty1);
    sum2 += a12*((lane_id < 2 )? tx2: ((lane_id < 28)? ty2: tz2));
    sum3 += a12*((lane_id < 6 )? tx3: ty3);
    sum4 += a12*((lane_id < 8 )? rx0: ry0);
    sum5 += a12*((lane_id < 12)? rx1: ry1);
    sum6 += a12*((lane_id < 16)? rx2: ry2);
    sum7 += a12*((lane_id < 18)? rx3: ry3);

    friend_id0 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+23+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+15+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tz2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    tz3 = __shfl(threadInput9, friend_id3);
    rx0 = __shfl(threadInput9 , friend_id4);
    ry0 = __shfl(threadInput10, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += a13*((lane_id < 21)? tx0: ty0);
    sum1 += a13*((lane_id < 24)? tx1: ty1);
    sum2 += a13*((lane_id < 1 )? tx2: ((lane_id < 27)? ty2: tz2));
    sum3 += a13*((lane_id < 5 )? tx3: ((lane_id < 31)? ty3: tz3));
    sum4 += a13*((lane_id < 8 )? rx0: ry0);
    sum5 += a13*((lane_id < 11)? rx1: ry1);
    sum6 += a13*((lane_id < 15)? rx2: ry2);
    sum7 += a13*((lane_id < 17)? rx3: ry3);

    friend_id0 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    tz3 = __shfl(threadInput9, friend_id3);
    rx0 = __shfl(threadInput9 , friend_id4);
    ry0 = __shfl(threadInput10, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += a14*((lane_id < 20)? tx0: ty0);
    sum1 += a14*((lane_id < 24)? tx1: ty1);
    sum2 += a14*((lane_id < 26)? tx2: ty2);
    sum3 += a14*((lane_id < 4 )? tx3: ((lane_id < 30)? ty3: tz3));
    sum4 += a14*((lane_id < 8 )? rx0: ry0);
    sum5 += a14*((lane_id < 10)? rx1: ry1);
    sum6 += a14*((lane_id < 14)? rx2: ry2);
    sum7 += a14*((lane_id < 16)? rx3: ry3);

    friend_id0 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+0+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput8, friend_id3);
    ty3 = __shfl(threadInput9, friend_id3);
    rx0 = __shfl(threadInput10, friend_id4);
    ry0 = __shfl(threadInput11, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rz1 = __shfl(threadInput13, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += a15*((lane_id < 14)? tx0: ty0);
    sum1 += a15*((lane_id < 16)? tx1: ty1);
    sum2 += a15*((lane_id < 20)? tx2: ty2);
    sum3 += a15*((lane_id < 24)? tx3: ty3);
    sum4 += a15*((lane_id < 26)? rx0: ry0);
    sum5 += a15*((lane_id < 4 )? rx1: ((lane_id < 30)? ry1: rz1));
    sum6 += a15*((lane_id < 8 )? rx2: ry2);
    sum7 += a15*((lane_id < 10)? rx3: ry3);

    friend_id0 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+1+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput8, friend_id3);
    ty3 = __shfl(threadInput9, friend_id3);
    rx0 = __shfl(threadInput10, friend_id4);
    ry0 = __shfl(threadInput11, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rz1 = __shfl(threadInput13, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += a16*((lane_id < 13)? tx0: ty0);
    sum1 += a16*((lane_id < 16)? tx1: ty1);
    sum2 += a16*((lane_id < 19)? tx2: ty2);
    sum3 += a16*((lane_id < 23)? tx3: ty3);
    sum4 += a16*((lane_id < 25)? rx0: ry0);
    sum5 += a16*((lane_id < 3 )? rx1: ((lane_id < 29)? ry1: rz1));
    sum6 += a16*((lane_id < 7 )? rx2: ry2);
    sum7 += a16*((lane_id < 9 )? rx3: ry3);

    friend_id0 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+2+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput8, friend_id3);
    ty3 = __shfl(threadInput9, friend_id3);
    rx0 = __shfl(threadInput10, friend_id4);
    ry0 = __shfl(threadInput11, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rz1 = __shfl(threadInput13, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += a17*((lane_id < 12)? tx0: ty0);
    sum1 += a17*((lane_id < 16)? tx1: ty1);
    sum2 += a17*((lane_id < 18)? tx2: ty2);
    sum3 += a17*((lane_id < 22)? tx3: ty3);
    sum4 += a17*((lane_id < 24)? rx0: ry0);
    sum5 += a17*((lane_id < 2 )? rx1: ((lane_id < 28)? ry1: rz1));
    sum6 += a17*((lane_id < 6 )? rx2: ry2);
    sum7 += a17*((lane_id < 8 )? rx3: ry3);

    friend_id0 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput16, friend_id7);
    ry3 = __shfl(threadInput17, friend_id7);
    rz3 = __shfl(threadInput18, friend_id7);
    sum0 += a18*((lane_id < 8 )? tx0: ty0);
    sum1 += a18*((lane_id < 10)? tx1: ty1);
    sum2 += a18*((lane_id < 14)? tx2: ty2);
    sum3 += a18*((lane_id < 16)? tx3: ty3);
    sum4 += a18*((lane_id < 20)? rx0: ry0);
    sum5 += a18*((lane_id < 24)? rx1: ry1);
    sum6 += a18*((lane_id < 26)? rx2: ry2);
    sum7 += a18*((lane_id < 4 )? rx3: ((lane_id < 30)? ry3: rz3));

    friend_id0 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput16, friend_id7);
    ry3 = __shfl(threadInput17, friend_id7);
    rz3 = __shfl(threadInput18, friend_id7);
    sum0 += a19*((lane_id < 7 )? tx0: ty0);
    sum1 += a19*((lane_id < 9 )? tx1: ty1);
    sum2 += a19*((lane_id < 13)? tx2: ty2);
    sum3 += a19*((lane_id < 16)? tx3: ty3);
    sum4 += a19*((lane_id < 19)? rx0: ry0);
    sum5 += a19*((lane_id < 23)? rx1: ry1);
    sum6 += a19*((lane_id < 25)? rx2: ry2);
    sum7 += a19*((lane_id < 3 )? rx3: ((lane_id < 29)? ry3: rz3));

    friend_id0 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput16, friend_id7);
    ry3 = __shfl(threadInput17, friend_id7);
    rz3 = __shfl(threadInput18, friend_id7);
    sum0 += a20*((lane_id < 6 )? tx0: ty0);
    sum1 += a20*((lane_id < 8 )? tx1: ty1);
    sum2 += a20*((lane_id < 12)? tx2: ty2);
    sum3 += a20*((lane_id < 16)? tx3: ty3);
    sum4 += a20*((lane_id < 18)? rx0: ry0);
    sum5 += a20*((lane_id < 22)? rx1: ry1);
    sum6 += a20*((lane_id < 24)? rx2: ry2);
    sum7 += a20*((lane_id < 2 )? rx3: ((lane_id < 28)? ry3: rz3));

    friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tz1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput17, friend_id7);
    ry3 = __shfl(threadInput18, friend_id7);
    sum0 += a21*((lane_id < 24)? tx0: ty0);
    sum1 += a21*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
    sum2 += a21*((lane_id < 6 )? tx2: ty2);
    sum3 += a21*((lane_id < 8 )? tx3: ty3);
    sum4 += a21*((lane_id < 12)? rx0: ry0);
    sum5 += a21*((lane_id < 16)? rx1: ry1);
    sum6 += a21*((lane_id < 18)? rx2: ry2);
    sum7 += a21*((lane_id < 22)? rx3: ry3);

    friend_id0 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+23+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+15+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tz1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tz2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput17, friend_id7);
    ry3 = __shfl(threadInput18, friend_id7);
    sum0 += a22*((lane_id < 24)? tx0: ty0);
    sum1 += a22*((lane_id < 1 )? tx1: ((lane_id < 27)? ty1: tz1));
    sum2 += a22*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
    sum3 += a22*((lane_id < 8 )? tx3: ty3);
    sum4 += a22*((lane_id < 11)? rx0: ry0);
    sum5 += a22*((lane_id < 15)? rx1: ry1);
    sum6 += a22*((lane_id < 17)? rx2: ry2);
    sum7 += a22*((lane_id < 21)? rx3: ry3);

    friend_id0 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tz2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput17, friend_id7);
    ry3 = __shfl(threadInput18, friend_id7);
    sum0 += a23*((lane_id < 24)? tx0: ty0);
    sum1 += a23*((lane_id < 26)? tx1: ty1);
    sum2 += a23*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
    sum3 += a23*((lane_id < 8 )? tx3: ty3);
    sum4 += a23*((lane_id < 10)? rx0: ry0);
    sum5 += a23*((lane_id < 14)? rx1: ry1);
    sum6 += a23*((lane_id < 16)? rx2: ry2);
    sum7 += a23*((lane_id < 20)? rx3: ry3);

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput8, friend_id2);
    ty2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput10, friend_id3);
    ty3 = __shfl(threadInput11, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rz0 = __shfl(threadInput13, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput17, friend_id7);
    ry3 = __shfl(threadInput18, friend_id7);
    sum0 += a24*((lane_id < 16)? tx0: ty0);
    sum1 += a24*((lane_id < 20)? tx1: ty1);
    sum2 += a24*((lane_id < 24)? tx2: ty2);
    sum3 += a24*((lane_id < 26)? tx3: ty3);
    sum4 += a24*((lane_id < 4 )? rx0: ((lane_id < 30)? ry0: rz0));
    sum5 += a24*((lane_id < 8 )? rx1: ry1);
    sum6 += a24*((lane_id < 10)? rx2: ry2);
    sum7 += a24*((lane_id < 14)? rx3: ry3);

    friend_id0 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput8, friend_id2);
    ty2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput10, friend_id3);
    ty3 = __shfl(threadInput11, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rz0 = __shfl(threadInput13, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput17, friend_id7);
    ry3 = __shfl(threadInput18, friend_id7);
    sum0 += a25*((lane_id < 16)? tx0: ty0);
    sum1 += a25*((lane_id < 19)? tx1: ty1);
    sum2 += a25*((lane_id < 23)? tx2: ty2);
    sum3 += a25*((lane_id < 25)? tx3: ty3);
    sum4 += a25*((lane_id < 3 )? rx0: ((lane_id < 29)? ry0: rz0));
    sum5 += a25*((lane_id < 7 )? rx1: ry1);
    sum6 += a25*((lane_id < 9 )? rx2: ry2);
    sum7 += a25*((lane_id < 13)? rx3: ry3);

    friend_id0 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput8, friend_id2);
    ty2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput10, friend_id3);
    ty3 = __shfl(threadInput11, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rz0 = __shfl(threadInput13, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput17, friend_id7);
    ry3 = __shfl(threadInput18, friend_id7);
    sum0 += a26*((lane_id < 16)? tx0: ty0);
    sum1 += a26*((lane_id < 18)? tx1: ty1);
    sum2 += a26*((lane_id < 22)? tx2: ty2);
    sum3 += a26*((lane_id < 24)? tx3: ty3);
    sum4 += a26*((lane_id < 2 )? rx0: ((lane_id < 28)? ry0: rz0));
    sum5 += a26*((lane_id < 6 )? rx1: ry1);
    sum6 += a26*((lane_id < 8 )? rx2: ry2);
    sum7 += a26*((lane_id < 12)? rx3: ry3);


    OUT_3D(k  ,j,i) = sum0;
    OUT_3D(k+1,j,i) = sum1;
    OUT_3D(k+2,j,i) = sum2;
    OUT_3D(k+3,j,i) = sum3;
    OUT_3D(k+4,j,i) = sum4;
    OUT_3D(k+5,j,i) = sum5;
    OUT_3D(k+6,j,i) = sum6;
    OUT_3D(k+7,j,i) = sum7;
}

__global__ void Stencil_Cuda_Sweep_Shfl(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, 
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, DATA_TYPE a7, 
        DATA_TYPE a8, DATA_TYPE a9, DATA_TYPE a10, DATA_TYPE a11, DATA_TYPE a12, 
        DATA_TYPE a13, DATA_TYPE a14, DATA_TYPE a15, DATA_TYPE a16, DATA_TYPE a17, 
        DATA_TYPE a18, DATA_TYPE a19, DATA_TYPE a20, DATA_TYPE a21, DATA_TYPE a22, 
        DATA_TYPE a23, DATA_TYPE a24, DATA_TYPE a25, DATA_TYPE a26 , 
        int Z, int M, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    const int block_z = Z / gridDim.z;
    int k = block_z * blockIdx.z ;
    const int k_end = k + block_z;
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int lane_id = tid % warpSize;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    DATA_TYPE tx, ty;
    int friend_id;
    int new_i, new_j;
    DATA_TYPE t3_threadInput0, t3_threadInput1;
    DATA_TYPE t2_threadInput0, t2_threadInput1;
    DATA_TYPE t1_threadInput0, t1_threadInput1;

#define SM_2D_M2 32 
#define SM_2D_N2 8 

    DATA_TYPE sum = 0.0;

    int b = (k == 0)      ? k : k - 1;
    // t3 is current layer; t2 is previous layer
    new_i = (warp_id_x<<3) + lane_id%10-1;     // 10 is extended dimension of i
    new_j = (warp_id_y<<2) + lane_id/10-1;     
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput0 = IN_3D(k  , new_j, new_i);
    t2_threadInput0 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput1 = IN_3D(k  , new_j, new_i);
    t2_threadInput1 = IN_3D(b  , new_j, new_i);

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        sum = 0.0;
        // move the current storage down 
        t1_threadInput0 = t2_threadInput0;
        t1_threadInput1 = t2_threadInput1;
        t2_threadInput0 = t3_threadInput0;
        t2_threadInput1 = t3_threadInput1;

        int t = (k == Z-1)    ? k : k + 1;
        new_i = (warp_id_x<<3) + lane_id%10-1;  
        new_j = (warp_id_y<<2) + lane_id/10-1;     
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput0 = IN_3D(t, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput1 = IN_3D(t, new_j, new_i);

        friend_id = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += a0*((lane_id < 26)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += a9*((lane_id < 26)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += a18*((lane_id < 26)? tx: ty);

        friend_id = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += a1*((lane_id < 25)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += a10*((lane_id < 25)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += a19*((lane_id < 25)? tx: ty);

        friend_id = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += a2*((lane_id < 24)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += a11*((lane_id < 24)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += a20*((lane_id < 24)? tx: ty);

        friend_id = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += a3*((lane_id < 18)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += a12*((lane_id < 18)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += a21*((lane_id < 18)? tx: ty);

        friend_id = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += a4*((lane_id < 17)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += a13*((lane_id < 17)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += a22*((lane_id < 17)? tx: ty);

        friend_id = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += a5*((lane_id < 16)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += a14*((lane_id < 16)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += a23*((lane_id < 16)? tx: ty);

        friend_id = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += a6*((lane_id < 10)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += a15*((lane_id < 10)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += a24*((lane_id < 10)? tx: ty);

        friend_id = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += a7*((lane_id < 9 )? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += a16*((lane_id < 9 )? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += a25*((lane_id < 9 )? tx: ty);
        
        friend_id = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += a8*((lane_id < 8 )? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += a17*((lane_id < 8 )? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += a26*((lane_id < 8 )? tx: ty);


        OUT_3D(k,j,i) = sum;
    }
}

__global__ void Stencil_Cuda_Sweep_Shfl2(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, 
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, DATA_TYPE a7, 
        DATA_TYPE a8, DATA_TYPE a9, DATA_TYPE a10, DATA_TYPE a11, DATA_TYPE a12, 
        DATA_TYPE a13, DATA_TYPE a14, DATA_TYPE a15, DATA_TYPE a16, DATA_TYPE a17, 
        DATA_TYPE a18, DATA_TYPE a19, DATA_TYPE a20, DATA_TYPE a21, DATA_TYPE a22, 
        DATA_TYPE a23, DATA_TYPE a24, DATA_TYPE a25, DATA_TYPE a26 , 
        int Z, int M, int N)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = (((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<3) + (lane_id>>3) ;

    const int block_z = Z / gridDim.z;
    int k = block_z * blockIdx.z ;
    const int k_end = k + block_z;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = ((((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<3) + (lane_id>>3))>>2; // 1x4x8, warp_ids are division of 
    DATA_TYPE tx0, ty0;
    DATA_TYPE tx1, ty1, tz1;
    int friend_id0, friend_id1;
    int new_i, new_j;
    DATA_TYPE t3_threadInput0, t3_threadInput1, t3_threadInput2, t3_threadInput3;
    DATA_TYPE t2_threadInput0, t2_threadInput1, t2_threadInput2, t2_threadInput3;
    DATA_TYPE t1_threadInput0, t1_threadInput1, t1_threadInput2, t1_threadInput3;

#define SM_2D_M2 32 
#define SM_2D_N2 8 

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;

    // t3 is current layer; t2 is previous layer
    int b = (k == 0)      ? k : k - 1;
    new_i = (warp_id_x<<3) + lane_id%10-1;     // 10 is extended dimension of i
    new_j = (warp_id_y<<2) + lane_id/10-1;     
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput0 = IN_3D(k  , new_j, new_i);
    t2_threadInput0 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput1 = IN_3D(k  , new_j, new_i);
    t2_threadInput1 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+64)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput2 = IN_3D(k  , new_j, new_i);
    t2_threadInput2 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+96)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput3 = IN_3D(k  , new_j, new_i);
    t2_threadInput3 = IN_3D(b  , new_j, new_i);

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        sum0 = 0.0;
        sum1 = 0.0;
        // move the current storage down 
        t1_threadInput0 = t2_threadInput0;
        t1_threadInput1 = t2_threadInput1;
        t1_threadInput2 = t2_threadInput2;
        t1_threadInput3 = t2_threadInput3;
        t2_threadInput0 = t3_threadInput0;
        t2_threadInput1 = t3_threadInput1;
        t2_threadInput2 = t3_threadInput2;
        t2_threadInput3 = t3_threadInput3;

        int t = (k == Z-1)    ? k : k + 1;
        new_i = (warp_id_x<<3) + lane_id%10-1;  
        new_j = (warp_id_y<<2) + lane_id/10-1;     
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput0 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput1 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+64)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput2 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+96)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput3 = IN_3D(t  , new_j, new_i);

        friend_id0 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        sum0 += a0*((lane_id < 26)? tx0: ty0);
        sum1 += a0*((lane_id < 20)? tx1: ty1);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        sum0 += a9*((lane_id < 26)? tx0: ty0);
        sum1 += a9*((lane_id < 20)? tx1: ty1);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        sum0 += a18*((lane_id < 26)? tx0: ty0);
        sum1 += a18*((lane_id < 20)? tx1: ty1);

        friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        sum0 += a1*((lane_id < 25)? tx0: ty0);
        sum1 += a1*((lane_id < 19)? tx1: ty1);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        sum0 += a10*((lane_id < 25)? tx0: ty0);
        sum1 += a10*((lane_id < 19)? tx1: ty1);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        sum0 += a19*((lane_id < 25)? tx0: ty0);
        sum1 += a19*((lane_id < 19)? tx1: ty1);

        friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        sum0 += a2*((lane_id < 24)? tx0: ty0);
        sum1 += a2*((lane_id < 18)? tx1: ty1);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        sum0 += a11*((lane_id < 24)? tx0: ty0);
        sum1 += a11*((lane_id < 18)? tx1: ty1);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        sum0 += a20*((lane_id < 24)? tx0: ty0);
        sum1 += a20*((lane_id < 18)? tx1: ty1);

        friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        sum0 += a3*((lane_id < 18)? tx0: ty0);
        sum1 += a3*((lane_id < 12)? tx1: ty1);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        sum0 += a12*((lane_id < 18)? tx0: ty0);
        sum1 += a12*((lane_id < 12)? tx1: ty1);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        sum0 += a21*((lane_id < 18)? tx0: ty0);
        sum1 += a21*((lane_id < 12)? tx1: ty1);

        friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        sum0 += a4*((lane_id < 17)? tx0: ty0);
        sum1 += a4*((lane_id < 11)? tx1: ty1);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        sum0 += a13*((lane_id < 17)? tx0: ty0);
        sum1 += a13*((lane_id < 11)? tx1: ty1);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        sum0 += a22*((lane_id < 17)? tx0: ty0);
        sum1 += a22*((lane_id < 11)? tx1: ty1);

        friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        sum0 += a5*((lane_id < 16)? tx0: ty0);
        sum1 += a5*((lane_id < 10)? tx1: ty1);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        sum0 += a14*((lane_id < 16)? tx0: ty0);
        sum1 += a14*((lane_id < 10)? tx1: ty1);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        sum0 += a23*((lane_id < 16)? tx0: ty0);
        sum1 += a23*((lane_id < 10)? tx1: ty1);

        friend_id0 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tz1 = __shfl(t1_threadInput3, friend_id1);
        sum0 += a6*((lane_id < 10)? tx0: ty0);
        sum1 += a6*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tz1 = __shfl(t2_threadInput3, friend_id1);
        sum0 += a15*((lane_id < 10)? tx0: ty0);
        sum1 += a15*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tz1 = __shfl(t3_threadInput3, friend_id1);
        sum0 += a24*((lane_id < 10)? tx0: ty0);
        sum1 += a24*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));

        friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tz1 = __shfl(t1_threadInput3, friend_id1);
        sum0 += a7*((lane_id < 9 )? tx0: ty0);
        sum1 += a7*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tz1 = __shfl(t2_threadInput3, friend_id1);
        sum0 += a16*((lane_id < 9 )? tx0: ty0);
        sum1 += a16*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tz1 = __shfl(t3_threadInput3, friend_id1);
        sum0 += a25*((lane_id < 9 )? tx0: ty0);
        sum1 += a25*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        
        friend_id0 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tz1 = __shfl(t1_threadInput3, friend_id1);
        sum0 += a8*((lane_id < 8 )? tx0: ty0);
        sum1 += a8*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tz1 = __shfl(t2_threadInput3, friend_id1);
        sum0 += a17*((lane_id < 8 )? tx0: ty0);
        sum1 += a17*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tz1 = __shfl(t3_threadInput3, friend_id1);
        sum0 += a26*((lane_id < 8 )? tx0: ty0);
        sum1 += a26*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        


        OUT_3D(k,j  ,i) = sum0;
        OUT_3D(k,j+4,i) = sum1;
    }
}

__global__ void Stencil_Cuda_Sweep_Shfl4(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, 
        DATA_TYPE a3, DATA_TYPE a4, DATA_TYPE a5, DATA_TYPE a6, DATA_TYPE a7, 
        DATA_TYPE a8, DATA_TYPE a9, DATA_TYPE a10, DATA_TYPE a11, DATA_TYPE a12, 
        DATA_TYPE a13, DATA_TYPE a14, DATA_TYPE a15, DATA_TYPE a16, DATA_TYPE a17, 
        DATA_TYPE a18, DATA_TYPE a19, DATA_TYPE a20, DATA_TYPE a21, DATA_TYPE a22, 
        DATA_TYPE a23, DATA_TYPE a24, DATA_TYPE a25, DATA_TYPE a26 , 
        int Z, int M, int N)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = (((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<4) + (lane_id>>3) ;

    const int block_z = Z / gridDim.z;
    int k = block_z * blockIdx.z ;
    const int k_end = k + block_z;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = ((((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<4) + (lane_id>>3))>>2; // 1x4x8, warp_ids are division of 
    DATA_TYPE tx0, ty0;
    DATA_TYPE tx1, ty1, tz1;
    DATA_TYPE tx2, ty2, tz2;
    DATA_TYPE tx3, ty3, tz3;
    int friend_id0, friend_id1;
    int friend_id2, friend_id3;
    int new_i, new_j;
    DATA_TYPE t3_threadInput0, t3_threadInput1, t3_threadInput2, t3_threadInput3, t3_threadInput4, t3_threadInput5;
    DATA_TYPE t2_threadInput0, t2_threadInput1, t2_threadInput2, t2_threadInput3, t2_threadInput4, t2_threadInput5;
    DATA_TYPE t1_threadInput0, t1_threadInput1, t1_threadInput2, t1_threadInput3, t1_threadInput4, t1_threadInput5;

#define SM_2D_M2 32 
#define SM_2D_N2 8 

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;

    // t3 is current layer; t2 is previous layer
    int b = (k == 0)      ? k : k - 1;
    new_i = (warp_id_x<<3) + lane_id%10-1;     // 10 is extended dimension of i
    new_j = (warp_id_y<<2) + lane_id/10-1;     
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput0 = IN_3D(k  , new_j, new_i);
    t2_threadInput0 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput1 = IN_3D(k  , new_j, new_i);
    t2_threadInput1 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+64)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput2 = IN_3D(k  , new_j, new_i);
    t2_threadInput2 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+96)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput3 = IN_3D(k  , new_j, new_i);
    t2_threadInput3 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+128)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput4 = IN_3D(k  , new_j, new_i);
    t2_threadInput4 = IN_3D(b  , new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+160)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    t3_threadInput5 = IN_3D(k  , new_j, new_i);
    t2_threadInput5 = IN_3D(b  , new_j, new_i);

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        sum0 = 0.0;
        sum1 = 0.0;
        sum2 = 0.0;
        sum3 = 0.0;
        // move the current storage down 
        t1_threadInput0 = t2_threadInput0;
        t1_threadInput1 = t2_threadInput1;
        t1_threadInput2 = t2_threadInput2;
        t1_threadInput3 = t2_threadInput3;
        t1_threadInput4 = t2_threadInput4;
        t1_threadInput5 = t2_threadInput5;

        t2_threadInput0 = t3_threadInput0;
        t2_threadInput1 = t3_threadInput1;
        t2_threadInput2 = t3_threadInput2;
        t2_threadInput3 = t3_threadInput3;
        t2_threadInput4 = t3_threadInput4;
        t2_threadInput5 = t3_threadInput5;

        int t = (k == Z-1)    ? k : k + 1;
        new_i = (warp_id_x<<3) + lane_id%10-1;  
        new_j = (warp_id_y<<2) + lane_id/10-1;     
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput0 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput1 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+64)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput2 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+96)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput3 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+128)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput4 = IN_3D(t  , new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
        new_j = (warp_id_y<<2) + (lane_id+160)/10-1;
        new_j = (new_j == -1)   ? 0   : new_j;
        new_j = (new_j >= M)    ? M-1 : new_j;
        new_i = (new_i == -1)   ? 0   : new_i;
        new_i = (new_i >= N)    ? N-1 : new_i;
        t3_threadInput5 = IN_3D(t  , new_j, new_i);

        friend_id0 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tx3 = __shfl(t1_threadInput3, friend_id3);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        sum0 += a0*((lane_id < 26)? tx0: ty0);
        sum1 += a0*((lane_id < 20)? tx1: ty1);
        sum2 += a0*((lane_id < 14)? tx2: ty2);
        sum3 += a0*((lane_id < 8 )? tx3: ty3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tx3 = __shfl(t2_threadInput3, friend_id3);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        sum0 += a9*((lane_id < 26)? tx0: ty0);
        sum1 += a9*((lane_id < 20)? tx1: ty1);
        sum2 += a9*((lane_id < 14)? tx2: ty2);
        sum3 += a9*((lane_id < 8 )? tx3: ty3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tx3 = __shfl(t3_threadInput3, friend_id3);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        sum0 += a18*((lane_id < 26)? tx0: ty0);
        sum1 += a18*((lane_id < 20)? tx1: ty1);
        sum2 += a18*((lane_id < 14)? tx2: ty2);
        sum3 += a18*((lane_id < 8 )? tx3: ty3);

        friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tx3 = __shfl(t1_threadInput3, friend_id3);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        sum0 += a1*((lane_id < 25)? tx0: ty0);
        sum1 += a1*((lane_id < 19)? tx1: ty1);
        sum2 += a1*((lane_id < 13)? tx2: ty2);
        sum3 += a1*((lane_id < 7 )? tx3: ty3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tx3 = __shfl(t2_threadInput3, friend_id3);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        sum0 += a10*((lane_id < 25)? tx0: ty0);
        sum1 += a10*((lane_id < 19)? tx1: ty1);
        sum2 += a10*((lane_id < 13)? tx2: ty2);
        sum3 += a10*((lane_id < 7 )? tx3: ty3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tx3 = __shfl(t3_threadInput3, friend_id3);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        sum0 += a19*((lane_id < 25)? tx0: ty0);
        sum1 += a19*((lane_id < 19)? tx1: ty1);
        sum2 += a19*((lane_id < 13)? tx2: ty2);
        sum3 += a19*((lane_id < 7 )? tx3: ty3);

        friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tx3 = __shfl(t1_threadInput3, friend_id3);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        sum0 += a2*((lane_id < 24)? tx0: ty0);
        sum1 += a2*((lane_id < 18)? tx1: ty1);
        sum2 += a2*((lane_id < 12)? tx2: ty2);
        sum3 += a2*((lane_id < 6 )? tx3: ty3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tx3 = __shfl(t2_threadInput3, friend_id3);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        sum0 += a11*((lane_id < 24)? tx0: ty0);
        sum1 += a11*((lane_id < 18)? tx1: ty1);
        sum2 += a11*((lane_id < 12)? tx2: ty2);
        sum3 += a11*((lane_id < 6 )? tx3: ty3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tx3 = __shfl(t3_threadInput3, friend_id3);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        sum0 += a20*((lane_id < 24)? tx0: ty0);
        sum1 += a20*((lane_id < 18)? tx1: ty1);
        sum2 += a20*((lane_id < 12)? tx2: ty2);
        sum3 += a20*((lane_id < 6 )? tx3: ty3);

        friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        tz3 = __shfl(t1_threadInput5, friend_id3);
        sum0 += a3*((lane_id < 18)? tx0: ty0);
        sum1 += a3*((lane_id < 12)? tx1: ty1);
        sum2 += a3*((lane_id < 6 )? tx2: ty2);
        sum3 += a3*((lane_id < 24)? ty3: tz3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        tz3 = __shfl(t2_threadInput5, friend_id3);
        sum0 += a12*((lane_id < 18)? tx0: ty0);
        sum1 += a12*((lane_id < 12)? tx1: ty1);
        sum2 += a12*((lane_id < 6 )? tx2: ty2);
        sum3 += a12*((lane_id < 24)? ty3: tz3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        tz3 = __shfl(t3_threadInput5, friend_id3);
        sum0 += a21*((lane_id < 18)? tx0: ty0);
        sum1 += a21*((lane_id < 12)? tx1: ty1);
        sum2 += a21*((lane_id < 6 )? tx2: ty2);
        sum3 += a21*((lane_id < 24)? ty3: tz3);

        friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tz2 = __shfl(t1_threadInput4, friend_id2);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        tz3 = __shfl(t1_threadInput5, friend_id3);
        sum0 += a4*((lane_id < 17)? tx0: ty0);
        sum1 += a4*((lane_id < 11)? tx1: ty1);
        sum2 += a4*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
        sum3 += a4*((lane_id < 24)? ty3: tz3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tz2 = __shfl(t2_threadInput4, friend_id2);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        tz3 = __shfl(t2_threadInput5, friend_id3);
        sum0 += a13*((lane_id < 17)? tx0: ty0);
        sum1 += a13*((lane_id < 11)? tx1: ty1);
        sum2 += a13*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
        sum3 += a13*((lane_id < 24)? ty3: tz3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tz2 = __shfl(t3_threadInput4, friend_id2);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        tz3 = __shfl(t3_threadInput5, friend_id3);
        sum0 += a22*((lane_id < 17)? tx0: ty0);
        sum1 += a22*((lane_id < 11)? tx1: ty1);
        sum2 += a22*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
        sum3 += a22*((lane_id < 24)? ty3: tz3);

        friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tz2 = __shfl(t1_threadInput4, friend_id2);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        tz3 = __shfl(t1_threadInput5, friend_id3);
        sum0 += a5*((lane_id < 16)? tx0: ty0);
        sum1 += a5*((lane_id < 10)? tx1: ty1);
        sum2 += a5*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
        sum3 += a5*((lane_id < 24)? ty3: tz3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tz2 = __shfl(t2_threadInput4, friend_id2);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        tz3 = __shfl(t2_threadInput5, friend_id3);
        sum0 += a14*((lane_id < 16)? tx0: ty0);
        sum1 += a14*((lane_id < 10)? tx1: ty1);
        sum2 += a14*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
        sum3 += a14*((lane_id < 24)? ty3: tz3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tz2 = __shfl(t3_threadInput4, friend_id2);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        tz3 = __shfl(t3_threadInput5, friend_id3);
        sum0 += a23*((lane_id < 16)? tx0: ty0);
        sum1 += a23*((lane_id < 10)? tx1: ty1);
        sum2 += a23*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
        sum3 += a23*((lane_id < 24)? ty3: tz3);

        friend_id0 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tz1 = __shfl(t1_threadInput3, friend_id1);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tz2 = __shfl(t1_threadInput4, friend_id2);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        tz3 = __shfl(t1_threadInput5, friend_id3);
        sum0 += a6*((lane_id < 10)? tx0: ty0);
        sum1 += a6*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        sum2 += a6*((lane_id < 24)? ty2: tz2);
        sum3 += a6*((lane_id < 16)? ty3: tz3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tz1 = __shfl(t2_threadInput3, friend_id1);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tz2 = __shfl(t2_threadInput4, friend_id2);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        tz3 = __shfl(t2_threadInput5, friend_id3);
        sum0 += a15*((lane_id < 10)? tx0: ty0);
        sum1 += a15*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        sum2 += a15*((lane_id < 24)? ty2: tz2);
        sum3 += a15*((lane_id < 16)? ty3: tz3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tz1 = __shfl(t3_threadInput3, friend_id1);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tz2 = __shfl(t3_threadInput4, friend_id2);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        tz3 = __shfl(t3_threadInput5, friend_id3);
        sum0 += a24*((lane_id < 10)? tx0: ty0);
        sum1 += a24*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        sum2 += a24*((lane_id < 24)? ty2: tz2);
        sum3 += a24*((lane_id < 16)? ty3: tz3);

        friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tz1 = __shfl(t1_threadInput3, friend_id1);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tz2 = __shfl(t1_threadInput4, friend_id2);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        tz3 = __shfl(t1_threadInput5, friend_id3);
        sum0 += a7*((lane_id < 9 )? tx0: ty0);
        sum1 += a7*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        sum2 += a7*((lane_id < 23)? ty2: tz2);
        sum3 += a7*((lane_id < 16)? ty3: tz3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tz1 = __shfl(t2_threadInput3, friend_id1);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tz2 = __shfl(t2_threadInput4, friend_id2);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        tz3 = __shfl(t2_threadInput5, friend_id3);
        sum0 += a16*((lane_id < 9 )? tx0: ty0);
        sum1 += a16*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        sum2 += a16*((lane_id < 23)? ty2: tz2);
        sum3 += a16*((lane_id < 16)? ty3: tz3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tz1 = __shfl(t3_threadInput3, friend_id1);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tz2 = __shfl(t3_threadInput4, friend_id2);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        tz3 = __shfl(t3_threadInput5, friend_id3);
        sum0 += a25*((lane_id < 9 )? tx0: ty0);
        sum1 += a25*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        sum2 += a25*((lane_id < 23)? ty2: tz2);
        sum3 += a25*((lane_id < 16)? ty3: tz3);
        
        friend_id0 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tz1 = __shfl(t1_threadInput3, friend_id1);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tz2 = __shfl(t1_threadInput4, friend_id2);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        tz3 = __shfl(t1_threadInput5, friend_id3);
        sum0 += a8*((lane_id < 8 )? tx0: ty0);
        sum1 += a8*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        sum2 += a8*((lane_id < 22)? ty2: tz2);
        sum3 += a8*((lane_id < 16)? ty3: tz3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tz1 = __shfl(t2_threadInput3, friend_id1);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tz2 = __shfl(t2_threadInput4, friend_id2);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        tz3 = __shfl(t2_threadInput5, friend_id3);
        sum0 += a17*((lane_id < 8 )? tx0: ty0);
        sum1 += a17*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        sum2 += a17*((lane_id < 22)? ty2: tz2);
        sum3 += a17*((lane_id < 16)? ty3: tz3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tz1 = __shfl(t3_threadInput3, friend_id1);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tz2 = __shfl(t3_threadInput4, friend_id2);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        tz3 = __shfl(t3_threadInput5, friend_id3);
        sum0 += a26*((lane_id < 8 )? tx0: ty0);
        sum1 += a26*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        sum2 += a26*((lane_id < 22)? ty2: tz2);
        sum3 += a26*((lane_id < 16)? ty3: tz3);


        OUT_3D(k,j   ,i) = sum0;
        OUT_3D(k,j+4 ,i) = sum1;
        OUT_3D(k,j+8 ,i) = sum2;
        OUT_3D(k,j+12,i) = sum3;
    }
}

int main(int argc, char **argv)
{
#ifdef __DEBUG
    int z = 4;
    int m = 128;
    int n = 8;
#else
    int z = 256; 
    int m = 256;
    int n = 256; 
#endif
    int total = (z)*(m)*(n);
    const int K = 27;
#ifdef __DEBUG
    DATA_TYPE args[K] = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
                         1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,  
                         1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
#else
    DATA_TYPE args[K] = {0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 
                         0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037,  
                         0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037};
#endif
    DATA_TYPE *in = new DATA_TYPE[total];
    DATA_TYPE *out_ref = new DATA_TYPE[total];
    unsigned int seed = time(NULL);
    Init_Input_3D(in, z, m, n, seed);

    // Show_Me(in, z, m, n, "Input:");
    for(int i = 0; i < ITER; i++)
    {
        Stencil_Seq(in, out_ref, args[0], args[1], args[2], args[3], args[4],
                args[5], args[6], args[7], args[8], args[9], args[10],
                args[11], args[12], args[13], args[14], args[15], args[16],
                args[17], args[18], args[19], args[20], args[21], args[22], 
                args[23], args[24], args[25], args[26] , z, m, n);
        swap(in, out_ref);
    }
    swap(in, out_ref);
    // Show_Me(out_ref, z, m, n, "Output:");


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    float time_wo_pci;

    DATA_TYPE *in_d;
    DATA_TYPE *out_d;
    DATA_TYPE *out = new DATA_TYPE[total];
    hipMalloc((void**)&in_d, total*sizeof(DATA_TYPE));
    hipMalloc((void**)&out_d, total*sizeof(DATA_TYPE));

    // Cuda version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, seed);
    Clear_Output_3D(out, z, m, n);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid((n)/8, (m)/4, (z)/8);
    dim3 dimBlock(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda<<<dimGrid, dimBlock>>>(in_d, out_d, args[0], args[1], args[2], args[3], args[4],
                args[5], args[6], args[7], args[8], args[9], args[10],
                args[11], args[12], args[13], args[14], args[15], args[16],
                args[17], args[18], args[19], args[20], args[21], args[22], 
                args[23], args[24], args[25], args[26] , z, m, n); 
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z, m, n, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Sweep version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, seed); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid1((n)/64, (m)/4, 4);
    dim3 dimBlock1(64, 4, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep<<<dimGrid1, dimBlock1>>>(in_d, out_d, args[0], args[1], args[2], args[3], args[4],
                args[5], args[6], args[7], args[8], args[9], args[10],
                args[11], args[12], args[13], args[14], args[15], args[16],
                args[17], args[18], args[19], args[20], args[21], args[22], 
                args[23], args[24], args[25], args[26] , z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Sweep):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sweep: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sweep Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z, m, n, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 3D Block SM version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, seed); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid2((n)/8, (m)/4, (z)/8);
    dim3 dimBlock2(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sm<<<dimGrid2, dimBlock2, ((SM_Z)*(SM_M)*(SM_N))*sizeof(DATA_TYPE)>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4],
                args[5], args[6], args[7], args[8], args[9], args[10],
                args[11], args[12], args[13], args[14], args[15], args[16],
                args[17], args[18], args[19], args[20], args[21], args[22], 
                args[23], args[24], args[25], args[26] , z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Sm):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sm: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sm Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z, m, n, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 2D Block SM version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, seed); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid3((n)/64, (m)/4, 4);
    dim3 dimBlock3(64, 4, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Sm<<<dimGrid3, dimBlock3, ((SM_2D_M)*(SM_2D_N)*3*sizeof(DATA_TYPE))>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4],
                args[5], args[6], args[7], args[8], args[9], args[10],
                args[11], args[12], args[13], args[14], args[15], args[16],
                args[17], args[18], args[19], args[20], args[21], args[22], 
                args[23], args[24], args[25], args[26] , z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Sweep_Sm):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sweep_Sm: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sweep_Sm Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z, m, n, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 3D Block Shfl version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, seed); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid4((n)/8, (m)/4, (z)/8);
    dim3 dimBlock4(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl<<<dimGrid4, dimBlock4>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4],
                args[5], args[6], args[7], args[8], args[9], args[10],
                args[11], args[12], args[13], args[14], args[15], args[16],
                args[17], args[18], args[19], args[20], args[21], args[22], 
                args[23], args[24], args[25], args[26] , z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Shfl):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z, m, n, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 3D Block Shfl2 version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, seed); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid5((n)/8, (m)/4, (z)/(8*2));
    dim3 dimBlock5(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl2<<<dimGrid5, dimBlock5>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4],
                args[5], args[6], args[7], args[8], args[9], args[10],
                args[11], args[12], args[13], args[14], args[15], args[16],
                args[17], args[18], args[19], args[20], args[21], args[22], 
                args[23], args[24], args[25], args[26] , z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Shfl2):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl2: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl2 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z, m, n, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 3D Block Shfl4 version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, seed); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid6((n)/8, (m)/4, (z)/(8*4));
    dim3 dimBlock6(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl4<<<dimGrid6, dimBlock6>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4],
                args[5], args[6], args[7], args[8], args[9], args[10],
                args[11], args[12], args[13], args[14], args[15], args[16],
                args[17], args[18], args[19], args[20], args[21], args[22], 
                args[23], args[24], args[25], args[26] , z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Shfl4):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl4: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl4 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z, m, n, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 3D Block Shfl8 version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, seed); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid7((n)/8, (m)/4, (z)/(8*8));
    dim3 dimBlock7(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl8<<<dimGrid7, dimBlock7>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4],
                args[5], args[6], args[7], args[8], args[9], args[10],
                args[11], args[12], args[13], args[14], args[15], args[16],
                args[17], args[18], args[19], args[20], args[21], args[22], 
                args[23], args[24], args[25], args[26] , z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Shfl8):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl8: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl8 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z, m, n, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 2D Block Shfl version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, seed); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid8((n)/8, (m)/32, 4);
    dim3 dimBlock8(8, 32, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Shfl<<<dimGrid8, dimBlock8>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4],
                args[5], args[6], args[7], args[8], args[9], args[10],
                args[11], args[12], args[13], args[14], args[15], args[16],
                args[17], args[18], args[19], args[20], args[21], args[22], 
                args[23], args[24], args[25], args[26] , z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Sweep_Shfl):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sweep_Shfl: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sweep_Shfl Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z, m, n, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 2D Block Shfl2 version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, seed); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid9((n)/8, (m)/(32*2), 4);
    dim3 dimBlock9(8, 32, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Shfl2<<<dimGrid9, dimBlock9>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4],
                args[5], args[6], args[7], args[8], args[9], args[10],
                args[11], args[12], args[13], args[14], args[15], args[16],
                args[17], args[18], args[19], args[20], args[21], args[22], 
                args[23], args[24], args[25], args[26] , z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Sweep_Shfl2):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sweep_Shfl2: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sweep_Shfl2 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z, m, n, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda 2D Block Shfl4 version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, seed); // reset input
    Clear_Output_3D(out, z, m, n); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid10((n)/8, (m)/(32*4), 4);
    dim3 dimBlock10(8, 32, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Shfl4<<<dimGrid10, dimBlock10>>>(
            in_d, out_d, args[0], args[1], args[2], args[3], args[4],
                args[5], args[6], args[7], args[8], args[9], args[10],
                args[11], args[12], args[13], args[14], args[15], args[16],
                args[17], args[18], args[19], args[20], args[21], args[22], 
                args[23], args[24], args[25], args[26] , z, m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Sweep_Shfl4):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sweep_Shfl4: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sweep_Shfl4 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z, m, n, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z, m, n, ITER, time_wo_pci, sizeof(DATA_TYPE)));
    // cout << out[0] << endl;

    hipFree(in_d);
    hipFree(out_d);


    delete[] in;
    delete[] out;
    delete[] out_ref;

}
