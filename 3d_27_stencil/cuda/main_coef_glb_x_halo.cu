#include "hip/hip_runtime.h"
#include <iostream>
#include <metrics.h>
using namespace std;
#define IN_3D(_z,_y,_x)   in[(_z)*(m+2*halo)*(n+2*halo)+(_y)*(n+2*halo)+(_x)]
#define OUT_3D(_z,_y,_x) out[(_z)*(m+2*halo)*(n+2*halo)+(_y)*(n+2*halo)+(_x)]
#define ARG_3D(_l,_w,_x,_y)   args[(_l)]

#define SM_Z (8+2)
#define SM_M (4+2)
#define SM_N (8+2)
#define LOC_3D(_z,_y,_x) local[(_z)*(SM_M)*(SM_N)+(_y)*(SM_N)+(_x)]

#define SM_2D_M (4+2)
#define SM_2D_N (64+2)
#define LOC_2D(_y,_x) local[(_y)*(SM_2D_N)+(_x)]
// #define LOC_2D2(_y,_x) local[(_y)*(SM_2D_N2+2*halo)+(_x)]
#define LOC_L_2D(_z,_y,_x) local[(_z)*(SM_2D_M*SM_2D_N)+(_y)*(SM_2D_N)+(_x)]

// #define DATA_TYPE float
// #define DATA_TYPE double
#define warpSize 32 

// #define __DEBUG

#ifdef __DEBUG
#define ITER 1
#else
#define ITER 100
#endif


// #define TEMP
#define SPAC1

__device__ __inline__ double shfl(double x, int lane)
{
    // Split the double number into 2 32b registers.
    int lo, hi;
    asm volatile( "mov.b32 {%0,%1}, %2;" : "=r"(lo), "=r"(hi) : "d"(x));
    // Shuffle the two 32b registers.
    lo = __shfl(lo, lane);
    hi = __shfl(hi, lane);
    // Recreate the 64b number.
    asm volatile( "mov.b64 %0, {%1,%2};" : "=d"(x) : "r"(lo), "r"(hi));
    return x;
}

void Init_Input_3D(DATA_TYPE *in, int z, int m, int n, int halo, unsigned int seed)
{
    srand(seed);

    for(int k = halo; k < z+halo; k++)
        for(int j = halo; j < m+halo; j++)
            for(int i = halo; i < n+halo; i++)
#ifdef __DEBUG
                IN_3D(k,j,i) = 1; 
                // IN_3D(k,j,i) = (DATA_TYPE)rand()*100.0 / ((long)RAND_MAX);
#else
                IN_3D(k,j,i) = (DATA_TYPE)rand()*10.0 / ((long)RAND_MAX);
#endif
}

void Init_Args_3D(DATA_TYPE *args, int l, int z, int m, int n, int halo, DATA_TYPE val)
{
    for(int k = 0; k < l; k++)
    {
        for(int w = 0; w < z+2*halo; w++)
        {
            for(int i = 0; i < m+2*halo; i++)
            {
                for(int j = 0; j < n+2*halo; j++)
                {
                    ARG_3D(k,w,i,j) = val; 
                }
            }
        }
    }
}

void Clear_Output_3D(DATA_TYPE *in, int z, int m, int n, int halo)
{
    for(int k = 0; k < z+2*halo; k++)
        for(int j = 0; j < m+2*halo; j++)
            for(int i = 0; i < n+2*halo; i++)
                IN_3D(k,j,i) = 0;
}

void Show_Me(DATA_TYPE *in, int z, int m, int n, int halo, std::string prompt)
{
    std::cout << prompt << std::endl;
    for(int k = 0; k < z+2*halo; k++)
    {
        for(int j = 0; j < m+2*halo; j++)
        {
            for(int i = 0; i < n+2*halo; i++)
                std::cout << IN_3D(k,j,i) << ",";
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
}

void Stencil_Seq(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo)
{

    for(int k = halo; k < z+halo; k++)
    {
        for(int j = halo; j < m+halo; j++)
        {
            for(int i = halo; i < n+halo; i++)
            {
                OUT_3D(k,j,i) = ARG_3D(0 ,k,j,i) * IN_3D(k-1,j-1,i-1) +
                                ARG_3D(1 ,k,j,i) * IN_3D(k-1,j-1,i  ) +
                                ARG_3D(2 ,k,j,i) * IN_3D(k-1,j-1,i+1) +
                                ARG_3D(3 ,k,j,i) * IN_3D(k-1,j  ,i-1) +
                                ARG_3D(4 ,k,j,i) * IN_3D(k-1,j  ,i  ) +
                                ARG_3D(5 ,k,j,i) * IN_3D(k-1,j  ,i+1) +
                                ARG_3D(6 ,k,j,i) * IN_3D(k-1,j+1,i-1) + 
                                ARG_3D(7 ,k,j,i) * IN_3D(k-1,j+1,i  ) + 
                                ARG_3D(8 ,k,j,i) * IN_3D(k-1,j+1,i+1) + 
                                ARG_3D(9 ,k,j,i) * IN_3D(k  ,j-1,i-1) + 
                                ARG_3D(10,k,j,i) * IN_3D(k  ,j-1,i  ) + 
                                ARG_3D(11,k,j,i) * IN_3D(k  ,j-1,i+1) + 
                                ARG_3D(12,k,j,i) * IN_3D(k  ,j  ,i-1) + 
                                ARG_3D(13,k,j,i) * IN_3D(k  ,j  ,i  ) + 
                                ARG_3D(14,k,j,i) * IN_3D(k  ,j  ,i+1) + 
                                ARG_3D(15,k,j,i) * IN_3D(k  ,j+1,i-1) + 
                                ARG_3D(16,k,j,i) * IN_3D(k  ,j+1,i  ) + 
                                ARG_3D(17,k,j,i) * IN_3D(k  ,j+1,i+1) + 
                                ARG_3D(18,k,j,i) * IN_3D(k+1,j-1,i-1) + 
                                ARG_3D(19,k,j,i) * IN_3D(k+1,j-1,i  ) + 
                                ARG_3D(20,k,j,i) * IN_3D(k+1,j-1,i+1) + 
                                ARG_3D(21,k,j,i) * IN_3D(k+1,j  ,i-1) + 
                                ARG_3D(22,k,j,i) * IN_3D(k+1,j  ,i  ) + 
                                ARG_3D(23,k,j,i) * IN_3D(k+1,j  ,i+1) + 
                                ARG_3D(24,k,j,i) * IN_3D(k+1,j+1,i-1) + 
                                ARG_3D(25,k,j,i) * IN_3D(k+1,j+1,i  ) + 
                                ARG_3D(26,k,j,i) * IN_3D(k+1,j+1,i+1) ;
            }
        }
    }
    
}

inline double tol_finder(int error_tol)
{
    double val = 1.0;
    for(; error_tol > 0; error_tol--)
        val *= 10;
    return 1.0/(double)val;
}

bool Verify(DATA_TYPE *test, DATA_TYPE *ref, int n)
{
    bool flag = true;
    double precision = tol_finder(2);

    for(int i = 0; i < n; i++)
    {
        if(fabs(test[i]-ref[i]) > precision)
        {
            std::cout << "difference: " << fabs(test[i]-ref[i])-precision << std::endl;
            std::cout << "wrong at " << i << " test:" << test[i] << " (ref: " << ref[i] << ")";
            std::cout << std::endl;
            flag = false;
            break;
        }
    }
    return flag;
}

__global__ void Stencil_Cuda(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = threadIdx.y + blockIdx.y * blockDim.y + halo;
    int k = threadIdx.z + blockIdx.z * blockDim.z + halo;

    OUT_3D(k,j,i) = ARG_3D(0 ,k,j,i) * IN_3D(k-1,j-1,i-1) +
                    ARG_3D(1 ,k,j,i) * IN_3D(k-1,j-1,i  ) +
                    ARG_3D(2 ,k,j,i) * IN_3D(k-1,j-1,i+1) +
                    ARG_3D(3 ,k,j,i) * IN_3D(k-1,j  ,i-1) +
                    ARG_3D(4 ,k,j,i) * IN_3D(k-1,j  ,i  ) +
                    ARG_3D(5 ,k,j,i) * IN_3D(k-1,j  ,i+1) +
                    ARG_3D(6 ,k,j,i) * IN_3D(k-1,j+1,i-1) + 
                    ARG_3D(7 ,k,j,i) * IN_3D(k-1,j+1,i  ) + 
                    ARG_3D(8 ,k,j,i) * IN_3D(k-1,j+1,i+1) + 
                    ARG_3D(9 ,k,j,i) * IN_3D(k  ,j-1,i-1) + 
                    ARG_3D(10,k,j,i) * IN_3D(k  ,j-1,i  ) + 
                    ARG_3D(11,k,j,i) * IN_3D(k  ,j-1,i+1) + 
                    ARG_3D(12,k,j,i) * IN_3D(k  ,j  ,i-1) + 
                    ARG_3D(13,k,j,i) * IN_3D(k  ,j  ,i  ) + 
                    ARG_3D(14,k,j,i) * IN_3D(k  ,j  ,i+1) + 
                    ARG_3D(15,k,j,i) * IN_3D(k  ,j+1,i-1) + 
                    ARG_3D(16,k,j,i) * IN_3D(k  ,j+1,i  ) + 
                    ARG_3D(17,k,j,i) * IN_3D(k  ,j+1,i+1) + 
                    ARG_3D(18,k,j,i) * IN_3D(k+1,j-1,i-1) + 
                    ARG_3D(19,k,j,i) * IN_3D(k+1,j-1,i  ) + 
                    ARG_3D(20,k,j,i) * IN_3D(k+1,j-1,i+1) + 
                    ARG_3D(21,k,j,i) * IN_3D(k+1,j  ,i-1) + 
                    ARG_3D(22,k,j,i) * IN_3D(k+1,j  ,i  ) + 
                    ARG_3D(23,k,j,i) * IN_3D(k+1,j  ,i+1) + 
                    ARG_3D(24,k,j,i) * IN_3D(k+1,j+1,i-1) + 
                    ARG_3D(25,k,j,i) * IN_3D(k+1,j+1,i  ) + 
                    ARG_3D(26,k,j,i) * IN_3D(k+1,j+1,i+1) ;
}

__global__ void Stencil_Cuda_Sweep(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = threadIdx.y + blockIdx.y * blockDim.y + halo;

    const int block_z = z / gridDim.z;
    int k = block_z * blockIdx.z + halo;
    const int k_end = k + block_z;

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        OUT_3D(k,j,i) = ARG_3D(0 ,k,j,i) * IN_3D(k-1,j-1,i-1) +
                        ARG_3D(1 ,k,j,i) * IN_3D(k-1,j-1,i  ) +
                        ARG_3D(2 ,k,j,i) * IN_3D(k-1,j-1,i+1) +
                        ARG_3D(3 ,k,j,i) * IN_3D(k-1,j  ,i-1) +
                        ARG_3D(4 ,k,j,i) * IN_3D(k-1,j  ,i  ) +
                        ARG_3D(5 ,k,j,i) * IN_3D(k-1,j  ,i+1) +
                        ARG_3D(6 ,k,j,i) * IN_3D(k-1,j+1,i-1) + 
                        ARG_3D(7 ,k,j,i) * IN_3D(k-1,j+1,i  ) + 
                        ARG_3D(8 ,k,j,i) * IN_3D(k-1,j+1,i+1) + 
                        ARG_3D(9 ,k,j,i) * IN_3D(k  ,j-1,i-1) + 
                        ARG_3D(10,k,j,i) * IN_3D(k  ,j-1,i  ) + 
                        ARG_3D(11,k,j,i) * IN_3D(k  ,j-1,i+1) + 
                        ARG_3D(12,k,j,i) * IN_3D(k  ,j  ,i-1) + 
                        ARG_3D(13,k,j,i) * IN_3D(k  ,j  ,i  ) + 
                        ARG_3D(14,k,j,i) * IN_3D(k  ,j  ,i+1) + 
                        ARG_3D(15,k,j,i) * IN_3D(k  ,j+1,i-1) + 
                        ARG_3D(16,k,j,i) * IN_3D(k  ,j+1,i  ) + 
                        ARG_3D(17,k,j,i) * IN_3D(k  ,j+1,i+1) + 
                        ARG_3D(18,k,j,i) * IN_3D(k+1,j-1,i-1) + 
                        ARG_3D(19,k,j,i) * IN_3D(k+1,j-1,i  ) + 
                        ARG_3D(20,k,j,i) * IN_3D(k+1,j-1,i+1) + 
                        ARG_3D(21,k,j,i) * IN_3D(k+1,j  ,i-1) + 
                        ARG_3D(22,k,j,i) * IN_3D(k+1,j  ,i  ) + 
                        ARG_3D(23,k,j,i) * IN_3D(k+1,j  ,i+1) + 
                        ARG_3D(24,k,j,i) * IN_3D(k+1,j+1,i-1) + 
                        ARG_3D(25,k,j,i) * IN_3D(k+1,j+1,i  ) + 
                        ARG_3D(26,k,j,i) * IN_3D(k+1,j+1,i+1) ;
    }
}

__global__ void Stencil_Cuda_Sweep_Sm(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = threadIdx.y + blockIdx.y * blockDim.y + halo;
    const int block_z = z / gridDim.z;
    int k = block_z * blockIdx.z + halo;
    const int k_end = k + block_z;
    int li = threadIdx.x + 1;
    int lj = threadIdx.y + 1;

    extern __shared__ DATA_TYPE local[];

    int t1, t2, t3;
    t3 = 2; t2 = 1;
    LOC_L_2D(t3,lj,li) = IN_3D(k,j,i);
    LOC_L_2D(t2,lj,li) = IN_3D(k-1,j,i);
    if(li == halo)                                   
    {
        LOC_L_2D(t3,lj  ,li-1) = IN_3D(k  ,j  ,i-1); 
        LOC_L_2D(t2,lj  ,li-1) = IN_3D(k-1,j  ,i-1);
    }
    if(li == SM_2D_N-2)                         
    {
        LOC_L_2D(t3,lj  ,li+1) = IN_3D(k  ,j  ,i+1); 
        LOC_L_2D(t2,lj  ,li+1) = IN_3D(k-1,j  ,i+1);
    }
    if(lj == halo)                                   
    {
        LOC_L_2D(t3,lj-1,li  ) = IN_3D(k  ,j-1,i  ); 
        LOC_L_2D(t2,lj-1,li  ) = IN_3D(k-1,j-1,i  );
    }
    if(lj == SM_2D_M-2)                         
    {
        LOC_L_2D(t3,lj+1,li  ) = IN_3D(k  ,j+1,i  ); 
        LOC_L_2D(t2,lj+1,li  ) = IN_3D(k-1,j+1,i  );
    }
    if(li == halo && lj == halo)                     
    {
        LOC_L_2D(t3,lj-1,li-1) = IN_3D(k  ,j-1,i-1); 
        LOC_L_2D(t2,lj-1,li-1) = IN_3D(k-1,j-1,i-1);
    }
    if(li == SM_2D_N-2 && lj == halo)           
    {
        LOC_L_2D(t3,lj-1,li+1) = IN_3D(k  ,j-1,i+1); 
        LOC_L_2D(t2,lj-1,li+1) = IN_3D(k-1,j-1,i+1);
    }
    if(li == halo && lj == SM_2D_M-2)           
    { 
        LOC_L_2D(t3,lj+1,li-1) = IN_3D(k  ,j+1,i-1); 
        LOC_L_2D(t2,lj+1,li-1) = IN_3D(k-1,j+1,i-1);
    }
    if(li == SM_2D_N-2 && lj == SM_2D_M-2) 
    {
        LOC_L_2D(t3,lj+1,li+1) = IN_3D(k  ,j+1,i+1); 
        LOC_L_2D(t2,lj+1,li+1) = IN_3D(k-1,j+1,i+1);
    }

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        t1 = t2;
        t2 = t3;
        t3 = (t3+1)%3;
        LOC_L_2D(t3,lj,li) = IN_3D(k+1,j,i);
        if(li == halo)                         {LOC_L_2D(t3,lj  ,li-1) = IN_3D(k+1,j  ,i-1);}
        if(li == SM_2D_N-2)                    {LOC_L_2D(t3,lj  ,li+1) = IN_3D(k+1,j  ,i+1);}
        if(lj == halo)                         {LOC_L_2D(t3,lj-1,li  ) = IN_3D(k+1,j-1,i  );}
        if(lj == SM_2D_M-2)                    {LOC_L_2D(t3,lj+1,li  ) = IN_3D(k+1,j+1,i  );}
        if(li == halo && lj == halo)           {LOC_L_2D(t3,lj-1,li-1) = IN_3D(k+1,j-1,i-1);}
        if(li == SM_2D_N-2 && lj == halo)      {LOC_L_2D(t3,lj-1,li+1) = IN_3D(k+1,j-1,i+1);}
        if(li == halo && lj == SM_2D_M-2)      {LOC_L_2D(t3,lj+1,li-1) = IN_3D(k+1,j+1,i-1);}
        if(li == SM_2D_N-2 && lj == SM_2D_M-2) {LOC_L_2D(t3,lj+1,li+1) = IN_3D(k+1,j+1,i+1);}
        __syncthreads();

        OUT_3D(k,j,i) = ARG_3D(0 ,k,j,i) * LOC_L_2D(t1,lj-1,li-1) +
                        ARG_3D(1 ,k,j,i) * LOC_L_2D(t1,lj-1,li  ) +
                        ARG_3D(2 ,k,j,i) * LOC_L_2D(t1,lj-1,li+1) +
                        ARG_3D(3 ,k,j,i) * LOC_L_2D(t1,lj  ,li-1) +
                        ARG_3D(4 ,k,j,i) * LOC_L_2D(t1,lj  ,li  ) +
                        ARG_3D(5 ,k,j,i) * LOC_L_2D(t1,lj  ,li+1) +
                        ARG_3D(6 ,k,j,i) * LOC_L_2D(t1,lj+1,li-1) + 
                        ARG_3D(7 ,k,j,i) * LOC_L_2D(t1,lj+1,li  ) + 
                        ARG_3D(8 ,k,j,i) * LOC_L_2D(t1,lj+1,li+1) + 
                        ARG_3D(9 ,k,j,i) * LOC_L_2D(t2,lj-1,li-1) + 
                        ARG_3D(10,k,j,i) * LOC_L_2D(t2,lj-1,li  ) + 
                        ARG_3D(11,k,j,i) * LOC_L_2D(t2,lj-1,li+1) + 
                        ARG_3D(12,k,j,i) * LOC_L_2D(t2,lj  ,li-1) + 
                        ARG_3D(13,k,j,i) * LOC_L_2D(t2,lj  ,li  ) + 
                        ARG_3D(14,k,j,i) * LOC_L_2D(t2,lj  ,li+1) + 
                        ARG_3D(15,k,j,i) * LOC_L_2D(t2,lj+1,li-1) + 
                        ARG_3D(16,k,j,i) * LOC_L_2D(t2,lj+1,li  ) + 
                        ARG_3D(17,k,j,i) * LOC_L_2D(t2,lj+1,li+1) + 
                        ARG_3D(18,k,j,i) * LOC_L_2D(t3,lj-1,li-1) + 
                        ARG_3D(19,k,j,i) * LOC_L_2D(t3,lj-1,li  ) + 
                        ARG_3D(20,k,j,i) * LOC_L_2D(t3,lj-1,li+1) + 
                        ARG_3D(21,k,j,i) * LOC_L_2D(t3,lj  ,li-1) + 
                        ARG_3D(22,k,j,i) * LOC_L_2D(t3,lj  ,li  ) + 
                        ARG_3D(23,k,j,i) * LOC_L_2D(t3,lj  ,li+1) + 
                        ARG_3D(24,k,j,i) * LOC_L_2D(t3,lj+1,li-1) + 
                        ARG_3D(25,k,j,i) * LOC_L_2D(t3,lj+1,li  ) + 
                        ARG_3D(26,k,j,i) * LOC_L_2D(t3,lj+1,li+1) ;
        
        __syncthreads();
    }

}

__global__ void Stencil_Cuda_Sm(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = threadIdx.y + blockIdx.y * blockDim.y + halo;
    int k = threadIdx.z + blockIdx.z * blockDim.z + halo;

    int li = threadIdx.x + 1;
    int lj = threadIdx.y + 1;
    int lk = threadIdx.z + 1;

    __shared__ DATA_TYPE local[SM_Z*SM_M*SM_N];

    LOC_3D(lk,lj,li) = IN_3D(k,j,i);

    if(li == halo  ) LOC_3D(lk,lj,li-1) = IN_3D(k,j,i-1);
    if(li == SM_N-2) LOC_3D(lk,lj,li+1) = IN_3D(k,j,i+1);
    if(lj == halo  ) LOC_3D(lk,lj-1,li) = IN_3D(k,j-1,i);
    if(lj == SM_M-2) LOC_3D(lk,lj+1,li) = IN_3D(k,j+1,i);
    if(lk == halo  ) LOC_3D(lk-1,lj,li) = IN_3D(k-1,j,i);
    if(lk == SM_Z-2) LOC_3D(lk+1,lj,li) = IN_3D(k+1,j,i);
    if(li == halo   && lj == halo  ) LOC_3D(lk  ,lj-1,li-1) = IN_3D(k  ,j-1,i-1);
    if(li == halo   && lk == halo  ) LOC_3D(lk-1,lj  ,li-1) = IN_3D(k-1,j  ,i-1);
    if(lj == halo   && lk == halo  ) LOC_3D(lk-1,lj-1,li  ) = IN_3D(k-1,j-1,i  );
    if(li == SM_N-2 && lj == halo  ) LOC_3D(lk  ,lj-1,li+1) = IN_3D(k  ,j-1,i+1);
    if(li == SM_N-2 && lk == halo  ) LOC_3D(lk-1,lj  ,li+1) = IN_3D(k-1,j  ,i+1);
    if(lj == SM_M-2 && lk == halo  ) LOC_3D(lk-1,lj+1,li  ) = IN_3D(k-1,j+1,i  );
    if(li == halo   && lj == SM_M-2) LOC_3D(lk  ,lj+1,li-1) = IN_3D(k  ,j+1,i-1);
    if(li == halo   && lk == SM_Z-2) LOC_3D(lk+1,lj  ,li-1) = IN_3D(k+1,j  ,i-1);
    if(lj == halo   && lk == SM_Z-2) LOC_3D(lk+1,lj-1,li  ) = IN_3D(k+1,j-1,i  );
    if(li == SM_N-2 && lj == SM_M-2) LOC_3D(lk  ,lj+1,li+1) = IN_3D(k  ,j+1,i+1);
    if(li == SM_N-2 && lk == SM_Z-2) LOC_3D(lk+1,lj  ,li+1) = IN_3D(k+1,j  ,i+1);
    if(lj == SM_M-2 && lk == SM_Z-2) LOC_3D(lk+1,lj+1,li  ) = IN_3D(k+1,j+1,i  );
    if(li == halo   && lj == halo   && lk == halo  ) LOC_3D(lk-1,lj-1,li-1) = IN_3D(k-1,j-1,i-1);
    if(li == halo   && lj == halo   && lk == SM_Z-2) LOC_3D(lk+1,lj-1,li-1) = IN_3D(k+1,j-1,i-1);
    if(li == halo   && lj == SM_M-2 && lk == halo  ) LOC_3D(lk-1,lj+1,li-1) = IN_3D(k-1,j+1,i-1);
    if(li == halo   && lj == SM_M-2 && lk == SM_Z-2) LOC_3D(lk+1,lj+1,li-1) = IN_3D(k+1,j+1,i-1);
    if(li == SM_N-2 && lj == halo   && lk == halo  ) LOC_3D(lk-1,lj-1,li+1) = IN_3D(k-1,j-1,i+1);
    if(li == SM_N-2 && lj == halo   && lk == SM_Z-2) LOC_3D(lk+1,lj-1,li+1) = IN_3D(k+1,j-1,i+1);
    if(li == SM_N-2 && lj == SM_M-2 && lk == halo  ) LOC_3D(lk-1,lj+1,li+1) = IN_3D(k-1,j+1,i+1);
    if(li == SM_N-2 && lj == SM_M-2 && lk == SM_Z-2) LOC_3D(lk+1,lj+1,li+1) = IN_3D(k+1,j+1,i+1);

    __syncthreads();


    OUT_3D(k,j,i) = ARG_3D(0 ,k,j,i) * LOC_3D(lk-1,lj-1,li-1) +
                    ARG_3D(1 ,k,j,i) * LOC_3D(lk-1,lj-1,li  ) +
                    ARG_3D(2 ,k,j,i) * LOC_3D(lk-1,lj-1,li+1) +
                    ARG_3D(3 ,k,j,i) * LOC_3D(lk-1,lj  ,li-1) +
                    ARG_3D(4 ,k,j,i) * LOC_3D(lk-1,lj  ,li  ) +
                    ARG_3D(5 ,k,j,i) * LOC_3D(lk-1,lj  ,li+1) +
                    ARG_3D(6 ,k,j,i) * LOC_3D(lk-1,lj+1,li-1) + 
                    ARG_3D(7 ,k,j,i) * LOC_3D(lk-1,lj+1,li  ) + 
                    ARG_3D(8 ,k,j,i) * LOC_3D(lk-1,lj+1,li+1) + 
                    ARG_3D(9 ,k,j,i) * LOC_3D(lk  ,lj-1,li-1) + 
                    ARG_3D(10,k,j,i) * LOC_3D(lk  ,lj-1,li  ) + 
                    ARG_3D(11,k,j,i) * LOC_3D(lk  ,lj-1,li+1) + 
                    ARG_3D(12,k,j,i) * LOC_3D(lk  ,lj  ,li-1) + 
                    ARG_3D(13,k,j,i) * LOC_3D(lk  ,lj  ,li  ) + 
                    ARG_3D(14,k,j,i) * LOC_3D(lk  ,lj  ,li+1) + 
                    ARG_3D(15,k,j,i) * LOC_3D(lk  ,lj+1,li-1) + 
                    ARG_3D(16,k,j,i) * LOC_3D(lk  ,lj+1,li  ) + 
                    ARG_3D(17,k,j,i) * LOC_3D(lk  ,lj+1,li+1) + 
                    ARG_3D(18,k,j,i) * LOC_3D(lk+1,lj-1,li-1) + 
                    ARG_3D(19,k,j,i) * LOC_3D(lk+1,lj-1,li  ) + 
                    ARG_3D(20,k,j,i) * LOC_3D(lk+1,lj-1,li+1) + 
                    ARG_3D(21,k,j,i) * LOC_3D(lk+1,lj  ,li-1) + 
                    ARG_3D(22,k,j,i) * LOC_3D(lk+1,lj  ,li  ) + 
                    ARG_3D(23,k,j,i) * LOC_3D(lk+1,lj  ,li+1) + 
                    ARG_3D(24,k,j,i) * LOC_3D(lk+1,lj+1,li-1) + 
                    ARG_3D(25,k,j,i) * LOC_3D(lk+1,lj+1,li  ) + 
                    ARG_3D(26,k,j,i) * LOC_3D(lk+1,lj+1,li+1) ;
}

__global__ void Stencil_Cuda_Sm2(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = threadIdx.y + blockIdx.y * blockDim.y + halo;
    int k = (((threadIdx.z + blockIdx.z * blockDim.z)>>3)<<4) + threadIdx.z + halo;

    int li = threadIdx.x + 1;
    int lj = threadIdx.y + 1;
    int lk = threadIdx.z + 1;

    __shared__ DATA_TYPE local[SM_Z*SM_M*SM_N];

    LOC_3D(lk,lj,li) = IN_3D(k,j,i);

    if(li == halo  ) LOC_3D(lk,lj,li-1) = IN_3D(k,j,i-1);
    if(li == SM_N-2) LOC_3D(lk,lj,li+1) = IN_3D(k,j,i+1);
    if(lj == halo  ) LOC_3D(lk,lj-1,li) = IN_3D(k,j-1,i);
    if(lj == SM_M-2) LOC_3D(lk,lj+1,li) = IN_3D(k,j+1,i);
    if(lk == halo  ) LOC_3D(lk-1,lj,li) = IN_3D(k-1,j,i);
    if(lk == SM_Z-2) LOC_3D(lk+1,lj,li) = IN_3D(k+1,j,i);
    if(li == halo   && lj == halo  ) LOC_3D(lk  ,lj-1,li-1) = IN_3D(k  ,j-1,i-1);
    if(li == halo   && lk == halo  ) LOC_3D(lk-1,lj  ,li-1) = IN_3D(k-1,j  ,i-1);
    if(lj == halo   && lk == halo  ) LOC_3D(lk-1,lj-1,li  ) = IN_3D(k-1,j-1,i  );
    if(li == SM_N-2 && lj == halo  ) LOC_3D(lk  ,lj-1,li+1) = IN_3D(k  ,j-1,i+1);
    if(li == SM_N-2 && lk == halo  ) LOC_3D(lk-1,lj  ,li+1) = IN_3D(k-1,j  ,i+1);
    if(lj == SM_M-2 && lk == halo  ) LOC_3D(lk-1,lj+1,li  ) = IN_3D(k-1,j+1,i  );
    if(li == halo   && lj == SM_M-2) LOC_3D(lk  ,lj+1,li-1) = IN_3D(k  ,j+1,i-1);
    if(li == halo   && lk == SM_Z-2) LOC_3D(lk+1,lj  ,li-1) = IN_3D(k+1,j  ,i-1);
    if(lj == halo   && lk == SM_Z-2) LOC_3D(lk+1,lj-1,li  ) = IN_3D(k+1,j-1,i  );
    if(li == SM_N-2 && lj == SM_M-2) LOC_3D(lk  ,lj+1,li+1) = IN_3D(k  ,j+1,i+1);
    if(li == SM_N-2 && lk == SM_Z-2) LOC_3D(lk+1,lj  ,li+1) = IN_3D(k+1,j  ,i+1);
    if(lj == SM_M-2 && lk == SM_Z-2) LOC_3D(lk+1,lj+1,li  ) = IN_3D(k+1,j+1,i  );
    if(li == halo   && lj == halo   && lk == halo  ) LOC_3D(lk-1,lj-1,li-1) = IN_3D(k-1,j-1,i-1);
    if(li == halo   && lj == halo   && lk == SM_Z-2) LOC_3D(lk+1,lj-1,li-1) = IN_3D(k+1,j-1,i-1);
    if(li == halo   && lj == SM_M-2 && lk == halo  ) LOC_3D(lk-1,lj+1,li-1) = IN_3D(k-1,j+1,i-1);
    if(li == halo   && lj == SM_M-2 && lk == SM_Z-2) LOC_3D(lk+1,lj+1,li-1) = IN_3D(k+1,j+1,i-1);
    if(li == SM_N-2 && lj == halo   && lk == halo  ) LOC_3D(lk-1,lj-1,li+1) = IN_3D(k-1,j-1,i+1);
    if(li == SM_N-2 && lj == halo   && lk == SM_Z-2) LOC_3D(lk+1,lj-1,li+1) = IN_3D(k+1,j-1,i+1);
    if(li == SM_N-2 && lj == SM_M-2 && lk == halo  ) LOC_3D(lk-1,lj+1,li+1) = IN_3D(k-1,j+1,i+1);
    if(li == SM_N-2 && lj == SM_M-2 && lk == SM_Z-2) LOC_3D(lk+1,lj+1,li+1) = IN_3D(k+1,j+1,i+1);

    __syncthreads();


    OUT_3D(k,j,i) = ARG_3D(0 ,k,j,i) * LOC_3D(lk-1,lj-1,li-1) +
                    ARG_3D(1 ,k,j,i) * LOC_3D(lk-1,lj-1,li  ) +
                    ARG_3D(2 ,k,j,i) * LOC_3D(lk-1,lj-1,li+1) +
                    ARG_3D(3 ,k,j,i) * LOC_3D(lk-1,lj  ,li-1) +
                    ARG_3D(4 ,k,j,i) * LOC_3D(lk-1,lj  ,li  ) +
                    ARG_3D(5 ,k,j,i) * LOC_3D(lk-1,lj  ,li+1) +
                    ARG_3D(6 ,k,j,i) * LOC_3D(lk-1,lj+1,li-1) + 
                    ARG_3D(7 ,k,j,i) * LOC_3D(lk-1,lj+1,li  ) + 
                    ARG_3D(8 ,k,j,i) * LOC_3D(lk-1,lj+1,li+1) + 
                    ARG_3D(9 ,k,j,i) * LOC_3D(lk  ,lj-1,li-1) + 
                    ARG_3D(10,k,j,i) * LOC_3D(lk  ,lj-1,li  ) + 
                    ARG_3D(11,k,j,i) * LOC_3D(lk  ,lj-1,li+1) + 
                    ARG_3D(12,k,j,i) * LOC_3D(lk  ,lj  ,li-1) + 
                    ARG_3D(13,k,j,i) * LOC_3D(lk  ,lj  ,li  ) + 
                    ARG_3D(14,k,j,i) * LOC_3D(lk  ,lj  ,li+1) + 
                    ARG_3D(15,k,j,i) * LOC_3D(lk  ,lj+1,li-1) + 
                    ARG_3D(16,k,j,i) * LOC_3D(lk  ,lj+1,li  ) + 
                    ARG_3D(17,k,j,i) * LOC_3D(lk  ,lj+1,li+1) + 
                    ARG_3D(18,k,j,i) * LOC_3D(lk+1,lj-1,li-1) + 
                    ARG_3D(19,k,j,i) * LOC_3D(lk+1,lj-1,li  ) + 
                    ARG_3D(20,k,j,i) * LOC_3D(lk+1,lj-1,li+1) + 
                    ARG_3D(21,k,j,i) * LOC_3D(lk+1,lj  ,li-1) + 
                    ARG_3D(22,k,j,i) * LOC_3D(lk+1,lj  ,li  ) + 
                    ARG_3D(23,k,j,i) * LOC_3D(lk+1,lj  ,li+1) + 
                    ARG_3D(24,k,j,i) * LOC_3D(lk+1,lj+1,li-1) + 
                    ARG_3D(25,k,j,i) * LOC_3D(lk+1,lj+1,li  ) + 
                    ARG_3D(26,k,j,i) * LOC_3D(lk+1,lj+1,li+1) ;

    __syncthreads();
    k+=8;
    LOC_3D(lk,lj,li) = IN_3D(k,j,i);

    if(li == halo  ) LOC_3D(lk,lj,li-1) = IN_3D(k,j,i-1);
    if(li == SM_N-2) LOC_3D(lk,lj,li+1) = IN_3D(k,j,i+1);
    if(lj == halo  ) LOC_3D(lk,lj-1,li) = IN_3D(k,j-1,i);
    if(lj == SM_M-2) LOC_3D(lk,lj+1,li) = IN_3D(k,j+1,i);
    if(lk == halo  ) LOC_3D(lk-1,lj,li) = IN_3D(k-1,j,i);
    if(lk == SM_Z-2) LOC_3D(lk+1,lj,li) = IN_3D(k+1,j,i);
    if(li == halo   && lj == halo  ) LOC_3D(lk  ,lj-1,li-1) = IN_3D(k  ,j-1,i-1);
    if(li == halo   && lk == halo  ) LOC_3D(lk-1,lj  ,li-1) = IN_3D(k-1,j  ,i-1);
    if(lj == halo   && lk == halo  ) LOC_3D(lk-1,lj-1,li  ) = IN_3D(k-1,j-1,i  );
    if(li == SM_N-2 && lj == halo  ) LOC_3D(lk  ,lj-1,li+1) = IN_3D(k  ,j-1,i+1);
    if(li == SM_N-2 && lk == halo  ) LOC_3D(lk-1,lj  ,li+1) = IN_3D(k-1,j  ,i+1);
    if(lj == SM_M-2 && lk == halo  ) LOC_3D(lk-1,lj+1,li  ) = IN_3D(k-1,j+1,i  );
    if(li == halo   && lj == SM_M-2) LOC_3D(lk  ,lj+1,li-1) = IN_3D(k  ,j+1,i-1);
    if(li == halo   && lk == SM_Z-2) LOC_3D(lk+1,lj  ,li-1) = IN_3D(k+1,j  ,i-1);
    if(lj == halo   && lk == SM_Z-2) LOC_3D(lk+1,lj-1,li  ) = IN_3D(k+1,j-1,i  );
    if(li == SM_N-2 && lj == SM_M-2) LOC_3D(lk  ,lj+1,li+1) = IN_3D(k  ,j+1,i+1);
    if(li == SM_N-2 && lk == SM_Z-2) LOC_3D(lk+1,lj  ,li+1) = IN_3D(k+1,j  ,i+1);
    if(lj == SM_M-2 && lk == SM_Z-2) LOC_3D(lk+1,lj+1,li  ) = IN_3D(k+1,j+1,i  );
    if(li == halo   && lj == halo   && lk == halo  ) LOC_3D(lk-1,lj-1,li-1) = IN_3D(k-1,j-1,i-1);
    if(li == halo   && lj == halo   && lk == SM_Z-2) LOC_3D(lk+1,lj-1,li-1) = IN_3D(k+1,j-1,i-1);
    if(li == halo   && lj == SM_M-2 && lk == halo  ) LOC_3D(lk-1,lj+1,li-1) = IN_3D(k-1,j+1,i-1);
    if(li == halo   && lj == SM_M-2 && lk == SM_Z-2) LOC_3D(lk+1,lj+1,li-1) = IN_3D(k+1,j+1,i-1);
    if(li == SM_N-2 && lj == halo   && lk == halo  ) LOC_3D(lk-1,lj-1,li+1) = IN_3D(k-1,j-1,i+1);
    if(li == SM_N-2 && lj == halo   && lk == SM_Z-2) LOC_3D(lk+1,lj-1,li+1) = IN_3D(k+1,j-1,i+1);
    if(li == SM_N-2 && lj == SM_M-2 && lk == halo  ) LOC_3D(lk-1,lj+1,li+1) = IN_3D(k-1,j+1,i+1);
    if(li == SM_N-2 && lj == SM_M-2 && lk == SM_Z-2) LOC_3D(lk+1,lj+1,li+1) = IN_3D(k+1,j+1,i+1);

    __syncthreads();


    OUT_3D(k,j,i) = ARG_3D(0 ,k,j,i) * LOC_3D(lk-1,lj-1,li-1) +
                    ARG_3D(1 ,k,j,i) * LOC_3D(lk-1,lj-1,li  ) +
                    ARG_3D(2 ,k,j,i) * LOC_3D(lk-1,lj-1,li+1) +
                    ARG_3D(3 ,k,j,i) * LOC_3D(lk-1,lj  ,li-1) +
                    ARG_3D(4 ,k,j,i) * LOC_3D(lk-1,lj  ,li  ) +
                    ARG_3D(5 ,k,j,i) * LOC_3D(lk-1,lj  ,li+1) +
                    ARG_3D(6 ,k,j,i) * LOC_3D(lk-1,lj+1,li-1) + 
                    ARG_3D(7 ,k,j,i) * LOC_3D(lk-1,lj+1,li  ) + 
                    ARG_3D(8 ,k,j,i) * LOC_3D(lk-1,lj+1,li+1) + 
                    ARG_3D(9 ,k,j,i) * LOC_3D(lk  ,lj-1,li-1) + 
                    ARG_3D(10,k,j,i) * LOC_3D(lk  ,lj-1,li  ) + 
                    ARG_3D(11,k,j,i) * LOC_3D(lk  ,lj-1,li+1) + 
                    ARG_3D(12,k,j,i) * LOC_3D(lk  ,lj  ,li-1) + 
                    ARG_3D(13,k,j,i) * LOC_3D(lk  ,lj  ,li  ) + 
                    ARG_3D(14,k,j,i) * LOC_3D(lk  ,lj  ,li+1) + 
                    ARG_3D(15,k,j,i) * LOC_3D(lk  ,lj+1,li-1) + 
                    ARG_3D(16,k,j,i) * LOC_3D(lk  ,lj+1,li  ) + 
                    ARG_3D(17,k,j,i) * LOC_3D(lk  ,lj+1,li+1) + 
                    ARG_3D(18,k,j,i) * LOC_3D(lk+1,lj-1,li-1) + 
                    ARG_3D(19,k,j,i) * LOC_3D(lk+1,lj-1,li  ) + 
                    ARG_3D(20,k,j,i) * LOC_3D(lk+1,lj-1,li+1) + 
                    ARG_3D(21,k,j,i) * LOC_3D(lk+1,lj  ,li-1) + 
                    ARG_3D(22,k,j,i) * LOC_3D(lk+1,lj  ,li  ) + 
                    ARG_3D(23,k,j,i) * LOC_3D(lk+1,lj  ,li+1) + 
                    ARG_3D(24,k,j,i) * LOC_3D(lk+1,lj+1,li-1) + 
                    ARG_3D(25,k,j,i) * LOC_3D(lk+1,lj+1,li  ) + 
                    ARG_3D(26,k,j,i) * LOC_3D(lk+1,lj+1,li+1) ;
}

__global__ void Stencil_Cuda_SmX(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo) 
{
    __shared__ DATA_TYPE local[SM_Z*SM_M*SM_N];
    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = threadIdx.y + blockIdx.y * blockDim.y + halo;
    int k = threadIdx.z + blockIdx.z * blockDim.z + halo;

    int li = threadIdx.x + 1;
    int lj = threadIdx.y + 1;
    int lk = threadIdx.z + 1;

    int lane_id = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;

    int blk_id_x = blockIdx.x; //(threadIdx.x + blockIdx.x * blockDim.x)>>3; 
    int blk_id_y = blockIdx.y; //(threadIdx.y + blockIdx.y * blockDim.y)>>2;
    int blk_id_z = blockIdx.z; //(threadIdx.z + blockIdx.z * blockDim.z)>>3; 

    int new_i = (blk_id_x<<3) + lane_id%10 ;     
    int new_j = (blk_id_y<<2) + (lane_id/10)%6 ; 
    int new_k = (blk_id_z<<3) + lane_id/60 ;     
    int new_li = lane_id%10;
    int new_lj = (lane_id/10)%6;
    int new_lk = lane_id/60;
    LOC_3D(new_lk,new_lj,new_li) = IN_3D(new_k,new_j,new_i);
    new_i = (blk_id_x<<3) + (lane_id+256)%10 ;
    new_j = (blk_id_y<<2) + ((lane_id+256)/10)%6 ;
    new_k = (blk_id_z<<3) + (lane_id+256)/60 ;
    new_li = (lane_id+256)%10;
    new_lj = ((lane_id+256)/10)%6;
    new_lk = (lane_id+256)/60; 
    LOC_3D(new_lk,new_lj,new_li) = IN_3D(new_k,new_j,new_i);
    new_i = (blk_id_x<<3) + (lane_id+512)%10 ;
    new_j = (blk_id_y<<2) + ((lane_id+512)/10)%6 ;
    new_k = (blk_id_z<<3) + (lane_id+512)/60 ;
    new_li = (lane_id+512)%10;
    new_lj = ((lane_id+512)/10)%6;
    new_lk = (lane_id+512)/60; 
    if(new_li < 10 &&  new_lj < 6 && new_lk < 10 )
        LOC_3D(new_lk,new_lj,new_li) = IN_3D(new_k,new_j,new_i);

    __syncthreads();

    OUT_3D(k,j,i) = ARG_3D(0 ,k,j,i) * LOC_3D(lk-1,lj-1,li-1) +
                    ARG_3D(1 ,k,j,i) * LOC_3D(lk-1,lj-1,li  ) +
                    ARG_3D(2 ,k,j,i) * LOC_3D(lk-1,lj-1,li+1) +
                    ARG_3D(3 ,k,j,i) * LOC_3D(lk-1,lj  ,li-1) +
                    ARG_3D(4 ,k,j,i) * LOC_3D(lk-1,lj  ,li  ) +
                    ARG_3D(5 ,k,j,i) * LOC_3D(lk-1,lj  ,li+1) +
                    ARG_3D(6 ,k,j,i) * LOC_3D(lk-1,lj+1,li-1) + 
                    ARG_3D(7 ,k,j,i) * LOC_3D(lk-1,lj+1,li  ) + 
                    ARG_3D(8 ,k,j,i) * LOC_3D(lk-1,lj+1,li+1) + 
                    ARG_3D(9 ,k,j,i) * LOC_3D(lk  ,lj-1,li-1) + 
                    ARG_3D(10,k,j,i) * LOC_3D(lk  ,lj-1,li  ) + 
                    ARG_3D(11,k,j,i) * LOC_3D(lk  ,lj-1,li+1) + 
                    ARG_3D(12,k,j,i) * LOC_3D(lk  ,lj  ,li-1) + 
                    ARG_3D(13,k,j,i) * LOC_3D(lk  ,lj  ,li  ) + 
                    ARG_3D(14,k,j,i) * LOC_3D(lk  ,lj  ,li+1) + 
                    ARG_3D(15,k,j,i) * LOC_3D(lk  ,lj+1,li-1) + 
                    ARG_3D(16,k,j,i) * LOC_3D(lk  ,lj+1,li  ) + 
                    ARG_3D(17,k,j,i) * LOC_3D(lk  ,lj+1,li+1) + 
                    ARG_3D(18,k,j,i) * LOC_3D(lk+1,lj-1,li-1) + 
                    ARG_3D(19,k,j,i) * LOC_3D(lk+1,lj-1,li  ) + 
                    ARG_3D(20,k,j,i) * LOC_3D(lk+1,lj-1,li+1) + 
                    ARG_3D(21,k,j,i) * LOC_3D(lk+1,lj  ,li-1) + 
                    ARG_3D(22,k,j,i) * LOC_3D(lk+1,lj  ,li  ) + 
                    ARG_3D(23,k,j,i) * LOC_3D(lk+1,lj  ,li+1) + 
                    ARG_3D(24,k,j,i) * LOC_3D(lk+1,lj+1,li-1) + 
                    ARG_3D(25,k,j,i) * LOC_3D(lk+1,lj+1,li  ) + 
                    ARG_3D(26,k,j,i) * LOC_3D(lk+1,lj+1,li+1) ;

}


__global__ void Stencil_Cuda_Shfl(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = threadIdx.y + blockIdx.y * blockDim.y + halo;
    int k = threadIdx.z + blockIdx.z * blockDim.z + halo;
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int lane_id = tid % warpSize;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    int warp_id_z = (threadIdx.z + blockIdx.z * blockDim.z)>>0; // there numbers
    int new_i = (warp_id_x<<3) + lane_id%10;     // 10 is extended dimension of i
    int new_j = (warp_id_y<<2) + (lane_id/10)%6; // 6  is extended dimension of j 
    int new_k = (warp_id_z<<0) + lane_id/60;     // 60 is extended area of ixj = 10x6
    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5;
    threadInput0 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10;
    new_j = (warp_id_y<<2) + ((lane_id+32)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+32)/60;
    threadInput1 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10;
    new_j = (warp_id_y<<2) + ((lane_id+64)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+64)/60;
    threadInput2 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10;
    new_j = (warp_id_y<<2) + ((lane_id+96)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+96)/60;
    threadInput3 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10;
    new_j = (warp_id_y<<2) + ((lane_id+128)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+128)/60;
    threadInput4 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10;
    new_j = (warp_id_y<<2) + ((lane_id+160)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+160)/60;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    new_j = (new_j < m+2*halo)? new_j: m+2*halo-1;
    new_k = (new_k < z+2*halo)? new_k: z+2*halo-1;
    threadInput5 = IN_3D(new_k, new_j, new_i);

    DATA_TYPE sum = 0.0;
    int friend_id;
    DATA_TYPE tx, ty, tz;

    friend_id = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += ARG_3D(0,k,j,i)*((lane_id < 26)? tx: ty);

    friend_id = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += ARG_3D(1,k,j,i)*((lane_id < 25)? tx: ty);

    friend_id = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += ARG_3D(2,k,j,i)*((lane_id < 24)? tx: ty);

    friend_id = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += ARG_3D(3,k,j,i)*((lane_id < 18)? tx: ty);

    friend_id = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += ARG_3D(4,k,j,i)*((lane_id < 17)? tx: ty);

    friend_id = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += ARG_3D(5,k,j,i)*((lane_id < 16)? tx: ty);

    friend_id = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += ARG_3D(6,k,j,i)*((lane_id < 10)? tx: ty);

    friend_id = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += ARG_3D(7,k,j,i)*((lane_id < 9 )? tx: ty);

    friend_id = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += ARG_3D(8,k,j,i)*((lane_id < 8 )? tx: ty);

    friend_id = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput1, friend_id);
    ty = __shfl(threadInput2, friend_id);
    tz = __shfl(threadInput3, friend_id);
    sum += ARG_3D(9,k,j,i)*((lane_id < 4 )? tx: ((lane_id < 30)? ty: tz));

    friend_id = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput1, friend_id);
    ty = __shfl(threadInput2, friend_id);
    tz = __shfl(threadInput3, friend_id);
    sum += ARG_3D(10,k,j,i)*((lane_id < 3 )? tx: ((lane_id < 29)? ty: tz));

    friend_id = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput1, friend_id);
    ty = __shfl(threadInput2, friend_id);
    tz = __shfl(threadInput3, friend_id);
    sum += ARG_3D(11,k,j,i)*((lane_id < 2 )? tx: ((lane_id < 28)? ty: tz));

    friend_id = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput2, friend_id);
    ty = __shfl(threadInput3, friend_id);
    sum += ARG_3D(12,k,j,i)*((lane_id < 22)? tx: ty);

    friend_id = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput2, friend_id);
    ty = __shfl(threadInput3, friend_id);
    sum += ARG_3D(13,k,j,i)*((lane_id < 21)? tx: ty);

    friend_id = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput2, friend_id);
    ty = __shfl(threadInput3, friend_id);
    sum += ARG_3D(14,k,j,i)*((lane_id < 20)? tx: ty);

    friend_id = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput2, friend_id);
    ty = __shfl(threadInput3, friend_id);
    sum += ARG_3D(15,k,j,i)*((lane_id < 14)? tx: ty);

    friend_id = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput2, friend_id);
    ty = __shfl(threadInput3, friend_id);
    sum += ARG_3D(16,k,j,i)*((lane_id < 13)? tx: ty);

    friend_id = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput2, friend_id);
    ty = __shfl(threadInput3, friend_id);
    sum += ARG_3D(17,k,j,i)*((lane_id < 12)? tx: ty);

    friend_id = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput3, friend_id);
    ty = __shfl(threadInput4, friend_id);
    sum += ARG_3D(18,k,j,i)*((lane_id < 8 )? tx: ty);

    friend_id = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput3, friend_id);
    ty = __shfl(threadInput4, friend_id);
    sum += ARG_3D(19,k,j,i)*((lane_id < 7 )? tx: ty);

    friend_id = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput3, friend_id);
    ty = __shfl(threadInput4, friend_id);
    sum += ARG_3D(20,k,j,i)*((lane_id < 6 )? tx: ty);

    friend_id = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput4, friend_id);
    ty = __shfl(threadInput5, friend_id);
    sum += ARG_3D(21,k,j,i)*((lane_id < 24)? tx: ty);

    friend_id = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput4, friend_id);
    ty = __shfl(threadInput5, friend_id);
    sum += ARG_3D(22,k,j,i)*((lane_id < 24)? tx: ty);

    friend_id = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput4, friend_id);
    ty = __shfl(threadInput5, friend_id);
    sum += ARG_3D(23,k,j,i)*((lane_id < 24)? tx: ty);

    friend_id = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput4, friend_id);
    ty = __shfl(threadInput5, friend_id);
    sum += ARG_3D(24,k,j,i)*((lane_id < 16)? tx: ty);

    friend_id = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput4, friend_id);
    ty = __shfl(threadInput5, friend_id);
    sum += ARG_3D(25,k,j,i)*((lane_id < 16)? tx: ty);

    friend_id = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput4, friend_id);
    ty = __shfl(threadInput5, friend_id);
    sum += ARG_3D(26,k,j,i)*((lane_id < 16)? tx: ty);

    
    OUT_3D(k,j,i) = sum;
}

__global__ void Stencil_Cuda_Shfl2(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = threadIdx.y + blockIdx.y * blockDim.y + halo;
    int k = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<1) + (lane_id>>5) + halo; 
    // thread coarsening: related to warp dimensions 1x4x8. 
    // We coarsen from dimension z from 2^0 to 2^1, also need to know there are how many values in dimension z

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    int warp_id_z = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<1) + (lane_id>>5); // these numbers
    int new_i = (warp_id_x<<3) + lane_id%10;     // 10 is extended dimension of i
    int new_j = (warp_id_y<<2) + (lane_id/10)%6; // 6  is extended dimension of j 
    int new_k = (warp_id_z<<0) + lane_id/60;     // 60 is extended area of ixj = 10x6
    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5,
              threadInput6, threadInput7;
    threadInput0 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10;
    new_j = (warp_id_y<<2) + ((lane_id+32)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+32)/60;
    threadInput1 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10;
    new_j = (warp_id_y<<2) + ((lane_id+64)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+64)/60;
    threadInput2 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10;
    new_j = (warp_id_y<<2) + ((lane_id+96)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+96)/60;
    threadInput3 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10;
    new_j = (warp_id_y<<2) + ((lane_id+128)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+128)/60;
    threadInput4 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10;
    new_j = (warp_id_y<<2) + ((lane_id+160)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+160)/60;
    threadInput5 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+192)%10;
    new_j = (warp_id_y<<2) + ((lane_id+192)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+192)/60;
    threadInput6 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+224)%10;
    new_j = (warp_id_y<<2) + ((lane_id+224)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+224)/60;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    new_j = (new_j < m+2*halo)? new_j: m+2*halo-1;
    new_k = (new_k < z+2*halo)? new_k: z+2*halo-1;
    threadInput7 = IN_3D(new_k, new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    int friend_id0, friend_id1;
    DATA_TYPE tx0, ty0, tz0, tx1, ty1, tz1;

    friend_id0 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    sum0 += ARG_3D(0,k  ,j,i)*((lane_id < 26)? tx0: ty0);
    sum1 += ARG_3D(0,k+1,j,i)*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));

    friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    sum0 += ARG_3D(1,k  ,j,i)*((lane_id < 25)? tx0: ty0);
    sum1 += ARG_3D(1,k+1,j,i)*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));

    friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    sum0 += ARG_3D(2,k  ,j,i)*((lane_id < 24)? tx0: ty0);
    sum1 += ARG_3D(2,k+1,j,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));

    friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    sum0 += ARG_3D(3,k  ,j,i)*((lane_id < 18)? tx0: ty0);
    sum1 += ARG_3D(3,k+1,j,i)*((lane_id < 22)? tx1: ty1);

    friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    sum0 += ARG_3D(4,k  ,j,i)*((lane_id < 17)? tx0: ty0);
    sum1 += ARG_3D(4,k+1,j,i)*((lane_id < 21)? tx1: ty1);

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    sum0 += ARG_3D(5,k  ,j,i)*((lane_id < 16)? tx0: ty0);
    sum1 += ARG_3D(5,k+1,j,i)*((lane_id < 20)? tx1: ty1);

    friend_id0 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    sum0 += ARG_3D(6,k  ,j,i)*((lane_id < 10)? tx0: ty0);
    sum1 += ARG_3D(6,k+1,j,i)*((lane_id < 14)? tx1: ty1);

    friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    sum0 += ARG_3D(7,k  ,j,i)*((lane_id < 9 )? tx0: ty0);
    sum1 += ARG_3D(7,k+1,j,i)*((lane_id < 13)? tx1: ty1);

    friend_id0 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    sum0 += ARG_3D(8,k  ,j,i)*((lane_id < 8 )? tx0: ty0);
    sum1 += ARG_3D(8,k+1,j,i)*((lane_id < 12)? tx1: ty1);

    friend_id0 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    sum0 += ARG_3D(9,k  ,j,i)*((lane_id < 4 )? tx0: ((lane_id < 30)? ty0: tz0));
    sum1 += ARG_3D(9,k+1,j,i)*((lane_id < 8)? tx1: ty1);

    friend_id0 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    sum0 += ARG_3D(10,k  ,j,i)*((lane_id < 3 )? tx0: ((lane_id < 29)? ty0: tz0));
    sum1 += ARG_3D(10,k+1,j,i)*((lane_id < 7)? tx1: ty1);

    friend_id0 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    sum0 += ARG_3D(11,k  ,j,i)*((lane_id < 2 )? tx0: ((lane_id < 28)? ty0: tz0));
    sum1 += ARG_3D(11,k+1,j,i)*((lane_id < 6)? tx1: ty1);

    friend_id0 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    sum0 += ARG_3D(12,k  ,j,i)*((lane_id < 22)? tx0: ty0);
    sum1 += ARG_3D(12,k+1,j,i)*((lane_id < 24)? tx1: ty1);

    friend_id0 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    sum0 += ARG_3D(13,k  ,j,i)*((lane_id < 21)? tx0: ty0);
    sum1 += ARG_3D(13,k+1,j,i)*((lane_id < 24)? tx1: ty1);

    friend_id0 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    sum0 += ARG_3D(14,k  ,j,i)*((lane_id < 20)? tx0: ty0);
    sum1 += ARG_3D(14,k+1,j,i)*((lane_id < 24)? tx1: ty1);

    friend_id0 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    sum0 += ARG_3D(15,k  ,j,i)*((lane_id < 14)? tx0: ty0);
    sum1 += ARG_3D(15,k+1,j,i)*((lane_id < 16)? tx1: ty1);

    friend_id0 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    sum0 += ARG_3D(16,k  ,j,i)*((lane_id < 13)? tx0: ty0);
    sum1 += ARG_3D(16,k+1,j,i)*((lane_id < 16)? tx1: ty1);

    friend_id0 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    sum0 += ARG_3D(17,k  ,j,i)*((lane_id < 12)? tx0: ty0);
    sum1 += ARG_3D(17,k+1,j,i)*((lane_id < 16)? tx1: ty1);

    friend_id0 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    sum0 += ARG_3D(18,k  ,j,i)*((lane_id < 8 )? tx0: ty0);
    sum1 += ARG_3D(18,k+1,j,i)*((lane_id < 10)? tx1: ty1);

    friend_id0 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    sum0 += ARG_3D(19,k  ,j,i)*((lane_id < 7 )? tx0: ty0);
    sum1 += ARG_3D(19,k+1,j,i)*((lane_id < 9 )? tx1: ty1);

    friend_id0 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    sum0 += ARG_3D(20,k  ,j,i)*((lane_id < 6 )? tx0: ty0);
    sum1 += ARG_3D(20,k+1,j,i)*((lane_id < 8 )? tx1: ty1);

    friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tz1 = __shfl(threadInput7, friend_id1);
    sum0 += ARG_3D(21,k  ,j,i)*((lane_id < 24)? tx0: ty0);
    sum1 += ARG_3D(21,k+1,j,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));

    friend_id0 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tz1 = __shfl(threadInput7, friend_id1);
    sum0 += ARG_3D(22,k  ,j,i)*((lane_id < 24)? tx0: ty0);
    sum1 += ARG_3D(22,k+1,j,i)*((lane_id < 1 )? tx1: ((lane_id < 27)? ty1: tz1));

    friend_id0 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    sum0 += ARG_3D(23,k  ,j,i)*((lane_id < 24)? tx0: ty0);
    sum1 += ARG_3D(23,k+1,j,i)*((lane_id < 26)? tx1: ty1);

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    sum0 += ARG_3D(24,k  ,j,i)*((lane_id < 16)? tx0: ty0);
    sum1 += ARG_3D(24,k+1,j,i)*((lane_id < 20)? tx1: ty1);

    friend_id0 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    sum0 += ARG_3D(25,k  ,j,i)*((lane_id < 16)? tx0: ty0);
    sum1 += ARG_3D(25,k+1,j,i)*((lane_id < 19)? tx1: ty1);

    friend_id0 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    sum0 += ARG_3D(26,k  ,j,i)*((lane_id < 16)? tx0: ty0);
    sum1 += ARG_3D(26,k+1,j,i)*((lane_id < 18)? tx1: ty1);


    OUT_3D(k  ,j,i) = sum0;
    OUT_3D(k+1,j,i) = sum1;

}

__global__ void Stencil_Cuda_Shfl4(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args,
        int z, int m, int n, int halo)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = threadIdx.y + blockIdx.y * blockDim.y + halo;
    int k = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<2) + (lane_id>>5) + halo; 
    // Thread coarsening: related to warp dimensions 1x4x8. 
    // We coarsen from dimension z from 2^0 to 2^2, also need to know there are how many values in dimension z,
    // which is (lane_id>>5) 

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    int warp_id_z = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<2) + (lane_id>>5); // these numbers
    int new_i = (warp_id_x<<3) + lane_id%10;     // 10 is extended dimension of i
    int new_j = (warp_id_y<<2) + (lane_id/10)%6; // 6  is extended dimension of j 
    int new_k = (warp_id_z<<0) + lane_id/60;     // 60 is extended area of ixj = 10x6
    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5,
              threadInput6, threadInput7, threadInput8, threadInput9, threadInput10, threadInput11;
    threadInput0 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10;
    new_j = (warp_id_y<<2) + ((lane_id+32)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+32)/60;
    threadInput1 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10;
    new_j = (warp_id_y<<2) + ((lane_id+64)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+64)/60;
    threadInput2 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10;
    new_j = (warp_id_y<<2) + ((lane_id+96)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+96)/60;
    threadInput3 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10;
    new_j = (warp_id_y<<2) + ((lane_id+128)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+128)/60;
    threadInput4 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10;
    new_j = (warp_id_y<<2) + ((lane_id+160)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+160)/60;
    threadInput5 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+192)%10;
    new_j = (warp_id_y<<2) + ((lane_id+192)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+192)/60;
    threadInput6 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+224)%10;
    new_j = (warp_id_y<<2) + ((lane_id+224)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+224)/60;
    threadInput7 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+256)%10;
    new_j = (warp_id_y<<2) + ((lane_id+256)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+256)/60;
    threadInput8 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+288)%10;
    new_j = (warp_id_y<<2) + ((lane_id+288)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+288)/60;
    threadInput9 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+320)%10;
    new_j = (warp_id_y<<2) + ((lane_id+320)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+320)/60;
    threadInput10 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+352)%10;
    new_j = (warp_id_y<<2) + ((lane_id+352)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+352)/60;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    new_j = (new_j < m+2*halo)? new_j: m+2*halo-1;
    new_k = (new_k < z+2*halo)? new_k: z+2*halo-1;
    threadInput11 = IN_3D(new_k, new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    int friend_id0, friend_id1, friend_id2, friend_id3;
    DATA_TYPE tx0, ty0, tz0, tx1, ty1, tz1, tx2, ty2, tz2, tx3, ty3, tz3;

    friend_id0 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    sum0 += ARG_3D(0,k  ,j,i)*((lane_id < 26)? tx0: ty0);
    sum1 += ARG_3D(0,k+1,j,i)*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
    sum2 += ARG_3D(0,k+2,j,i)*((lane_id < 8 )? tx2: ty2);
    sum3 += ARG_3D(0,k+3,j,i)*((lane_id < 10)? tx3: ty3);

    friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    sum0 += ARG_3D(1,k  ,j,i)*((lane_id < 25)? tx0: ty0);
    sum1 += ARG_3D(1,k+1,j,i)*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
    sum2 += ARG_3D(1,k+2,j,i)*((lane_id < 7 )? tx2: ty2);
    sum3 += ARG_3D(1,k+3,j,i)*((lane_id < 9 )? tx3: ty3);

    friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    sum0 += ARG_3D(2,k  ,j,i)*((lane_id < 24)? tx0: ty0);
    sum1 += ARG_3D(2,k+1,j,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
    sum2 += ARG_3D(2,k+2,j,i)*((lane_id < 6 )? tx2: ty2);
    sum3 += ARG_3D(2,k+3,j,i)*((lane_id < 8 )? tx3: ty3);

    friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    tz3 = __shfl(threadInput7, friend_id3);
    sum0 += ARG_3D(3,k  ,j,i)*((lane_id < 18)? tx0: ty0);
    sum1 += ARG_3D(3,k+1,j,i)*((lane_id < 22)? tx1: ty1);
    sum2 += ARG_3D(3,k+2,j,i)*((lane_id < 24)? tx2: ty2);
    sum3 += ARG_3D(3,k+3,j,i)*((lane_id < 2 )? tx3: ((lane_id < 28)? ty3: tz3));

    friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    tz3 = __shfl(threadInput7, friend_id3);
    sum0 += ARG_3D(4,k  ,j,i)*((lane_id < 17)? tx0: ty0);
    sum1 += ARG_3D(4,k+1,j,i)*((lane_id < 21)? tx1: ty1);
    sum2 += ARG_3D(4,k+2,j,i)*((lane_id < 24)? tx2: ty2);
    sum3 += ARG_3D(4,k+3,j,i)*((lane_id < 1 )? tx3: ((lane_id < 27)? ty3: tz3));

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    sum0 += ARG_3D(5,k  ,j,i)*((lane_id < 16)? tx0: ty0);
    sum1 += ARG_3D(5,k+1,j,i)*((lane_id < 20)? tx1: ty1);
    sum2 += ARG_3D(5,k+2,j,i)*((lane_id < 24)? tx2: ty2);
    sum3 += ARG_3D(5,k+3,j,i)*((lane_id < 26)? tx3: ty3);

    friend_id0 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    sum0 += ARG_3D(6,k  ,j,i)*((lane_id < 10)? tx0: ty0);
    sum1 += ARG_3D(6,k+1,j,i)*((lane_id < 14)? tx1: ty1);
    sum2 += ARG_3D(6,k+2,j,i)*((lane_id < 16)? tx2: ty2);
    sum3 += ARG_3D(6,k+3,j,i)*((lane_id < 20)? tx3: ty3);

    friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    sum0 += ARG_3D(7,k  ,j,i)*((lane_id < 9 )? tx0: ty0);
    sum1 += ARG_3D(7,k+1,j,i)*((lane_id < 13)? tx1: ty1);
    sum2 += ARG_3D(7,k+2,j,i)*((lane_id < 16)? tx2: ty2);
    sum3 += ARG_3D(7,k+3,j,i)*((lane_id < 19)? tx3: ty3);

    friend_id0 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    sum0 += ARG_3D(8,k  ,j,i)*((lane_id < 8 )? tx0: ty0);
    sum1 += ARG_3D(8,k+1,j,i)*((lane_id < 12)? tx1: ty1);
    sum2 += ARG_3D(8,k+2,j,i)*((lane_id < 16)? tx2: ty2);
    sum3 += ARG_3D(8,k+3,j,i)*((lane_id < 18)? tx3: ty3);

    friend_id0 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    sum0 += ARG_3D(9,k  ,j,i)*((lane_id < 4 )? tx0: ((lane_id < 30)? ty0: tz0));
    sum1 += ARG_3D(9,k+1,j,i)*((lane_id < 8)? tx1: ty1);
    sum2 += ARG_3D(9,k+2,j,i)*((lane_id < 10)? tx2: ty2);
    sum3 += ARG_3D(9,k+3,j,i)*((lane_id < 14)? tx3: ty3);

    friend_id0 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    sum0 += ARG_3D(10,k  ,j,i)*((lane_id < 3 )? tx0: ((lane_id < 29)? ty0: tz0));
    sum1 += ARG_3D(10,k+1,j,i)*((lane_id < 7 )? tx1: ty1);
    sum2 += ARG_3D(10,k+2,j,i)*((lane_id < 9 )? tx2: ty2);
    sum3 += ARG_3D(10,k+3,j,i)*((lane_id < 13)? tx3: ty3);

    friend_id0 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    sum0 += ARG_3D(11,k  ,j,i)*((lane_id < 2 )? tx0: ((lane_id < 28)? ty0: tz0));
    sum1 += ARG_3D(11,k+1,j,i)*((lane_id < 6 )? tx1: ty1);
    sum2 += ARG_3D(11,k+2,j,i)*((lane_id < 8 )? tx2: ty2);
    sum3 += ARG_3D(11,k+3,j,i)*((lane_id < 12)? tx3: ty3);

    friend_id0 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tz2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    sum0 += ARG_3D(12,k  ,j,i)*((lane_id < 22)? tx0: ty0);
    sum1 += ARG_3D(12,k+1,j,i)*((lane_id < 24)? tx1: ty1);
    sum2 += ARG_3D(12,k+2,j,i)*((lane_id < 2 )? tx2: ((lane_id < 28)? ty2: tz2));
    sum3 += ARG_3D(12,k+3,j,i)*((lane_id < 6 )? tx3: ty3);

    friend_id0 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tz2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    tz3 = __shfl(threadInput9, friend_id3);
    sum0 += ARG_3D(13,k  ,j,i)*((lane_id < 21)? tx0: ty0);
    sum1 += ARG_3D(13,k+1,j,i)*((lane_id < 24)? tx1: ty1);
    sum2 += ARG_3D(13,k+2,j,i)*((lane_id < 1 )? tx2: ((lane_id < 27)? ty2: tz2));
    sum3 += ARG_3D(13,k+3,j,i)*((lane_id < 5 )? tx3: ((lane_id < 31)? ty3: tz3));

    friend_id0 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    tz3 = __shfl(threadInput9, friend_id3);
    sum0 += ARG_3D(14,k  ,j,i)*((lane_id < 20)? tx0: ty0);
    sum1 += ARG_3D(14,k+1,j,i)*((lane_id < 24)? tx1: ty1);
    sum2 += ARG_3D(14,k+2,j,i)*((lane_id < 26)? tx2: ty2);
    sum3 += ARG_3D(14,k+3,j,i)*((lane_id < 4 )? tx3: ((lane_id < 30)? ty3: tz3));

    friend_id0 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput8, friend_id3);
    ty3 = __shfl(threadInput9, friend_id3);
    sum0 += ARG_3D(15,k  ,j,i)*((lane_id < 14)? tx0: ty0);
    sum1 += ARG_3D(15,k+1,j,i)*((lane_id < 16)? tx1: ty1);
    sum2 += ARG_3D(15,k+2,j,i)*((lane_id < 20)? tx2: ty2);
    sum3 += ARG_3D(15,k+3,j,i)*((lane_id < 24)? tx3: ty3);

    friend_id0 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput8, friend_id3);
    ty3 = __shfl(threadInput9, friend_id3);
    sum0 += ARG_3D(16,k  ,j,i)*((lane_id < 13)? tx0: ty0);
    sum1 += ARG_3D(16,k+1,j,i)*((lane_id < 16)? tx1: ty1);
    sum2 += ARG_3D(16,k+2,j,i)*((lane_id < 19)? tx2: ty2);
    sum3 += ARG_3D(16,k+3,j,i)*((lane_id < 23)? tx3: ty3);

    friend_id0 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput8, friend_id3);
    ty3 = __shfl(threadInput9, friend_id3);
    sum0 += ARG_3D(17,k  ,j,i)*((lane_id < 12)? tx0: ty0);
    sum1 += ARG_3D(17,k+1,j,i)*((lane_id < 16)? tx1: ty1);
    sum2 += ARG_3D(17,k+2,j,i)*((lane_id < 18)? tx2: ty2);
    sum3 += ARG_3D(17,k+3,j,i)*((lane_id < 22)? tx3: ty3);

    friend_id0 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    sum0 += ARG_3D(18,k  ,j,i)*((lane_id < 8 )? tx0: ty0);
    sum1 += ARG_3D(18,k+1,j,i)*((lane_id < 10)? tx1: ty1);
    sum2 += ARG_3D(18,k+2,j,i)*((lane_id < 14)? tx2: ty2);
    sum3 += ARG_3D(18,k+3,j,i)*((lane_id < 16)? tx3: ty3);

    friend_id0 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    sum0 += ARG_3D(19,k  ,j,i)*((lane_id < 7 )? tx0: ty0);
    sum1 += ARG_3D(19,k+1,j,i)*((lane_id < 9 )? tx1: ty1);
    sum2 += ARG_3D(19,k+2,j,i)*((lane_id < 13)? tx2: ty2);
    sum3 += ARG_3D(19,k+3,j,i)*((lane_id < 16)? tx3: ty3);

    friend_id0 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    sum0 += ARG_3D(20,k  ,j,i)*((lane_id < 6 )? tx0: ty0);
    sum1 += ARG_3D(20,k+1,j,i)*((lane_id < 8 )? tx1: ty1);
    sum2 += ARG_3D(20,k+2,j,i)*((lane_id < 12)? tx2: ty2);
    sum3 += ARG_3D(20,k+3,j,i)*((lane_id < 16)? tx3: ty3);

    friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tz1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    sum0 += ARG_3D(21,k  ,j,i)*((lane_id < 24)? tx0: ty0);
    sum1 += ARG_3D(21,k+1,j,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
    sum2 += ARG_3D(21,k+2,j,i)*((lane_id < 6 )? tx2: ty2);
    sum3 += ARG_3D(21,k+3,j,i)*((lane_id < 8 )? tx3: ty3);

    friend_id0 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+23+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tz1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tz2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    sum0 += ARG_3D(22,k  ,j,i)*((lane_id < 24)? tx0: ty0);
    sum1 += ARG_3D(22,k+1,j,i)*((lane_id < 1 )? tx1: ((lane_id < 27)? ty1: tz1));
    sum2 += ARG_3D(22,k+2,j,i)*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
    sum3 += ARG_3D(22,k+3,j,i)*((lane_id < 8 )? tx3: ty3);

    friend_id0 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tz2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    sum0 += ARG_3D(23,k  ,j,i)*((lane_id < 24)? tx0: ty0);
    sum1 += ARG_3D(23,k+1,j,i)*((lane_id < 26)? tx1: ty1);
    sum2 += ARG_3D(23,k+2,j,i)*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
    sum3 += ARG_3D(23,k+3,j,i)*((lane_id < 8 )? tx3: ty3);

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput8, friend_id2);
    ty2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput10, friend_id3);
    ty3 = __shfl(threadInput11, friend_id3);
    sum0 += ARG_3D(24,k  ,j,i)*((lane_id < 16)? tx0: ty0);
    sum1 += ARG_3D(24,k+1,j,i)*((lane_id < 20)? tx1: ty1);
    sum2 += ARG_3D(24,k+2,j,i)*((lane_id < 24)? tx2: ty2);
    sum3 += ARG_3D(24,k+3,j,i)*((lane_id < 26)? tx3: ty3);

    friend_id0 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput8, friend_id2);
    ty2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput10, friend_id3);
    ty3 = __shfl(threadInput11, friend_id3);
    sum0 += ARG_3D(25,k  ,j,i)*((lane_id < 16)? tx0: ty0);
    sum1 += ARG_3D(25,k+1,j,i)*((lane_id < 19)? tx1: ty1);
    sum2 += ARG_3D(25,k+2,j,i)*((lane_id < 23)? tx2: ty2);
    sum3 += ARG_3D(25,k+3,j,i)*((lane_id < 25)? tx3: ty3);

    friend_id0 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput8, friend_id2);
    ty2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput10, friend_id3);
    ty3 = __shfl(threadInput11, friend_id3);
    sum0 += ARG_3D(26,k  ,j,i)*((lane_id < 16)? tx0: ty0);
    sum1 += ARG_3D(26,k+1,j,i)*((lane_id < 18)? tx1: ty1);
    sum2 += ARG_3D(26,k+2,j,i)*((lane_id < 22)? tx2: ty2);
    sum3 += ARG_3D(26,k+3,j,i)*((lane_id < 24)? tx3: ty3);


    OUT_3D(k  ,j,i) = sum0;
    OUT_3D(k+1,j,i) = sum1;
    OUT_3D(k+2,j,i) = sum2;
    OUT_3D(k+3,j,i) = sum3;
}

__global__ void Stencil_Cuda_Shfl8(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = threadIdx.y + blockIdx.y * blockDim.y + halo;
    int k = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<3) + (lane_id>>5) + halo; 
    // Thread coarsening: related to warp dimensions 1x4x8. 
    // We coarsen from dimension z from 2^0 to 2^3, also need to know there are how many values in dimension z,
    // which is (lane_id>>5) 

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    int warp_id_z = (((threadIdx.z + blockIdx.z * blockDim.z)>>0)<<3) + (lane_id>>5); // these numbers
    int new_i = (warp_id_x<<3) + lane_id%10;     // 10 is extended dimension of i
    int new_j = (warp_id_y<<2) + (lane_id/10)%6; // 6  is extended dimension of j 
    int new_k = (warp_id_z<<0) + lane_id/60;     // 60 is extended area of ixj = 10x6
    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5,
              threadInput6, threadInput7, threadInput8, threadInput9, threadInput10, threadInput11,
              threadInput12, threadInput13, threadInput14, threadInput15, threadInput16, threadInput17,
              threadInput18;
    threadInput0 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10;
    new_j = (warp_id_y<<2) + ((lane_id+32)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+32)/60;
    threadInput1 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10;
    new_j = (warp_id_y<<2) + ((lane_id+64)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+64)/60;
    threadInput2 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10;
    new_j = (warp_id_y<<2) + ((lane_id+96)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+96)/60;
    threadInput3 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10;
    new_j = (warp_id_y<<2) + ((lane_id+128)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+128)/60;
    threadInput4 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10;
    new_j = (warp_id_y<<2) + ((lane_id+160)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+160)/60;
    threadInput5 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+192)%10;
    new_j = (warp_id_y<<2) + ((lane_id+192)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+192)/60;
    threadInput6 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+224)%10;
    new_j = (warp_id_y<<2) + ((lane_id+224)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+224)/60;
    threadInput7 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+256)%10;
    new_j = (warp_id_y<<2) + ((lane_id+256)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+256)/60;
    threadInput8 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+288)%10;
    new_j = (warp_id_y<<2) + ((lane_id+288)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+288)/60;
    threadInput9 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+320)%10;
    new_j = (warp_id_y<<2) + ((lane_id+320)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+320)/60;
    threadInput10 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+352)%10;
    new_j = (warp_id_y<<2) + ((lane_id+352)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+352)/60;
    threadInput11 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+384)%10;
    new_j = (warp_id_y<<2) + ((lane_id+384)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+384)/60;
    threadInput12 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+416)%10;
    new_j = (warp_id_y<<2) + ((lane_id+416)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+416)/60;
    threadInput13 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+448)%10;
    new_j = (warp_id_y<<2) + ((lane_id+448)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+448)/60;
    threadInput14 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+480)%10;
    new_j = (warp_id_y<<2) + ((lane_id+480)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+480)/60;
    threadInput15 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+512)%10;
    new_j = (warp_id_y<<2) + ((lane_id+512)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+512)/60;
    threadInput16 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+544)%10;
    new_j = (warp_id_y<<2) + ((lane_id+544)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+544)/60;
    threadInput17 = IN_3D(new_k, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+576)%10;
    new_j = (warp_id_y<<2) + ((lane_id+576)/10)%6;
    new_k = (warp_id_z<<0) + (lane_id+576)/60;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    new_j = (new_j < m+2*halo)? new_j: m+2*halo-1;
    new_k = (new_k < z+2*halo)? new_k: z+2*halo-1;
    threadInput18 = IN_3D(new_k, new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    DATA_TYPE sum4 = 0.0;
    DATA_TYPE sum5 = 0.0;
    DATA_TYPE sum6 = 0.0;
    DATA_TYPE sum7 = 0.0;
    int friend_id0, friend_id1, friend_id2, friend_id3;
    int friend_id4, friend_id5, friend_id6, friend_id7;
    DATA_TYPE tx0, ty0, tz0, tx1, ty1, tz1, tx2, ty2, tz2, tx3, ty3, tz3;
    DATA_TYPE rx0, ry0, rz0, rx1, ry1, rz1, rx2, ry2, rz2, rx3, ry3, rz3;

    friend_id0 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    rx0 = __shfl(threadInput7, friend_id4);
    ry0 = __shfl(threadInput8, friend_id4);
    rx1 = __shfl(threadInput9 , friend_id5);
    ry1 = __shfl(threadInput10, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += ARG_3D(0,k  ,j,i)*((lane_id < 26)? tx0: ty0);
    sum1 += ARG_3D(0,k+1,j,i)*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
    sum2 += ARG_3D(0,k+2,j,i)*((lane_id < 8 )? tx2: ty2);
    sum3 += ARG_3D(0,k+3,j,i)*((lane_id < 10)? tx3: ty3);
    sum4 += ARG_3D(0,k+4,j,i)*((lane_id < 14)? rx0: ry0);
    sum5 += ARG_3D(0,k+5,j,i)*((lane_id < 16)? rx1: ry1);
    sum6 += ARG_3D(0,k+6,j,i)*((lane_id < 20)? rx2: ry2);
    sum7 += ARG_3D(0,k+7,j,i)*((lane_id < 24)? rx3: ry3);

    friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    rx0 = __shfl(threadInput7, friend_id4);
    ry0 = __shfl(threadInput8, friend_id4);
    rx1 = __shfl(threadInput9 , friend_id5);
    ry1 = __shfl(threadInput10, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += ARG_3D(1,k  ,j,i)*((lane_id < 25)? tx0: ty0);
    sum1 += ARG_3D(1,k+1,j,i)*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
    sum2 += ARG_3D(1,k+2,j,i)*((lane_id < 7 )? tx2: ty2);
    sum3 += ARG_3D(1,k+3,j,i)*((lane_id < 9 )? tx3: ty3);
    sum4 += ARG_3D(1,k+4,j,i)*((lane_id < 13)? rx0: ry0);
    sum5 += ARG_3D(1,k+5,j,i)*((lane_id < 16)? rx1: ry1);
    sum6 += ARG_3D(1,k+6,j,i)*((lane_id < 19)? rx2: ry2);
    sum7 += ARG_3D(1,k+7,j,i)*((lane_id < 23)? rx3: ry3);

    friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    rx0 = __shfl(threadInput7, friend_id4);
    ry0 = __shfl(threadInput8, friend_id4);
    rx1 = __shfl(threadInput9 , friend_id5);
    ry1 = __shfl(threadInput10, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += ARG_3D(2,k  ,j,i)*((lane_id < 24)? tx0: ty0);
    sum1 += ARG_3D(2,k+1,j,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
    sum2 += ARG_3D(2,k+2,j,i)*((lane_id < 6 )? tx2: ty2);
    sum3 += ARG_3D(2,k+3,j,i)*((lane_id < 8 )? tx3: ty3);
    sum4 += ARG_3D(2,k+4,j,i)*((lane_id < 12)? rx0: ry0);
    sum5 += ARG_3D(2,k+5,j,i)*((lane_id < 16)? rx1: ry1);
    sum6 += ARG_3D(2,k+6,j,i)*((lane_id < 18)? rx2: ry2);
    sum7 += ARG_3D(2,k+7,j,i)*((lane_id < 22)? rx3: ry3);

    friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    tz3 = __shfl(threadInput7, friend_id3);
    rx0 = __shfl(threadInput7, friend_id4);
    ry0 = __shfl(threadInput8, friend_id4);
    rx1 = __shfl(threadInput9 , friend_id5);
    ry1 = __shfl(threadInput10, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += ARG_3D(3,k  ,j,i)*((lane_id < 18)? tx0: ty0);
    sum1 += ARG_3D(3,k+1,j,i)*((lane_id < 22)? tx1: ty1);
    sum2 += ARG_3D(3,k+2,j,i)*((lane_id < 24)? tx2: ty2);
    sum3 += ARG_3D(3,k+3,j,i)*((lane_id < 2 )? tx3: ((lane_id < 28)? ty3: tz3));
    sum4 += ARG_3D(3,k+4,j,i)*((lane_id < 6 )? rx0: ry0);
    sum5 += ARG_3D(3,k+5,j,i)*((lane_id < 8 )? rx1: ry1);
    sum6 += ARG_3D(3,k+6,j,i)*((lane_id < 12)? rx2: ry2);
    sum7 += ARG_3D(3,k+7,j,i)*((lane_id < 16)? rx3: ry3);

    friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+23+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+15+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput5, friend_id3);
    ty3 = __shfl(threadInput6, friend_id3);
    tz3 = __shfl(threadInput7, friend_id3);
    rx0 = __shfl(threadInput7, friend_id4);
    ry0 = __shfl(threadInput8, friend_id4);
    rz0 = __shfl(threadInput9, friend_id4);
    rx1 = __shfl(threadInput9 , friend_id5);
    ry1 = __shfl(threadInput10, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += ARG_3D(4,k  ,j,i)*((lane_id < 17)? tx0: ty0);
    sum1 += ARG_3D(4,k+1,j,i)*((lane_id < 21)? tx1: ty1);
    sum2 += ARG_3D(4,k+2,j,i)*((lane_id < 24)? tx2: ty2);
    sum3 += ARG_3D(4,k+3,j,i)*((lane_id < 1 )? tx3: ((lane_id < 27)? ty3: tz3));
    sum4 += ARG_3D(4,k+4,j,i)*((lane_id < 5 )? rx0: ((lane_id < 31)? ry0: rz0));
    sum5 += ARG_3D(4,k+5,j,i)*((lane_id < 8 )? rx1: ry1);
    sum6 += ARG_3D(4,k+6,j,i)*((lane_id < 11)? rx2: ry2);
    sum7 += ARG_3D(4,k+7,j,i)*((lane_id < 15)? rx3: ry3);

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    rx0 = __shfl(threadInput7, friend_id4);
    ry0 = __shfl(threadInput8, friend_id4);
    rz0 = __shfl(threadInput9, friend_id4);
    rx1 = __shfl(threadInput9 , friend_id5);
    ry1 = __shfl(threadInput10, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += ARG_3D(5,k  ,j,i)*((lane_id < 16)? tx0: ty0);
    sum1 += ARG_3D(5,k+1,j,i)*((lane_id < 20)? tx1: ty1);
    sum2 += ARG_3D(5,k+2,j,i)*((lane_id < 24)? tx2: ty2);
    sum3 += ARG_3D(5,k+3,j,i)*((lane_id < 26)? tx3: ty3);
    sum4 += ARG_3D(5,k+4,j,i)*((lane_id < 4 )? rx0: ((lane_id < 30)? ry0: rz0));
    sum5 += ARG_3D(5,k+5,j,i)*((lane_id < 8 )? rx1: ry1);
    sum6 += ARG_3D(5,k+6,j,i)*((lane_id < 10)? rx2: ry2);
    sum7 += ARG_3D(5,k+7,j,i)*((lane_id < 14)? rx3: ry3);

    friend_id0 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    rx0 = __shfl(threadInput8, friend_id4);
    ry0 = __shfl(threadInput9, friend_id4);
    rx1 = __shfl(threadInput10, friend_id5);
    ry1 = __shfl(threadInput11, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rz2 = __shfl(threadInput13, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += ARG_3D(6,k  ,j,i)*((lane_id < 10)? tx0: ty0);
    sum1 += ARG_3D(6,k+1,j,i)*((lane_id < 14)? tx1: ty1);
    sum2 += ARG_3D(6,k+2,j,i)*((lane_id < 16)? tx2: ty2);
    sum3 += ARG_3D(6,k+3,j,i)*((lane_id < 20)? tx3: ty3);
    sum4 += ARG_3D(6,k+4,j,i)*((lane_id < 24)? rx0: ry0);
    sum5 += ARG_3D(6,k+5,j,i)*((lane_id < 26)? rx1: ry1);
    sum6 += ARG_3D(6,k+6,j,i)*((lane_id < 4 )? rx2: ((lane_id < 30)? ry2: rz2));
    sum7 += ARG_3D(6,k+7,j,i)*((lane_id < 8 )? rx3: ry3);

    friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    rx0 = __shfl(threadInput8, friend_id4);
    ry0 = __shfl(threadInput9, friend_id4);
    rx1 = __shfl(threadInput10, friend_id5);
    ry1 = __shfl(threadInput11, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rz2 = __shfl(threadInput13, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += ARG_3D(7,k  ,j,i)*((lane_id < 9 )? tx0: ty0);
    sum1 += ARG_3D(7,k+1,j,i)*((lane_id < 13)? tx1: ty1);
    sum2 += ARG_3D(7,k+2,j,i)*((lane_id < 16)? tx2: ty2);
    sum3 += ARG_3D(7,k+3,j,i)*((lane_id < 19)? tx3: ty3);
    sum4 += ARG_3D(7,k+4,j,i)*((lane_id < 23)? rx0: ry0);
    sum5 += ARG_3D(7,k+5,j,i)*((lane_id < 25)? rx1: ry1);
    sum6 += ARG_3D(7,k+6,j,i)*((lane_id < 3 )? rx2: ((lane_id < 29)? ry2: rz2));
    sum7 += ARG_3D(7,k+7,j,i)*((lane_id < 7 )? rx3: ry3);

    friend_id0 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput2, friend_id1);
    ty1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput4, friend_id2);
    ty2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput6, friend_id3);
    ty3 = __shfl(threadInput7, friend_id3);
    rx0 = __shfl(threadInput8, friend_id4);
    ry0 = __shfl(threadInput9, friend_id4);
    rx1 = __shfl(threadInput10, friend_id5);
    ry1 = __shfl(threadInput11, friend_id5);
    rx2 = __shfl(threadInput11, friend_id6);
    ry2 = __shfl(threadInput12, friend_id6);
    rz2 = __shfl(threadInput13, friend_id6);
    rx3 = __shfl(threadInput13, friend_id7);
    ry3 = __shfl(threadInput14, friend_id7);
    sum0 += ARG_3D(8,k  ,j,i)*((lane_id < 8 )? tx0: ty0);
    sum1 += ARG_3D(8,k+1,j,i)*((lane_id < 12)? tx1: ty1);
    sum2 += ARG_3D(8,k+2,j,i)*((lane_id < 16)? tx2: ty2);
    sum3 += ARG_3D(8,k+3,j,i)*((lane_id < 18)? tx3: ty3);
    sum4 += ARG_3D(8,k+4,j,i)*((lane_id < 22)? rx0: ry0);
    sum5 += ARG_3D(8,k+5,j,i)*((lane_id < 24)? rx1: ry1);
    sum6 += ARG_3D(8,k+6,j,i)*((lane_id < 2 )? rx2: ((lane_id < 28)? ry2: rz2));
    sum7 += ARG_3D(8,k+7,j,i)*((lane_id < 6 )? rx3: ry3);

    friend_id0 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    rx0 = __shfl(threadInput9 , friend_id4);
    ry0 = __shfl(threadInput10, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += ARG_3D(9,k  ,j,i)*((lane_id < 4 )? tx0: ((lane_id < 30)? ty0: tz0));
    sum1 += ARG_3D(9,k+1,j,i)*((lane_id < 8)? tx1: ty1);
    sum2 += ARG_3D(9,k+2,j,i)*((lane_id < 10)? tx2: ty2);
    sum3 += ARG_3D(9,k+3,j,i)*((lane_id < 14)? tx3: ty3);
    sum4 += ARG_3D(9,k+4,j,i)*((lane_id < 16)? rx0: ry0);
    sum5 += ARG_3D(9,k+5,j,i)*((lane_id < 20)? rx1: ry1);
    sum6 += ARG_3D(9,k+6,j,i)*((lane_id < 24)? rx2: ry2);
    sum7 += ARG_3D(9,k+7,j,i)*((lane_id < 26)? rx3: ry3);

    friend_id0 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    rx0 = __shfl(threadInput9 , friend_id4);
    ry0 = __shfl(threadInput10, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += ARG_3D(10,k  ,j,i)*((lane_id < 3 )? tx0: ((lane_id < 29)? ty0: tz0));
    sum1 += ARG_3D(10,k+1,j,i)*((lane_id < 7 )? tx1: ty1);
    sum2 += ARG_3D(10,k+2,j,i)*((lane_id < 9 )? tx2: ty2);
    sum3 += ARG_3D(10,k+3,j,i)*((lane_id < 13)? tx3: ty3);
    sum4 += ARG_3D(10,k+4,j,i)*((lane_id < 16)? rx0: ry0);
    sum5 += ARG_3D(10,k+5,j,i)*((lane_id < 19)? rx1: ry1);
    sum6 += ARG_3D(10,k+6,j,i)*((lane_id < 23)? rx2: ry2);
    sum7 += ARG_3D(10,k+7,j,i)*((lane_id < 25)? rx3: ry3);

    friend_id0 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput3, friend_id1);
    ty1 = __shfl(threadInput4, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    rx0 = __shfl(threadInput9 , friend_id4);
    ry0 = __shfl(threadInput10, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += ARG_3D(11,k  ,j,i)*((lane_id < 2 )? tx0: ((lane_id < 28)? ty0: tz0));
    sum1 += ARG_3D(11,k+1,j,i)*((lane_id < 6 )? tx1: ty1);
    sum2 += ARG_3D(11,k+2,j,i)*((lane_id < 8 )? tx2: ty2);
    sum3 += ARG_3D(11,k+3,j,i)*((lane_id < 12)? tx3: ty3);
    sum4 += ARG_3D(11,k+4,j,i)*((lane_id < 16)? rx0: ry0);
    sum5 += ARG_3D(11,k+5,j,i)*((lane_id < 18)? rx1: ry1);
    sum6 += ARG_3D(11,k+6,j,i)*((lane_id < 22)? rx2: ry2);
    sum7 += ARG_3D(11,k+7,j,i)*((lane_id < 24)? rx3: ry3);

    friend_id0 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tz2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    rx0 = __shfl(threadInput9 , friend_id4);
    ry0 = __shfl(threadInput10, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += ARG_3D(12,k  ,j,i)*((lane_id < 22)? tx0: ty0);
    sum1 += ARG_3D(12,k+1,j,i)*((lane_id < 24)? tx1: ty1);
    sum2 += ARG_3D(12,k+2,j,i)*((lane_id < 2 )? tx2: ((lane_id < 28)? ty2: tz2));
    sum3 += ARG_3D(12,k+3,j,i)*((lane_id < 6 )? tx3: ty3);
    sum4 += ARG_3D(12,k+4,j,i)*((lane_id < 8 )? rx0: ry0);
    sum5 += ARG_3D(12,k+5,j,i)*((lane_id < 12)? rx1: ry1);
    sum6 += ARG_3D(12,k+6,j,i)*((lane_id < 16)? rx2: ry2);
    sum7 += ARG_3D(12,k+7,j,i)*((lane_id < 18)? rx3: ry3);

    friend_id0 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+23+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+15+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput5, friend_id2);
    ty2 = __shfl(threadInput6, friend_id2);
    tz2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    tz3 = __shfl(threadInput9, friend_id3);
    rx0 = __shfl(threadInput9 , friend_id4);
    ry0 = __shfl(threadInput10, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += ARG_3D(13,k  ,j,i)*((lane_id < 21)? tx0: ty0);
    sum1 += ARG_3D(13,k+1,j,i)*((lane_id < 24)? tx1: ty1);
    sum2 += ARG_3D(13,k+2,j,i)*((lane_id < 1 )? tx2: ((lane_id < 27)? ty2: tz2));
    sum3 += ARG_3D(13,k+3,j,i)*((lane_id < 5 )? tx3: ((lane_id < 31)? ty3: tz3));
    sum4 += ARG_3D(13,k+4,j,i)*((lane_id < 8 )? rx0: ry0);
    sum5 += ARG_3D(13,k+5,j,i)*((lane_id < 11)? rx1: ry1);
    sum6 += ARG_3D(13,k+6,j,i)*((lane_id < 15)? rx2: ry2);
    sum7 += ARG_3D(13,k+7,j,i)*((lane_id < 17)? rx3: ry3);

    friend_id0 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput7, friend_id3);
    ty3 = __shfl(threadInput8, friend_id3);
    tz3 = __shfl(threadInput9, friend_id3);
    rx0 = __shfl(threadInput9 , friend_id4);
    ry0 = __shfl(threadInput10, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += ARG_3D(14,k  ,j,i)*((lane_id < 20)? tx0: ty0);
    sum1 += ARG_3D(14,k+1,j,i)*((lane_id < 24)? tx1: ty1);
    sum2 += ARG_3D(14,k+2,j,i)*((lane_id < 26)? tx2: ty2);
    sum3 += ARG_3D(14,k+3,j,i)*((lane_id < 4 )? tx3: ((lane_id < 30)? ty3: tz3));
    sum4 += ARG_3D(14,k+4,j,i)*((lane_id < 8 )? rx0: ry0);
    sum5 += ARG_3D(14,k+5,j,i)*((lane_id < 10)? rx1: ry1);
    sum6 += ARG_3D(14,k+6,j,i)*((lane_id < 14)? rx2: ry2);
    sum7 += ARG_3D(14,k+7,j,i)*((lane_id < 16)? rx3: ry3);

    friend_id0 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+0+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput8, friend_id3);
    ty3 = __shfl(threadInput9, friend_id3);
    rx0 = __shfl(threadInput10, friend_id4);
    ry0 = __shfl(threadInput11, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rz1 = __shfl(threadInput13, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += ARG_3D(15,k  ,j,i)*((lane_id < 14)? tx0: ty0);
    sum1 += ARG_3D(15,k+1,j,i)*((lane_id < 16)? tx1: ty1);
    sum2 += ARG_3D(15,k+2,j,i)*((lane_id < 20)? tx2: ty2);
    sum3 += ARG_3D(15,k+3,j,i)*((lane_id < 24)? tx3: ty3);
    sum4 += ARG_3D(15,k+4,j,i)*((lane_id < 26)? rx0: ry0);
    sum5 += ARG_3D(15,k+5,j,i)*((lane_id < 4 )? rx1: ((lane_id < 30)? ry1: rz1));
    sum6 += ARG_3D(15,k+6,j,i)*((lane_id < 8 )? rx2: ry2);
    sum7 += ARG_3D(15,k+7,j,i)*((lane_id < 10)? rx3: ry3);

    friend_id0 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+1+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput8, friend_id3);
    ty3 = __shfl(threadInput9, friend_id3);
    rx0 = __shfl(threadInput10, friend_id4);
    ry0 = __shfl(threadInput11, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rz1 = __shfl(threadInput13, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += ARG_3D(16,k  ,j,i)*((lane_id < 13)? tx0: ty0);
    sum1 += ARG_3D(16,k+1,j,i)*((lane_id < 16)? tx1: ty1);
    sum2 += ARG_3D(16,k+2,j,i)*((lane_id < 19)? tx2: ty2);
    sum3 += ARG_3D(16,k+3,j,i)*((lane_id < 23)? tx3: ty3);
    sum4 += ARG_3D(16,k+4,j,i)*((lane_id < 25)? rx0: ry0);
    sum5 += ARG_3D(16,k+5,j,i)*((lane_id < 3 )? rx1: ((lane_id < 29)? ry1: rz1));
    sum6 += ARG_3D(16,k+6,j,i)*((lane_id < 7 )? rx2: ry2);
    sum7 += ARG_3D(16,k+7,j,i)*((lane_id < 9 )? rx3: ry3);

    friend_id0 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+2+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput2, friend_id0);
    ty0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput4, friend_id1);
    ty1 = __shfl(threadInput5, friend_id1);
    tx2 = __shfl(threadInput6, friend_id2);
    ty2 = __shfl(threadInput7, friend_id2);
    tx3 = __shfl(threadInput8, friend_id3);
    ty3 = __shfl(threadInput9, friend_id3);
    rx0 = __shfl(threadInput10, friend_id4);
    ry0 = __shfl(threadInput11, friend_id4);
    rx1 = __shfl(threadInput11, friend_id5);
    ry1 = __shfl(threadInput12, friend_id5);
    rz1 = __shfl(threadInput13, friend_id5);
    rx2 = __shfl(threadInput13, friend_id6);
    ry2 = __shfl(threadInput14, friend_id6);
    rx3 = __shfl(threadInput15, friend_id7);
    ry3 = __shfl(threadInput16, friend_id7);
    sum0 += ARG_3D(17,k  ,j,i)*((lane_id < 12)? tx0: ty0);
    sum1 += ARG_3D(17,k+1,j,i)*((lane_id < 16)? tx1: ty1);
    sum2 += ARG_3D(17,k+2,j,i)*((lane_id < 18)? tx2: ty2);
    sum3 += ARG_3D(17,k+3,j,i)*((lane_id < 22)? tx3: ty3);
    sum4 += ARG_3D(17,k+4,j,i)*((lane_id < 24)? rx0: ry0);
    sum5 += ARG_3D(17,k+5,j,i)*((lane_id < 2 )? rx1: ((lane_id < 28)? ry1: rz1));
    sum6 += ARG_3D(17,k+6,j,i)*((lane_id < 6 )? rx2: ry2);
    sum7 += ARG_3D(17,k+7,j,i)*((lane_id < 8 )? rx3: ry3);

    friend_id0 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput16, friend_id7);
    ry3 = __shfl(threadInput17, friend_id7);
    rz3 = __shfl(threadInput18, friend_id7);
    sum0 += ARG_3D(18,k  ,j,i)*((lane_id < 8 )? tx0: ty0);
    sum1 += ARG_3D(18,k+1,j,i)*((lane_id < 10)? tx1: ty1);
    sum2 += ARG_3D(18,k+2,j,i)*((lane_id < 14)? tx2: ty2);
    sum3 += ARG_3D(18,k+3,j,i)*((lane_id < 16)? tx3: ty3);
    sum4 += ARG_3D(18,k+4,j,i)*((lane_id < 20)? rx0: ry0);
    sum5 += ARG_3D(18,k+5,j,i)*((lane_id < 24)? rx1: ry1);
    sum6 += ARG_3D(18,k+6,j,i)*((lane_id < 26)? rx2: ry2);
    sum7 += ARG_3D(18,k+7,j,i)*((lane_id < 4 )? rx3: ((lane_id < 30)? ry3: rz3));

    friend_id0 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput16, friend_id7);
    ry3 = __shfl(threadInput17, friend_id7);
    rz3 = __shfl(threadInput18, friend_id7);
    sum0 += ARG_3D(19,k  ,j,i)*((lane_id < 7 )? tx0: ty0);
    sum1 += ARG_3D(19,k+1,j,i)*((lane_id < 9 )? tx1: ty1);
    sum2 += ARG_3D(19,k+2,j,i)*((lane_id < 13)? tx2: ty2);
    sum3 += ARG_3D(19,k+3,j,i)*((lane_id < 16)? tx3: ty3);
    sum4 += ARG_3D(19,k+4,j,i)*((lane_id < 19)? rx0: ry0);
    sum5 += ARG_3D(19,k+5,j,i)*((lane_id < 23)? rx1: ry1);
    sum6 += ARG_3D(19,k+6,j,i)*((lane_id < 25)? rx2: ry2);
    sum7 += ARG_3D(19,k+7,j,i)*((lane_id < 3 )? rx3: ((lane_id < 29)? ry3: rz3));

    friend_id0 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput3, friend_id0);
    ty0 = __shfl(threadInput4, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput16, friend_id7);
    ry3 = __shfl(threadInput17, friend_id7);
    rz3 = __shfl(threadInput18, friend_id7);
    sum0 += ARG_3D(20,k  ,j,i)*((lane_id < 6 )? tx0: ty0);
    sum1 += ARG_3D(20,k+1,j,i)*((lane_id < 8 )? tx1: ty1);
    sum2 += ARG_3D(20,k+2,j,i)*((lane_id < 12)? tx2: ty2);
    sum3 += ARG_3D(20,k+3,j,i)*((lane_id < 16)? tx3: ty3);
    sum4 += ARG_3D(20,k+4,j,i)*((lane_id < 18)? rx0: ry0);
    sum5 += ARG_3D(20,k+5,j,i)*((lane_id < 22)? rx1: ry1);
    sum6 += ARG_3D(20,k+6,j,i)*((lane_id < 24)? rx2: ry2);
    sum7 += ARG_3D(20,k+7,j,i)*((lane_id < 2 )? rx3: ((lane_id < 28)? ry3: rz3));

    friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tz1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput17, friend_id7);
    ry3 = __shfl(threadInput18, friend_id7);
    sum0 += ARG_3D(21,k  ,j,i)*((lane_id < 24)? tx0: ty0);
    sum1 += ARG_3D(21,k+1,j,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
    sum2 += ARG_3D(21,k+2,j,i)*((lane_id < 6 )? tx2: ty2);
    sum3 += ARG_3D(21,k+3,j,i)*((lane_id < 8 )? tx3: ty3);
    sum4 += ARG_3D(21,k+4,j,i)*((lane_id < 12)? rx0: ry0);
    sum5 += ARG_3D(21,k+5,j,i)*((lane_id < 16)? rx1: ry1);
    sum6 += ARG_3D(21,k+6,j,i)*((lane_id < 18)? rx2: ry2);
    sum7 += ARG_3D(21,k+7,j,i)*((lane_id < 22)? rx3: ry3);

    friend_id0 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+31+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+23+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+15+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+7 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput5, friend_id1);
    ty1 = __shfl(threadInput6, friend_id1);
    tz1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tz2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput17, friend_id7);
    ry3 = __shfl(threadInput18, friend_id7);
    sum0 += ARG_3D(22,k  ,j,i)*((lane_id < 24)? tx0: ty0);
    sum1 += ARG_3D(22,k+1,j,i)*((lane_id < 1 )? tx1: ((lane_id < 27)? ty1: tz1));
    sum2 += ARG_3D(22,k+2,j,i)*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
    sum3 += ARG_3D(22,k+3,j,i)*((lane_id < 8 )? tx3: ty3);
    sum4 += ARG_3D(22,k+4,j,i)*((lane_id < 11)? rx0: ry0);
    sum5 += ARG_3D(22,k+5,j,i)*((lane_id < 15)? rx1: ry1);
    sum6 += ARG_3D(22,k+6,j,i)*((lane_id < 17)? rx2: ry2);
    sum7 += ARG_3D(22,k+7,j,i)*((lane_id < 21)? rx3: ry3);

    friend_id0 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput7, friend_id2);
    ty2 = __shfl(threadInput8, friend_id2);
    tz2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput9 , friend_id3);
    ty3 = __shfl(threadInput10, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput17, friend_id7);
    ry3 = __shfl(threadInput18, friend_id7);
    sum0 += ARG_3D(23,k  ,j,i)*((lane_id < 24)? tx0: ty0);
    sum1 += ARG_3D(23,k+1,j,i)*((lane_id < 26)? tx1: ty1);
    sum2 += ARG_3D(23,k+2,j,i)*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
    sum3 += ARG_3D(23,k+3,j,i)*((lane_id < 8 )? tx3: ty3);
    sum4 += ARG_3D(23,k+4,j,i)*((lane_id < 10)? rx0: ry0);
    sum5 += ARG_3D(23,k+5,j,i)*((lane_id < 14)? rx1: ry1);
    sum6 += ARG_3D(23,k+6,j,i)*((lane_id < 16)? rx2: ry2);
    sum7 += ARG_3D(23,k+7,j,i)*((lane_id < 20)? rx3: ry3);

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput8, friend_id2);
    ty2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput10, friend_id3);
    ty3 = __shfl(threadInput11, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rz0 = __shfl(threadInput13, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput17, friend_id7);
    ry3 = __shfl(threadInput18, friend_id7);
    sum0 += ARG_3D(24,k  ,j,i)*((lane_id < 16)? tx0: ty0);
    sum1 += ARG_3D(24,k+1,j,i)*((lane_id < 20)? tx1: ty1);
    sum2 += ARG_3D(24,k+2,j,i)*((lane_id < 24)? tx2: ty2);
    sum3 += ARG_3D(24,k+3,j,i)*((lane_id < 26)? tx3: ty3);
    sum4 += ARG_3D(24,k+4,j,i)*((lane_id < 4 )? rx0: ((lane_id < 30)? ry0: rz0));
    sum5 += ARG_3D(24,k+5,j,i)*((lane_id < 8 )? rx1: ry1);
    sum6 += ARG_3D(24,k+6,j,i)*((lane_id < 10)? rx2: ry2);
    sum7 += ARG_3D(24,k+7,j,i)*((lane_id < 14)? rx3: ry3);

    friend_id0 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput8, friend_id2);
    ty2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput10, friend_id3);
    ty3 = __shfl(threadInput11, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rz0 = __shfl(threadInput13, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput17, friend_id7);
    ry3 = __shfl(threadInput18, friend_id7);
    sum0 += ARG_3D(25,k  ,j,i)*((lane_id < 16)? tx0: ty0);
    sum1 += ARG_3D(25,k+1,j,i)*((lane_id < 19)? tx1: ty1);
    sum2 += ARG_3D(25,k+2,j,i)*((lane_id < 23)? tx2: ty2);
    sum3 += ARG_3D(25,k+3,j,i)*((lane_id < 25)? tx3: ty3);
    sum4 += ARG_3D(25,k+4,j,i)*((lane_id < 3 )? rx0: ((lane_id < 29)? ry0: rz0));
    sum5 += ARG_3D(25,k+5,j,i)*((lane_id < 7 )? rx1: ry1);
    sum6 += ARG_3D(25,k+6,j,i)*((lane_id < 9 )? rx2: ry2);
    sum7 += ARG_3D(25,k+7,j,i)*((lane_id < 13)? rx3: ry3);

    friend_id0 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id4 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
    friend_id5 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id6 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
    friend_id7 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput4, friend_id0);
    ty0 = __shfl(threadInput5, friend_id0);
    tx1 = __shfl(threadInput6, friend_id1);
    ty1 = __shfl(threadInput7, friend_id1);
    tx2 = __shfl(threadInput8, friend_id2);
    ty2 = __shfl(threadInput9, friend_id2);
    tx3 = __shfl(threadInput10, friend_id3);
    ty3 = __shfl(threadInput11, friend_id3);
    rx0 = __shfl(threadInput11, friend_id4);
    ry0 = __shfl(threadInput12, friend_id4);
    rz0 = __shfl(threadInput13, friend_id4);
    rx1 = __shfl(threadInput13, friend_id5);
    ry1 = __shfl(threadInput14, friend_id5);
    rx2 = __shfl(threadInput15, friend_id6);
    ry2 = __shfl(threadInput16, friend_id6);
    rx3 = __shfl(threadInput17, friend_id7);
    ry3 = __shfl(threadInput18, friend_id7);
    sum0 += ARG_3D(26,k  ,j,i)*((lane_id < 16)? tx0: ty0);
    sum1 += ARG_3D(26,k+1,j,i)*((lane_id < 18)? tx1: ty1);
    sum2 += ARG_3D(26,k+2,j,i)*((lane_id < 22)? tx2: ty2);
    sum3 += ARG_3D(26,k+3,j,i)*((lane_id < 24)? tx3: ty3);
    sum4 += ARG_3D(26,k+4,j,i)*((lane_id < 2 )? rx0: ((lane_id < 28)? ry0: rz0));
    sum5 += ARG_3D(26,k+5,j,i)*((lane_id < 6 )? rx1: ry1);
    sum6 += ARG_3D(26,k+6,j,i)*((lane_id < 8 )? rx2: ry2);
    sum7 += ARG_3D(26,k+7,j,i)*((lane_id < 12)? rx3: ry3);


    OUT_3D(k  ,j,i) = sum0;
    OUT_3D(k+1,j,i) = sum1;
    OUT_3D(k+2,j,i) = sum2;
    OUT_3D(k+3,j,i) = sum3;
    OUT_3D(k+4,j,i) = sum4;
    OUT_3D(k+5,j,i) = sum5;
    OUT_3D(k+6,j,i) = sum6;
    OUT_3D(k+7,j,i) = sum7;
}

__global__ void Stencil_Cuda_Sweep_Shfl(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = threadIdx.y + blockIdx.y * blockDim.y + halo;
    const int block_z = z / gridDim.z;
    int k = block_z * blockIdx.z + halo;
    const int k_end = k + block_z;
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int lane_id = tid % warpSize;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2; // 1x4x8, warp_ids are division of 
    DATA_TYPE tx, ty;
    int friend_id;
    int new_i, new_j;
    DATA_TYPE t3_threadInput0, t3_threadInput1;
    DATA_TYPE t2_threadInput0, t2_threadInput1;
    DATA_TYPE t1_threadInput0, t1_threadInput1;

#define SM_2D_M2 32 
#define SM_2D_N2 8 

    DATA_TYPE sum = 0.0;

    // t3 is current layer; t2 is previous layer
    new_i = (warp_id_x<<3) + lane_id%10;     // 10 is extended dimension of i
    new_j = (warp_id_y<<2) + lane_id/10;     
    t3_threadInput0 = IN_3D(k  , new_j, new_i);
    t2_threadInput0 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10;
    new_j = (warp_id_y<<2) + (lane_id+32)/10;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    new_j = (new_j < m+2*halo)? new_j: m+2*halo-1;
    t3_threadInput1 = IN_3D(k  , new_j, new_i);
    t2_threadInput1 = IN_3D(k-1, new_j, new_i);

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        sum = 0.0;
        // move the current storage down 
        t1_threadInput0 = t2_threadInput0;
        t1_threadInput1 = t2_threadInput1;
        t2_threadInput0 = t3_threadInput0;
        t2_threadInput1 = t3_threadInput1;

        new_i = (warp_id_x<<3) + lane_id%10;  
        new_j = (warp_id_y<<2) + lane_id/10;     
        t3_threadInput0 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+32)%10;
        new_j = (warp_id_y<<2) + (lane_id+32)/10;
        new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
        new_j = (new_j < m+2*halo)? new_j: m+2*halo-1;
        t3_threadInput1 = IN_3D(k+1, new_j, new_i);

        friend_id = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += ARG_3D(0,k,j,i)*((lane_id < 26)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += ARG_3D(9,k,j,i)*((lane_id < 26)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += ARG_3D(18,k,j,i)*((lane_id < 26)? tx: ty);

        friend_id = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += ARG_3D(1,k,j,i)*((lane_id < 25)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += ARG_3D(10,k,j,i)*((lane_id < 25)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += ARG_3D(19,k,j,i)*((lane_id < 25)? tx: ty);

        friend_id = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += ARG_3D(2,k,j,i)*((lane_id < 24)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += ARG_3D(11,k,j,i)*((lane_id < 24)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += ARG_3D(20,k,j,i)*((lane_id < 24)? tx: ty);

        friend_id = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += ARG_3D(3,k,j,i)*((lane_id < 18)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += ARG_3D(12,k,j,i)*((lane_id < 18)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += ARG_3D(21,k,j,i)*((lane_id < 18)? tx: ty);

        friend_id = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += ARG_3D(4,k,j,i)*((lane_id < 17)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += ARG_3D(13,k,j,i)*((lane_id < 17)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += ARG_3D(22,k,j,i)*((lane_id < 17)? tx: ty);

        friend_id = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += ARG_3D(5,k,j,i)*((lane_id < 16)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += ARG_3D(14,k,j,i)*((lane_id < 16)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += ARG_3D(23,k,j,i)*((lane_id < 16)? tx: ty);

        friend_id = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += ARG_3D(6,k,j,i)*((lane_id < 10)? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += ARG_3D(15,k,j,i)*((lane_id < 10)? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += ARG_3D(24,k,j,i)*((lane_id < 10)? tx: ty);

        friend_id = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += ARG_3D(7,k,j,i)*((lane_id < 9 )? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += ARG_3D(16,k,j,i)*((lane_id < 9 )? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += ARG_3D(25,k,j,i)*((lane_id < 9 )? tx: ty);
        
        friend_id = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
        tx = __shfl(t1_threadInput0, friend_id);
        ty = __shfl(t1_threadInput1, friend_id);
        sum += ARG_3D(8,k,j,i)*((lane_id < 8 )? tx: ty);
        tx = __shfl(t2_threadInput0, friend_id);
        ty = __shfl(t2_threadInput1, friend_id);
        sum += ARG_3D(17,k,j,i)*((lane_id < 8 )? tx: ty);
        tx = __shfl(t3_threadInput0, friend_id);
        ty = __shfl(t3_threadInput1, friend_id);
        sum += ARG_3D(26,k,j,i)*((lane_id < 8 )? tx: ty);

        OUT_3D(k,j,i) = sum;
    }
}

__global__ void Stencil_Cuda_Sweep_Shfl2(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = (((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<3) + (lane_id>>3) + halo;

    const int block_z = z / gridDim.z;
    int k = block_z * blockIdx.z + halo;
    const int k_end = k + block_z;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = ((((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<3) + (lane_id>>3))>>2; // 1x4x8, warp_ids are division of 
    DATA_TYPE tx0, ty0;
    DATA_TYPE tx1, ty1, tz1;
    int friend_id0, friend_id1;
    int new_i, new_j;
    DATA_TYPE t3_threadInput0, t3_threadInput1, t3_threadInput2, t3_threadInput3;
    DATA_TYPE t2_threadInput0, t2_threadInput1, t2_threadInput2, t2_threadInput3;
    DATA_TYPE t1_threadInput0, t1_threadInput1, t1_threadInput2, t1_threadInput3;

#define SM_2D_M2 32 
#define SM_2D_N2 8 

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;

    // t3 is current layer; t2 is previous layer
    new_i = (warp_id_x<<3) + lane_id%10;     // 10 is extended dimension of i
    new_j = (warp_id_y<<2) + lane_id/10;     
    t3_threadInput0 = IN_3D(k  , new_j, new_i);
    t2_threadInput0 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10;
    new_j = (warp_id_y<<2) + (lane_id+32)/10;
    t3_threadInput1 = IN_3D(k  , new_j, new_i);
    t2_threadInput1 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10;
    new_j = (warp_id_y<<2) + (lane_id+64)/10;
    t3_threadInput2 = IN_3D(k  , new_j, new_i);
    t2_threadInput2 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10;
    new_j = (warp_id_y<<2) + (lane_id+96)/10;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    new_j = (new_j < m+2*halo)? new_j: m+2*halo-1;
    t3_threadInput3 = IN_3D(k  , new_j, new_i);
    t2_threadInput3 = IN_3D(k-1, new_j, new_i);

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        sum0 = 0.0;
        sum1 = 0.0;
        // move the current storage down 
        t1_threadInput0 = t2_threadInput0;
        t1_threadInput1 = t2_threadInput1;
        t1_threadInput2 = t2_threadInput2;
        t1_threadInput3 = t2_threadInput3;
        t2_threadInput0 = t3_threadInput0;
        t2_threadInput1 = t3_threadInput1;
        t2_threadInput2 = t3_threadInput2;
        t2_threadInput3 = t3_threadInput3;

        new_i = (warp_id_x<<3) + lane_id%10;  
        new_j = (warp_id_y<<2) + lane_id/10;     
        t3_threadInput0 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+32)%10;
        new_j = (warp_id_y<<2) + (lane_id+32)/10;
        t3_threadInput1 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+64)%10;
        new_j = (warp_id_y<<2) + (lane_id+64)/10;
        t3_threadInput2 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+96)%10;
        new_j = (warp_id_y<<2) + (lane_id+96)/10;
        new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
        new_j = (new_j < m+2*halo)? new_j: m+2*halo-1;
        t3_threadInput3 = IN_3D(k+1, new_j, new_i);

        friend_id0 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        sum0 += ARG_3D(0,k,j  ,i)*((lane_id < 26)? tx0: ty0);
        sum1 += ARG_3D(0,k,j+4,i)*((lane_id < 20)? tx1: ty1);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        sum0 += ARG_3D(9,k,j  ,i)*((lane_id < 26)? tx0: ty0);
        sum1 += ARG_3D(9,k,j+4,i)*((lane_id < 20)? tx1: ty1);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        sum0 += ARG_3D(18,k,j  ,i)*((lane_id < 26)? tx0: ty0);
        sum1 += ARG_3D(18,k,j+4,i)*((lane_id < 20)? tx1: ty1);

        friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        sum0 += ARG_3D(1,k,j  ,i)*((lane_id < 25)? tx0: ty0);
        sum1 += ARG_3D(1,k,j+4,i)*((lane_id < 19)? tx1: ty1);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        sum0 += ARG_3D(10,k,j  ,i)*((lane_id < 25)? tx0: ty0);
        sum1 += ARG_3D(10,k,j+4,i)*((lane_id < 19)? tx1: ty1);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        sum0 += ARG_3D(19,k,j  ,i)*((lane_id < 25)? tx0: ty0);
        sum1 += ARG_3D(19,k,j+4,i)*((lane_id < 19)? tx1: ty1);

        friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        sum0 += ARG_3D(2,k,j  ,i)*((lane_id < 24)? tx0: ty0);
        sum1 += ARG_3D(2,k,j+4,i)*((lane_id < 18)? tx1: ty1);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        sum0 += ARG_3D(11,k,j  ,i)*((lane_id < 24)? tx0: ty0);
        sum1 += ARG_3D(11,k,j+4,i)*((lane_id < 18)? tx1: ty1);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        sum0 += ARG_3D(20,k,j  ,i)*((lane_id < 24)? tx0: ty0);
        sum1 += ARG_3D(20,k,j+4,i)*((lane_id < 18)? tx1: ty1);

        friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        sum0 += ARG_3D(3,k,j  ,i)*((lane_id < 18)? tx0: ty0);
        sum1 += ARG_3D(3,k,j+4,i)*((lane_id < 12)? tx1: ty1);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        sum0 += ARG_3D(12,k,j  ,i)*((lane_id < 18)? tx0: ty0);
        sum1 += ARG_3D(12,k,j+4,i)*((lane_id < 12)? tx1: ty1);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        sum0 += ARG_3D(21,k,j  ,i)*((lane_id < 18)? tx0: ty0);
        sum1 += ARG_3D(21,k,j+4,i)*((lane_id < 12)? tx1: ty1);

        friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        sum0 += ARG_3D(4,k,j  ,i)*((lane_id < 17)? tx0: ty0);
        sum1 += ARG_3D(4,k,j+4,i)*((lane_id < 11)? tx1: ty1);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        sum0 += ARG_3D(13,k,j  ,i)*((lane_id < 17)? tx0: ty0);
        sum1 += ARG_3D(13,k,j+4,i)*((lane_id < 11)? tx1: ty1);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        sum0 += ARG_3D(22,k,j  ,i)*((lane_id < 17)? tx0: ty0);
        sum1 += ARG_3D(22,k,j+4,i)*((lane_id < 11)? tx1: ty1);

        friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        sum0 += ARG_3D(5,k,j  ,i)*((lane_id < 16)? tx0: ty0);
        sum1 += ARG_3D(5,k,j+4,i)*((lane_id < 10)? tx1: ty1);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        sum0 += ARG_3D(14,k,j  ,i)*((lane_id < 16)? tx0: ty0);
        sum1 += ARG_3D(14,k,j+4,i)*((lane_id < 10)? tx1: ty1);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        sum0 += ARG_3D(23,k,j  ,i)*((lane_id < 16)? tx0: ty0);
        sum1 += ARG_3D(23,k,j+4,i)*((lane_id < 10)? tx1: ty1);

        friend_id0 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tz1 = __shfl(t1_threadInput3, friend_id1);
        sum0 += ARG_3D(6,k,j  ,i)*((lane_id < 10)? tx0: ty0);
        sum1 += ARG_3D(6,k,j+4,i)*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tz1 = __shfl(t2_threadInput3, friend_id1);
        sum0 += ARG_3D(15,k,j  ,i)*((lane_id < 10)? tx0: ty0);
        sum1 += ARG_3D(15,k,j+4,i)*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tz1 = __shfl(t3_threadInput3, friend_id1);
        sum0 += ARG_3D(24,k,j  ,i)*((lane_id < 10)? tx0: ty0);
        sum1 += ARG_3D(24,k,j+4,i)*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));

        friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tz1 = __shfl(t1_threadInput3, friend_id1);
        sum0 += ARG_3D(7,k,j  ,i)*((lane_id < 9 )? tx0: ty0);
        sum1 += ARG_3D(7,k,j+4,i)*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tz1 = __shfl(t2_threadInput3, friend_id1);
        sum0 += ARG_3D(16,k,j  ,i)*((lane_id < 9 )? tx0: ty0);
        sum1 += ARG_3D(16,k,j+4,i)*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tz1 = __shfl(t3_threadInput3, friend_id1);
        sum0 += ARG_3D(25,k,j  ,i)*((lane_id < 9 )? tx0: ty0);
        sum1 += ARG_3D(25,k,j+4,i)*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        
        friend_id0 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tz1 = __shfl(t1_threadInput3, friend_id1);
        sum0 += ARG_3D(8,k,j  ,i)*((lane_id < 8 )? tx0: ty0);
        sum1 += ARG_3D(8,k,j+4,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tz1 = __shfl(t2_threadInput3, friend_id1);
        sum0 += ARG_3D(17,k,j  ,i)*((lane_id < 8 )? tx0: ty0);
        sum1 += ARG_3D(17,k,j+4,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tz1 = __shfl(t3_threadInput3, friend_id1);
        sum0 += ARG_3D(26,k,j  ,i)*((lane_id < 8 )? tx0: ty0);
        sum1 += ARG_3D(26,k,j+4,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        

        OUT_3D(k,j  ,i) = sum0;
        OUT_3D(k,j+4,i) = sum1;
    }
}

__global__ void Stencil_Cuda_Sweep_Shfl4(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = (((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<4) + (lane_id>>3) + halo;

    const int block_z = z / gridDim.z;
    int k = block_z * blockIdx.z + halo;
    const int k_end = k + block_z;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = ((((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<4) + (lane_id>>3))>>2; // 1x4x8, warp_ids are division of 
    DATA_TYPE tx0, ty0;
    DATA_TYPE tx1, ty1, tz1;
    DATA_TYPE tx2, ty2, tz2;
    DATA_TYPE tx3, ty3, tz3;
    int friend_id0, friend_id1;
    int friend_id2, friend_id3;
    int new_i, new_j;
    DATA_TYPE t3_threadInput0, t3_threadInput1, t3_threadInput2, t3_threadInput3, t3_threadInput4, t3_threadInput5;
    DATA_TYPE t2_threadInput0, t2_threadInput1, t2_threadInput2, t2_threadInput3, t2_threadInput4, t2_threadInput5;
    DATA_TYPE t1_threadInput0, t1_threadInput1, t1_threadInput2, t1_threadInput3, t1_threadInput4, t1_threadInput5;

#define SM_2D_M2 32 
#define SM_2D_N2 8 

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;

    // t3 is current layer; t2 is previous layer
    new_i = (warp_id_x<<3) + lane_id%10;     // 10 is extended dimension of i
    new_j = (warp_id_y<<2) + lane_id/10;     
    t3_threadInput0 = IN_3D(k  , new_j, new_i);
    t2_threadInput0 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10;
    new_j = (warp_id_y<<2) + (lane_id+32)/10;
    t3_threadInput1 = IN_3D(k  , new_j, new_i);
    t2_threadInput1 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10;
    new_j = (warp_id_y<<2) + (lane_id+64)/10;
    t3_threadInput2 = IN_3D(k  , new_j, new_i);
    t2_threadInput2 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10;
    new_j = (warp_id_y<<2) + (lane_id+96)/10;
    t3_threadInput3 = IN_3D(k  , new_j, new_i);
    t2_threadInput3 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10;
    new_j = (warp_id_y<<2) + (lane_id+128)/10;
    t3_threadInput4 = IN_3D(k  , new_j, new_i);
    t2_threadInput4 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10;
    new_j = (warp_id_y<<2) + (lane_id+160)/10;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    new_j = (new_j < m+2*halo)? new_j: m+2*halo-1;
    t3_threadInput5 = IN_3D(k  , new_j, new_i);
    t2_threadInput5 = IN_3D(k-1, new_j, new_i);

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        sum0 = 0.0;
        sum1 = 0.0;
        sum2 = 0.0;
        sum3 = 0.0;
        // move the current storage down 
        t1_threadInput0 = t2_threadInput0;
        t1_threadInput1 = t2_threadInput1;
        t1_threadInput2 = t2_threadInput2;
        t1_threadInput3 = t2_threadInput3;
        t1_threadInput4 = t2_threadInput4;
        t1_threadInput5 = t2_threadInput5;

        t2_threadInput0 = t3_threadInput0;
        t2_threadInput1 = t3_threadInput1;
        t2_threadInput2 = t3_threadInput2;
        t2_threadInput3 = t3_threadInput3;
        t2_threadInput4 = t3_threadInput4;
        t2_threadInput5 = t3_threadInput5;

        new_i = (warp_id_x<<3) + lane_id%10;  
        new_j = (warp_id_y<<2) + lane_id/10;     
        t3_threadInput0 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+32)%10;
        new_j = (warp_id_y<<2) + (lane_id+32)/10;
        t3_threadInput1 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+64)%10;
        new_j = (warp_id_y<<2) + (lane_id+64)/10;
        t3_threadInput2 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+96)%10;
        new_j = (warp_id_y<<2) + (lane_id+96)/10;
        t3_threadInput3 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+128)%10;
        new_j = (warp_id_y<<2) + (lane_id+128)/10;
        t3_threadInput4 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+160)%10;
        new_j = (warp_id_y<<2) + (lane_id+160)/10;
        new_i = (new_i < n+2*halo)? new_i: n+2*halo;
        new_j = (new_j < m+2*halo)? new_j: m+2*halo;
        t3_threadInput5 = IN_3D(k+1, new_j, new_i);


        friend_id0 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tx3 = __shfl(t1_threadInput3, friend_id3);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        sum0 += ARG_3D(0,k,j   ,i)*((lane_id < 26)? tx0: ty0);
        sum1 += ARG_3D(0,k,j+4 ,i)*((lane_id < 20)? tx1: ty1);
        sum2 += ARG_3D(0,k,j+8 ,i)*((lane_id < 14)? tx2: ty2);
        sum3 += ARG_3D(0,k,j+12,i)*((lane_id < 8 )? tx3: ty3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tx3 = __shfl(t2_threadInput3, friend_id3);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        sum0 += ARG_3D(9,k,j   ,i)*((lane_id < 26)? tx0: ty0);
        sum1 += ARG_3D(9,k,j+4 ,i)*((lane_id < 20)? tx1: ty1);
        sum2 += ARG_3D(9,k,j+8 ,i)*((lane_id < 14)? tx2: ty2);
        sum3 += ARG_3D(9,k,j+12,i)*((lane_id < 8 )? tx3: ty3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tx3 = __shfl(t3_threadInput3, friend_id3);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        sum0 += ARG_3D(18,k,j   ,i)*((lane_id < 26)? tx0: ty0);
        sum1 += ARG_3D(18,k,j+4 ,i)*((lane_id < 20)? tx1: ty1);
        sum2 += ARG_3D(18,k,j+8 ,i)*((lane_id < 14)? tx2: ty2);
        sum3 += ARG_3D(18,k,j+12,i)*((lane_id < 8 )? tx3: ty3);

        friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tx3 = __shfl(t1_threadInput3, friend_id3);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        sum0 += ARG_3D(1,k,j   ,i)*((lane_id < 25)? tx0: ty0);
        sum1 += ARG_3D(1,k,j+4 ,i)*((lane_id < 19)? tx1: ty1);
        sum2 += ARG_3D(1,k,j+8 ,i)*((lane_id < 13)? tx2: ty2);
        sum3 += ARG_3D(1,k,j+12,i)*((lane_id < 7 )? tx3: ty3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tx3 = __shfl(t2_threadInput3, friend_id3);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        sum0 += ARG_3D(10,k,j   ,i)*((lane_id < 25)? tx0: ty0);
        sum1 += ARG_3D(10,k,j+4 ,i)*((lane_id < 19)? tx1: ty1);
        sum2 += ARG_3D(10,k,j+8 ,i)*((lane_id < 13)? tx2: ty2);
        sum3 += ARG_3D(10,k,j+12,i)*((lane_id < 7 )? tx3: ty3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tx3 = __shfl(t3_threadInput3, friend_id3);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        sum0 += ARG_3D(19,k,j   ,i)*((lane_id < 25)? tx0: ty0);
        sum1 += ARG_3D(19,k,j+4 ,i)*((lane_id < 19)? tx1: ty1);
        sum2 += ARG_3D(19,k,j+8 ,i)*((lane_id < 13)? tx2: ty2);
        sum3 += ARG_3D(19,k,j+12,i)*((lane_id < 7 )? tx3: ty3);

        friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tx3 = __shfl(t1_threadInput3, friend_id3);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        sum0 += ARG_3D(2,k,j   ,i)*((lane_id < 24)? tx0: ty0);
        sum1 += ARG_3D(2,k,j+4 ,i)*((lane_id < 18)? tx1: ty1);
        sum2 += ARG_3D(2,k,j+8 ,i)*((lane_id < 12)? tx2: ty2);
        sum3 += ARG_3D(2,k,j+12,i)*((lane_id < 6 )? tx3: ty3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tx3 = __shfl(t2_threadInput3, friend_id3);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        sum0 += ARG_3D(11,k,j   ,i)*((lane_id < 24)? tx0: ty0);
        sum1 += ARG_3D(11,k,j+4 ,i)*((lane_id < 18)? tx1: ty1);
        sum2 += ARG_3D(11,k,j+8 ,i)*((lane_id < 12)? tx2: ty2);
        sum3 += ARG_3D(11,k,j+12,i)*((lane_id < 6 )? tx3: ty3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tx3 = __shfl(t3_threadInput3, friend_id3);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        sum0 += ARG_3D(20,k,j   ,i)*((lane_id < 24)? tx0: ty0);
        sum1 += ARG_3D(20,k,j+4 ,i)*((lane_id < 18)? tx1: ty1);
        sum2 += ARG_3D(20,k,j+8 ,i)*((lane_id < 12)? tx2: ty2);
        sum3 += ARG_3D(20,k,j+12,i)*((lane_id < 6 )? tx3: ty3);

        friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        tz3 = __shfl(t1_threadInput5, friend_id3);
        sum0 += ARG_3D(3,k,j   ,i)*((lane_id < 18)? tx0: ty0);
        sum1 += ARG_3D(3,k,j+4 ,i)*((lane_id < 12)? tx1: ty1);
        sum2 += ARG_3D(3,k,j+8 ,i)*((lane_id < 6 )? tx2: ty2);
        sum3 += ARG_3D(3,k,j+12,i)*((lane_id < 24)? ty3: tz3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        tz3 = __shfl(t2_threadInput5, friend_id3);
        sum0 += ARG_3D(12,k,j   ,i)*((lane_id < 18)? tx0: ty0);
        sum1 += ARG_3D(12,k,j+4 ,i)*((lane_id < 12)? tx1: ty1);
        sum2 += ARG_3D(12,k,j+8 ,i)*((lane_id < 6 )? tx2: ty2);
        sum3 += ARG_3D(12,k,j+12,i)*((lane_id < 24)? ty3: tz3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        tz3 = __shfl(t3_threadInput5, friend_id3);
        sum0 += ARG_3D(21,k,j   ,i)*((lane_id < 18)? tx0: ty0);
        sum1 += ARG_3D(21,k,j+4 ,i)*((lane_id < 12)? tx1: ty1);
        sum2 += ARG_3D(21,k,j+8 ,i)*((lane_id < 6 )? tx2: ty2);
        sum3 += ARG_3D(21,k,j+12,i)*((lane_id < 24)? ty3: tz3);

        friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tz2 = __shfl(t1_threadInput4, friend_id2);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        tz3 = __shfl(t1_threadInput5, friend_id3);
        sum0 += ARG_3D(4,k,j   ,i)*((lane_id < 17)? tx0: ty0);
        sum1 += ARG_3D(4,k,j+4 ,i)*((lane_id < 11)? tx1: ty1);
        sum2 += ARG_3D(4,k,j+8 ,i)*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
        sum3 += ARG_3D(4,k,j+12,i)*((lane_id < 24)? ty3: tz3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tz2 = __shfl(t2_threadInput4, friend_id2);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        tz3 = __shfl(t2_threadInput5, friend_id3);
        sum0 += ARG_3D(13,k,j   ,i)*((lane_id < 17)? tx0: ty0);
        sum1 += ARG_3D(13,k,j+4 ,i)*((lane_id < 11)? tx1: ty1);
        sum2 += ARG_3D(13,k,j+8 ,i)*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
        sum3 += ARG_3D(13,k,j+12,i)*((lane_id < 24)? ty3: tz3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tz2 = __shfl(t3_threadInput4, friend_id2);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        tz3 = __shfl(t3_threadInput5, friend_id3);
        sum0 += ARG_3D(22,k,j   ,i)*((lane_id < 17)? tx0: ty0);
        sum1 += ARG_3D(22,k,j+4 ,i)*((lane_id < 11)? tx1: ty1);
        sum2 += ARG_3D(22,k,j+8 ,i)*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
        sum3 += ARG_3D(22,k,j+12,i)*((lane_id < 24)? ty3: tz3);

        friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tz2 = __shfl(t1_threadInput4, friend_id2);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        tz3 = __shfl(t1_threadInput5, friend_id3);
        sum0 += ARG_3D(5,k,j   ,i)*((lane_id < 16)? tx0: ty0);
        sum1 += ARG_3D(5,k,j+4 ,i)*((lane_id < 10)? tx1: ty1);
        sum2 += ARG_3D(5,k,j+8 ,i)*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
        sum3 += ARG_3D(5,k,j+12,i)*((lane_id < 24)? ty3: tz3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tz2 = __shfl(t2_threadInput4, friend_id2);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        tz3 = __shfl(t2_threadInput5, friend_id3);
        sum0 += ARG_3D(14,k,j   ,i)*((lane_id < 16)? tx0: ty0);
        sum1 += ARG_3D(14,k,j+4 ,i)*((lane_id < 10)? tx1: ty1);
        sum2 += ARG_3D(14,k,j+8 ,i)*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
        sum3 += ARG_3D(14,k,j+12,i)*((lane_id < 24)? ty3: tz3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tz2 = __shfl(t3_threadInput4, friend_id2);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        tz3 = __shfl(t3_threadInput5, friend_id3);
        sum0 += ARG_3D(23,k,j   ,i)*((lane_id < 16)? tx0: ty0);
        sum1 += ARG_3D(23,k,j+4 ,i)*((lane_id < 10)? tx1: ty1);
        sum2 += ARG_3D(23,k,j+8 ,i)*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
        sum3 += ARG_3D(23,k,j+12,i)*((lane_id < 24)? ty3: tz3);

        friend_id0 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tz1 = __shfl(t1_threadInput3, friend_id1);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tz2 = __shfl(t1_threadInput4, friend_id2);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        tz3 = __shfl(t1_threadInput5, friend_id3);
        sum0 += ARG_3D(6,k,j   ,i)*((lane_id < 10)? tx0: ty0);
        sum1 += ARG_3D(6,k,j+4 ,i)*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        sum2 += ARG_3D(6,k,j+8 ,i)*((lane_id < 24)? ty2: tz2);
        sum3 += ARG_3D(6,k,j+12,i)*((lane_id < 16)? ty3: tz3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tz1 = __shfl(t2_threadInput3, friend_id1);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tz2 = __shfl(t2_threadInput4, friend_id2);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        tz3 = __shfl(t2_threadInput5, friend_id3);
        sum0 += ARG_3D(15,k,j   ,i)*((lane_id < 10)? tx0: ty0);
        sum1 += ARG_3D(15,k,j+4 ,i)*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        sum2 += ARG_3D(15,k,j+8 ,i)*((lane_id < 24)? ty2: tz2);
        sum3 += ARG_3D(15,k,j+12,i)*((lane_id < 16)? ty3: tz3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tz1 = __shfl(t3_threadInput3, friend_id1);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tz2 = __shfl(t3_threadInput4, friend_id2);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        tz3 = __shfl(t3_threadInput5, friend_id3);
        sum0 += ARG_3D(24,k,j   ,i)*((lane_id < 10)? tx0: ty0);
        sum1 += ARG_3D(24,k,j+4 ,i)*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        sum2 += ARG_3D(24,k,j+8 ,i)*((lane_id < 24)? ty2: tz2);
        sum3 += ARG_3D(24,k,j+12,i)*((lane_id < 16)? ty3: tz3);

        friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tz1 = __shfl(t1_threadInput3, friend_id1);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tz2 = __shfl(t1_threadInput4, friend_id2);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        tz3 = __shfl(t1_threadInput5, friend_id3);
        sum0 += ARG_3D(7,k,j   ,i)*((lane_id < 9 )? tx0: ty0);
        sum1 += ARG_3D(7,k,j+4 ,i)*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        sum2 += ARG_3D(7,k,j+8 ,i)*((lane_id < 23)? ty2: tz2);
        sum3 += ARG_3D(7,k,j+12,i)*((lane_id < 16)? ty3: tz3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tz1 = __shfl(t2_threadInput3, friend_id1);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tz2 = __shfl(t2_threadInput4, friend_id2);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        tz3 = __shfl(t2_threadInput5, friend_id3);
        sum0 += ARG_3D(16,k,j   ,i)*((lane_id < 9 )? tx0: ty0);
        sum1 += ARG_3D(16,k,j+4 ,i)*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        sum2 += ARG_3D(16,k,j+8 ,i)*((lane_id < 23)? ty2: tz2);
        sum3 += ARG_3D(16,k,j+12,i)*((lane_id < 16)? ty3: tz3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tz1 = __shfl(t3_threadInput3, friend_id1);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tz2 = __shfl(t3_threadInput4, friend_id2);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        tz3 = __shfl(t3_threadInput5, friend_id3);
        sum0 += ARG_3D(25,k,j   ,i)*((lane_id < 9 )? tx0: ty0);
        sum1 += ARG_3D(25,k,j+4 ,i)*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        sum2 += ARG_3D(25,k,j+8 ,i)*((lane_id < 23)? ty2: tz2);
        sum3 += ARG_3D(25,k,j+12,i)*((lane_id < 16)? ty3: tz3);
        
        friend_id0 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tz1 = __shfl(t1_threadInput3, friend_id1);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tz2 = __shfl(t1_threadInput4, friend_id2);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        tz3 = __shfl(t1_threadInput5, friend_id3);
        sum0 += ARG_3D(8,k,j   ,i)*((lane_id < 8 )? tx0: ty0);
        sum1 += ARG_3D(8,k,j+4 ,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        sum2 += ARG_3D(8,k,j+8 ,i)*((lane_id < 22)? ty2: tz2);
        sum3 += ARG_3D(8,k,j+12,i)*((lane_id < 16)? ty3: tz3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tz1 = __shfl(t2_threadInput3, friend_id1);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tz2 = __shfl(t2_threadInput4, friend_id2);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        tz3 = __shfl(t2_threadInput5, friend_id3);
        sum0 += ARG_3D(17,k,j   ,i)*((lane_id < 8 )? tx0: ty0);
        sum1 += ARG_3D(17,k,j+4 ,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        sum2 += ARG_3D(17,k,j+8 ,i)*((lane_id < 22)? ty2: tz2);
        sum3 += ARG_3D(17,k,j+12,i)*((lane_id < 16)? ty3: tz3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tz1 = __shfl(t3_threadInput3, friend_id1);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tz2 = __shfl(t3_threadInput4, friend_id2);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        tz3 = __shfl(t3_threadInput5, friend_id3);
        sum0 += ARG_3D(26,k,j   ,i)*((lane_id < 8 )? tx0: ty0);
        sum1 += ARG_3D(26,k,j+4 ,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        sum2 += ARG_3D(26,k,j+8 ,i)*((lane_id < 22)? ty2: tz2);
        sum3 += ARG_3D(26,k,j+12,i)*((lane_id < 16)? ty3: tz3);

        
        OUT_3D(k,j   ,i) = sum0;
        OUT_3D(k,j+4 ,i) = sum1;
        OUT_3D(k,j+8 ,i) = sum2;
        OUT_3D(k,j+12,i) = sum3;
    }
}

__global__ void Stencil_Cuda_Sweep_Shfl8(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, 
        int z, int m, int n, int halo)
{
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;
    int j = (((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<5) + (lane_id>>3) + halo;

    const int block_z = z / gridDim.z;
    int k = block_z * blockIdx.z + halo;
    const int k_end = k + block_z;
    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3; // because the warp dimensions are 
    int warp_id_y = ((((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<5) + (lane_id>>3))>>2; // 1x4x8, warp_ids are division of 
    DATA_TYPE tx0, ty0;
    DATA_TYPE tx1, ty1, tz1;
    DATA_TYPE tx2, ty2, tz2;
    DATA_TYPE tx3, ty3, tz3;
    DATA_TYPE rx0, ry0, rx1, ry1, rz1, rx2, ry2, rz2, rx3, ry3, rz3;
    int friend_id0, friend_id1;
    int friend_id2, friend_id3;
    int new_i, new_j;
    DATA_TYPE t3_threadInput0, t3_threadInput1, t3_threadInput2, t3_threadInput3, t3_threadInput4, t3_threadInput5;
    DATA_TYPE t3_threadInput6, t3_threadInput7, t3_threadInput8, t3_threadInput9, t3_threadInput10;
    DATA_TYPE t2_threadInput0, t2_threadInput1, t2_threadInput2, t2_threadInput3, t2_threadInput4, t2_threadInput5;
    DATA_TYPE t2_threadInput6, t2_threadInput7, t2_threadInput8, t2_threadInput9, t2_threadInput10;
    DATA_TYPE t1_threadInput0, t1_threadInput1, t1_threadInput2, t1_threadInput3, t1_threadInput4, t1_threadInput5;
    DATA_TYPE t1_threadInput6, t1_threadInput7, t1_threadInput8, t1_threadInput9, t1_threadInput10;

#define SM_2D_M2 32 
#define SM_2D_N2 8 

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    DATA_TYPE sum4 = 0.0;
    DATA_TYPE sum5 = 0.0;
    DATA_TYPE sum6 = 0.0;
    DATA_TYPE sum7 = 0.0;

    // t3 is current layer; t2 is previous layer
    new_i = (warp_id_x<<3) + lane_id%10;     // 10 is extended dimension of i
    new_j = (warp_id_y<<2) + lane_id/10;     
    t3_threadInput0 = IN_3D(k  , new_j, new_i);
    t2_threadInput0 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10;
    new_j = (warp_id_y<<2) + (lane_id+32)/10;
    t3_threadInput1 = IN_3D(k  , new_j, new_i);
    t2_threadInput1 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10;
    new_j = (warp_id_y<<2) + (lane_id+64)/10;
    t3_threadInput2 = IN_3D(k  , new_j, new_i);
    t2_threadInput2 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10;
    new_j = (warp_id_y<<2) + (lane_id+96)/10;
    t3_threadInput3 = IN_3D(k  , new_j, new_i);
    t2_threadInput3 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10;
    new_j = (warp_id_y<<2) + (lane_id+128)/10;
    t3_threadInput4 = IN_3D(k  , new_j, new_i);
    t2_threadInput4 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10;
    new_j = (warp_id_y<<2) + (lane_id+160)/10;
    t3_threadInput5 = IN_3D(k  , new_j, new_i);
    t2_threadInput5 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+192)%10;
    new_j = (warp_id_y<<2) + (lane_id+192)/10;
    t3_threadInput6 = IN_3D(k  , new_j, new_i);
    t2_threadInput6 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+224)%10;
    new_j = (warp_id_y<<2) + (lane_id+224)/10;
    t3_threadInput7 = IN_3D(k  , new_j, new_i);
    t2_threadInput7 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+256)%10;
    new_j = (warp_id_y<<2) + (lane_id+256)/10;
    t3_threadInput8 = IN_3D(k  , new_j, new_i);
    t2_threadInput8 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+288)%10;
    new_j = (warp_id_y<<2) + (lane_id+288)/10;
    t3_threadInput9 = IN_3D(k  , new_j, new_i);
    t2_threadInput9 = IN_3D(k-1, new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+320)%10;
    new_j = (warp_id_y<<2) + (lane_id+320)/10;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    new_j = (new_j < m+2*halo)? new_j: m+2*halo-1;
    t3_threadInput10 = IN_3D(k  , new_j, new_i);
    t2_threadInput10 = IN_3D(k-1, new_j, new_i);

#pragma unroll // it seems the loop-unroll is useless to performance
    for(; k < k_end; ++k)
    {
        sum0 = 0.0;
        sum1 = 0.0;
        sum2 = 0.0;
        sum3 = 0.0;
        sum4 = 0.0;
        sum5 = 0.0;
        sum6 = 0.0;
        sum7 = 0.0;
        // move the current storage down 
        t1_threadInput0  = t2_threadInput0 ;
        t1_threadInput1  = t2_threadInput1 ;
        t1_threadInput2  = t2_threadInput2 ;
        t1_threadInput3  = t2_threadInput3 ;
        t1_threadInput4  = t2_threadInput4 ;
        t1_threadInput5  = t2_threadInput5 ;
        t1_threadInput6  = t2_threadInput6 ;
        t1_threadInput7  = t2_threadInput7 ;
        t1_threadInput8  = t2_threadInput8 ;
        t1_threadInput9  = t2_threadInput9 ;
        t1_threadInput10 = t2_threadInput10;

        t2_threadInput0  = t3_threadInput0 ;
        t2_threadInput1  = t3_threadInput1 ;
        t2_threadInput2  = t3_threadInput2 ;
        t2_threadInput3  = t3_threadInput3 ;
        t2_threadInput4  = t3_threadInput4 ;
        t2_threadInput5  = t3_threadInput5 ;
        t2_threadInput6  = t3_threadInput6 ;
        t2_threadInput7  = t3_threadInput7 ;
        t2_threadInput8  = t3_threadInput8 ;
        t2_threadInput9  = t3_threadInput9 ;
        t2_threadInput10 = t3_threadInput10;

        new_i = (warp_id_x<<3) + lane_id%10;  
        new_j = (warp_id_y<<2) + lane_id/10;     
        t3_threadInput0 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+32)%10;
        new_j = (warp_id_y<<2) + (lane_id+32)/10;
        t3_threadInput1 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+64)%10;
        new_j = (warp_id_y<<2) + (lane_id+64)/10;
        t3_threadInput2 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+96)%10;
        new_j = (warp_id_y<<2) + (lane_id+96)/10;
        t3_threadInput3 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+128)%10;
        new_j = (warp_id_y<<2) + (lane_id+128)/10;
        t3_threadInput4 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+160)%10;
        new_j = (warp_id_y<<2) + (lane_id+160)/10;
        t3_threadInput5 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+192)%10;
        new_j = (warp_id_y<<2) + (lane_id+192)/10;
        t3_threadInput6 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+224)%10;
        new_j = (warp_id_y<<2) + (lane_id+224)/10;
        t3_threadInput7 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+256)%10;
        new_j = (warp_id_y<<2) + (lane_id+256)/10;
        t3_threadInput8 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+288)%10;
        new_j = (warp_id_y<<2) + (lane_id+288)/10;
        t3_threadInput9 = IN_3D(k+1, new_j, new_i);
        new_i = (warp_id_x<<3) + (lane_id+320)%10;
        new_j = (warp_id_y<<2) + (lane_id+320)/10;
        new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
        new_j = (new_j < m+2*halo)? new_j: m+2*halo-1;
        t3_threadInput10 = IN_3D(k+1, new_j, new_i);


        friend_id0 = (lane_id+0 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+8 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+16+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+24+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tx3 = __shfl(t1_threadInput3, friend_id3);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        rx0 = __shfl(t1_threadInput5, friend_id0);
        ry0 = __shfl(t1_threadInput6, friend_id0);
        rx1 = __shfl(t1_threadInput6, friend_id1);
        ry1 = __shfl(t1_threadInput7, friend_id1);
        rx2 = __shfl(t1_threadInput7, friend_id2);
        ry2 = __shfl(t1_threadInput8, friend_id2);
        rx3 = __shfl(t1_threadInput8, friend_id3);
        ry3 = __shfl(t1_threadInput9, friend_id3);
        sum0 += ARG_3D(0,k,j,i)*((lane_id < 26)? tx0: ty0);
        sum1 += ARG_3D(0,k,j,i)*((lane_id < 20)? tx1: ty1);
        sum2 += ARG_3D(0,k,j,i)*((lane_id < 14)? tx2: ty2);
        sum3 += ARG_3D(0,k,j,i)*((lane_id < 8 )? tx3: ty3);
        sum4 += ARG_3D(0,k,j,i)*((lane_id < 26)? rx0: ry0);
        sum5 += ARG_3D(0,k,j,i)*((lane_id < 20)? rx1: ry1);
        sum6 += ARG_3D(0,k,j,i)*((lane_id < 14)? rx2: ry2);
        sum7 += ARG_3D(0,k,j,i)*((lane_id < 8 )? rx3: ry3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tx3 = __shfl(t2_threadInput3, friend_id3);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        rx0 = __shfl(t2_threadInput5, friend_id0);
        ry0 = __shfl(t2_threadInput6, friend_id0);
        rx1 = __shfl(t2_threadInput6, friend_id1);
        ry1 = __shfl(t2_threadInput7, friend_id1);
        rx2 = __shfl(t2_threadInput7, friend_id2);
        ry2 = __shfl(t2_threadInput8, friend_id2);
        rx3 = __shfl(t2_threadInput8, friend_id3);
        ry3 = __shfl(t2_threadInput9, friend_id3);
        sum0 += ARG_3D(9,k,j,i)*((lane_id < 26)? tx0: ty0);
        sum1 += ARG_3D(9,k,j,i)*((lane_id < 20)? tx1: ty1);
        sum2 += ARG_3D(9,k,j,i)*((lane_id < 14)? tx2: ty2);
        sum3 += ARG_3D(9,k,j,i)*((lane_id < 8 )? tx3: ty3);
        sum4 += ARG_3D(9,k,j,i)*((lane_id < 26)? rx0: ry0);
        sum5 += ARG_3D(9,k,j,i)*((lane_id < 20)? rx1: ry1);
        sum6 += ARG_3D(9,k,j,i)*((lane_id < 14)? rx2: ry2);
        sum7 += ARG_3D(9,k,j,i)*((lane_id < 8 )? rx3: ry3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tx3 = __shfl(t3_threadInput3, friend_id3);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        rx0 = __shfl(t3_threadInput5, friend_id0);
        ry0 = __shfl(t3_threadInput6, friend_id0);
        rx1 = __shfl(t3_threadInput6, friend_id1);
        ry1 = __shfl(t3_threadInput7, friend_id1);
        rx2 = __shfl(t3_threadInput7, friend_id2);
        ry2 = __shfl(t3_threadInput8, friend_id2);
        rx3 = __shfl(t3_threadInput8, friend_id3);
        ry3 = __shfl(t3_threadInput9, friend_id3);
        sum0 += ARG_3D(18,k,j,i)*((lane_id < 26)? tx0: ty0);
        sum1 += ARG_3D(18,k,j,i)*((lane_id < 20)? tx1: ty1);
        sum2 += ARG_3D(18,k,j,i)*((lane_id < 14)? tx2: ty2);
        sum3 += ARG_3D(18,k,j,i)*((lane_id < 8 )? tx3: ty3);
        sum4 += ARG_3D(18,k,j,i)*((lane_id < 26)? rx0: ry0);
        sum5 += ARG_3D(18,k,j,i)*((lane_id < 20)? rx1: ry1);
        sum6 += ARG_3D(18,k,j,i)*((lane_id < 14)? rx2: ry2);
        sum7 += ARG_3D(18,k,j,i)*((lane_id < 8 )? rx3: ry3);

        friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tx3 = __shfl(t1_threadInput3, friend_id3);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        rx0 = __shfl(t1_threadInput5, friend_id0);
        ry0 = __shfl(t1_threadInput6, friend_id0);
        rx1 = __shfl(t1_threadInput6, friend_id1);
        ry1 = __shfl(t1_threadInput7, friend_id1);
        rx2 = __shfl(t1_threadInput7, friend_id2);
        ry2 = __shfl(t1_threadInput8, friend_id2);
        rx3 = __shfl(t1_threadInput8, friend_id3);
        ry3 = __shfl(t1_threadInput9, friend_id3);
        sum0 += ARG_3D(1,k,j,i)*((lane_id < 25)? tx0: ty0);
        sum1 += ARG_3D(1,k,j,i)*((lane_id < 19)? tx1: ty1);
        sum2 += ARG_3D(1,k,j,i)*((lane_id < 13)? tx2: ty2);
        sum3 += ARG_3D(1,k,j,i)*((lane_id < 7 )? tx3: ty3);
        sum4 += ARG_3D(1,k,j,i)*((lane_id < 25)? rx0: ry0);
        sum5 += ARG_3D(1,k,j,i)*((lane_id < 19)? rx1: ry1);
        sum6 += ARG_3D(1,k,j,i)*((lane_id < 13)? rx2: ry2);
        sum7 += ARG_3D(1,k,j,i)*((lane_id < 7 )? rx3: ry3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tx3 = __shfl(t2_threadInput3, friend_id3);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        rx0 = __shfl(t2_threadInput5, friend_id0);
        ry0 = __shfl(t2_threadInput6, friend_id0);
        rx1 = __shfl(t2_threadInput6, friend_id1);
        ry1 = __shfl(t2_threadInput7, friend_id1);
        rx2 = __shfl(t2_threadInput7, friend_id2);
        ry2 = __shfl(t2_threadInput8, friend_id2);
        rx3 = __shfl(t2_threadInput8, friend_id3);
        ry3 = __shfl(t2_threadInput9, friend_id3);
        sum0 += ARG_3D(10,k,j,i)*((lane_id < 25)? tx0: ty0);
        sum1 += ARG_3D(10,k,j,i)*((lane_id < 19)? tx1: ty1);
        sum2 += ARG_3D(10,k,j,i)*((lane_id < 13)? tx2: ty2);
        sum3 += ARG_3D(10,k,j,i)*((lane_id < 7 )? tx3: ty3);
        sum4 += ARG_3D(10,k,j,i)*((lane_id < 25)? rx0: ry0);
        sum5 += ARG_3D(10,k,j,i)*((lane_id < 19)? rx1: ry1);
        sum6 += ARG_3D(10,k,j,i)*((lane_id < 13)? rx2: ry2);
        sum7 += ARG_3D(10,k,j,i)*((lane_id < 7 )? rx3: ry3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tx3 = __shfl(t3_threadInput3, friend_id3);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        rx0 = __shfl(t3_threadInput5, friend_id0);
        ry0 = __shfl(t3_threadInput6, friend_id0);
        rx1 = __shfl(t3_threadInput6, friend_id1);
        ry1 = __shfl(t3_threadInput7, friend_id1);
        rx2 = __shfl(t3_threadInput7, friend_id2);
        ry2 = __shfl(t3_threadInput8, friend_id2);
        rx3 = __shfl(t3_threadInput8, friend_id3);
        ry3 = __shfl(t3_threadInput9, friend_id3);
        sum0 += ARG_3D(19,k,j,i)*((lane_id < 25)? tx0: ty0);
        sum1 += ARG_3D(19,k,j,i)*((lane_id < 19)? tx1: ty1);
        sum2 += ARG_3D(19,k,j,i)*((lane_id < 13)? tx2: ty2);
        sum3 += ARG_3D(19,k,j,i)*((lane_id < 7 )? tx3: ty3);
        sum4 += ARG_3D(19,k,j,i)*((lane_id < 25)? rx0: ry0);
        sum5 += ARG_3D(19,k,j,i)*((lane_id < 19)? rx1: ry1);
        sum6 += ARG_3D(19,k,j,i)*((lane_id < 13)? rx2: ry2);
        sum7 += ARG_3D(19,k,j,i)*((lane_id < 7 )? rx3: ry3);

        friend_id0 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0, friend_id0);
        ty0 = __shfl(t1_threadInput1, friend_id0);
        tx1 = __shfl(t1_threadInput1, friend_id1);
        ty1 = __shfl(t1_threadInput2, friend_id1);
        tx2 = __shfl(t1_threadInput2, friend_id2);
        ty2 = __shfl(t1_threadInput3, friend_id2);
        tx3 = __shfl(t1_threadInput3, friend_id3);
        ty3 = __shfl(t1_threadInput4, friend_id3);
        rx0 = __shfl(t1_threadInput5, friend_id0);
        ry0 = __shfl(t1_threadInput6, friend_id0);
        rx1 = __shfl(t1_threadInput6, friend_id1);
        ry1 = __shfl(t1_threadInput7, friend_id1);
        rx2 = __shfl(t1_threadInput7, friend_id2);
        ry2 = __shfl(t1_threadInput8, friend_id2);
        rx3 = __shfl(t1_threadInput8, friend_id3);
        ry3 = __shfl(t1_threadInput9, friend_id3);
        sum0 += ARG_3D(2,k,j,i)*((lane_id < 24)? tx0: ty0);
        sum1 += ARG_3D(2,k,j,i)*((lane_id < 18)? tx1: ty1);
        sum2 += ARG_3D(2,k,j,i)*((lane_id < 12)? tx2: ty2);
        sum3 += ARG_3D(2,k,j,i)*((lane_id < 6 )? tx3: ty3);
        sum4 += ARG_3D(2,k,j,i)*((lane_id < 24)? rx0: ry0);
        sum5 += ARG_3D(2,k,j,i)*((lane_id < 18)? rx1: ry1);
        sum6 += ARG_3D(2,k,j,i)*((lane_id < 12)? rx2: ry2);
        sum7 += ARG_3D(2,k,j,i)*((lane_id < 6 )? rx3: ry3);
        tx0 = __shfl(t2_threadInput0, friend_id0);
        ty0 = __shfl(t2_threadInput1, friend_id0);
        tx1 = __shfl(t2_threadInput1, friend_id1);
        ty1 = __shfl(t2_threadInput2, friend_id1);
        tx2 = __shfl(t2_threadInput2, friend_id2);
        ty2 = __shfl(t2_threadInput3, friend_id2);
        tx3 = __shfl(t2_threadInput3, friend_id3);
        ty3 = __shfl(t2_threadInput4, friend_id3);
        rx0 = __shfl(t2_threadInput5, friend_id0);
        ry0 = __shfl(t2_threadInput6, friend_id0);
        rx1 = __shfl(t2_threadInput6, friend_id1);
        ry1 = __shfl(t2_threadInput7, friend_id1);
        rx2 = __shfl(t2_threadInput7, friend_id2);
        ry2 = __shfl(t2_threadInput8, friend_id2);
        rx3 = __shfl(t2_threadInput8, friend_id3);
        ry3 = __shfl(t2_threadInput9, friend_id3);
        sum0 += ARG_3D(11,k,j,i)*((lane_id < 24)? tx0: ty0);
        sum1 += ARG_3D(11,k,j,i)*((lane_id < 18)? tx1: ty1);
        sum2 += ARG_3D(11,k,j,i)*((lane_id < 12)? tx2: ty2);
        sum3 += ARG_3D(11,k,j,i)*((lane_id < 6 )? tx3: ty3);
        sum4 += ARG_3D(11,k,j,i)*((lane_id < 24)? rx0: ry0);
        sum5 += ARG_3D(11,k,j,i)*((lane_id < 18)? rx1: ry1);
        sum6 += ARG_3D(11,k,j,i)*((lane_id < 12)? rx2: ry2);
        sum7 += ARG_3D(11,k,j,i)*((lane_id < 6 )? rx3: ry3);
        tx0 = __shfl(t3_threadInput0, friend_id0);
        ty0 = __shfl(t3_threadInput1, friend_id0);
        tx1 = __shfl(t3_threadInput1, friend_id1);
        ty1 = __shfl(t3_threadInput2, friend_id1);
        tx2 = __shfl(t3_threadInput2, friend_id2);
        ty2 = __shfl(t3_threadInput3, friend_id2);
        tx3 = __shfl(t3_threadInput3, friend_id3);
        ty3 = __shfl(t3_threadInput4, friend_id3);
        rx0 = __shfl(t3_threadInput5, friend_id0);
        ry0 = __shfl(t3_threadInput6, friend_id0);
        rx1 = __shfl(t3_threadInput6, friend_id1);
        ry1 = __shfl(t3_threadInput7, friend_id1);
        rx2 = __shfl(t3_threadInput7, friend_id2);
        ry2 = __shfl(t3_threadInput8, friend_id2);
        rx3 = __shfl(t3_threadInput8, friend_id3);
        ry3 = __shfl(t3_threadInput9, friend_id3);
        sum0 += ARG_3D(20,k,j,i)*((lane_id < 24)? tx0: ty0);
        sum1 += ARG_3D(20,k,j,i)*((lane_id < 18)? tx1: ty1);
        sum2 += ARG_3D(20,k,j,i)*((lane_id < 12)? tx2: ty2);
        sum3 += ARG_3D(20,k,j,i)*((lane_id < 6 )? tx3: ty3);
        sum4 += ARG_3D(20,k,j,i)*((lane_id < 24)? rx0: ry0);
        sum5 += ARG_3D(20,k,j,i)*((lane_id < 18)? rx1: ry1);
        sum6 += ARG_3D(20,k,j,i)*((lane_id < 12)? rx2: ry2);
        sum7 += ARG_3D(20,k,j,i)*((lane_id < 6 )? rx3: ry3);

        friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0 , friend_id0);
        ty0 = __shfl(t1_threadInput1 , friend_id0);
        tx1 = __shfl(t1_threadInput1 , friend_id1);
        ty1 = __shfl(t1_threadInput2 , friend_id1);
        tx2 = __shfl(t1_threadInput2 , friend_id2);
        ty2 = __shfl(t1_threadInput3 , friend_id2);
        ty3 = __shfl(t1_threadInput4 , friend_id3);
        tz3 = __shfl(t1_threadInput5 , friend_id3);
        rx0 = __shfl(t1_threadInput5 , friend_id0);
        ry0 = __shfl(t1_threadInput6 , friend_id0);
        rx1 = __shfl(t1_threadInput6 , friend_id1);
        ry1 = __shfl(t1_threadInput7 , friend_id1);
        rx2 = __shfl(t1_threadInput7 , friend_id2);
        ry2 = __shfl(t1_threadInput8 , friend_id2);
        ry3 = __shfl(t1_threadInput9 , friend_id3);
        rz3 = __shfl(t1_threadInput10, friend_id3);
        sum0 += ARG_3D(3,k,j,i)*((lane_id < 18)? tx0: ty0);
        sum1 += ARG_3D(3,k,j,i)*((lane_id < 12)? tx1: ty1);
        sum2 += ARG_3D(3,k,j,i)*((lane_id < 6 )? tx2: ty2);
        sum3 += ARG_3D(3,k,j,i)*((lane_id < 24)? ty3: tz3);
        sum4 += ARG_3D(3,k,j,i)*((lane_id < 18)? rx0: ry0);
        sum5 += ARG_3D(3,k,j,i)*((lane_id < 12)? rx1: ry1);
        sum6 += ARG_3D(3,k,j,i)*((lane_id < 6 )? rx2: ry2);
        sum7 += ARG_3D(3,k,j,i)*((lane_id < 24)? ry3: rz3);
        tx0 = __shfl(t2_threadInput0 , friend_id0);
        ty0 = __shfl(t2_threadInput1 , friend_id0);
        tx1 = __shfl(t2_threadInput1 , friend_id1);
        ty1 = __shfl(t2_threadInput2 , friend_id1);
        tx2 = __shfl(t2_threadInput2 , friend_id2);
        ty2 = __shfl(t2_threadInput3 , friend_id2);
        ty3 = __shfl(t2_threadInput4 , friend_id3);
        tz3 = __shfl(t2_threadInput5 , friend_id3);
        rx0 = __shfl(t2_threadInput5 , friend_id0);
        ry0 = __shfl(t2_threadInput6 , friend_id0);
        rx1 = __shfl(t2_threadInput6 , friend_id1);
        ry1 = __shfl(t2_threadInput7 , friend_id1);
        rx2 = __shfl(t2_threadInput7 , friend_id2);
        ry2 = __shfl(t2_threadInput8 , friend_id2);
        ry3 = __shfl(t2_threadInput9 , friend_id3);
        rz3 = __shfl(t2_threadInput10, friend_id3);
        sum0 += ARG_3D(12,k,j,i)*((lane_id < 18)? tx0: ty0);
        sum1 += ARG_3D(12,k,j,i)*((lane_id < 12)? tx1: ty1);
        sum2 += ARG_3D(12,k,j,i)*((lane_id < 6 )? tx2: ty2);
        sum3 += ARG_3D(12,k,j,i)*((lane_id < 24)? ty3: tz3);
        sum4 += ARG_3D(12,k,j,i)*((lane_id < 18)? rx0: ry0);
        sum5 += ARG_3D(12,k,j,i)*((lane_id < 12)? rx1: ry1);
        sum6 += ARG_3D(12,k,j,i)*((lane_id < 6 )? rx2: ry2);
        sum7 += ARG_3D(12,k,j,i)*((lane_id < 24)? ry3: rz3);
        tx0 = __shfl(t3_threadInput0 , friend_id0);
        ty0 = __shfl(t3_threadInput1 , friend_id0);
        tx1 = __shfl(t3_threadInput1 , friend_id1);
        ty1 = __shfl(t3_threadInput2 , friend_id1);
        tx2 = __shfl(t3_threadInput2 , friend_id2);
        ty2 = __shfl(t3_threadInput3 , friend_id2);
        ty3 = __shfl(t3_threadInput4 , friend_id3);
        tz3 = __shfl(t3_threadInput5 , friend_id3);
        rx0 = __shfl(t3_threadInput5 , friend_id0);
        ry0 = __shfl(t3_threadInput6 , friend_id0);
        rx1 = __shfl(t3_threadInput6 , friend_id1);
        ry1 = __shfl(t3_threadInput7 , friend_id1);
        rx2 = __shfl(t3_threadInput7 , friend_id2);
        ry2 = __shfl(t3_threadInput8 , friend_id2);
        ry3 = __shfl(t3_threadInput9 , friend_id3);
        rz3 = __shfl(t3_threadInput10, friend_id3);
        sum0 += ARG_3D(21,k,j,i)*((lane_id < 18)? tx0: ty0);
        sum1 += ARG_3D(21,k,j,i)*((lane_id < 12)? tx1: ty1);
        sum2 += ARG_3D(21,k,j,i)*((lane_id < 6 )? tx2: ty2);
        sum3 += ARG_3D(21,k,j,i)*((lane_id < 24)? ty3: tz3);
        sum4 += ARG_3D(21,k,j,i)*((lane_id < 18)? rx0: ry0);
        sum5 += ARG_3D(21,k,j,i)*((lane_id < 12)? rx1: ry1);
        sum6 += ARG_3D(21,k,j,i)*((lane_id < 6 )? rx2: ry2);
        sum7 += ARG_3D(21,k,j,i)*((lane_id < 24)? ry3: rz3);

        friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0 , friend_id0);
        ty0 = __shfl(t1_threadInput1 , friend_id0);
        tx1 = __shfl(t1_threadInput1 , friend_id1);
        ty1 = __shfl(t1_threadInput2 , friend_id1);
        tx2 = __shfl(t1_threadInput2 , friend_id2);
        ty2 = __shfl(t1_threadInput3 , friend_id2);
        tz2 = __shfl(t1_threadInput4 , friend_id2);
        ty3 = __shfl(t1_threadInput4 , friend_id3);
        tz3 = __shfl(t1_threadInput5 , friend_id3);
        rx0 = __shfl(t1_threadInput5 , friend_id0);
        ry0 = __shfl(t1_threadInput6 , friend_id0);
        rx1 = __shfl(t1_threadInput6 , friend_id1);
        ry1 = __shfl(t1_threadInput7 , friend_id1);
        rx2 = __shfl(t1_threadInput7 , friend_id2);
        ry2 = __shfl(t1_threadInput8 , friend_id2);
        rz2 = __shfl(t1_threadInput9 , friend_id2);
        ry3 = __shfl(t1_threadInput9 , friend_id3);
        rz3 = __shfl(t1_threadInput10, friend_id3);
        sum0 += ARG_3D(4,k,j,i)*((lane_id < 17)? tx0: ty0);
        sum1 += ARG_3D(4,k,j,i)*((lane_id < 11)? tx1: ty1);
        sum2 += ARG_3D(4,k,j,i)*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
        sum3 += ARG_3D(4,k,j,i)*((lane_id < 24)? ty3: tz3);
        sum4 += ARG_3D(4,k,j,i)*((lane_id < 17)? rx0: ry0);
        sum5 += ARG_3D(4,k,j,i)*((lane_id < 11)? rx1: ry1);
        sum6 += ARG_3D(4,k,j,i)*((lane_id < 5 )? rx2: ((lane_id < 31)? ry2: rz2));
        sum7 += ARG_3D(4,k,j,i)*((lane_id < 24)? ry3: rz3);
        tx0 = __shfl(t2_threadInput0 , friend_id0);
        ty0 = __shfl(t2_threadInput1 , friend_id0);
        tx1 = __shfl(t2_threadInput1 , friend_id1);
        ty1 = __shfl(t2_threadInput2 , friend_id1);
        tx2 = __shfl(t2_threadInput2 , friend_id2);
        ty2 = __shfl(t2_threadInput3 , friend_id2);
        tz2 = __shfl(t2_threadInput4 , friend_id2);
        ty3 = __shfl(t2_threadInput4 , friend_id3);
        tz3 = __shfl(t2_threadInput5 , friend_id3);
        rx0 = __shfl(t2_threadInput5 , friend_id0);
        ry0 = __shfl(t2_threadInput6 , friend_id0);
        rx1 = __shfl(t2_threadInput6 , friend_id1);
        ry1 = __shfl(t2_threadInput7 , friend_id1);
        rx2 = __shfl(t2_threadInput7 , friend_id2);
        ry2 = __shfl(t2_threadInput8 , friend_id2);
        rz2 = __shfl(t2_threadInput9 , friend_id2);
        ry3 = __shfl(t2_threadInput9 , friend_id3);
        rz3 = __shfl(t2_threadInput10, friend_id3);
        sum0 += ARG_3D(13,k,j,i)*((lane_id < 17)? tx0: ty0);
        sum1 += ARG_3D(13,k,j,i)*((lane_id < 11)? tx1: ty1);
        sum2 += ARG_3D(13,k,j,i)*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
        sum3 += ARG_3D(13,k,j,i)*((lane_id < 24)? ty3: tz3);
        sum4 += ARG_3D(13,k,j,i)*((lane_id < 17)? rx0: ry0);
        sum5 += ARG_3D(13,k,j,i)*((lane_id < 11)? rx1: ry1);
        sum6 += ARG_3D(13,k,j,i)*((lane_id < 5 )? rx2: ((lane_id < 31)? ry2: rz2));
        sum7 += ARG_3D(13,k,j,i)*((lane_id < 24)? ry3: rz3);
        tx0 = __shfl(t3_threadInput0 , friend_id0);
        ty0 = __shfl(t3_threadInput1 , friend_id0);
        tx1 = __shfl(t3_threadInput1 , friend_id1);
        ty1 = __shfl(t3_threadInput2 , friend_id1);
        tx2 = __shfl(t3_threadInput2 , friend_id2);
        ty2 = __shfl(t3_threadInput3 , friend_id2);
        tz2 = __shfl(t3_threadInput4 , friend_id2);
        ty3 = __shfl(t3_threadInput4 , friend_id3);
        tz3 = __shfl(t3_threadInput5 , friend_id3);
        rx0 = __shfl(t3_threadInput5 , friend_id0);
        ry0 = __shfl(t3_threadInput6 , friend_id0);
        rx1 = __shfl(t3_threadInput6 , friend_id1);
        ry1 = __shfl(t3_threadInput7 , friend_id1);
        rx2 = __shfl(t3_threadInput7 , friend_id2);
        ry2 = __shfl(t3_threadInput8 , friend_id2);
        rz2 = __shfl(t3_threadInput9 , friend_id2);
        ry3 = __shfl(t3_threadInput9 , friend_id3);
        rz3 = __shfl(t3_threadInput10, friend_id3);
        sum0 += ARG_3D(22,k,j,i)*((lane_id < 17)? tx0: ty0);
        sum1 += ARG_3D(22,k,j,i)*((lane_id < 11)? tx1: ty1);
        sum2 += ARG_3D(22,k,j,i)*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
        sum3 += ARG_3D(22,k,j,i)*((lane_id < 24)? ty3: tz3);
        sum4 += ARG_3D(22,k,j,i)*((lane_id < 17)? rx0: ry0);
        sum5 += ARG_3D(22,k,j,i)*((lane_id < 11)? rx1: ry1);
        sum6 += ARG_3D(22,k,j,i)*((lane_id < 5 )? rx2: ((lane_id < 31)? ry2: rz2));
        sum7 += ARG_3D(22,k,j,i)*((lane_id < 24)? ry3: rz3);

        friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0 , friend_id0);
        ty0 = __shfl(t1_threadInput1 , friend_id0);
        tx1 = __shfl(t1_threadInput1 , friend_id1);
        ty1 = __shfl(t1_threadInput2 , friend_id1);
        tx2 = __shfl(t1_threadInput2 , friend_id2);
        ty2 = __shfl(t1_threadInput3 , friend_id2);
        tz2 = __shfl(t1_threadInput4 , friend_id2);
        ty3 = __shfl(t1_threadInput4 , friend_id3);
        tz3 = __shfl(t1_threadInput5 , friend_id3);
        rx0 = __shfl(t1_threadInput5 , friend_id0);
        ry0 = __shfl(t1_threadInput6 , friend_id0);
        rx1 = __shfl(t1_threadInput6 , friend_id1);
        ry1 = __shfl(t1_threadInput7 , friend_id1);
        rx2 = __shfl(t1_threadInput7 , friend_id2);
        ry2 = __shfl(t1_threadInput8 , friend_id2);
        rz2 = __shfl(t1_threadInput9 , friend_id2);
        ry3 = __shfl(t1_threadInput9 , friend_id3);
        rz3 = __shfl(t1_threadInput10, friend_id3);
        sum0 += ARG_3D(5,k,j,i)*((lane_id < 16)? tx0: ty0);
        sum1 += ARG_3D(5,k,j,i)*((lane_id < 10)? tx1: ty1);
        sum2 += ARG_3D(5,k,j,i)*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
        sum3 += ARG_3D(5,k,j,i)*((lane_id < 24)? ty3: tz3);
        sum4 += ARG_3D(5,k,j,i)*((lane_id < 16)? rx0: ry0);
        sum5 += ARG_3D(5,k,j,i)*((lane_id < 10)? rx1: ry1);
        sum6 += ARG_3D(5,k,j,i)*((lane_id < 4 )? rx2: ((lane_id < 30)? ry2: rz2));
        sum7 += ARG_3D(5,k,j,i)*((lane_id < 24)? ry3: rz3);
        tx0 = __shfl(t2_threadInput0 , friend_id0);
        ty0 = __shfl(t2_threadInput1 , friend_id0);
        tx1 = __shfl(t2_threadInput1 , friend_id1);
        ty1 = __shfl(t2_threadInput2 , friend_id1);
        tx2 = __shfl(t2_threadInput2 , friend_id2);
        ty2 = __shfl(t2_threadInput3 , friend_id2);
        tz2 = __shfl(t2_threadInput4 , friend_id2);
        ty3 = __shfl(t2_threadInput4 , friend_id3);
        tz3 = __shfl(t2_threadInput5 , friend_id3);
        rx0 = __shfl(t2_threadInput5 , friend_id0);
        ry0 = __shfl(t2_threadInput6 , friend_id0);
        rx1 = __shfl(t2_threadInput6 , friend_id1);
        ry1 = __shfl(t2_threadInput7 , friend_id1);
        rx2 = __shfl(t2_threadInput7 , friend_id2);
        ry2 = __shfl(t2_threadInput8 , friend_id2);
        rz2 = __shfl(t2_threadInput9 , friend_id2);
        ry3 = __shfl(t2_threadInput9 , friend_id3);
        rz3 = __shfl(t2_threadInput10, friend_id3);
        sum0 += ARG_3D(14,k,j,i)*((lane_id < 16)? tx0: ty0);
        sum1 += ARG_3D(14,k,j,i)*((lane_id < 10)? tx1: ty1);
        sum2 += ARG_3D(14,k,j,i)*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
        sum3 += ARG_3D(14,k,j,i)*((lane_id < 24)? ty3: tz3);
        sum4 += ARG_3D(14,k,j,i)*((lane_id < 16)? rx0: ry0);
        sum5 += ARG_3D(14,k,j,i)*((lane_id < 10)? rx1: ry1);
        sum6 += ARG_3D(14,k,j,i)*((lane_id < 4 )? rx2: ((lane_id < 30)? ry2: rz2));
        sum7 += ARG_3D(14,k,j,i)*((lane_id < 24)? ry3: rz3);
        tx0 = __shfl(t3_threadInput0 , friend_id0);
        ty0 = __shfl(t3_threadInput1 , friend_id0);
        tx1 = __shfl(t3_threadInput1 , friend_id1);
        ty1 = __shfl(t3_threadInput2 , friend_id1);
        tx2 = __shfl(t3_threadInput2 , friend_id2);
        ty2 = __shfl(t3_threadInput3 , friend_id2);
        tz2 = __shfl(t3_threadInput4 , friend_id2);
        ty3 = __shfl(t3_threadInput4 , friend_id3);
        tz3 = __shfl(t3_threadInput5 , friend_id3);
        rx0 = __shfl(t3_threadInput5 , friend_id0);
        ry0 = __shfl(t3_threadInput6 , friend_id0);
        rx1 = __shfl(t3_threadInput6 , friend_id1);
        ry1 = __shfl(t3_threadInput7 , friend_id1);
        rx2 = __shfl(t3_threadInput7 , friend_id2);
        ry2 = __shfl(t3_threadInput8 , friend_id2);
        rz2 = __shfl(t3_threadInput9 , friend_id2);
        ry3 = __shfl(t3_threadInput9 , friend_id3);
        rz3 = __shfl(t3_threadInput10, friend_id3);
        sum0 += ARG_3D(23,k,j,i)*((lane_id < 16)? tx0: ty0);
        sum1 += ARG_3D(23,k,j,i)*((lane_id < 10)? tx1: ty1);
        sum2 += ARG_3D(23,k,j,i)*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
        sum3 += ARG_3D(23,k,j,i)*((lane_id < 24)? ty3: tz3);
        sum4 += ARG_3D(23,k,j,i)*((lane_id < 16)? rx0: ry0);
        sum5 += ARG_3D(23,k,j,i)*((lane_id < 10)? rx1: ry1);
        sum6 += ARG_3D(23,k,j,i)*((lane_id < 4 )? rx2: ((lane_id < 30)? ry2: rz2));
        sum7 += ARG_3D(23,k,j,i)*((lane_id < 24)? ry3: rz3);

        friend_id0 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0 , friend_id0);
        ty0 = __shfl(t1_threadInput1 , friend_id0);
        tx1 = __shfl(t1_threadInput1 , friend_id1);
        ty1 = __shfl(t1_threadInput2 , friend_id1);
        tz1 = __shfl(t1_threadInput3 , friend_id1);
        ty2 = __shfl(t1_threadInput3 , friend_id2);
        tz2 = __shfl(t1_threadInput4 , friend_id2);
        ty3 = __shfl(t1_threadInput4 , friend_id3);
        tz3 = __shfl(t1_threadInput5 , friend_id3);
        rx0 = __shfl(t1_threadInput5 , friend_id0);
        ry0 = __shfl(t1_threadInput6 , friend_id0);
        rx1 = __shfl(t1_threadInput6 , friend_id1);
        ry1 = __shfl(t1_threadInput7 , friend_id1);
        rz1 = __shfl(t1_threadInput8 , friend_id1);
        ry2 = __shfl(t1_threadInput8 , friend_id2);
        rz2 = __shfl(t1_threadInput9 , friend_id2);
        ry3 = __shfl(t1_threadInput9 , friend_id3);
        rz3 = __shfl(t1_threadInput10, friend_id3);
        sum0 += ARG_3D(6,k,j,i)*((lane_id < 10)? tx0: ty0);
        sum1 += ARG_3D(6,k,j,i)*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        sum2 += ARG_3D(6,k,j,i)*((lane_id < 24)? ty2: tz2);
        sum3 += ARG_3D(6,k,j,i)*((lane_id < 16)? ty3: tz3);
        sum4 += ARG_3D(6,k,j,i)*((lane_id < 10)? rx0: ry0);
        sum5 += ARG_3D(6,k,j,i)*((lane_id < 4 )? rx1: ((lane_id < 30)? ry1: rz1));
        sum6 += ARG_3D(6,k,j,i)*((lane_id < 24)? ry2: rz2);
        sum7 += ARG_3D(6,k,j,i)*((lane_id < 16)? ry3: rz3);
        tx0 = __shfl(t2_threadInput0 , friend_id0);
        ty0 = __shfl(t2_threadInput1 , friend_id0);
        tx1 = __shfl(t2_threadInput1 , friend_id1);
        ty1 = __shfl(t2_threadInput2 , friend_id1);
        tz1 = __shfl(t2_threadInput3 , friend_id1);
        ty2 = __shfl(t2_threadInput3 , friend_id2);
        tz2 = __shfl(t2_threadInput4 , friend_id2);
        ty3 = __shfl(t2_threadInput4 , friend_id3);
        tz3 = __shfl(t2_threadInput5 , friend_id3);
        rx0 = __shfl(t2_threadInput5 , friend_id0);
        ry0 = __shfl(t2_threadInput6 , friend_id0);
        rx1 = __shfl(t2_threadInput6 , friend_id1);
        ry1 = __shfl(t2_threadInput7 , friend_id1);
        rz1 = __shfl(t2_threadInput8 , friend_id1);
        ry2 = __shfl(t2_threadInput8 , friend_id2);
        rz2 = __shfl(t2_threadInput9 , friend_id2);
        ry3 = __shfl(t2_threadInput9 , friend_id3);
        rz3 = __shfl(t2_threadInput10, friend_id3);
        sum0 += ARG_3D(15,k,j,i)*((lane_id < 10)? tx0: ty0);
        sum1 += ARG_3D(15,k,j,i)*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        sum2 += ARG_3D(15,k,j,i)*((lane_id < 24)? ty2: tz2);
        sum3 += ARG_3D(15,k,j,i)*((lane_id < 16)? ty3: tz3);
        sum4 += ARG_3D(15,k,j,i)*((lane_id < 10)? rx0: ry0);
        sum5 += ARG_3D(15,k,j,i)*((lane_id < 4 )? rx1: ((lane_id < 30)? ry1: rz1));
        sum6 += ARG_3D(15,k,j,i)*((lane_id < 24)? ry2: rz2);
        sum7 += ARG_3D(15,k,j,i)*((lane_id < 16)? ry3: rz3);
        tx0 = __shfl(t3_threadInput0 , friend_id0);
        ty0 = __shfl(t3_threadInput1 , friend_id0);
        tx1 = __shfl(t3_threadInput1 , friend_id1);
        ty1 = __shfl(t3_threadInput2 , friend_id1);
        tz1 = __shfl(t3_threadInput3 , friend_id1);
        ty2 = __shfl(t3_threadInput3 , friend_id2);
        tz2 = __shfl(t3_threadInput4 , friend_id2);
        ty3 = __shfl(t3_threadInput4 , friend_id3);
        tz3 = __shfl(t3_threadInput5 , friend_id3);
        rx0 = __shfl(t3_threadInput5 , friend_id0);
        ry0 = __shfl(t3_threadInput6 , friend_id0);
        rx1 = __shfl(t3_threadInput6 , friend_id1);
        ry1 = __shfl(t3_threadInput7 , friend_id1);
        rz1 = __shfl(t3_threadInput8 , friend_id1);
        ry2 = __shfl(t3_threadInput8 , friend_id2);
        rz2 = __shfl(t3_threadInput9 , friend_id2);
        ry3 = __shfl(t3_threadInput9 , friend_id3);
        rz3 = __shfl(t3_threadInput10, friend_id3);
        sum0 += ARG_3D(24,k,j,i)*((lane_id < 10)? tx0: ty0);
        sum1 += ARG_3D(24,k,j,i)*((lane_id < 4 )? tx1: ((lane_id < 30)? ty1: tz1));
        sum2 += ARG_3D(24,k,j,i)*((lane_id < 24)? ty2: tz2);
        sum3 += ARG_3D(24,k,j,i)*((lane_id < 16)? ty3: tz3);
        sum4 += ARG_3D(24,k,j,i)*((lane_id < 10)? rx0: ry0);
        sum5 += ARG_3D(24,k,j,i)*((lane_id < 4 )? rx1: ((lane_id < 30)? ry1: rz1));
        sum6 += ARG_3D(24,k,j,i)*((lane_id < 24)? ry2: rz2);
        sum7 += ARG_3D(24,k,j,i)*((lane_id < 16)? ry3: rz3);

        friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0 , friend_id0);
        ty0 = __shfl(t1_threadInput1 , friend_id0);
        tx1 = __shfl(t1_threadInput1 , friend_id1);
        ty1 = __shfl(t1_threadInput2 , friend_id1);
        tz1 = __shfl(t1_threadInput3 , friend_id1);
        ty2 = __shfl(t1_threadInput3 , friend_id2);
        tz2 = __shfl(t1_threadInput4 , friend_id2);
        ty3 = __shfl(t1_threadInput4 , friend_id3);
        tz3 = __shfl(t1_threadInput5 , friend_id3);
        rx0 = __shfl(t1_threadInput5 , friend_id0);
        ry0 = __shfl(t1_threadInput6 , friend_id0);
        rx1 = __shfl(t1_threadInput6 , friend_id1);
        ry1 = __shfl(t1_threadInput7 , friend_id1);
        rz1 = __shfl(t1_threadInput8 , friend_id1);
        ry2 = __shfl(t1_threadInput8 , friend_id2);
        rz2 = __shfl(t1_threadInput9 , friend_id2);
        ry3 = __shfl(t1_threadInput9 , friend_id3);
        rz3 = __shfl(t1_threadInput10, friend_id3);
        sum0 += ARG_3D(7,k,j,i)*((lane_id < 9 )? tx0: ty0);
        sum1 += ARG_3D(7,k,j,i)*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        sum2 += ARG_3D(7,k,j,i)*((lane_id < 23)? ty2: tz2);
        sum3 += ARG_3D(7,k,j,i)*((lane_id < 16)? ty3: tz3);
        sum4 += ARG_3D(7,k,j,i)*((lane_id < 9 )? rx0: ry0);
        sum5 += ARG_3D(7,k,j,i)*((lane_id < 3 )? rx1: ((lane_id < 29)? ry1: rz1));
        sum6 += ARG_3D(7,k,j,i)*((lane_id < 23)? ry2: rz2);
        sum7 += ARG_3D(7,k,j,i)*((lane_id < 16)? ry3: rz3);
        tx0 = __shfl(t2_threadInput0 , friend_id0);
        ty0 = __shfl(t2_threadInput1 , friend_id0);
        tx1 = __shfl(t2_threadInput1 , friend_id1);
        ty1 = __shfl(t2_threadInput2 , friend_id1);
        tz1 = __shfl(t2_threadInput3 , friend_id1);
        ty2 = __shfl(t2_threadInput3 , friend_id2);
        tz2 = __shfl(t2_threadInput4 , friend_id2);
        ty3 = __shfl(t2_threadInput4 , friend_id3);
        tz3 = __shfl(t2_threadInput5 , friend_id3);
        rx0 = __shfl(t2_threadInput5 , friend_id0);
        ry0 = __shfl(t2_threadInput6 , friend_id0);
        rx1 = __shfl(t2_threadInput6 , friend_id1);
        ry1 = __shfl(t2_threadInput7 , friend_id1);
        rz1 = __shfl(t2_threadInput8 , friend_id1);
        ry2 = __shfl(t2_threadInput8 , friend_id2);
        rz2 = __shfl(t2_threadInput9 , friend_id2);
        ry3 = __shfl(t2_threadInput9 , friend_id3);
        rz3 = __shfl(t2_threadInput10, friend_id3);
        sum0 += ARG_3D(16,k,j,i)*((lane_id < 9 )? tx0: ty0);
        sum1 += ARG_3D(16,k,j,i)*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        sum2 += ARG_3D(16,k,j,i)*((lane_id < 23)? ty2: tz2);
        sum3 += ARG_3D(16,k,j,i)*((lane_id < 16)? ty3: tz3);
        sum4 += ARG_3D(16,k,j,i)*((lane_id < 9 )? rx0: ry0);
        sum5 += ARG_3D(16,k,j,i)*((lane_id < 3 )? rx1: ((lane_id < 29)? ry1: rz1));
        sum6 += ARG_3D(16,k,j,i)*((lane_id < 23)? ry2: rz2);
        sum7 += ARG_3D(16,k,j,i)*((lane_id < 16)? ry3: rz3);
        tx0 = __shfl(t3_threadInput0 , friend_id0);
        ty0 = __shfl(t3_threadInput1 , friend_id0);
        tx1 = __shfl(t3_threadInput1 , friend_id1);
        ty1 = __shfl(t3_threadInput2 , friend_id1);
        tz1 = __shfl(t3_threadInput3 , friend_id1);
        ty2 = __shfl(t3_threadInput3 , friend_id2);
        tz2 = __shfl(t3_threadInput4 , friend_id2);
        ty3 = __shfl(t3_threadInput4 , friend_id3);
        tz3 = __shfl(t3_threadInput5 , friend_id3);
        rx0 = __shfl(t3_threadInput5 , friend_id0);
        ry0 = __shfl(t3_threadInput6 , friend_id0);
        rx1 = __shfl(t3_threadInput6 , friend_id1);
        ry1 = __shfl(t3_threadInput7 , friend_id1);
        rz1 = __shfl(t3_threadInput8 , friend_id1);
        ry2 = __shfl(t3_threadInput8 , friend_id2);
        rz2 = __shfl(t3_threadInput9 , friend_id2);
        ry3 = __shfl(t3_threadInput9 , friend_id3);
        rz3 = __shfl(t3_threadInput10, friend_id3);
        sum0 += ARG_3D(25,k,j,i)*((lane_id < 9 )? tx0: ty0);
        sum1 += ARG_3D(25,k,j,i)*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
        sum2 += ARG_3D(25,k,j,i)*((lane_id < 23)? ty2: tz2);
        sum3 += ARG_3D(25,k,j,i)*((lane_id < 16)? ty3: tz3);
        sum4 += ARG_3D(25,k,j,i)*((lane_id < 9 )? rx0: ry0);
        sum5 += ARG_3D(25,k,j,i)*((lane_id < 3 )? rx1: ((lane_id < 29)? ry1: rz1));
        sum6 += ARG_3D(25,k,j,i)*((lane_id < 23)? ry2: rz2);
        sum7 += ARG_3D(25,k,j,i)*((lane_id < 16)? ry3: rz3);
        
        friend_id0 = (lane_id+22+((lane_id>>3)<<1))&(warpSize-1);
        friend_id1 = (lane_id+30+((lane_id>>3)<<1))&(warpSize-1);
        friend_id2 = (lane_id+6 +((lane_id>>3)<<1))&(warpSize-1);
        friend_id3 = (lane_id+14+((lane_id>>3)<<1))&(warpSize-1);
        tx0 = __shfl(t1_threadInput0 , friend_id0);
        ty0 = __shfl(t1_threadInput1 , friend_id0);
        tx1 = __shfl(t1_threadInput1 , friend_id1);
        ty1 = __shfl(t1_threadInput2 , friend_id1);
        tz1 = __shfl(t1_threadInput3 , friend_id1);
        ty2 = __shfl(t1_threadInput3 , friend_id2);
        tz2 = __shfl(t1_threadInput4 , friend_id2);
        ty3 = __shfl(t1_threadInput4 , friend_id3);
        tz3 = __shfl(t1_threadInput5 , friend_id3);
        rx0 = __shfl(t1_threadInput5 , friend_id0);
        ry0 = __shfl(t1_threadInput6 , friend_id0);
        rx1 = __shfl(t1_threadInput6 , friend_id1);
        ry1 = __shfl(t1_threadInput7 , friend_id1);
        rz1 = __shfl(t1_threadInput8 , friend_id1);
        ry2 = __shfl(t1_threadInput8 , friend_id2);
        rz2 = __shfl(t1_threadInput9 , friend_id2);
        ry3 = __shfl(t1_threadInput9 , friend_id3);
        rz3 = __shfl(t1_threadInput10, friend_id3);
        sum0 += ARG_3D(8,k,j,i)*((lane_id < 8 )? tx0: ty0);
        sum1 += ARG_3D(8,k,j,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        sum2 += ARG_3D(8,k,j,i)*((lane_id < 22)? ty2: tz2);
        sum3 += ARG_3D(8,k,j,i)*((lane_id < 16)? ty3: tz3);
        sum4 += ARG_3D(8,k,j,i)*((lane_id < 8 )? rx0: ry0);
        sum5 += ARG_3D(8,k,j,i)*((lane_id < 2 )? rx1: ((lane_id < 28)? ry1: rz1));
        sum6 += ARG_3D(8,k,j,i)*((lane_id < 22)? ry2: rz2);
        sum7 += ARG_3D(8,k,j,i)*((lane_id < 16)? ry3: rz3);
        tx0 = __shfl(t2_threadInput0 , friend_id0);
        ty0 = __shfl(t2_threadInput1 , friend_id0);
        tx1 = __shfl(t2_threadInput1 , friend_id1);
        ty1 = __shfl(t2_threadInput2 , friend_id1);
        tz1 = __shfl(t2_threadInput3 , friend_id1);
        ty2 = __shfl(t2_threadInput3 , friend_id2);
        tz2 = __shfl(t2_threadInput4 , friend_id2);
        ty3 = __shfl(t2_threadInput4 , friend_id3);
        tz3 = __shfl(t2_threadInput5 , friend_id3);
        rx0 = __shfl(t2_threadInput5 , friend_id0);
        ry0 = __shfl(t2_threadInput6 , friend_id0);
        rx1 = __shfl(t2_threadInput6 , friend_id1);
        ry1 = __shfl(t2_threadInput7 , friend_id1);
        rz1 = __shfl(t2_threadInput8 , friend_id1);
        ry2 = __shfl(t2_threadInput8 , friend_id2);
        rz2 = __shfl(t2_threadInput9 , friend_id2);
        ry3 = __shfl(t2_threadInput9 , friend_id3);
        rz3 = __shfl(t2_threadInput10, friend_id3);
        sum0 += ARG_3D(17,k,j,i)*((lane_id < 8 )? tx0: ty0);
        sum1 += ARG_3D(17,k,j,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        sum2 += ARG_3D(17,k,j,i)*((lane_id < 22)? ty2: tz2);
        sum3 += ARG_3D(17,k,j,i)*((lane_id < 16)? ty3: tz3);
        sum4 += ARG_3D(17,k,j,i)*((lane_id < 8 )? rx0: ry0);
        sum5 += ARG_3D(17,k,j,i)*((lane_id < 2 )? rx1: ((lane_id < 28)? ry1: rz1));
        sum6 += ARG_3D(17,k,j,i)*((lane_id < 22)? ry2: rz2);
        sum7 += ARG_3D(17,k,j,i)*((lane_id < 16)? ry3: rz3);
        tx0 = __shfl(t3_threadInput0 , friend_id0);
        ty0 = __shfl(t3_threadInput1 , friend_id0);
        tx1 = __shfl(t3_threadInput1 , friend_id1);
        ty1 = __shfl(t3_threadInput2 , friend_id1);
        tz1 = __shfl(t3_threadInput3 , friend_id1);
        ty2 = __shfl(t3_threadInput3 , friend_id2);
        tz2 = __shfl(t3_threadInput4 , friend_id2);
        ty3 = __shfl(t3_threadInput4 , friend_id3);
        tz3 = __shfl(t3_threadInput5 , friend_id3);
        rx0 = __shfl(t3_threadInput5 , friend_id0);
        ry0 = __shfl(t3_threadInput6 , friend_id0);
        rx1 = __shfl(t3_threadInput6 , friend_id1);
        ry1 = __shfl(t3_threadInput7 , friend_id1);
        rz1 = __shfl(t3_threadInput8 , friend_id1);
        ry2 = __shfl(t3_threadInput8 , friend_id2);
        rz2 = __shfl(t3_threadInput9 , friend_id2);
        ry3 = __shfl(t3_threadInput9 , friend_id3);
        rz3 = __shfl(t3_threadInput10, friend_id3);
        sum0 += ARG_3D(26,k,j,i)*((lane_id < 8 )? tx0: ty0);
        sum1 += ARG_3D(26,k,j,i)*((lane_id < 2 )? tx1: ((lane_id < 28)? ty1: tz1));
        sum2 += ARG_3D(26,k,j,i)*((lane_id < 22)? ty2: tz2);
        sum3 += ARG_3D(26,k,j,i)*((lane_id < 16)? ty3: tz3);
        sum4 += ARG_3D(26,k,j,i)*((lane_id < 8 )? rx0: ry0);
        sum5 += ARG_3D(26,k,j,i)*((lane_id < 2 )? rx1: ((lane_id < 28)? ry1: rz1));
        sum6 += ARG_3D(26,k,j,i)*((lane_id < 22)? ry2: rz2);
        sum7 += ARG_3D(26,k,j,i)*((lane_id < 16)? ry3: rz3);

        
        OUT_3D(k,j   ,i) = sum0;
        OUT_3D(k,j+4 ,i) = sum1;
        OUT_3D(k,j+8 ,i) = sum2;
        OUT_3D(k,j+12,i) = sum3;
        OUT_3D(k,j+16,i) = sum4;
        OUT_3D(k,j+20,i) = sum5;
        OUT_3D(k,j+24,i) = sum6;
        OUT_3D(k,j+28,i) = sum7;
}
}int main(int argc, char **argv)
{
#ifdef __DEBUG
    int z = 64;
    int m = 8;
    int n = 8;
#else
    int z = 256; 
    int m = 256;
    int n = 256; 
#endif
    int halo = 1;
    int total = (z+2*halo)*(m+2*halo)*(n+2*halo);

    const int K = 27;
#ifdef __DEBUG
    DATA_TYPE args[K] = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
                         1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0,  
                         1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
#else
    DATA_TYPE args[K] = {0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 
                         0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037,  
                         0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037, 0.037};
#endif

    DATA_TYPE *in = new DATA_TYPE[total];
    DATA_TYPE *out_ref = new DATA_TYPE[total];
    unsigned int seed = time(NULL);
    Init_Input_3D(in, z, m, n, halo, seed);

    // Show_Me(in, z, m, n, halo, "Input:");
    for(int i = 0; i < ITER; i++)
    {
        Stencil_Seq(in, out_ref, args, z, m, n, halo);
        swap(in, out_ref);
    }
    swap(in, out_ref);
    // Show_Me(out_ref, z, m, n, halo, "Output:");


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    float time_wo_pci;

    DATA_TYPE *in_d;
    DATA_TYPE *args_d;
    DATA_TYPE *out_d;
    DATA_TYPE *out = new DATA_TYPE[total];
    hipMalloc((void**)&in_d, total*sizeof(DATA_TYPE));
    hipMalloc((void**)&out_d, total*sizeof(DATA_TYPE));
    hipMalloc((void**)&args_d, (K)*sizeof(DATA_TYPE));
    hipMemcpy(args_d, args, (K)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    // Cuda version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, halo, seed);
    Clear_Output_3D(out, z, m, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid((n)/8, (m)/4, (z)/8);
    dim3 dimBlock(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda<<<dimGrid, dimBlock>>>(in_d, out_d, args_d, z, m, n, halo); 
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, halo, "Output(Cuda):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z+2*halo, m+2*halo, n+2*halo, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z+2*halo, m+2*halo, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Sweep version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, halo, seed); // reset input
    Clear_Output_3D(out, z, m, n, halo); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid1((n)/64, (m)/4, 4);
    dim3 dimBlock1(64, 4, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep<<<dimGrid1, dimBlock1>>>(in_d, out_d, args_d, z, m, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Sweep):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sweep: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sweep Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z+2*halo, m+2*halo, n+2*halo, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z+2*halo, m+2*halo, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 3D Block SM version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, halo, seed); // reset input
    Clear_Output_3D(out, z, m, n, halo); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid2((n)/8, (m)/4, (z)/8);
    dim3 dimBlock2(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sm<<<dimGrid2, dimBlock2>>>(
            in_d, out_d, args_d, z, m, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Sm):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sm: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sm Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z+2*halo, m+2*halo, n+2*halo, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z+2*halo, m+2*halo, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 2D Block SM version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, halo, seed); // reset input
    Clear_Output_3D(out, z, m, n, halo); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid3((n)/64, (m)/4, 4);
    dim3 dimBlock3(64, 4, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Sm<<<dimGrid3, dimBlock3, ((SM_2D_M)*(SM_2D_N)*3*sizeof(DATA_TYPE))>>>(
            in_d, out_d, args_d, z, m, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Sweep_Sm):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sweep_Sm: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sweep_Sm Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z+2*halo, m+2*halo, n+2*halo, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z+2*halo, m+2*halo, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 3D Block Shfl version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, halo, seed); // reset input
    Clear_Output_3D(out, z, m, n, halo); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid4((n)/8, (m)/4, (z)/8);
    dim3 dimBlock4(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl<<<dimGrid4, dimBlock4>>>(
            in_d, out_d, args_d, z, m, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Shfl):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z+2*halo, m+2*halo, n+2*halo, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z+2*halo, m+2*halo, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 3D Block Shfl2 version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, halo, seed); // reset input
    Clear_Output_3D(out, z, m, n, halo); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid5((n)/8, (m)/4, (z)/(8*2));
    dim3 dimBlock5(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl2<<<dimGrid5, dimBlock5>>>(
            in_d, out_d, args_d, z, m, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Shfl2):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl2: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl2 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z+2*halo, m+2*halo, n+2*halo, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z+2*halo, m+2*halo, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 3D Block Shfl4 version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, halo, seed); // reset input
    Clear_Output_3D(out, z, m, n, halo); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid6((n)/8, (m)/4, (z)/(8*4));
    dim3 dimBlock6(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl4<<<dimGrid6, dimBlock6>>>(
            in_d, out_d, args_d, z, m, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Shfl4):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl4: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl4 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z+2*halo, m+2*halo, n+2*halo, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z+2*halo, m+2*halo, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 3D Block Shfl8 version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, halo, seed); // reset input
    Clear_Output_3D(out, z, m, n, halo); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid7((n)/8, (m)/4, (z)/(8*8));
    dim3 dimBlock7(8, 4, 8);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl8<<<dimGrid7, dimBlock7>>>(
            in_d, out_d, args_d, z, m, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, halo, "Output(Cuda_Shfl8):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl8: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl8 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z+2*halo, m+2*halo, n+2*halo, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z+2*halo, m+2*halo, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    // Cuda 2D Block Shfl version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, halo, seed); // reset input
    Clear_Output_3D(out, z, m, n, halo); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid8((n)/8, (m)/32, 4);
    dim3 dimBlock8(8, 32, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Shfl<<<dimGrid8, dimBlock8>>>(
            in_d, out_d, args_d, z, m, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Sweep_Shfl):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sweep_Shfl: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sweep_Shfl Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z+2*halo, m+2*halo, n+2*halo, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z+2*halo, m+2*halo, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda 2D Block Shfl2 version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, halo, seed); // reset input
    Clear_Output_3D(out, z, m, n, halo); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid9((n)/8, (m)/(32*2), 4);
    dim3 dimBlock9(8, 32, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Shfl2<<<dimGrid9, dimBlock9>>>(
            in_d, out_d, args_d, z, m, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Sweep_Shfl2):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sweep_Shfl2: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sweep_Shfl2 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z+2*halo, m+2*halo, n+2*halo, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z+2*halo, m+2*halo, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda 2D Block Shfl4 version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, halo, seed); // reset input
    Clear_Output_3D(out, z, m, n, halo); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid10((n)/8, (m)/(32*4), 4);
    dim3 dimBlock10(8, 32, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Shfl4<<<dimGrid10, dimBlock10>>>(
            in_d, out_d, args_d, z, m, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Sweep_Shfl4):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sweep_Shfl4: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sweep_Shfl4 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z+2*halo, m+2*halo, n+2*halo, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z+2*halo, m+2*halo, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda 2D Block Shfl8 version
    /////////////////////////////////////////////////////////
    Init_Input_3D(in, z, m, n, halo, seed); // reset input
    Clear_Output_3D(out, z, m, n, halo); // flush output
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, (total)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid11((n)/8, (m)/(32*8), 4);
    dim3 dimBlock11(8, 32, 1);
    hipEventRecord(start);
    for(int i =0; i< ITER; i++)
    {
        Stencil_Cuda_Sweep_Shfl8<<<dimGrid11, dimBlock11>>>(
            in_d, out_d, args_d, z, m, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, z, m, n, "Output(Cuda_Sweep_Shfl8):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sweep_Shfl8: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sweep_Shfl8 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(z+2*halo, m+2*halo, n+2*halo, ITER, OPS_3D27, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, z+2*halo, m+2*halo, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    hipFree(in_d);
    hipFree(out_d);

    delete[] in;
    delete[] out;
    delete[] out_ref;

}
