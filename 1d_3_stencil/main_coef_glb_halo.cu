#include "hip/hip_runtime.h"
#include <iostream>
#include "metrics.h"
using namespace std;
// #define DATA_TYPE float
// #define DATA_TYPE double
#define warpSize 32 

#define IN_1D(_x) in[_x]
#define OUT_1D(_x) out[_x]
#define ARG_1D(_l,_x) args[(_l)*(n+2*halo)+(_x)]

// #define __DEBUG

#ifdef __DEBUG
#define ITER 1
#else
#define ITER 100
#endif

void Init_Args_1D(DATA_TYPE *args, int l, int n, int halo, DATA_TYPE val)
{
    for(int k = 0; k < l; k++)
    {
        for(int i = 0; i < n+2*halo; i++)
        {
            ARG_1D(k,i) = val; 
        }
    }
}

void Init_Input_1D(DATA_TYPE *in, int n, int halo, unsigned int seed)
{
    srand(seed);
    for(int i = 0; i < n+2*halo; i++)
    {
        if(i < halo || i >= n+halo)
            IN_1D(i) = 0.0;
        else
#ifdef __DEBUG
            IN_1D(i) = 1.0; 
                // IN_2D(i,j) = (DATA_TYPE)rand()*100.0 / ((long)RAND_MAX);
#else
            IN_1D(i) = (DATA_TYPE)rand()*10.0 / ((long)RAND_MAX);
#endif
    }
}

void Clear_Output_1D(DATA_TYPE *in, int n, int halo)
{
    for(int i = 0; i < n+2*halo; i++)
    {
        IN_1D(i) = 0.0;
    }
}

void Show_Me(DATA_TYPE *in, int n, int halo, string prompt)
{
    cout << prompt << endl;
    for(int i = 0; i < n+2*halo; i++)
    {
        std::cout << IN_1D(i) << ",";
    }
    std::cout << std::endl;
}

inline double tol_finder(int error_tol)
{
    double val = 1.0;
    for(; error_tol > 0; error_tol--)
        val *= 10;
    return 1.0/(double)val;
}

bool Verify(DATA_TYPE *test, DATA_TYPE *ref, int n)
{
    bool flag = true;
    double precision = tol_finder(2);

    for(int i = 0; i < n; i++)
    {
        if(fabs(test[i]-ref[i]) > precision)
        {
            std::cout << "wrong at " << i << " test:" << test[i] << " (ref: " << ref[i] << ")";
            std::cout << std::endl;
            flag = false;
            break;
        }
    }
    return flag;
}


void Stencil_Seq(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, int n, int halo)
{
    for(int i = halo; i < n+halo; i++)
    {
        OUT_1D(i) = ARG_1D(0,i)*IN_1D(i-1) + 
                    ARG_1D(1,i)*IN_1D(i  ) + 
                    ARG_1D(2,i)*IN_1D(i+1) ;
    }
}

__global__ void Stencil_Cuda(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, int n, int halo) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;  
    OUT_1D(i) = ARG_1D(0,i)*IN_1D(i-1) + 
                ARG_1D(1,i)*IN_1D(i  ) + 
                ARG_1D(2,i)*IN_1D(i+1) ;
}

__global__ void Stencil_Cuda_Sm(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, int n, int halo) 
{
    __shared__ DATA_TYPE local[256+2];
    unsigned int tid = threadIdx.x;
    // unsigned int lane_id = tid % warpSize;
    unsigned int gid = threadIdx.x + blockIdx.x * blockDim.x + halo;  
    int local_id = tid + halo;
    local[local_id] = IN_1D(gid);
    if(tid == 0)
        local[local_id-halo] = IN_1D(gid-halo);
    if(tid == blockDim.x - 1)
        local[local_id+halo] = IN_1D(gid+halo);
    __syncthreads();

    OUT_1D(gid) = ARG_1D(0,gid)*local[local_id-1] + 
                  ARG_1D(1,gid)*local[local_id  ] + 
                  ARG_1D(2,gid)*local[local_id+1] ;
}

__global__ void Stencil_Cuda_Shfl(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, int n, int halo) 
{
    unsigned int gid = threadIdx.x + blockIdx.x * blockDim.x + halo;  
    unsigned int tid = threadIdx.x;
    unsigned int lane_id = tid % warpSize;

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>5;

    DATA_TYPE threadInput0, threadInput1;
    int new_i = (warp_id_x<<5) + lane_id%34;
    threadInput0 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + (lane_id+32)%34;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    threadInput1 = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    int friend_id0;
    DATA_TYPE tx0, ty0;

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    sum0 += ARG_1D(0,gid)*tx0;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    sum0 += ARG_1D(1,gid)*((lane_id < 31)? tx0: ty0);

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    sum0 += ARG_1D(2,gid)*((lane_id < 30)? tx0: ty0);

    OUT_1D(gid) = sum0; 
}

__global__ void Stencil_Cuda_Shfl2(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, int n, int halo) 
{
    unsigned int tid = threadIdx.x;
    unsigned int lane_id = tid % warpSize;
    unsigned int gid = (((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<6) + lane_id + halo;  
    int warp_id_x = ((((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<6) + lane_id)>>5;

    DATA_TYPE threadInput0, threadInput1, threadInput2;
    int new_i = (warp_id_x<<5) + lane_id%34;
    threadInput0 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+32)/34)*34 + (lane_id+32)%34;
    threadInput1 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+64)/34)*34 + (lane_id+64)%34;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    threadInput2 = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    int friend_id0;
    // int friend_id1;
    DATA_TYPE tx0, ty0, tx1, ty1;

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    sum0 += ARG_1D(0,gid   )*tx0;
    sum1 += ARG_1D(0,gid+32)*tx1;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    ty1 = __shfl(threadInput2, friend_id0);
    sum0 += ARG_1D(1,gid   )*((lane_id < 31)? tx0: ty0);
    sum1 += ARG_1D(1,gid+32)*((lane_id < 31)? tx1: ty1);

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    ty1 = __shfl(threadInput2, friend_id0);
    sum0 += ARG_1D(2,gid   )*((lane_id < 30)? tx0: ty0);
    sum1 += ARG_1D(2,gid+32)*((lane_id < 30)? tx1: ty1);

    OUT_1D(gid   ) = sum0; 
    OUT_1D(gid+32) = sum1; 
}

__global__ void Stencil_Cuda_Shfl4(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, int n, int halo) 
{
    unsigned int tid = threadIdx.x;
    unsigned int lane_id = tid % warpSize;
    unsigned int gid = (((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<7) + lane_id + halo;  
    int warp_id_x = ((((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<7) + lane_id)>>5;

    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4; 
    int new_i = (warp_id_x<<5) + lane_id%34;
    threadInput0 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+32)/34)*34 + (lane_id+32)%34;
    threadInput1 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+64)/34)*34 + (lane_id+64)%34;
    threadInput2 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+96)/34)*34 + (lane_id+96)%34;
    threadInput3 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+128)/34)*34 + (lane_id+128)%34;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    threadInput4 = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    int friend_id0;
    // int friend_id1;
    DATA_TYPE tx0, ty0, tx1, ty1;
    DATA_TYPE tx2, ty2, tx3, ty3;

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    tx2 = __shfl(threadInput2, friend_id0);
    tx3 = __shfl(threadInput3, friend_id0);
    sum0 += ARG_1D(0,gid   )*tx0;
    sum1 += ARG_1D(0,gid+32)*tx1;
    sum2 += ARG_1D(0,gid+64)*tx2;
    sum3 += ARG_1D(0,gid+96)*tx3;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    ty1 = __shfl(threadInput2, friend_id0);
    tx2 = __shfl(threadInput2, friend_id0);
    ty2 = __shfl(threadInput3, friend_id0);
    tx3 = __shfl(threadInput3, friend_id0);
    ty3 = __shfl(threadInput4, friend_id0);
    sum0 += ARG_1D(1,gid   )*((lane_id < 31)? tx0: ty0);
    sum1 += ARG_1D(1,gid+32)*((lane_id < 31)? tx1: ty1);
    sum2 += ARG_1D(1,gid+64)*((lane_id < 31)? tx2: ty2);
    sum3 += ARG_1D(1,gid+96)*((lane_id < 31)? tx3: ty3);

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    ty1 = __shfl(threadInput2, friend_id0);
    tx2 = __shfl(threadInput2, friend_id0);
    ty2 = __shfl(threadInput3, friend_id0);
    tx3 = __shfl(threadInput3, friend_id0);
    ty3 = __shfl(threadInput4, friend_id0);
    sum0 += ARG_1D(2,gid   )*((lane_id < 30)? tx0: ty0);
    sum1 += ARG_1D(2,gid+32)*((lane_id < 30)? tx1: ty1);
    sum2 += ARG_1D(2,gid+64)*((lane_id < 30)? tx2: ty2);
    sum3 += ARG_1D(2,gid+96)*((lane_id < 30)? tx3: ty3);

    OUT_1D(gid   ) = sum0; 
    OUT_1D(gid+32) = sum1; 
    OUT_1D(gid+64) = sum2; 
    OUT_1D(gid+96) = sum3; 
}

__global__ void Stencil_Cuda_Shfl8(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE *args, int n, int halo) 
{
    unsigned int tid = threadIdx.x;
    unsigned int lane_id = tid % warpSize;
    unsigned int gid = (((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<8) + lane_id + halo;  
    int warp_id_x = ((((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<8) + lane_id)>>5;

    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5,
              threadInput6, threadInput7, threadInput8;
    int new_i = (warp_id_x<<5) + lane_id%34;
    threadInput0 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+32)/34)*34 + (lane_id+32)%34;
    threadInput1 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+64)/34)*34 + (lane_id+64)%34;
    threadInput2 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+96)/34)*34 + (lane_id+96)%34;
    threadInput3 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+128)/34)*34 + (lane_id+128)%34;
    threadInput4 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+160)/34)*34 + (lane_id+160)%34;
    threadInput5 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+192)/34)*34 + (lane_id+192)%34;
    threadInput6 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+224)/34)*34 + (lane_id+224)%34;
    threadInput7 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+256)/34)*34 + (lane_id+256)%34;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    threadInput8 = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    DATA_TYPE sum4 = 0.0;
    DATA_TYPE sum5 = 0.0;
    DATA_TYPE sum6 = 0.0;
    DATA_TYPE sum7 = 0.0;
    int friend_id0;
    // int friend_id1;
    DATA_TYPE tx0, ty0, tx1, ty1;
    DATA_TYPE tx2, ty2, tx3, ty3;
    DATA_TYPE tx4, ty4, tx5, ty5;
    DATA_TYPE tx6, ty6, tx7, ty7;

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    tx2 = __shfl(threadInput2, friend_id0);
    tx3 = __shfl(threadInput3, friend_id0);
    tx4 = __shfl(threadInput4, friend_id0);
    tx5 = __shfl(threadInput5, friend_id0);
    tx6 = __shfl(threadInput6, friend_id0);
    tx7 = __shfl(threadInput7, friend_id0);
    sum0 += ARG_1D(0,gid    )*tx0;
    sum1 += ARG_1D(0,gid+32 )*tx1;
    sum2 += ARG_1D(0,gid+64 )*tx2;
    sum3 += ARG_1D(0,gid+96 )*tx3;
    sum4 += ARG_1D(0,gid+128)*tx4;
    sum5 += ARG_1D(0,gid+160)*tx5;
    sum6 += ARG_1D(0,gid+192)*tx6;
    sum7 += ARG_1D(0,gid+224)*tx7;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    ty1 = __shfl(threadInput2, friend_id0);
    tx2 = __shfl(threadInput2, friend_id0);
    ty2 = __shfl(threadInput3, friend_id0);
    tx3 = __shfl(threadInput3, friend_id0);
    ty3 = __shfl(threadInput4, friend_id0);
    tx4 = __shfl(threadInput4, friend_id0);
    ty4 = __shfl(threadInput5, friend_id0);
    tx5 = __shfl(threadInput5, friend_id0);
    ty5 = __shfl(threadInput6, friend_id0);
    tx6 = __shfl(threadInput6, friend_id0);
    ty6 = __shfl(threadInput7, friend_id0);
    tx7 = __shfl(threadInput7, friend_id0);
    ty7 = __shfl(threadInput8, friend_id0);
    sum0 += ARG_1D(1,gid    )*((lane_id < 31)? tx0: ty0);
    sum1 += ARG_1D(1,gid+32 )*((lane_id < 31)? tx1: ty1);
    sum2 += ARG_1D(1,gid+64 )*((lane_id < 31)? tx2: ty2);
    sum3 += ARG_1D(1,gid+96 )*((lane_id < 31)? tx3: ty3);
    sum4 += ARG_1D(1,gid+128)*((lane_id < 31)? tx4: ty4);
    sum5 += ARG_1D(1,gid+160)*((lane_id < 31)? tx5: ty5);
    sum6 += ARG_1D(1,gid+192)*((lane_id < 31)? tx6: ty6);
    sum7 += ARG_1D(1,gid+224)*((lane_id < 31)? tx7: ty7);

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    ty1 = __shfl(threadInput2, friend_id0);
    tx2 = __shfl(threadInput2, friend_id0);
    ty2 = __shfl(threadInput3, friend_id0);
    tx3 = __shfl(threadInput3, friend_id0);
    ty3 = __shfl(threadInput4, friend_id0);
    tx4 = __shfl(threadInput4, friend_id0);
    ty4 = __shfl(threadInput5, friend_id0);
    tx5 = __shfl(threadInput5, friend_id0);
    ty5 = __shfl(threadInput6, friend_id0);
    tx6 = __shfl(threadInput6, friend_id0);
    ty6 = __shfl(threadInput7, friend_id0);
    tx7 = __shfl(threadInput7, friend_id0);
    ty7 = __shfl(threadInput8, friend_id0);
    sum0 += ARG_1D(2,gid    )*((lane_id < 30)? tx0: ty0);
    sum1 += ARG_1D(2,gid+32 )*((lane_id < 30)? tx1: ty1);
    sum2 += ARG_1D(2,gid+64 )*((lane_id < 30)? tx2: ty2);
    sum3 += ARG_1D(2,gid+96 )*((lane_id < 30)? tx3: ty3);
    sum4 += ARG_1D(2,gid+128)*((lane_id < 30)? tx4: ty4);
    sum5 += ARG_1D(2,gid+160)*((lane_id < 30)? tx5: ty5);
    sum6 += ARG_1D(2,gid+192)*((lane_id < 30)? tx6: ty6);
    sum7 += ARG_1D(2,gid+224)*((lane_id < 30)? tx7: ty7);

    OUT_1D(gid    ) = sum0; 
    OUT_1D(gid+32 ) = sum1; 
    OUT_1D(gid+64 ) = sum2; 
    OUT_1D(gid+96 ) = sum3; 
    OUT_1D(gid+128) = sum4; 
    OUT_1D(gid+160) = sum5; 
    OUT_1D(gid+192) = sum6; 
    OUT_1D(gid+224) = sum7; 
}

int main(int argc, char **argv)
{
#ifdef __DEBUG
    int n = 512;
#else
    int n = 33554432; // 2^25
#endif
    int halo = 1; 
    int total = (n+2*halo);
    int K = total*3;
    DATA_TYPE *args = new DATA_TYPE[K];
#ifdef __DEBUG
    Init_Args_1D(args, 3, n, halo, 1.0);
#else
    Init_Args_1D(args, 3, n, halo, 0.33);
#endif
    DATA_TYPE *in = new DATA_TYPE[total];
    DATA_TYPE *out_ref = new DATA_TYPE[total];
    unsigned int seed = time(NULL);
    Init_Input_1D(in, n, halo, seed);

    // Show_Me(in, n, halo, "Input:");
    for(int i=0; i< ITER; i++)
    {
        Stencil_Seq(in, out_ref, args, n, halo);
        swap(in, out_ref);
    }
    swap(in, out_ref);
    // Show_Me(out_ref, n, halo, "Output:");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    float time_wo_pci;

    DATA_TYPE *in_d;
    DATA_TYPE *args_d;
    DATA_TYPE *out_d;
    DATA_TYPE *out = new DATA_TYPE[total];
    hipMalloc((void**)&in_d, total*sizeof(DATA_TYPE));
    hipMalloc((void**)&out_d, total*sizeof(DATA_TYPE));
    hipMalloc((void**)&args_d, (K)*sizeof(DATA_TYPE));
    hipMemcpy(args_d, args, (K)*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    
    // Cuda version
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid((n)/256, 1, 1);
    dim3 dimBlock(256, 1, 1);

    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda<<<dimGrid, dimBlock>>>(in_d, out_d, args_d, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, n, halo,  "Output(Device):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(3, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shared Memory version 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid1((n)/256, 1, 1);
    dim3 dimBlock1(256, 1, 1);

    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Sm<<<dimGrid1, dimBlock1>>>(in_d, out_d, args_d, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, n, halo,  "Output(Device):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sm: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(3, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shfl version 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid2((n)/256, 1, 1);
    dim3 dimBlock2(256, 1, 1);

    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl<<<dimGrid2, dimBlock2>>>(in_d, out_d, args_d, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, n, halo,  "Output(Device):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(3, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shfl2 version 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid3((n)/(256*2), 1, 1);
    dim3 dimBlock3(256, 1, 1);

    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl2<<<dimGrid3, dimBlock3>>>(in_d, out_d, args_d, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, n, halo,  "Output(Device):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl2: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(3, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shfl4 version 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid4((n)/(256*4), 1, 1);
    dim3 dimBlock4(256, 1, 1);

    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl4<<<dimGrid4, dimBlock4>>>(in_d, out_d, args_d, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, n, halo,  "Output(Device):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl4: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(3, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shfl8 version 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid5((n)/(256*8), 1, 1);
    dim3 dimBlock5(256, 1, 1);

    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl8<<<dimGrid5, dimBlock5>>>(in_d, out_d, args_d, n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, n, halo,  "Output(Device):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl8: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(3, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));


    hipFree(in_d);
    hipFree(out_d);

    delete[] in;
    delete[] out;
    delete[] out_ref;
}

