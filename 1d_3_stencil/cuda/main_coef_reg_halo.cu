#include "hip/hip_runtime.h"
#include <iostream>
#include "metrics.h"
using namespace std;
// #define DATA_TYPE float
// #define DATA_TYPE double
#define warpSize 32 

#define IN_1D(_x) in[_x]
#define OUT_1D(_x) out[_x]
#define ARG_1D(_l,_x) args[(_l)*(n+2*halo)+(_x)]

// #define __DEBUG

#ifdef __DEBUG
#define ITER 1
#else
#define ITER 100
#endif

void Init_Args_1D(DATA_TYPE *args, int l, int n, int halo, DATA_TYPE val)
{
    for(int k = 0; k < l; k++)
    {
        for(int i = 0; i < n+2*halo; i++)
        {
            ARG_1D(k,i) = val; 
        }
    }
}

void Init_Input_1D(DATA_TYPE *in, int n, int halo, unsigned int seed)
{
    srand(seed);
    for(int i = 0; i < n+2*halo; i++)
    {
        if(i < halo || i >= n+halo)
            IN_1D(i) = 0.0;
        else
#ifdef __DEBUG
            IN_1D(i) = 1.0; 
                // IN_2D(i,j) = (DATA_TYPE)rand()*100.0 / ((long)RAND_MAX);
#else
            IN_1D(i) = (DATA_TYPE)rand()*10.0 / ((long)RAND_MAX);
#endif
    }
}

void Clear_Output_1D(DATA_TYPE *in, int n, int halo)
{
    for(int i = 0; i < n+2*halo; i++)
    {
        IN_1D(i) = 0.0;
    }
}

void Show_Me(DATA_TYPE *in, int n, int halo, string prompt)
{
    cout << prompt << endl;
    for(int i = 0; i < n+2*halo; i++)
    {
        std::cout << IN_1D(i) << ",";
    }
    std::cout << std::endl;
}

inline double tol_finder(int error_tol)
{
    double val = 1.0;
    for(; error_tol > 0; error_tol--)
        val *= 10;
    return 1.0/(double)val;
}

bool Verify(DATA_TYPE *test, DATA_TYPE *ref, int n)
{
    bool flag = true;
    double precision = tol_finder(2);

    for(int i = 0; i < n; i++)
    {
        if(fabs(test[i]-ref[i]) > precision)
        {
            std::cout << "wrong at " << i << " test:" << test[i] << " (ref: " << ref[i] << ")";
            std::cout << std::endl;
            flag = false;
            break;
        }
    }
    return flag;
}


void Stencil_Seq(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, int n, int halo)
{
#pragma omp parallel for
    for(int i = halo; i < n+halo; i++)
    {
        OUT_1D(i) = a0*IN_1D(i-1) + 
                    a1*IN_1D(i  ) + 
                    a2*IN_1D(i+1) ;
    }
}

__global__ void Stencil_Cuda(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, int n, int halo) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x + halo;  
    OUT_1D(i) = a0*IN_1D(i-1) + 
                a1*IN_1D(i  ) + 
                a2*IN_1D(i+1) ;
}

__global__ void Stencil_Cuda_Sm_Branch(DATA_TYPE *in, DATA_TYPE *out, 
        DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, int n, int halo) 
{
    __shared__ DATA_TYPE local[256+2];
    unsigned int tid = threadIdx.x;
    unsigned int gid = threadIdx.x + blockIdx.x * blockDim.x + halo;  
    int local_id = tid + halo;

    local[local_id] = IN_1D(gid);
    if(tid == 0)
        local[local_id-1] = IN_1D(gid-1);
    if(tid == blockDim.x - 1)
        local[local_id+1] = IN_1D(gid+1);
    __syncthreads();

    OUT_1D(gid) = a0*local[local_id-1] + 
                  a1*local[local_id  ] + 
                  a2*local[local_id+1] ;
}

__global__ void Stencil_Cuda_Sm_Cyclic(DATA_TYPE *in, DATA_TYPE *out, 
        DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, int n, int halo) 
{
    __shared__ DATA_TYPE local[256+2];
    unsigned int tid = threadIdx.x;
    unsigned int gid = threadIdx.x + blockIdx.x * blockDim.x + halo;  
    int local_id = tid + halo;

    unsigned int lane_id = threadIdx.x;
    int lane_id_it = lane_id;
    int blk_id_x = blockIdx.x;
    int new_i  = (blk_id_x<<8) + lane_id_it%258;
    int new_li = lane_id_it%258;
    local[new_li] = IN_1D(new_i);
    lane_id_it += 256;
    new_i  = (blk_id_x<<8) + (lane_id_it/258)*258 + lane_id_it%258;
    new_li = (lane_id_it/258)*258 + lane_id_it%258;
    if(new_li < 258)
        local[new_li] = IN_1D(new_i);

    __syncthreads();

    OUT_1D(gid) = a0*local[local_id-1] + 
                  a1*local[local_id  ] + 
                  a2*local[local_id+1] ;
}

__global__ void Stencil_Cuda_Shfl(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, int n, int halo) 
{
    unsigned int gid = threadIdx.x + blockIdx.x * blockDim.x + halo;  
    unsigned int tid = threadIdx.x;
    unsigned int lane_id = tid % warpSize;

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>5;

    DATA_TYPE threadInput0, threadInput1;
    int new_i = (warp_id_x<<5) + lane_id%34;
    threadInput0 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + (lane_id+32)%34;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    threadInput1 = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    int friend_id0;
    DATA_TYPE tx0, ty0;

    /*
    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = threadInput0;
    ty0 = __shfl(tx0, friend_id0);
    sum0 += a0*ty0;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = (lane_id > 0)? threadInput0: threadInput1;
    ty0 = __shfl(tx0, friend_id0);
    sum0 += a1*ty0;

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = (lane_id > 1)? threadInput0: threadInput1;
    ty0 = __shfl(tx0, friend_id0);
    sum0 += a2*ty0;
    */

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    sum0 += a0*tx0;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    sum0 += a1*((lane_id < 31)? tx0: ty0);

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    sum0 += a2*((lane_id < 30)? tx0: ty0);

    OUT_1D(gid) = sum0; 
}

__global__ void Stencil_Cuda_Shfl2(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, int n, int halo) 
{
    unsigned int tid = threadIdx.x;
    unsigned int lane_id = tid % warpSize;
    unsigned int gid = (((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<6) + lane_id + halo;  
    int warp_id_x = ((((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<6) + lane_id)>>5;

    DATA_TYPE threadInput0, threadInput1, threadInput2;
    int new_i = (warp_id_x<<5) + lane_id%34;
    threadInput0 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+32)/34)*34 + (lane_id+32)%34;
    threadInput1 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+64)/34)*34 + (lane_id+64)%34;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    threadInput2 = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    int friend_id0;
    // int friend_id1;
    DATA_TYPE tx0, ty0, tx1, ty1;

    /*
    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = threadInput0;
    tx1 = threadInput1;
    ty0 = __shfl(tx0, friend_id0);
    ty1 = __shfl(tx1, friend_id0);
    sum0 += a0*ty0;
    sum1 += a0*ty1;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = (lane_id > 0)? threadInput0: threadInput1;
    tx1 = (lane_id > 0)? threadInput1: threadInput2;
    ty0 = __shfl(tx0, friend_id0);
    ty1 = __shfl(tx1, friend_id0);
    sum0 += a1*ty0;
    sum1 += a1*ty1;

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = (lane_id > 1)? threadInput0: threadInput1;
    tx1 = (lane_id > 1)? threadInput1: threadInput2;
    ty0 = __shfl(tx0, friend_id0);
    ty1 = __shfl(tx1, friend_id0);
    sum0 += a2*ty0;
    sum1 += a2*ty1;
    */

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    sum0 += a0*tx0;
    sum1 += a0*tx1;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    ty1 = __shfl(threadInput2, friend_id0);
    sum0 += a1*((lane_id < 31)? tx0: ty0);
    sum1 += a1*((lane_id < 31)? tx1: ty1);

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    ty1 = __shfl(threadInput2, friend_id0);
    sum0 += a2*((lane_id < 30)? tx0: ty0);
    sum1 += a2*((lane_id < 30)? tx1: ty1);

    OUT_1D(gid   ) = sum0; 
    OUT_1D(gid+32) = sum1; 
}

__global__ void Stencil_Cuda_Shfl4(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, int n, int halo) 
{
    unsigned int tid = threadIdx.x;
    unsigned int lane_id = tid % warpSize;
    unsigned int gid = (((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<7) + lane_id + halo;  
    int warp_id_x = ((((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<7) + lane_id)>>5;

    /*
    DATA_TYPE reg[5]; 
    int new_i = (warp_id_x<<5) + lane_id%34;
    reg[0] = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+32)/34)*34 + (lane_id+32)%34;
    reg[1] = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+64)/34)*34 + (lane_id+64)%34;
    reg[2] = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+96)/34)*34 + (lane_id+96)%34;
    reg[3] = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+128)/34)*34 + (lane_id+128)%34;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    reg[4] = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    int friend_id0;
    // int friend_id1;
    DATA_TYPE tx0, ty0, tx1, ty1;
    DATA_TYPE tx2, ty2, tx3, ty3;
    int reg_id0;
    int reg_id1;
    int reg_id2;
    int reg_id3;

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(reg[0], friend_id0);
    tx1 = __shfl(reg[1], friend_id0);
    tx2 = __shfl(reg[2], friend_id0);
    tx3 = __shfl(reg[3], friend_id0);
    sum0 += a0*tx0;
    sum1 += a0*tx1;
    sum2 += a0*tx2;
    sum3 += a0*tx3;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    reg_id0 = (lane_id > 0)? 0: 1;
    reg_id1 = (lane_id > 0)? 1: 2;
    reg_id2 = (lane_id > 0)? 2: 3;
    reg_id3 = (lane_id > 0)? 3: 4;
    tx0 = __shfl(reg[reg_id0], friend_id0);
    tx1 = __shfl(reg[reg_id1], friend_id0);
    tx2 = __shfl(reg[reg_id2], friend_id0);
    tx3 = __shfl(reg[reg_id3], friend_id0);
    sum0 += a1*tx0;
    sum1 += a1*tx1;
    sum2 += a1*tx2;
    sum3 += a1*tx3;

    friend_id0 = (lane_id+2 )&(warpSize-1);
    reg_id0 = (lane_id > 1)? 0: 1;
    reg_id1 = (lane_id > 1)? 1: 2;
    reg_id2 = (lane_id > 1)? 2: 3;
    reg_id3 = (lane_id > 1)? 3: 4;
    tx0 = __shfl(reg[reg_id0], friend_id0);
    tx1 = __shfl(reg[reg_id1], friend_id0);
    tx2 = __shfl(reg[reg_id2], friend_id0);
    tx3 = __shfl(reg[reg_id3], friend_id0);
    sum0 += a2*tx0;
    sum1 += a2*tx1;
    sum2 += a2*tx2;
    sum3 += a2*tx3;
    */

    /*
    DATA_TYPE reg0, reg1, reg2, reg3, reg4; 
    int new_i = (warp_id_x<<5) + lane_id%34;
    reg0 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+32)/34)*34 + (lane_id+32)%34;
    reg1 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+64)/34)*34 + (lane_id+64)%34;
    reg2 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+96)/34)*34 + (lane_id+96)%34;
    reg3 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+128)/34)*34 + (lane_id+128)%34;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    reg4 = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    int friend_id0;
    // int friend_id1;
    DATA_TYPE tx0, ty0, tx1, ty1;
    DATA_TYPE tx2, ty2, tx3, ty3;
    int reg_id0;
    int reg_id1;
    int reg_id2;
    int reg_id3;

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    sum0 += a0*tx0;
    sum1 += a0*tx1;
    sum2 += a0*tx2;
    sum3 += a0*tx3;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    reg_id0 = (lane_id > 0)? reg0: reg1;
    reg_id1 = (lane_id > 0)? reg1: reg2;
    reg_id2 = (lane_id > 0)? reg2: reg3;
    reg_id3 = (lane_id > 0)? reg3: reg4;
    tx0 = __shfl(reg_id0, friend_id0);
    tx1 = __shfl(reg_id1, friend_id0);
    tx2 = __shfl(reg_id2, friend_id0);
    tx3 = __shfl(reg_id3, friend_id0);
    sum0 += a1*tx0;
    sum1 += a1*tx1;
    sum2 += a1*tx2;
    sum3 += a1*tx3;

    friend_id0 = (lane_id+2 )&(warpSize-1);
    reg_id0 = (lane_id > 1)? reg0: reg1;
    reg_id1 = (lane_id > 1)? reg1: reg2;
    reg_id2 = (lane_id > 1)? reg2: reg3;
    reg_id3 = (lane_id > 1)? reg3: reg4;
    tx0 = __shfl(reg_id0, friend_id0);
    tx1 = __shfl(reg_id1, friend_id0);
    tx2 = __shfl(reg_id2, friend_id0);
    tx3 = __shfl(reg_id3, friend_id0);
    sum0 += a2*tx0;
    sum1 += a2*tx1;
    sum2 += a2*tx2;
    sum3 += a2*tx3;
    */

    DATA_TYPE reg0, reg1, reg2, reg3, reg4; 
    int new_i = (warp_id_x<<5) + lane_id%34;
    reg0 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+32)/34)*34 + (lane_id+32)%34;
    reg1 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+64)/34)*34 + (lane_id+64)%34;
    reg2 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+96)/34)*34 + (lane_id+96)%34;
    reg3 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+128)/34)*34 + (lane_id+128)%34;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    reg4 = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    int friend_id0;
    // int friend_id1;
    DATA_TYPE tx0, ty0, tx1, ty1;
    DATA_TYPE tx2, ty2, tx3, ty3;

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    sum0 += a0*tx0;
    sum1 += a0*tx1;
    sum2 += a0*tx2;
    sum3 += a0*tx3;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    sum0 += a1*((lane_id < 31)? tx0: ty0);
    sum1 += a1*((lane_id < 31)? tx1: ty1);
    sum2 += a1*((lane_id < 31)? tx2: ty2);
    sum3 += a1*((lane_id < 31)? tx3: ty3);

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = __shfl(reg0, friend_id0);
    ty0 = __shfl(reg1, friend_id0);
    tx1 = __shfl(reg1, friend_id0);
    ty1 = __shfl(reg2, friend_id0);
    tx2 = __shfl(reg2, friend_id0);
    ty2 = __shfl(reg3, friend_id0);
    tx3 = __shfl(reg3, friend_id0);
    ty3 = __shfl(reg4, friend_id0);
    sum0 += a2*((lane_id < 30)? tx0: ty0);
    sum1 += a2*((lane_id < 30)? tx1: ty1);
    sum2 += a2*((lane_id < 30)? tx2: ty2);
    sum3 += a2*((lane_id < 30)? tx3: ty3);

    OUT_1D(gid   ) = sum0; 
    OUT_1D(gid+32) = sum1; 
    OUT_1D(gid+64) = sum2; 
    OUT_1D(gid+96) = sum3; 
}

__global__ void Stencil_Cuda_Shfl8(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, DATA_TYPE a2, int n, int halo) 
{
    unsigned int tid = threadIdx.x;
    unsigned int lane_id = tid % warpSize;
    unsigned int gid = (((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<8) + lane_id + halo;  
    int warp_id_x = ((((threadIdx.x + blockIdx.x * blockDim.x)>>5)<<8) + lane_id)>>5;

    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5,
              threadInput6, threadInput7, threadInput8;
    int new_i = (warp_id_x<<5) + lane_id%34;
    threadInput0 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+32)/34)*34 + (lane_id+32)%34;
    threadInput1 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+64)/34)*34 + (lane_id+64)%34;
    threadInput2 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+96)/34)*34 + (lane_id+96)%34;
    threadInput3 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+128)/34)*34 + (lane_id+128)%34;
    threadInput4 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+160)/34)*34 + (lane_id+160)%34;
    threadInput5 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+192)/34)*34 + (lane_id+192)%34;
    threadInput6 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+224)/34)*34 + (lane_id+224)%34;
    threadInput7 = IN_1D(new_i);
    new_i = (warp_id_x<<5) + ((lane_id+256)/34)*34 + (lane_id+256)%34;
    new_i = (new_i < n+2*halo)? new_i: n+2*halo-1;
    threadInput8 = IN_1D(new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    DATA_TYPE sum4 = 0.0;
    DATA_TYPE sum5 = 0.0;
    DATA_TYPE sum6 = 0.0;
    DATA_TYPE sum7 = 0.0;
    int friend_id0;
    // int friend_id1;
    DATA_TYPE tx0, ty0, tx1, ty1;
    DATA_TYPE tx2, ty2, tx3, ty3;
    DATA_TYPE tx4, ty4, tx5, ty5;
    DATA_TYPE tx6, ty6, tx7, ty7;

    friend_id0 = (lane_id+0 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    tx2 = __shfl(threadInput2, friend_id0);
    tx3 = __shfl(threadInput3, friend_id0);
    tx4 = __shfl(threadInput4, friend_id0);
    tx5 = __shfl(threadInput5, friend_id0);
    tx6 = __shfl(threadInput6, friend_id0);
    tx7 = __shfl(threadInput7, friend_id0);
    sum0 += a0*tx0;
    sum1 += a0*tx1;
    sum2 += a0*tx2;
    sum3 += a0*tx3;
    sum4 += a0*tx4;
    sum5 += a0*tx5;
    sum6 += a0*tx6;
    sum7 += a0*tx7;

    friend_id0 = (lane_id+1 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    ty1 = __shfl(threadInput2, friend_id0);
    tx2 = __shfl(threadInput2, friend_id0);
    ty2 = __shfl(threadInput3, friend_id0);
    tx3 = __shfl(threadInput3, friend_id0);
    ty3 = __shfl(threadInput4, friend_id0);
    tx4 = __shfl(threadInput4, friend_id0);
    ty4 = __shfl(threadInput5, friend_id0);
    tx5 = __shfl(threadInput5, friend_id0);
    ty5 = __shfl(threadInput6, friend_id0);
    tx6 = __shfl(threadInput6, friend_id0);
    ty6 = __shfl(threadInput7, friend_id0);
    tx7 = __shfl(threadInput7, friend_id0);
    ty7 = __shfl(threadInput8, friend_id0);
    sum0 += a1*((lane_id < 31)? tx0: ty0);
    sum1 += a1*((lane_id < 31)? tx1: ty1);
    sum2 += a1*((lane_id < 31)? tx2: ty2);
    sum3 += a1*((lane_id < 31)? tx3: ty3);
    sum4 += a1*((lane_id < 31)? tx4: ty4);
    sum5 += a1*((lane_id < 31)? tx5: ty5);
    sum6 += a1*((lane_id < 31)? tx6: ty6);
    sum7 += a1*((lane_id < 31)? tx7: ty7);

    friend_id0 = (lane_id+2 )&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id0);
    ty1 = __shfl(threadInput2, friend_id0);
    tx2 = __shfl(threadInput2, friend_id0);
    ty2 = __shfl(threadInput3, friend_id0);
    tx3 = __shfl(threadInput3, friend_id0);
    ty3 = __shfl(threadInput4, friend_id0);
    tx4 = __shfl(threadInput4, friend_id0);
    ty4 = __shfl(threadInput5, friend_id0);
    tx5 = __shfl(threadInput5, friend_id0);
    ty5 = __shfl(threadInput6, friend_id0);
    tx6 = __shfl(threadInput6, friend_id0);
    ty6 = __shfl(threadInput7, friend_id0);
    tx7 = __shfl(threadInput7, friend_id0);
    ty7 = __shfl(threadInput8, friend_id0);
    sum0 += a2*((lane_id < 30)? tx0: ty0);
    sum1 += a2*((lane_id < 30)? tx1: ty1);
    sum2 += a2*((lane_id < 30)? tx2: ty2);
    sum3 += a2*((lane_id < 30)? tx3: ty3);
    sum4 += a2*((lane_id < 30)? tx4: ty4);
    sum5 += a2*((lane_id < 30)? tx5: ty5);
    sum6 += a2*((lane_id < 30)? tx6: ty6);
    sum7 += a2*((lane_id < 30)? tx7: ty7);

    OUT_1D(gid    ) = sum0; 
    OUT_1D(gid+32 ) = sum1; 
    OUT_1D(gid+64 ) = sum2; 
    OUT_1D(gid+96 ) = sum3; 
    OUT_1D(gid+128) = sum4; 
    OUT_1D(gid+160) = sum5; 
    OUT_1D(gid+192) = sum6; 
    OUT_1D(gid+224) = sum7; 
}

int main(int argc, char **argv)
{
#ifdef __DEBUG
    int n = 512;
#else
    int n = 33554432; // 2^25
#endif
    int halo = 1; 
    int total = (n+2*halo);
    const int K = 3;
#ifdef __DEBUG
    DATA_TYPE args[K] = {1.0, 1.0, 1.0};
#else
    DATA_TYPE args[K] = {0.33, 0.33, 0.33};
#endif
    DATA_TYPE *in = new DATA_TYPE[total];
    DATA_TYPE *out_ref = new DATA_TYPE[total];
    unsigned int seed = time(NULL);
    Clear_Output_1D(in, n, halo);
    Clear_Output_1D(out_ref, n, halo);
    Init_Input_1D(in, n, halo, seed);

    // Show_Me(in, n, halo, "Input:");
    for(int i=0; i< ITER; i++)
    {
        Stencil_Seq(in, out_ref, args[0], args[1], args[2], n, halo);
        swap(in, out_ref);
    }
    swap(in, out_ref);
    // Show_Me(out_ref, n, halo, "Output:");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    float time_wo_pci;

    DATA_TYPE *in_d;
    DATA_TYPE *out_d;
    DATA_TYPE *out = new DATA_TYPE[total];
    hipMalloc((void**)&in_d, total*sizeof(DATA_TYPE));
    hipMalloc((void**)&out_d, total*sizeof(DATA_TYPE));
    dim3 dimGrid;
    dim3 dimBlock;
    
    // Cuda version
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dimGrid.x = (n)/256;
    dimGrid.y = 1;
    dimGrid.z = 1;
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda<<<dimGrid, dimBlock>>>(in_d, out_d, args[0], args[1], args[2] , n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shared Memory with Branch
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dimGrid.x = (n)/256;
    dimGrid.y = 1;
    dimGrid.z = 1;
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Sm_Branch<<<dimGrid, dimBlock>>>(in_d, out_d, args[0], args[1], args[2] , n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sm_Branch: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sm_Branch Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shared Memory with Cyclic
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dimGrid.x = (n)/256;
    dimGrid.y = 1;
    dimGrid.z = 1;
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Sm_Cyclic<<<dimGrid, dimBlock>>>(in_d, out_d, args[0], args[1], args[2] , n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sm_Cyclic: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sm_Cyclic Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shfl 1D-Warp 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dimGrid.x = (n)/256;
    dimGrid.y = 1;
    dimGrid.z = 1;
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl<<<dimGrid, dimBlock>>>(in_d, out_d, args[0], args[1], args[2] , n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shfl2 1D-Warp 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dimGrid.x = (n)/(256*2);
    dimGrid.y = 1;
    dimGrid.z = 1;
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl2<<<dimGrid, dimBlock>>>(in_d, out_d, args[0], args[1], args[2] , n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl2: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl2 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    // Cuda Shfl4 1D-Warp 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dimGrid.x = (n)/(256*4);
    dimGrid.y = 1;
    dimGrid.z = 1;
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl4<<<dimGrid, dimBlock>>>(in_d, out_d, args[0], args[1], args[2] , n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl4: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl4 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));

    /*
    // Cuda Shfl8 1D-Warp 
    /////////////////////////////////////////////////////////
    Init_Input_1D(in, n, halo, seed);
    Clear_Output_1D(out, n, halo);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid5((n)/(256*8), 1, 1);
    dim3 dimBlock5(256, 1, 1);

    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl8<<<dimGrid5, dimBlock5>>>(in_d, out_d, args[0], args[1], args[2] , n, halo);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, n, halo,  "Output(Device):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl8: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, 1, n+2*halo, ITER, OPS_1D3, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(0, 1, 1, n+2*halo, ITER, time_wo_pci, sizeof(DATA_TYPE)));
    */


    hipFree(in_d);
    hipFree(out_d);

    delete[] in;
    delete[] out;
    delete[] out_ref;
}

