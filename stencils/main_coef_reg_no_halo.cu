

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
#define IN_2D(_x,_y) in[(_x)*(N)+(_y)]
#define OUT_2D(_x,_y) out[(_x)*(N)+(_y)]
#define LOC_2D(_x,_y) local[(_x)*(18)+(_y)]

// #define __DEBUG

#ifdef __DEBUG
#define ITER 1
#else
#define ITER 100
#endif

#define DATA_TYPE float
#define warpSize 32 
float GetGFLOPS(int z, int m, int n, int count, int ops, float time) 
{
    float f = (z*m*n)*(float)(ops)*(float)(count)/time * 1.0e-09;
    return f;
}

float GetThroughput(int z, int m, int n, int count, float time) 
{
    return (z*m*n) * sizeof(DATA_TYPE) * 2.0 * ((float)count)
            / time * 1.0e-09;    
          
}

void Init_Input_2D(DATA_TYPE *in, int M, int N, unsigned int seed)
{
    srand(seed);

    for(int i = 0; i < M; i++)
    {
        for(int j = 0; j < N; j++)
        {
#ifdef __DEBUG
                IN_2D(i,j) = 1; 
#else
                IN_2D(i,j) = (DATA_TYPE)rand()*100.0 / ((long)RAND_MAX);
#endif
        }
    }
}

void Clear_Output_2D(DATA_TYPE *in, int M, int N)
{
    for(int i = 0; i < M; i++)
    {
        for(int j = 0; j < N; j++)
        {
            IN_2D(i,j) = 0; 
        }
    }
}

__global__ void Stencil_Cuda(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, int M, int N) 
{
    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;
    int n = (j == 0)      ? j : j - 1;
    int s = (j == M-1)    ? j : j + 1;
    int w = (i == 0)      ? i : i - 1;
    int e = (i == N-1)    ? i : i + 1;
    OUT_2D(j,i) = a0*IN_2D(n  ,i  ) + 
                  a1*IN_2D(j  ,w  ) + 
                  a2*IN_2D(s  ,i  ) +
                  a3*IN_2D(j  ,e  ) + 
                  a4*IN_2D(j  ,i  ) ;
}

void Stencil_Seq(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, int M, int N)
{
    for(int j = 0; j < M; j++)
    {
        int n = (j == 0)      ? j : j - 1;
        int s = (j == M-1)    ? j : j + 1;
        for(int i = 0; i < N; i++)
        {
            int w = (i == 0)      ? i : i - 1;
            int e = (i == N-1)    ? i : i + 1;
            OUT_2D(j,i) = a0 * IN_2D(n  ,i  ) +
                          a1 * IN_2D(j  ,w  ) +
                          a2 * IN_2D(s  ,i  ) +
                          a3 * IN_2D(j  ,e  ) +
                          a4 * IN_2D(j  ,i  ) ;
        }
    }
}

void Show_Me(DATA_TYPE *in, int M, int N, std::string prompt)
{
    std::cout << prompt << std::endl;
    for(int j = 0; j < M; j++)
    {
        for(int i = 0; i < N; i++)
            std::cout << IN_2D(j,i) << ",";
        std::cout << std::endl;
    }
}

__global__ void Stencil_Cuda_Shfl4(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, int M, int N) 
{
    unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x;
    unsigned int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = (((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<4) + (lane_id>>3) ;

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3;
    int warp_id_y = ((((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<4) + (lane_id>>3))>>2;

    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5;
    int new_i = (warp_id_x<<3) + lane_id%10-1;
    int new_j = (warp_id_y<<2) + lane_id/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+64)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+96)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+128)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput4 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+160)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput5 = IN_2D(new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    int friend_id0, friend_id1;
    int friend_id2, friend_id3;
    DATA_TYPE tx0, ty0, tx1, ty1, tz1, tx2, ty2, tz2, tx3, ty3, tz3;
    
    friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    tx3 = __shfl(threadInput3, friend_id3);
    ty3 = __shfl(threadInput4, friend_id3);
    sum0 += a0*((lane_id < 25)? tx0: ty0);
    sum1 += a0*((lane_id < 19)? tx1: ty1);
    sum2 += a0*((lane_id < 13)? tx2: ty2);
    sum3 += a0*((lane_id < 7 )? tx3: ty3);

   
    friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    ty3 = __shfl(threadInput4, friend_id3);
    tz3 = __shfl(threadInput5, friend_id3);
    sum0 += a1*((lane_id < 18)? tx0: ty0);
    sum1 += a1*((lane_id < 12)? tx1: ty1);
    sum2 += a1*((lane_id < 6 )? tx2: ty2);
    sum3 += a1*((lane_id < 24)? ty3: tz3);

  
    friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    tz2 = __shfl(threadInput4, friend_id2);
    ty3 = __shfl(threadInput4, friend_id3);
    tz3 = __shfl(threadInput5, friend_id3);
    sum0 += a4*((lane_id < 17)? tx0: ty0);
    sum1 += a4*((lane_id < 11)? tx1: ty1);
    sum2 += a4*((lane_id < 5)? tx2: ((lane_id < 31)? ty2: tz2));
    sum3 += a4*((lane_id < 24)? ty3: tz3);

 
    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    tz2 = __shfl(threadInput4, friend_id2);
    ty3 = __shfl(threadInput4, friend_id3);
    tz3 = __shfl(threadInput5, friend_id3);
    sum0 += a3*((lane_id < 16)? tx0: ty0);
    sum1 += a3*((lane_id < 10)? tx1: ty1);
    sum2 += a3*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
    sum3 += a3*((lane_id < 24)? ty3: tz3);


    friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    ty2 = __shfl(threadInput3, friend_id2);
    tz2 = __shfl(threadInput4, friend_id2);
    ty3 = __shfl(threadInput4, friend_id3);
    tz3 = __shfl(threadInput5, friend_id3);
    sum0 += a2*((lane_id < 9 )? tx0: ty0);
    sum1 += a2*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
    sum2 += a2*((lane_id < 23)? ty2: tz2);
    sum3 += a2*((lane_id < 16)? ty3: tz3);


    OUT_2D(j   ,i) = sum0; 
    OUT_2D(j+4 ,i) = sum1; 
    OUT_2D(j+8 ,i) = sum2; 
    OUT_2D(j+12,i) = sum3; 
}

__global__ void Stencil_Cuda_Shfl8(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, int M, int N) 
{
    unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x;
    unsigned int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = (((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<5) + (lane_id>>3) ;

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3;
    int warp_id_y = ((((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<5) + (lane_id>>3))>>2;

    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5;
    DATA_TYPE threadInput6, threadInput7, threadInput8, threadInput9, threadInput10;
    int new_i = (warp_id_x<<3) + lane_id%10-1;
    int new_j = (warp_id_y<<2) + lane_id/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+64)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+96)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+128)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput4 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+160)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput5 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+192)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+192)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput6 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+224)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+224)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput7 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+256)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+256)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput8 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+288)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+288)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput9 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+320)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+320)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput10 = IN_2D(new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    DATA_TYPE sum4 = 0.0;
    DATA_TYPE sum5 = 0.0;
    DATA_TYPE sum6 = 0.0;
    DATA_TYPE sum7 = 0.0;
    int friend_id0, friend_id1, friend_id2, friend_id3;
    // int friend_id4, friend_id5, friend_id6, friend_id7;
    DATA_TYPE tx0, ty0, tx1, ty1, tz1, tx2, ty2, tz2, tx3, ty3, tz3;
    DATA_TYPE rx0, ry0, rx1, ry1, rz1, rx2, ry2, rz2, rx3, ry3, rz3;
    
    friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+17+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+25+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    tx3 = __shfl(threadInput3, friend_id3);
    ty3 = __shfl(threadInput4, friend_id3);
    rx0 = __shfl(threadInput5, friend_id0);
    ry0 = __shfl(threadInput6, friend_id0);
    rx1 = __shfl(threadInput6, friend_id1);
    ry1 = __shfl(threadInput7, friend_id1);
    rx2 = __shfl(threadInput7, friend_id2);
    ry2 = __shfl(threadInput8, friend_id2);
    rx3 = __shfl(threadInput8, friend_id3);
    ry3 = __shfl(threadInput9, friend_id3);
    sum0 += a0*((lane_id < 25)? tx0: ty0);
    sum1 += a0*((lane_id < 19)? tx1: ty1);
    sum2 += a0*((lane_id < 13)? tx2: ty2);
    sum3 += a0*((lane_id < 7 )? tx3: ty3);
    sum4 += a0*((lane_id < 25)? rx0: ry0);
    sum5 += a0*((lane_id < 19)? rx1: ry1);
    sum6 += a0*((lane_id < 13)? rx2: ry2);
    sum7 += a0*((lane_id < 7 )? rx3: ry3);

    friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+26+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+2 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    ty3 = __shfl(threadInput4, friend_id3);
    tz3 = __shfl(threadInput5, friend_id3);
    rx0 = __shfl(threadInput5, friend_id0);
    ry0 = __shfl(threadInput6, friend_id0);
    rx1 = __shfl(threadInput6, friend_id1);
    ry1 = __shfl(threadInput7, friend_id1);
    rx2 = __shfl(threadInput7, friend_id2);
    ry2 = __shfl(threadInput8, friend_id2);
    ry3 = __shfl(threadInput9, friend_id3);
    rz3 = __shfl(threadInput10, friend_id3);
    sum0 += a1*((lane_id < 18)? tx0: ty0);
    sum1 += a1*((lane_id < 12)? tx1: ty1);
    sum2 += a1*((lane_id < 6 )? tx2: ty2);
    sum3 += a1*((lane_id < 24)? ty3: tz3);
    sum4 += a1*((lane_id < 18)? rx0: ry0);
    sum5 += a1*((lane_id < 12)? rx1: ry1);
    sum6 += a1*((lane_id < 6 )? rx2: ry2);
    sum7 += a1*((lane_id < 24)? ry3: rz3);

    friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+27+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+3 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0 , friend_id0);
    ty0 = __shfl(threadInput1 , friend_id0);
    tx1 = __shfl(threadInput1 , friend_id1);
    ty1 = __shfl(threadInput2 , friend_id1);
    tx2 = __shfl(threadInput2 , friend_id2);
    ty2 = __shfl(threadInput3 , friend_id2);
    tz2 = __shfl(threadInput4 , friend_id2);
    ty3 = __shfl(threadInput4 , friend_id3);
    tz3 = __shfl(threadInput5 , friend_id3);
    rx0 = __shfl(threadInput5 , friend_id0);
    ry0 = __shfl(threadInput6 , friend_id0);
    rx1 = __shfl(threadInput6 , friend_id1);
    ry1 = __shfl(threadInput7 , friend_id1);
    rx2 = __shfl(threadInput7 , friend_id2);
    ry2 = __shfl(threadInput8 , friend_id2);
    rz2 = __shfl(threadInput9 , friend_id2);
    ry3 = __shfl(threadInput9 , friend_id3);
    rz3 = __shfl(threadInput10, friend_id3);
    sum0 += a4*((lane_id < 17)? tx0: ty0);
    sum1 += a4*((lane_id < 11)? tx1: ty1);
    sum2 += a4*((lane_id < 5 )? tx2: ((lane_id < 31)? ty2: tz2));
    sum3 += a4*((lane_id < 24)? ty3: tz3);
    sum4 += a4*((lane_id < 17)? rx0: ry0);
    sum5 += a4*((lane_id < 11)? rx1: ry1);
    sum6 += a4*((lane_id < 5 )? rx2: ((lane_id < 31)? ry2: rz2));
    sum7 += a4*((lane_id < 24)? ry3: rz3);

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+28+((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+4 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0 , friend_id0);
    ty0 = __shfl(threadInput1 , friend_id0);
    tx1 = __shfl(threadInput1 , friend_id1);
    ty1 = __shfl(threadInput2 , friend_id1);
    tx2 = __shfl(threadInput2 , friend_id2);
    ty2 = __shfl(threadInput3 , friend_id2);
    tz2 = __shfl(threadInput4 , friend_id2);
    ty3 = __shfl(threadInput4 , friend_id3);
    tz3 = __shfl(threadInput5 , friend_id3);
    rx0 = __shfl(threadInput5 , friend_id0);
    ry0 = __shfl(threadInput6 , friend_id0);
    rx1 = __shfl(threadInput6 , friend_id1);
    ry1 = __shfl(threadInput7 , friend_id1);
    rx2 = __shfl(threadInput7 , friend_id2);
    ry2 = __shfl(threadInput8 , friend_id2);
    rz2 = __shfl(threadInput9 , friend_id2);
    ry3 = __shfl(threadInput9 , friend_id3);
    rz3 = __shfl(threadInput10, friend_id3);
    sum0 += a3*((lane_id < 16)? tx0: ty0);
    sum1 += a3*((lane_id < 10)? tx1: ty1);
    sum2 += a3*((lane_id < 4 )? tx2: ((lane_id < 30)? ty2: tz2));
    sum3 += a3*((lane_id < 24)? ty3: tz3);
    sum4 += a3*((lane_id < 16)? rx0: ry0);
    sum5 += a3*((lane_id < 10)? rx1: ry1);
    sum6 += a3*((lane_id < 4 )? rx2: ((lane_id < 30)? ry2: rz2));
    sum7 += a3*((lane_id < 24)? ry3: rz3);

    friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    friend_id2 = (lane_id+5 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id3 = (lane_id+13+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0 , friend_id0);
    ty0 = __shfl(threadInput1 , friend_id0);
    tx1 = __shfl(threadInput1 , friend_id1);
    ty1 = __shfl(threadInput2 , friend_id1);
    tz1 = __shfl(threadInput3 , friend_id1);
    ty2 = __shfl(threadInput3 , friend_id2);
    tz2 = __shfl(threadInput4 , friend_id2);
    ty3 = __shfl(threadInput4 , friend_id3);
    tz3 = __shfl(threadInput5 , friend_id3);
    rx0 = __shfl(threadInput5 , friend_id0);
    ry0 = __shfl(threadInput6 , friend_id0);
    rx1 = __shfl(threadInput6 , friend_id1);
    ry1 = __shfl(threadInput7 , friend_id1);
    rz1 = __shfl(threadInput8 , friend_id1);
    ry2 = __shfl(threadInput8 , friend_id2);
    rz2 = __shfl(threadInput9 , friend_id2);
    ry3 = __shfl(threadInput9 , friend_id3);
    rz3 = __shfl(threadInput10, friend_id3);
    sum0 += a2*((lane_id < 9 )? tx0: ty0);
    sum1 += a2*((lane_id < 3 )? tx1: ((lane_id < 29)? ty1: tz1));
    sum2 += a2*((lane_id < 23)? ty2: tz2);
    sum3 += a2*((lane_id < 16)? ty3: tz3);
    sum4 += a2*((lane_id < 9 )? rx0: ry0);
    sum5 += a2*((lane_id < 3 )? rx1: ((lane_id < 29)? ry1: rz1));
    sum6 += a2*((lane_id < 23)? ry2: rz2);
    sum7 += a2*((lane_id < 16)? ry3: rz3);

    OUT_2D(j   ,i) = sum0; 
    OUT_2D(j+4 ,i) = sum1; 
    OUT_2D(j+8 ,i) = sum2; 
    OUT_2D(j+12,i) = sum3; 
    OUT_2D(j+16,i) = sum4; 
    OUT_2D(j+20,i) = sum5; 
    OUT_2D(j+24,i) = sum6; 
    OUT_2D(j+28,i) = sum7; 
}

__global__ void Stencil_Cuda_Shfl4_2(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, int M, int N) 
{
    unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x;
    unsigned int lane_id = tid % warpSize;

    int i = (((threadIdx.x + blockIdx.x * blockDim.x)>>3)<<4) + (lane_id&7)  ;
    int j = (((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<3) + (lane_id>>3) ;

    int warp_id_x = ((((threadIdx.x + blockIdx.x * blockDim.x)>>3)<<4) + (lane_id&7) )>>3;
    int warp_id_y = ((((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<3) + (lane_id>>3))>>2;

    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5;
    int new_i = (warp_id_x<<3) + lane_id%18-1;
    int new_j = (warp_id_y<<2) + lane_id/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+32)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+64)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+96)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+128)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput4 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+160)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput5 = IN_2D(new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    int friend_id0, friend_id1;
    int friend_id2, friend_id3;
    DATA_TYPE tx0, ty0, tz0, tx1, ty1, tz1, tx2, ty2, tz2, tx3, ty3, tz3;

    friend_id0 = (lane_id+1 +((lane_id>>3)*10))&(warpSize-1);
    friend_id1 = (lane_id+9 +((lane_id>>3)*10))&(warpSize-1);
    friend_id2 = (lane_id+9 +((lane_id>>3)*10))&(warpSize-1);
    friend_id3 = (lane_id+17+((lane_id>>3)*10))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput0, friend_id1);
    ty1 = __shfl(threadInput1, friend_id1);
    tz1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    tz2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput2, friend_id3);
    ty3 = __shfl(threadInput3, friend_id3);
    tz3 = __shfl(threadInput4, friend_id3);
    sum0 += a0*((lane_id < 16)? tx0: ty0);
    sum1 += a0*((lane_id < 13)? tx1: ((lane_id < 25)? ty1: tz1));
    sum2 += a0*((lane_id < 13)? tx2: ((lane_id < 25)? ty2: tz2));
    sum3 += a0*((lane_id < 8 )? tx3: ((lane_id < 24)? ty3: tz3));

    friend_id0 = (lane_id+18+((lane_id>>3)*10))&(warpSize-1);
    friend_id1 = (lane_id+26+((lane_id>>3)*10))&(warpSize-1);
    friend_id2 = (lane_id+26+((lane_id>>3)*10))&(warpSize-1);
    friend_id3 = (lane_id+2 +((lane_id>>3)*10))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tz0 = __shfl(threadInput2, friend_id0);
    tx1 = __shfl(threadInput0, friend_id1);
    ty1 = __shfl(threadInput1, friend_id1);
    tz1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    tz2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput3, friend_id3);
    ty3 = __shfl(threadInput4, friend_id3);
    sum0 += a1*((lane_id < 8 )? tx0: ((lane_id < 24)? ty0: tz0));
    sum1 += a1*((lane_id < 6 )? tx1: ((lane_id < 18)? ty1: tz1));
    sum2 += a1*((lane_id < 6 )? tx2: ((lane_id < 18)? ty2: tz2));
    sum3 += a1*((lane_id < 16)? tx3: ty3);

    friend_id0 = (lane_id+19+((lane_id>>3)*10))&(warpSize-1);
    friend_id1 = (lane_id+27+((lane_id>>3)*10))&(warpSize-1);
    friend_id2 = (lane_id+27+((lane_id>>3)*10))&(warpSize-1);
    friend_id3 = (lane_id+3 +((lane_id>>3)*10))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tz0 = __shfl(threadInput2, friend_id0);
    tx1 = __shfl(threadInput0, friend_id1);
    ty1 = __shfl(threadInput1, friend_id1);
    tz1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    tz2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput3, friend_id3);
    ty3 = __shfl(threadInput4, friend_id3);
    tz3 = __shfl(threadInput5, friend_id3);
    sum0 += a4*((lane_id < 8 )? tx0: ((lane_id < 24)? ty0: tz0));
    sum1 += a4*((lane_id < 5 )? tx1: ((lane_id < 17)? ty1: tz1));
    sum2 += a4*((lane_id < 5 )? tx2: ((lane_id < 17)? ty2: tz2));
    sum3 += a4*((lane_id < 16)? tx3: ((lane_id < 31)? ty3: tz3));

    friend_id0 = (lane_id+20+((lane_id>>3)*10))&(warpSize-1);
    friend_id1 = (lane_id+28+((lane_id>>3)*10))&(warpSize-1);
    friend_id2 = (lane_id+28+((lane_id>>3)*10))&(warpSize-1);
    friend_id3 = (lane_id+4 +((lane_id>>3)*10))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tz0 = __shfl(threadInput2, friend_id0);
    tx1 = __shfl(threadInput0, friend_id1);
    ty1 = __shfl(threadInput1, friend_id1);
    tz1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    tz2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput3, friend_id3);
    ty3 = __shfl(threadInput4, friend_id3);
    tz3 = __shfl(threadInput5, friend_id3);
    sum0 += a3*((lane_id < 8 )? tx0: ((lane_id < 24)? ty0: tz0));
    sum1 += a3*((lane_id < 4 )? tx1: ((lane_id < 16)? ty1: tz1));
    sum2 += a3*((lane_id < 4 )? tx2: ((lane_id < 16)? ty2: tz2));
    sum3 += a3*((lane_id < 16)? tx3: ((lane_id < 30)? ty3: tz3));


    friend_id0 = (lane_id+5 +((lane_id>>3)*10))&(warpSize-1);
    friend_id1 = (lane_id+13+((lane_id>>3)*10))&(warpSize-1);
    friend_id2 = (lane_id+13+((lane_id>>3)*10))&(warpSize-1);
    friend_id3 = (lane_id+21+((lane_id>>3)*10))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tz2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput3, friend_id3);
    ty3 = __shfl(threadInput4, friend_id3);
    tz3 = __shfl(threadInput5, friend_id3);
    sum0 += a2*((lane_id < 16)? tx0: ((lane_id < 29)? ty0: tz0));
    sum1 += a2*((lane_id < 9 )? tx1: ((lane_id < 24)? ty1: tz1));
    sum2 += a2*((lane_id < 9 )? tx2: ((lane_id < 24)? ty2: tz2));
    sum3 += a2*((lane_id < 8 )? tx3: ((lane_id < 23)? ty3: tz3));

    OUT_2D(j  ,i  ) = sum0; 
    OUT_2D(j  ,i+8) = sum1; 
    OUT_2D(j+4,i  ) = sum2; 
    OUT_2D(j+4,i+8) = sum3; 
}

__global__ void Stencil_Cuda_Shfl8_2(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, int M, int N) 
{
    unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x;
    unsigned int lane_id = tid % warpSize;

    int i = (((threadIdx.x + blockIdx.x * blockDim.x)>>3)<<4) + (lane_id&7)  ;
    int j = (((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<4) + (lane_id>>3) ;

    int warp_id_x = ((((threadIdx.x + blockIdx.x * blockDim.x)>>3)<<4) + (lane_id&7) )>>3;
    int warp_id_y = ((((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<4) + (lane_id>>3))>>2;

    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3, threadInput4, threadInput5;
    DATA_TYPE threadInput6, threadInput7, threadInput8, threadInput9, threadInput10;
    int new_i = (warp_id_x<<3) + lane_id%18-1;
    int new_j = (warp_id_y<<2) + lane_id/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+32)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+64)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+96)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+128)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+128)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput4 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+160)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+160)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput5 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+192)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+192)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput6 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+224)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+224)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput7 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+256)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+256)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput8 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+288)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+288)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput9 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+320)%18-1;
    new_j = (warp_id_y<<2) + (lane_id+320)/18-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput10 = IN_2D(new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    DATA_TYPE sum2 = 0.0;
    DATA_TYPE sum3 = 0.0;
    DATA_TYPE sum4 = 0.0;
    DATA_TYPE sum5 = 0.0;
    DATA_TYPE sum6 = 0.0;
    DATA_TYPE sum7 = 0.0;
    int friend_id0, friend_id1, friend_id2, friend_id3;
    int friend_id4, friend_id5, friend_id6, friend_id7;
    DATA_TYPE tx0, ty0, tz0, tx1, ty1, tz1, tx2, ty2, tz2, tx3, ty3, tz3;
    DATA_TYPE rx0, ry0, rz0, rx1, ry1, rz1, rx2, ry2, rz2, rx3, ry3, rz3;

    friend_id0 = (lane_id+1 +((lane_id>>3)*10))&(warpSize-1);
    friend_id1 = (lane_id+9 +((lane_id>>3)*10))&(warpSize-1);
    friend_id2 = (lane_id+9 +((lane_id>>3)*10))&(warpSize-1);
    friend_id3 = (lane_id+17+((lane_id>>3)*10))&(warpSize-1);
    friend_id4 = (lane_id+17+((lane_id>>3)*10))&(warpSize-1);
    friend_id5 = (lane_id+25+((lane_id>>3)*10))&(warpSize-1);
    friend_id6 = (lane_id+25+((lane_id>>3)*10))&(warpSize-1);
    friend_id7 = (lane_id+1 +((lane_id>>3)*10))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput0, friend_id1);
    ty1 = __shfl(threadInput1, friend_id1);
    tz1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    tz2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput2, friend_id3);
    ty3 = __shfl(threadInput3, friend_id3);
    tz3 = __shfl(threadInput4, friend_id3);
    rx0 = __shfl(threadInput4, friend_id4);
    ry0 = __shfl(threadInput5, friend_id4);
    rz0 = __shfl(threadInput6, friend_id4);
    rx1 = __shfl(threadInput4, friend_id5);
    ry1 = __shfl(threadInput5, friend_id5);
    rz1 = __shfl(threadInput6, friend_id5);
    rx2 = __shfl(threadInput6, friend_id6);
    ry2 = __shfl(threadInput7, friend_id6);
    rz2 = __shfl(threadInput8, friend_id6);
    rx3 = __shfl(threadInput7, friend_id7);
    ry3 = __shfl(threadInput8, friend_id7);
    sum0 += a0*((lane_id < 16)? tx0: ty0);
    sum1 += a0*((lane_id < 13)? tx1: ((lane_id < 25)? ty1: tz1));
    sum2 += a0*((lane_id < 13)? tx2: ((lane_id < 25)? ty2: tz2));
    sum3 += a0*((lane_id < 8 )? tx3: ((lane_id < 24)? ty3: tz3));
    sum4 += a0*((lane_id < 8 )? rx0: ((lane_id < 24)? ry0: rz0));
    sum5 += a0*((lane_id < 7 )? rx1: ((lane_id < 19)? ry1: rz1));
    sum6 += a0*((lane_id < 7 )? rx2: ((lane_id < 19)? ry2: rz2));
    sum7 += a0*((lane_id < 16)? rx3: ry3);

    friend_id0 = (lane_id+18+((lane_id>>3)*10))&(warpSize-1);
    friend_id1 = (lane_id+26+((lane_id>>3)*10))&(warpSize-1);
    friend_id2 = (lane_id+26+((lane_id>>3)*10))&(warpSize-1);
    friend_id3 = (lane_id+2 +((lane_id>>3)*10))&(warpSize-1);
    friend_id4 = (lane_id+2 +((lane_id>>3)*10))&(warpSize-1);
    friend_id5 = (lane_id+10+((lane_id>>3)*10))&(warpSize-1);
    friend_id6 = (lane_id+10+((lane_id>>3)*10))&(warpSize-1);
    friend_id7 = (lane_id+18+((lane_id>>3)*10))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tz0 = __shfl(threadInput2, friend_id0);
    tx1 = __shfl(threadInput0, friend_id1);
    ty1 = __shfl(threadInput1, friend_id1);
    tz1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    tz2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput3, friend_id3);
    ty3 = __shfl(threadInput4, friend_id3);
    rx0 = __shfl(threadInput5, friend_id4);
    ry0 = __shfl(threadInput6, friend_id4);
    rx1 = __shfl(threadInput5, friend_id5);
    ry1 = __shfl(threadInput6, friend_id5);
    rz1 = __shfl(threadInput7, friend_id5);
    rx2 = __shfl(threadInput7, friend_id6);
    ry2 = __shfl(threadInput8, friend_id6);
    rz2 = __shfl(threadInput9, friend_id6);
    rx3 = __shfl(threadInput7, friend_id7);
    ry3 = __shfl(threadInput8, friend_id7);
    rz3 = __shfl(threadInput9, friend_id7);
    sum0 += a1*((lane_id < 8 )? tx0: ((lane_id < 24)? ty0: tz0));
    sum1 += a1*((lane_id < 6 )? tx1: ((lane_id < 18)? ty1: tz1));
    sum2 += a1*((lane_id < 6 )? tx2: ((lane_id < 18)? ty2: tz2));
    sum3 += a1*((lane_id < 16)? tx3: ty3);
    sum4 += a1*((lane_id < 16)? rx0: ry0);
    sum5 += a1*((lane_id < 12)? rx1: ((lane_id < 24)? ry1: rz1));
    sum6 += a1*((lane_id < 12)? rx2: ((lane_id < 24)? ry2: rz2));
    sum7 += a1*((lane_id < 8 )? rx3: ((lane_id < 24)? ry3: rz3));

    friend_id0 = (lane_id+19+((lane_id>>3)*10))&(warpSize-1);
    friend_id1 = (lane_id+27+((lane_id>>3)*10))&(warpSize-1);
    friend_id2 = (lane_id+27+((lane_id>>3)*10))&(warpSize-1);
    friend_id3 = (lane_id+3 +((lane_id>>3)*10))&(warpSize-1);
    friend_id4 = (lane_id+3 +((lane_id>>3)*10))&(warpSize-1);
    friend_id5 = (lane_id+11+((lane_id>>3)*10))&(warpSize-1);
    friend_id6 = (lane_id+11+((lane_id>>3)*10))&(warpSize-1);
    friend_id7 = (lane_id+19+((lane_id>>3)*10))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tz0 = __shfl(threadInput2, friend_id0);
    tx1 = __shfl(threadInput0, friend_id1);
    ty1 = __shfl(threadInput1, friend_id1);
    tz1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    tz2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput3, friend_id3);
    ty3 = __shfl(threadInput4, friend_id3);
    tz3 = __shfl(threadInput5, friend_id3);
    rx0 = __shfl(threadInput5, friend_id4);
    ry0 = __shfl(threadInput6, friend_id4);
    rz0 = __shfl(threadInput7, friend_id4);
    rx1 = __shfl(threadInput5, friend_id5);
    ry1 = __shfl(threadInput6, friend_id5);
    rz1 = __shfl(threadInput7, friend_id5);
    rx2 = __shfl(threadInput7, friend_id6);
    ry2 = __shfl(threadInput8, friend_id6);
    rz2 = __shfl(threadInput9, friend_id6);
    rx3 = __shfl(threadInput7, friend_id7);
    ry3 = __shfl(threadInput8, friend_id7);
    rz3 = __shfl(threadInput9, friend_id7);
    sum0 += a4*((lane_id < 8 )? tx0: ((lane_id < 24)? ty0: tz0));
    sum1 += a4*((lane_id < 5 )? tx1: ((lane_id < 17)? ty1: tz1));
    sum2 += a4*((lane_id < 5 )? tx2: ((lane_id < 17)? ty2: tz2));
    sum3 += a4*((lane_id < 16)? tx3: ((lane_id < 31)? ty3: tz3));
    sum4 += a4*((lane_id < 16)? rx0: ((lane_id < 31)? ry0: rz0));
    sum5 += a4*((lane_id < 11)? rx1: ((lane_id < 24)? ry1: rz1));
    sum6 += a4*((lane_id < 11)? rx2: ((lane_id < 24)? ry2: rz2));
    sum7 += a4*((lane_id < 8 )? rx3: ((lane_id < 24)? ry3: rz3));

    friend_id0 = (lane_id+20+((lane_id>>3)*10))&(warpSize-1);
    friend_id1 = (lane_id+28+((lane_id>>3)*10))&(warpSize-1);
    friend_id2 = (lane_id+28+((lane_id>>3)*10))&(warpSize-1);
    friend_id3 = (lane_id+4 +((lane_id>>3)*10))&(warpSize-1);
    friend_id4 = (lane_id+4 +((lane_id>>3)*10))&(warpSize-1);
    friend_id5 = (lane_id+12+((lane_id>>3)*10))&(warpSize-1);
    friend_id6 = (lane_id+12+((lane_id>>3)*10))&(warpSize-1);
    friend_id7 = (lane_id+20+((lane_id>>3)*10))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tz0 = __shfl(threadInput2, friend_id0);
    tx1 = __shfl(threadInput0, friend_id1);
    ty1 = __shfl(threadInput1, friend_id1);
    tz1 = __shfl(threadInput2, friend_id1);
    tx2 = __shfl(threadInput2, friend_id2);
    ty2 = __shfl(threadInput3, friend_id2);
    tz2 = __shfl(threadInput4, friend_id2);
    tx3 = __shfl(threadInput3, friend_id3);
    ty3 = __shfl(threadInput4, friend_id3);
    tz3 = __shfl(threadInput5, friend_id3);
    rx0 = __shfl(threadInput5, friend_id4);
    ry0 = __shfl(threadInput6, friend_id4);
    rz0 = __shfl(threadInput7, friend_id4);
    rx1 = __shfl(threadInput5, friend_id5);
    ry1 = __shfl(threadInput6, friend_id5);
    rz1 = __shfl(threadInput7, friend_id5);
    rx2 = __shfl(threadInput7, friend_id6);
    ry2 = __shfl(threadInput8, friend_id6);
    rz2 = __shfl(threadInput9, friend_id6);
    rx3 = __shfl(threadInput7, friend_id7);
    ry3 = __shfl(threadInput8, friend_id7);
    rz3 = __shfl(threadInput9, friend_id7);
    sum0 += a3*((lane_id < 8 )? tx0: ((lane_id < 24)? ty0: tz0));
    sum1 += a3*((lane_id < 4 )? tx1: ((lane_id < 16)? ty1: tz1));
    sum2 += a3*((lane_id < 4 )? tx2: ((lane_id < 16)? ty2: tz2));
    sum3 += a3*((lane_id < 16)? tx3: ((lane_id < 30)? ty3: tz3));
    sum4 += a3*((lane_id < 16)? rx0: ((lane_id < 30)? ry0: rz0));
    sum5 += a3*((lane_id < 10)? rx1: ((lane_id < 24)? ry1: rz1));
    sum6 += a3*((lane_id < 10)? rx2: ((lane_id < 24)? ry2: rz2));
    sum7 += a3*((lane_id < 8 )? rx3: ((lane_id < 24)? ry3: rz3));

    friend_id0 = (lane_id+5 +((lane_id>>3)*10))&(warpSize-1);
    friend_id1 = (lane_id+13+((lane_id>>3)*10))&(warpSize-1);
    friend_id2 = (lane_id+13+((lane_id>>3)*10))&(warpSize-1);
    friend_id3 = (lane_id+21+((lane_id>>3)*10))&(warpSize-1);
    friend_id4 = (lane_id+21+((lane_id>>3)*10))&(warpSize-1);
    friend_id5 = (lane_id+29+((lane_id>>3)*10))&(warpSize-1);
    friend_id6 = (lane_id+29+((lane_id>>3)*10))&(warpSize-1);
    friend_id7 = (lane_id+5 +((lane_id>>3)*10))&(warpSize-1);
    tx0 = __shfl(threadInput1, friend_id0);
    ty0 = __shfl(threadInput2, friend_id0);
    tz0 = __shfl(threadInput3, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    tx2 = __shfl(threadInput3, friend_id2);
    ty2 = __shfl(threadInput4, friend_id2);
    tz2 = __shfl(threadInput5, friend_id2);
    tx3 = __shfl(threadInput3, friend_id3);
    ty3 = __shfl(threadInput4, friend_id3);
    tz3 = __shfl(threadInput5, friend_id3);
    rx0 = __shfl(threadInput5 , friend_id4);
    ry0 = __shfl(threadInput6 , friend_id4);
    rz0 = __shfl(threadInput7 , friend_id4);
    rx1 = __shfl(threadInput5 , friend_id5);
    ry1 = __shfl(threadInput6 , friend_id5);
    rz1 = __shfl(threadInput7 , friend_id5);
    rx2 = __shfl(threadInput7 , friend_id6);
    ry2 = __shfl(threadInput8 , friend_id6);
    rz2 = __shfl(threadInput9 , friend_id6);
    rx3 = __shfl(threadInput8 , friend_id7);
    ry3 = __shfl(threadInput9 , friend_id7);
    rz3 = __shfl(threadInput10, friend_id7);
    sum0 += a2*((lane_id < 16)? tx0: ((lane_id < 29)? ty0: tz0));
    sum1 += a2*((lane_id < 9 )? tx1: ((lane_id < 24)? ty1: tz1));
    sum2 += a2*((lane_id < 9 )? tx2: ((lane_id < 24)? ty2: tz2));
    sum3 += a2*((lane_id < 8 )? tx3: ((lane_id < 23)? ty3: tz3));
    sum4 += a2*((lane_id < 8 )? rx0: ((lane_id < 23)? ry0: rz0));
    sum5 += a2*((lane_id < 3 )? rx1: ((lane_id < 16)? ry1: rz1));
    sum6 += a2*((lane_id < 3 )? rx2: ((lane_id < 16)? ry2: rz2));
    sum7 += a2*((lane_id < 16)? rx3: ((lane_id < 29)? ry3: rz3));


    OUT_2D(j   ,i  ) = sum0; 
    OUT_2D(j   ,i+8) = sum1; 
    OUT_2D(j+4 ,i  ) = sum2; 
    OUT_2D(j+4 ,i+8) = sum3; 
    OUT_2D(j+8 ,i  ) = sum4; 
    OUT_2D(j+8 ,i+8) = sum5; 
    OUT_2D(j+12,i  ) = sum6; 
    OUT_2D(j+12,i+8) = sum7; 
}

__global__ void Stencil_Cuda_Sm(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, int M, int N) 
{
    __shared__ DATA_TYPE local[18*18];
    int i = threadIdx.x + blockIdx.x * blockDim.x ; 
    int j = threadIdx.y + blockIdx.y * blockDim.y ; 
    int li = threadIdx.x + 1;
    int lj = threadIdx.y + 1;

    LOC_2D(lj,li) = IN_2D(j,i);
    int n = (j == 0)      ? j : j - 1;
    int s = (j == M-1)    ? j : j + 1;
    int w = (i == 0)      ? i : i - 1;
    int e = (i == N-1)    ? i : i + 1;

    if(threadIdx.x == 0)              LOC_2D(lj,li-1) = IN_2D(j,w);

    if(threadIdx.x == blockDim.x-1)   LOC_2D(lj,li+1) = IN_2D(j,e);
            
    if(threadIdx.y == 0)              LOC_2D(lj-1,li) = IN_2D(n,i);
                                                
    if(threadIdx.y == blockDim.y-1)   LOC_2D(lj+1,li) = IN_2D(s,i);

    __syncthreads();

    OUT_2D(j,i) = a0 *LOC_2D(lj-1,li  ) + 
                  a1 *LOC_2D(lj  ,li-1) + 
                  a2 *LOC_2D(lj+1,li  ) +
                  a3 *LOC_2D(lj  ,li+1) + 
                  a4 *LOC_2D(lj  ,li  ) ;
}

inline double tol_finder(int error_tol)
{
    double val = 1.0;
    for(; error_tol > 0; error_tol--)
        val *= 10;
    return 1.0/(double)val;
}

bool Verify(DATA_TYPE *test, DATA_TYPE *ref, int n)
{
    bool flag = true;
    double precision = tol_finder(2);

    for(int i = 0; i < n; i++)
    {
        if(fabs(test[i]-ref[i]) > precision)
        {
            std::cout << "difference: " << fabs(test[i]-ref[i])-precision << std::endl;
            std::cout << "wrong at " << i << " test:" << test[i] << " (ref: " << ref[i] << ")";
            std::cout << std::endl;
            flag = false;
            break;
        }
    }
    return flag;
}

__global__ void Stencil_Cuda_Shfl(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, int M, int N) 
{
    unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x;
    unsigned int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = threadIdx.y + blockIdx.y * blockDim.y ;

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3;
    int warp_id_y = (threadIdx.y + blockIdx.y * blockDim.y)>>2;

    DATA_TYPE threadInput0, threadInput1;
    int new_i = (warp_id_x<<3) + lane_id%10-1;
    int new_j = (warp_id_y<<2) + lane_id/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_2D(new_j, new_i);

    DATA_TYPE sum = 0.0;
    int friend_id;
    DATA_TYPE tx, ty;
    
    friend_id = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a0*((lane_id < 25)? tx: ty);

    friend_id = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a1*((lane_id < 18)? tx: ty);

    friend_id = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a4*((lane_id < 17)? tx: ty);

    friend_id = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a3*((lane_id < 16)? tx: ty);

    friend_id = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    tx = __shfl(threadInput0, friend_id);
    ty = __shfl(threadInput1, friend_id);
    sum += a2*((lane_id < 9)? tx: ty);

    OUT_2D(j,i) = sum; 
}

__global__ void Stencil_Cuda_Shfl2(DATA_TYPE *in, DATA_TYPE *out, DATA_TYPE a0, DATA_TYPE a1, 
        DATA_TYPE a2, DATA_TYPE a3, DATA_TYPE a4, int M, int N) 
{
    unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x;
    unsigned int lane_id = tid % warpSize;

    int i = threadIdx.x + blockIdx.x * blockDim.x ;
    int j = (((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<3) + (lane_id>>3) ;

    int warp_id_x = (threadIdx.x + blockIdx.x * blockDim.x)>>3;
    int warp_id_y = ((((threadIdx.y + blockIdx.y * blockDim.y)>>2)<<3) + (lane_id>>3))>>2;

    DATA_TYPE threadInput0, threadInput1, threadInput2, threadInput3;
    int new_i = (warp_id_x<<3) + lane_id%10-1;
    int new_j = (warp_id_y<<2) + lane_id/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput0 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+32)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+32)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput1 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+64)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+64)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput2 = IN_2D(new_j, new_i);
    new_i = (warp_id_x<<3) + (lane_id+96)%10-1;
    new_j = (warp_id_y<<2) + (lane_id+96)/10-1;
    new_j = (new_j == -1)   ? 0   : new_j;
    new_j = (new_j >= M)    ? M-1 : new_j;
    new_i = (new_i == -1)   ? 0   : new_i;
    new_i = (new_i >= N)    ? N-1 : new_i;
    threadInput3 = IN_2D(new_j, new_i);

    DATA_TYPE sum0 = 0.0;
    DATA_TYPE sum1 = 0.0;
    int friend_id0, friend_id1;
    DATA_TYPE tx0, ty0, tx1, ty1, tz1;

    friend_id0 = (lane_id+1 +((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+9 +((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    sum0 += a0*((lane_id < 25)? tx0: ty0);
    sum1 += a0*((lane_id < 19)? tx1: ty1);

    friend_id0 = (lane_id+10+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+18+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    sum0 += a1*((lane_id < 18)? tx0: ty0);
    sum1 += a1*((lane_id < 12)? tx1: ty1);

    friend_id0 = (lane_id+11+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+19+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    sum0 += a4*((lane_id < 17)? tx0: ty0);
    sum1 += a4*((lane_id < 11)? tx1: ty1);

    friend_id0 = (lane_id+12+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+20+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    sum0 += a3*((lane_id < 16)? tx0: ty0);
    sum1 += a3*((lane_id < 10)? tx1: ty1);

    friend_id0 = (lane_id+21+((lane_id>>3)<<1))&(warpSize-1);
    friend_id1 = (lane_id+29+((lane_id>>3)<<1))&(warpSize-1);
    tx0 = __shfl(threadInput0, friend_id0);
    ty0 = __shfl(threadInput1, friend_id0);
    tx1 = __shfl(threadInput1, friend_id1);
    ty1 = __shfl(threadInput2, friend_id1);
    tz1 = __shfl(threadInput3, friend_id1);
    sum0 += a2*((lane_id < 9)? tx0: ty0);
    sum1 += a2*((lane_id < 3)? tx1: ((lane_id < 29)? ty1: tz1));

    OUT_2D(j  ,i) = sum0; 
    OUT_2D(j+4,i) = sum1; 
}

int main(int argc, char **argv)
{
#ifdef __DEBUG
    int m = 64;
    int n = 64;
#else
    int m = 4096;
    int n = 4096;
#endif
    int total = (m)*(n);
    const int K = 5;
#ifdef __DEBUG
    DATA_TYPE args[K] = {1.0, 1.0, 1.0, 1.0, 1.0};
#else
    DATA_TYPE args[K] = {0.20, 0.20, 0.20, 0.20, 0.20};
#endif
    DATA_TYPE *in = new DATA_TYPE[total];
    DATA_TYPE *out_ref = new DATA_TYPE[total];
    unsigned int seed = time(NULL);
    Init_Input_2D(in, m, n, seed);

    // Show_Me(in, m, n, "Input:");
    for(int i=0; i< ITER; i++)
    {
        Stencil_Seq(in, out_ref, args[0], args[1], args[2], args[3], args[4], m, n);
        swap(in, out_ref);
    }
    swap(in, out_ref);
    // Show_Me(out_ref, m, n, "Output:");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    float time_wo_pci;

    DATA_TYPE *in_d;
    DATA_TYPE *out_d;
    DATA_TYPE *out = new DATA_TYPE[total];
    hipMalloc((void**)&in_d, total*sizeof(DATA_TYPE));
    hipMalloc((void**)&out_d, total*sizeof(DATA_TYPE));
    // Cuda version
    /////////////////////////////////////////////////////////
    Init_Input_2D(in, m, n, seed);
    Clear_Output_2D(out, m, n);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid((n)/16, (m)/16, 1);
    dim3 dimBlock(16, 16, 1);
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda<<<dimGrid, dimBlock>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], m, n); 
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, m, n, "Output:");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, m, n, ITER, 9, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, m, n, ITER, time_wo_pci));

    // Cuda Shared Memory version 
    /////////////////////////////////////////////////////////
    Init_Input_2D(in, m, n, seed);
    Clear_Output_2D(out, m, n);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid1((n)/16, (m)/16, 1);
    dim3 dimBlock1(16, 16, 1);
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Sm<<<dimGrid1, dimBlock1>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], m, n); 
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, m, n, "Output(SM):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Sm: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Sm Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, m, n, ITER, 9, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, m, n, ITER, time_wo_pci));

    // Cuda Shfl version 
    /////////////////////////////////////////////////////////
    Init_Input_2D(in, m, n, seed);
    Clear_Output_2D(out, m, n);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid2((n)/8, (m)/32, 1);
    dim3 dimBlock2(8, 32, 1);
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl<<<dimGrid2, dimBlock2>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, m, n, "Output(Shfl):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, m, n, ITER, 9, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, m, n, ITER, time_wo_pci));

    // Cuda Shfl2 version 
    /////////////////////////////////////////////////////////
    Init_Input_2D(in, m, n, seed);
    Clear_Output_2D(out, m, n);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid3((n)/8, (m)/(32*2), 1);
    dim3 dimBlock3(8, 32, 1);
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl2<<<dimGrid3, dimBlock3>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, m, n, "Output(Shfl2):");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl2: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl2 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, m, n, ITER, 9, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, m, n, ITER, time_wo_pci));


    // Cuda Shfl4 version 
    /////////////////////////////////////////////////////////
    Init_Input_2D(in, m, n, seed);
    Clear_Output_2D(out, m, n);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid4((n)/8, (m)/(32*4), 1);
    dim3 dimBlock4(8, 32, 1);
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl4<<<dimGrid4, dimBlock4>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], m, n); 
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, m, n, "Output:");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl4: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Shfl4 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, m, n, ITER, 9, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, m, n, ITER, time_wo_pci));

    // Cuda Shfl8 version 
    /////////////////////////////////////////////////////////
    Init_Input_2D(in, m, n, seed);
    Clear_Output_2D(out, m, n);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid5((n)/8, (m)/(32*8), 1);
    dim3 dimBlock5(8, 32, 1);
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl8<<<dimGrid5, dimBlock5>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], m, n); 
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, m, n, "Output:");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl8: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Shfl8 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, m, n, ITER, 9, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, m, n, ITER, time_wo_pci));

    // Cuda Shfl4_2 version 
    /////////////////////////////////////////////////////////
    Init_Input_2D(in, m, n, seed);
    Clear_Output_2D(out, m, n);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid6((n)/(8*2), (m)/(32*2), 1);
    dim3 dimBlock6(8, 32, 1);
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl4_2<<<dimGrid6, dimBlock6>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], m, n); 
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, m, n, "Output:");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl4_2: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Shfl4_2 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, m, n, ITER, 9, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, m, n, ITER, time_wo_pci));

    // Cuda Shfl8_2 version 
    /////////////////////////////////////////////////////////
    Init_Input_2D(in, m, n, seed);
    Clear_Output_2D(out, m, n);
    hipMemcpy(in_d, in, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, total*sizeof(DATA_TYPE), hipMemcpyHostToDevice);

    dim3 dimGrid7((n)/(8*2), (m)/(32*4), 1);
    dim3 dimBlock7(8, 32, 1);
    hipEventRecord(start);
    for(int i=0; i< ITER; i++)
    {
        Stencil_Cuda_Shfl8_2<<<dimGrid7, dimBlock7>>>(in_d, out_d, 
                args[0], args[1], args[2], args[3], args[4], m, n);
        swap(in_d, out_d);
    }
    swap(in_d, out_d);
    hipEventRecord(stop);
    hipMemcpy(out, out_d, (total)*sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    // Show_Me(out, m, n, "Output:");
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Verify Cuda_Shfl8_2: " << boolalpha << Verify(out, out_ref, total) << endl;
    cout << "Cuda_Shfl8_2 Time: " << milliseconds << endl;
    time_wo_pci = milliseconds * 1.0e-03;
    printf("FLOPS        : %.3f (GFLOPS)\n", GetGFLOPS(1, m, n, ITER, 9, time_wo_pci));
    printf("Throughput   : %.3f (GB/s)\n", GetThroughput(1, m, n, ITER, time_wo_pci));
   
    hipFree(in_d);
    hipFree(out_d);

    delete[] in;
    delete[] out;
    delete[] out_ref;
}

